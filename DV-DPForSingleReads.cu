/*
 *
 *    DV-DPForSingleReads.cu
 *    Soap3(gpu)
 *
 *
 *    This program is free software; you can redistribute it and/or
 *    modify it under the terms of the GNU General Public License
 *    as published by the Free Software Foundation; either version 2
 *    of the License, or (at your option) any later version.
 *
 *    This program is distributed in the hope that it will be useful,
 *    but WITHOUT ANY WARRANTY; without even the implied warranty of
 *    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *    GNU General Public License for more details.
 *
 *    You should have received a copy of the GNU General Public License
 *    along with this program; if not, write to the Free Software
 *    Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */

#include "DV-DPForSingleReads.h"
#include "OutputDPResult.h"
#include "DV-DPfunctions.h"

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <semaphore.h>
#include <hip/hip_runtime.h>

#include <functional>
#include <vector>
using namespace std;

typedef unsigned char uchar;
typedef unsigned int uint;
typedef unsigned long long uint64;

using namespace SingleDP_Space;
class SingleDPWrapper
{
        UnalignedSinglesArrays * unalignedReads;
        uint * queries, *upkdReadLengths, *origReadIDs;
        char * upkdQueryNames, *upkdQualities;
        uint maxReadLength;
        Soap3Index * index;
        uint * _bwt, *_revBwt, *_occ, *_revOcc;
        int alignmentType;
        uint accumReadNum;
        int outputFormat;
        FILE * outputFile;
        samfile_t * samOutputDPFilePtr;

        DPParameters * dpParameters;
        SOAP3Wrapper<void> * soap3Wrapper;

    public:
        uint numDPAlignedRead, numDPAlignment;

        SingleDPWrapper ( UnalignedSinglesArrays * unalignedReads,
                          uint * queries, uint * upkdReadLengths, uint * origReadIDs,
                          char * upkdQueryNames, char * upkdQualities, uint maxReadLength,
                          Soap3Index * index,
                          uint * _bwt, uint * _revBwt,
                          uint * _occ, uint * _revOcc,
                          int alignmentType, DPParameters * dpParameters,
                          uint accumReadNum, int outputFormat,
                          FILE * outputFile, samfile_t * samOutputDPFilePtr )
        {
            MC_MemberCopy2 ( this->, , unalignedReads, dpParameters );
            MC_MemberCopy3 ( this->, , queries, upkdReadLengths, origReadIDs );
            MC_MemberCopy3 ( this->, , upkdQueryNames, upkdQualities, maxReadLength );
            MC_MemberCopy ( this->, , index );
            MC_MemberCopy4 ( this->, , _bwt, _revBwt, _occ, _revOcc );
            MC_MemberCopy2 ( this->, , alignmentType, accumReadNum );
            MC_MemberCopy3 ( this->, , outputFormat, outputFile, samOutputDPFilePtr );
            soap3Wrapper =
                new SOAP3Wrapper<void> ( index,
                                         _bwt, _revBwt,
                                         _occ, _revOcc );
        }
        ~SingleDPWrapper ()
        {
            delete soap3Wrapper;
        }

        void seeding ( QueryIDStream * inputStream,
                       CandidateStream * canStream, QueryIDStream * unseededIDStream )
        {
            SingleEndSeedingEngine::
            performSeeding (
                /* input */
                inputStream, dpParameters,
                queries, upkdReadLengths, maxReadLength,
                soap3Wrapper, index,
                /* output */
                canStream, unseededIDStream );
        }
        void alignment ( CandidateStream * canStream,
                         QueryIDStream * unalignedIDStream )
        {
            uint alignedRead = 0, alignment = 0;
            SingleEndAlignmentEngine::
            performAlignment (
                /* input */
                canStream, dpParameters,
                queries, upkdReadLengths, maxReadLength,
                upkdQueryNames, origReadIDs, upkdQualities,
                index,
                alignmentType,
                accumReadNum, outputFormat,
                outputFile, samOutputDPFilePtr,
                /* output */
                unalignedIDStream, alignedRead, alignment );
            numDPAlignedRead += alignedRead;
            numDPAlignment += alignment;
        }
        void singleDPOneRound ( QueryIDStream * inputStream,
                                QueryIDStream * unseededIDStream, QueryIDStream * unalignedIDStream )
        {
            CandidateStream * canStream = new CandidateStream;
            soap3Wrapper->copyIndex ();
            seeding ( inputStream, canStream, unseededIDStream );
            soap3Wrapper->freeIndex ();
            alignment ( canStream, unalignedIDStream );
            delete canStream;
        }
        void outputUnaligned ( QueryIDStream * unalignedIDStream )
        {
            DPSOutputUnalignedReads (
                unalignedIDStream,
                queries, upkdReadLengths, maxReadLength,
                index, upkdQueryNames, origReadIDs, upkdQualities,
                accumReadNum, outputFormat,
                outputFile, samOutputDPFilePtr
            );
        }

        void run ()
        {
            numDPAlignedRead = 0;
            numDPAlignment = 0;
            QueryIDStream * input = new QueryIDStream ( unalignedReads );
            QueryIDStream * unaligned_round1 = new QueryIDStream;
            singleDPOneRound ( input, unaligned_round1, unaligned_round1 );
            delete input;
            outputUnaligned ( unaligned_round1 );
            delete unaligned_round1;
        }
};


void DPForUnalignSingle2 ( UnalignedSinglesArrays * unalignedReads,
                           unsigned int * queries, unsigned int * upkdReadLengths, unsigned int * origReadIDs, char * upkdQueryNames, char * upkdQualities,
                           unsigned int maxReadLength,
                           Soap3Index * index,
                           unsigned int * _bwt, unsigned int * _revBwt,
                           unsigned int * _occ, unsigned int * _revOcc,
                           int alignmentType, DPParameters * dpParameters,
                           unsigned int & numDPAlignedRead, unsigned int & numDPAlignment,
                           unsigned int accumReadNum, int outputFormat,
                           FILE * outputFile, samfile_t * samOutputDPFilePtr )
{
    using namespace SingleDP_Space;
    SingleDPWrapper
    singleDPWrapper (
        unalignedReads,
        queries, upkdReadLengths, origReadIDs,
        upkdQueryNames, upkdQualities, maxReadLength,
        index, _bwt,  _revBwt,
        _occ,  _revOcc,
        alignmentType, dpParameters,
        accumReadNum, outputFormat,
        outputFile, samOutputDPFilePtr );
    singleDPWrapper.run ();
    numDPAlignedRead = singleDPWrapper.numDPAlignedRead;
    numDPAlignment = singleDPWrapper.numDPAlignment;
}
