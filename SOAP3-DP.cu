#include "hip/hip_runtime.h"
/*
 *
 *    SOAP3-DP.cu
 *    Soap3(gpu)
 *
 *    Copyright (C) 2011, HKU
 *
 *    This program is free software; you can redistribute it and/or
 *    modify it under the terms of the GNU General Public License
 *    as published by the Free Software Foundation; either version 2
 *    of the License, or (at your option) any later version.
 *
 *    This program is distributed in the hope that it will be useful,
 *    but WITHOUT ANY WARRANTY; without even the implied warranty of
 *    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *    GNU General Public License for more details.
 *
 *    You should have received a copy of the GNU General Public License
 *    along with this program; if not, write to the Free Software
 *    Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <pthread.h>
#include <zlib.h>


#include "Release.h"
#include "DV-Kernel.h"
#include "CPUfunctions.h"
#include "alignment.h"
#include "SAM.h"
#include "samtools-0.1.18/bam.h"

#include "BGS-IO.h"
#include "2bwt-flex/SRAArguments.h"
#include "PEAlgnmt.h"
#include "AlgnResult.h"
#include "OutputDPResult.h"
#include "DV-SemiDP.h"
#include "DV-DPForBothUnalign.h"
#include "DV-DPForSingleReads.h"

#include "aio_thread.h"

int main ( int argc, char ** argv )
{
    // ======================================================================================
    // | VARIABLE DECLARATION                                                               |
    // ======================================================================================
    // local variables used in main, like BWT indexes and such.
    int i;
    double startTime, indexLoadTime, readLoadTime;
    double lastEventTime;
    double totalReadLoadTime = 0.0;
    double totalAlignmentTime = 0.0;
    double totalTrimAlignmentTime = 0.0;
    char * queryFileName = "";
    char * queryFileName2 = "";
    FILE * outputFile;
    FILE * outputDPFile;
    FILE * outputDoneFile;
    // FILE *queryFile;
    // FILE *queryFile2;
    //gzFile * gzQueryFile;
    //gzFile * gzQueryFile2;
    gzFile gzQueryFile;
    gzFile gzQueryFile2;
    bamFile bamQueryFile;
    bam_header_t * bamHeader;
    bam1_t * bam;
    char isFastq = 0;
    // user-specified maximum read length
    // and number of words per query
    uint maxReadLength;
    uint wordPerQuery;
    // uint numQueries;
    ullint roundUp;
    ullint totalQueryLength;
    uint * queries;
    uint * readLengths;
    uint * readIDs;
    char * upkdQueryNames;
    char * upkdQualities;
    uint * queries0;
    uint * readLengths0;
    uint * readIDs0;
    char * upkdQueryNames0;
    char * upkdQualities0;
    uint * queries1;
    uint * readLengths1;
    uint * readIDs1;
    char * upkdQueryNames1;
    char * upkdQualities1;
    unsigned int * unAlignedPair;
    char queryFileBuffer[INPUT_BUFFER_SIZE];
    char queryFileBuffer2[INPUT_BUFFER_SIZE];
    unsigned long long numOfAnswer;
    unsigned int numOfAlignedRead;
    unsigned int numOfUnAlignedPairs;
    uint detected_read_length = 0;
    // for single-end reads
    // it represents the max read length for the first ten reads (i.e. 0, 1, ..., 9)
    // for paired-end reads
    // the max read length for the first ten reads with even readIDs (i.e. 0,2,...,18)
    uint detected_read_length2 = 0;
    // for paired-end reads only
    // the max read length for the first ten reads with odd readIDs (i.e. 1,3,...,19)
    // Declare variables and set up preference for device.
    // #ifdef BGS_GPU_CASE_BREAKDOWN_TIME
    // hipEvent_t start, stop;
    // float time1, totalDeviceTime;
    // #endif
    uint * _bwt, *_occ;
    uint * _revBwt, *_revOcc;
    // Accumulated number of reads aligned and alignments
    ullint totalReadsAlignedForInputReads = 0;
    ullint totalAnsForInputReads = 0;
    // Input parameters
    InputOptions input_options;
    // Indicate whether the index has been loaded to GPU
    uint indexLoadedToGPU = 0;
    // The inputs for multi option
    MultiInputItem * multiInput = NULL;
    int expNum = 0;
    int currExp = 0;
    // ======================================================================================
    // | Configuration on GPU functions                                                     |
    // ======================================================================================
    // not much effect, also this is not supported by old version of cuda library
    // like the machines in TJ
    // thus these are depreciated
    hipDeviceSetCacheConfig ( hipFuncCachePreferL1 );
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferShared);
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_4mismatch_1), hipFuncCachePreferShared);
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_4mismatch_2), hipFuncCachePreferShared);
    // ======================================================================================
    // | PARSING CONFIGURATION FILE                                                         |
    // ======================================================================================
    char * iniFileName;
    iniFileName = ( char * ) malloc ( strlen ( argv[0] ) + 5 );
    strcpy ( iniFileName, argv[0] );
    strcpy ( iniFileName + strlen ( argv[0] ), ".ini" );
    IniParams ini_params;

    if ( ParseIniFile ( iniFileName, ini_params ) != 0 )
    {
        fprintf ( stderr, "Failed to parse config file ... %s\n", iniFileName );
        return 1;
    }

    printf ( "\n[Main] %s v%d.%d.%d (%s)\n", PROJECT_NAME, PROJECT_MAJOR, PROJECT_MINOR, PROJECT_REV, PROJECT_SPECIAL );
    // printf("[Main] Finished parsing ini file %s.\n\n", iniFileName);
    free ( iniFileName );
    // ======================================================================================
    // | CHECK THE INPUT ARGUMENTS                                                          |
    // ======================================================================================
    bool inputValid = parseInputArgs ( argc, argv, input_options );

    if ( !inputValid )
    { exit ( 1 ); }

    // for multi mode
    if ( input_options.isReadList == 1 )
    {
        multiInput = loadMultiInputFile ( input_options.queryFileName, ( input_options.readType == PAIR_END_READ ) ? 1 : 0,
                                          input_options.isReadBAM, expNum );
        updateInputOption ( ( &input_options ), multiInput, currExp++ );
    }

    // get the name of the query file (and the second query file for pair-ended reads)
    queryFileName = input_options.queryFileName;

    if ( input_options.readType == PAIR_END_READ && input_options.isReadBAM == 0 )
    {
        queryFileName2 = input_options.queryFileName2;
    }

    maxReadLength = input_options.maxReadLength;

    if ( input_options.readType == SINGLE_READ )
    { printf ( "[Main] Loading read file %s\n", queryFileName ); }
    else if ( input_options.isReadBAM == 0 )
    { printf ( "[Main] Loading read files %s and %s\n", queryFileName, queryFileName2 ); }

    // ======================================================================================
    // | Restriction on GPU card with 3G memory                                             |
    // ======================================================================================
    // restriction on the maxReadLength when using GPU card with 3G memory
    if ( ( ini_params.Ini_GPUMemory == 3 ) && ( maxReadLength > 128 ) )
    {
        printf ( "For GPU card with 3G memory, the program cannot support maximum read length more than 128.\n" );
        exit ( 1 );
    }

    // restriction on the number of hits of each end for pairing
#ifdef NO_CONSTRAINT_SINGLE_READ_NUM_FOR_PAIRING

    if ( input_options.readType == PAIR_END_READ )
    { ini_params.Ini_MaxOutputPerRead = 0xFFFFFFFF; }

#endif

    // set the maximum number of mismatches allowed for soap3 alignment
    // if DP is enabled.
    if ( input_options.enableDP == 1 )
    {
        input_options.numMismatch = ini_params.Ini_Soap3MisMatchAllow;
    }

    // ======================================================================================
    // | Selection of GPU device                                                            |
    // ======================================================================================

    if ( input_options.GPUDeviceID > -1 )
    { hipSetDevice ( input_options.GPUDeviceID ); }

    // ======================================================================================
    // | VARIABLES SETTING AND INITIALISATION                                               |
    // ======================================================================================
    // determine number of words per query. rounded up to power of 2
    wordPerQuery = 1;

    while ( wordPerQuery < maxReadLength )
    { wordPerQuery *= 2; }

    wordPerQuery = wordPerQuery / CHAR_PER_WORD;
    uint maxNumQueries = MAX_NUM_BATCH * NUM_BLOCKS * THREADS_PER_BLOCK;
    // maxNumQueries has to be divible by 2
    maxNumQueries = maxNumQueries / 2 * 2;
    // For Output filenames
    char * outputFileName[MAX_NUM_CPU_THREADS];

    for ( i = 0; i < MAX_NUM_CPU_THREADS; i++ )
    { outputFileName[i] = ( char * ) malloc ( strlen ( input_options.outputPrefix ) + 10 ); }

    // For Output of the DP results
    char * outputDPFileName;
    outputDPFileName = ( char * ) malloc ( strlen ( input_options.outputPrefix ) + 9 );
    // For Output of the unpaired results
    char * outputUnpairFileName;
    outputUnpairFileName = ( char * ) malloc ( strlen ( input_options.outputPrefix ) + 8 );
    // For Output the DONE file
    char * outputDoneFileName;
    outputDoneFileName = ( char * ) malloc ( strlen ( input_options.outputPrefix ) + 6 );

    for ( i = 0; i < MAX_NUM_CPU_THREADS; i++ )
    {
        sprintf ( outputFileName[i], "%s.gout.%d", input_options.outputPrefix, i + 1 );
    }

    sprintf ( outputDPFileName, "%s.dpout.1", input_options.outputPrefix );
    sprintf ( outputUnpairFileName, "%s.unpair", input_options.outputPrefix );
    sprintf ( outputDoneFileName, "%s.done", input_options.outputPrefix );
    // ======================================================================================
    // | STRUCTURES FOR SEMI-GLOBAL ALIGNMENT DP                                            |
    // ======================================================================================
    // Declare the structure for storing the alignment results
    // for the pairs of reads with one end has no hit but another has.
    // The structure will be used for proceeding semi-global DP
    ReadInputForDPArrays readInputForDPall;
    readInputForDPall.inputArrays = ( ReadInputForDP ** ) malloc ( MAX_NUM_CPU_THREADS * sizeof ( ReadInputForDP * ) );
    readInputForDPall.numArrays = ini_params.Ini_NumOfCpuThreads;
    // The following structure will be used for proceeding NEW semi-global DP
    ReadInputForDPArrays readInputForNewDPall;
    readInputForNewDPall.inputArrays = ( ReadInputForDP ** ) malloc ( MAX_NUM_CPU_THREADS * sizeof ( ReadInputForDP * ) );
    readInputForNewDPall.numArrays = ini_params.Ini_NumOfCpuThreads;
    // Declare the structure for storing the read IDs of the first end of the pairs
    // which both ends cannot be aligned (if the input read is paired-end)
    BothUnalignedPairsArrays * bothUnalignedPairsArrays = constructBothUnalignedPairsArrays ( ini_params.Ini_NumOfCpuThreads + 1 );
    // OR for single end cannot be aligned (if the input read is single-end)
    UnalignedSinglesArrays * unalignedSinglesArrays = bothUnalignedPairsArrays; // same array but just different name
    // Declare the structure for storing the alignment results
    // for the pairs of reads with both ends have hits but do not have valid insert size or proper strands.
    ReadInputForDPArrays readAnsForNonValidInsert;
    readAnsForNonValidInsert.inputArrays = ( ReadInputForDP ** ) malloc ( MAX_NUM_CPU_THREADS * sizeof ( ReadInputForDP * ) );
    readAnsForNonValidInsert.numArrays = ini_params.Ini_NumOfCpuThreads;
    // Parameters for DP
    DPParameters dpParameters;
    getParameterForAllDP ( dpParameters, input_options, ini_params );
    // ======================================================================================
    // | INDEX LOADING                                                                      |
    // ======================================================================================
    //Start measuring runtime..
    startTime = setStartTime ();
    lastEventTime = startTime;
    Soap3Index * index = INDEXLoad ( &ini_params, input_options.indexName, ini_params.Ini_shareIndex );
    HSP * hsp = index->sraIndex->hsp;
    HSPAux * hspaux = index->sraIndex->hspaux;
    indexLoadTime = getElapsedTime ( startTime );
    printf ( "[Main] Finished loading index into host.\n" );
    printf ( "[Main] Loading time : %9.4f seconds\n", indexLoadTime );
    printf ( "[Main] Reference sequence length : %u\n\n", index->sraIndex->bwt->textLength );
    lastEventTime = indexLoadTime;
    roundUp = ( maxNumQueries + 31 ) / 32 * 32;
    totalQueryLength = roundUp * wordPerQuery;
    // ==============================================================
    // | QUALITY CONSTANT and DP MATCH SCORE and alignment type     |
    // ==============================================================
    int quality_constant = DEFAULT_QUAL_CONST;

    if ( input_options.isIlluminaQual == 1 )
    { quality_constant = ILLUMINA_QUAL_CONST; }

    hspaux->dpMatchScore = ini_params.Ini_MatchScore;
    hspaux->dpMisMatchScore = ini_params.Ini_MismatchScore;
    hspaux->alignmentType = input_options.alignmentType;
    hspaux->readType = input_options.readType;
    hspaux->peMaxOutputPerRead = ini_params.Ini_PEMaxOutputPerPair;

    if ( input_options.readType == SINGLE_READ )
    { hspaux->ProceedDPForTooManyHits = ini_params.Ini_proceedDPForTooManyHits; }
    else
    { hspaux->ProceedDPForTooManyHits = 0; }

    // For Mapping Quality Score Calculation
    hspaux->x0_array = ( int * ) malloc ( roundUp * sizeof ( int ) );
    hspaux->x1_array = ( int * ) malloc ( roundUp * sizeof ( int ) );
    hspaux->mismatch_array = ( int * ) malloc ( roundUp * sizeof ( int ) );
    hspaux->minMAPQ = ini_params.Ini_minMAPQ;
    hspaux->maxMAPQ = ini_params.Ini_maxMAPQ;
    hspaux->bwaLikeScore = ini_params.Ini_bwaLikeScore;
    hspaux->maxLenReadName = ini_params.Ini_maxReadNameLen;

    if ( hspaux->bwaLikeScore )
    { bwase_initialize ( hspaux->g_log_n ); }

    // For storing the results of the unpaired reads
    hspaux->soap3AnsArray = ( ReadInputForDP ** ) malloc ( roundUp * sizeof ( ReadInputForDP * ) );
    hspaux->sa_start = ( unsigned int * ) malloc ( roundUp * sizeof ( unsigned int ) );
    hspaux->occ_start = ( unsigned int * ) malloc ( roundUp * sizeof ( unsigned int ) );
    hspaux->sa_num = ( unsigned int * ) malloc ( roundUp * sizeof ( unsigned int ) );
    hspaux->occ_num = ( unsigned int * ) malloc ( roundUp * sizeof ( unsigned int ) );

    // print MD string and NM tag?
    hspaux->isPrintMDNM = input_options.isPrintMDNM;

    // For the SAM output information
    hspaux->readGroup = input_options.readGroup;

    if ( strlen ( hspaux->readGroup ) == 0 )
    { hspaux->readGroup = input_options.queryFileName; }

    hspaux->sampleName = input_options.sampleName;

    if ( strlen ( hspaux->sampleName ) == 0 )
    { hspaux->sampleName = DEFAULT_SAMPLE_NAME; }

    hspaux->readGrpOption = input_options.readGrpOption;
    // ==================================================================
    // | For construction of arrays to store the unaligned read IDs
    // | for proceeding single deep-dp on them
    // ==================================================================
#ifdef PERFORM_DEEP_DP_FOR_UNALIGN_READS
    hspaux->readsIDForSingleDP = ( BothUnalignedPairsArrays * ) constructBothUnalignedPairsArrays ( 1 );
    hspaux->allHits = ( AllHits * ) constructAllHits (); // for storing the corresponding algnments
#endif
    // ======================================================================================
    // | ALLOCATE MEMORY FOR THE ARRAYS                                                     |
    // ======================================================================================
    queries0 = ( uint * ) malloc ( totalQueryLength * sizeof ( uint ) ); // a large array to store all queries
    readLengths0 = ( uint * ) malloc ( roundUp * sizeof ( uint ) );
    readIDs0 = ( uint * ) malloc ( roundUp * sizeof ( uint ) );
    upkdQualities0 = ( char * ) malloc ( roundUp * maxReadLength * sizeof ( char ) );
    memset ( upkdQualities0, 0, roundUp * maxReadLength );
    upkdQueryNames0 = ( char * ) malloc ( roundUp * ini_params.Ini_maxReadNameLen * sizeof ( char ) );
    queries1 = ( uint * ) malloc ( totalQueryLength * sizeof ( uint ) ); // a large array to store all queries
    readLengths1 = ( uint * ) malloc ( roundUp * sizeof ( uint ) );
    readIDs1 = ( uint * ) malloc ( roundUp * sizeof ( uint ) );
    upkdQualities1 = ( char * ) malloc ( roundUp * maxReadLength * sizeof ( char ) );
    memset ( upkdQualities1, 0, roundUp * maxReadLength );
    upkdQueryNames1 = ( char * ) malloc ( roundUp * ini_params.Ini_maxReadNameLen * sizeof ( char ) );
    /*
    queries = (uint*) malloc(totalQueryLength * sizeof(uint)); // a large array to store all queries
    readLengths = (uint*) malloc(roundUp * sizeof(uint));
    readIDs = (uint*) malloc(roundUp * sizeof(uint));
    upkdQualities = (char*) malloc(roundUp*maxReadLength * sizeof(char));
    memset(upkdQualities,0,roundUp*maxReadLength);
    upkdQueryNames = (char*) malloc(roundUp*ini_params.Ini_maxReadNameLen * sizeof(char));
    */
    unAlignedPair = ( unsigned int * ) malloc ( roundUp * sizeof ( unsigned int ) );

    for ( int threadId = 0; threadId < ini_params.Ini_NumOfCpuThreads; ++threadId )
    {
        readInputForDPall.inputArrays[threadId] = constructReadInputForDP ( ini_params.Ini_NumOfCpuThreads );
        readInputForNewDPall.inputArrays[threadId] = constructReadInputForDP ( ini_params.Ini_NumOfCpuThreads );
        readAnsForNonValidInsert.inputArrays[threadId] = constructReadInputForDP ( ini_params.Ini_NumOfCpuThreads );
    }

    uint maxBatchSize = NUM_BLOCKS * THREADS_PER_BLOCK * QUERIES_PER_THREAD; // queries processed in one kernel call
    // maxBatchSize has to be divible by 2
    maxBatchSize = maxBatchSize / 2 * 2;
    // #ifdef BGS_GPU_CASE_BREAKDOWN_TIME
    // totalDeviceTime=0;
    // #endif
    // initialize the output files
    bam_header_t samOutputHeader;
    samfile_t * samOutputFilePtr[MAX_NUM_CPU_THREADS];
    samfile_t * samOutputDPFilePtr;
    samfile_t * samOutputUnpairFilePtr;

    switch ( input_options.outputFormat )
    {
        case SRA_OUTPUT_FORMAT_SAM_API:
            SAMOutputHeaderConstruct ( &samOutputHeader, hsp, hspaux, maxReadLength );

            for ( int i = 0; i < ini_params.Ini_NumOfCpuThreads; i++ )
            {
                if ( input_options.isOutputBinary == 1 )
                { samOutputFilePtr[i] = samopen ( outputFileName[i], "wb", &samOutputHeader ); }
                else
                { samOutputFilePtr[i] = samopen ( outputFileName[i], "wh", &samOutputHeader ); }

                if ( samOutputFilePtr[i] == NULL )
                {
                    fprintf ( stderr, "Could not open the output file %s\n", outputFileName[i] );
                    exit ( 1 );
                }
            }

            break;

        default:
            for ( int i = 0; i < ini_params.Ini_NumOfCpuThreads; i++ )
            {
                outputFile = ( FILE * ) fopen ( outputFileName[i], "w" );

                if ( outputFile == NULL ) { fprintf ( stderr, "Could not open the output file %s\n", outputFileName[i] ); exit ( 1 );}

                OCCWriteOutputHeader ( hsp, outputFile, maxReadLength, 1, input_options.outputFormat ); // will modify the number of reads later
                fclose ( outputFile );
            }

            break;
    }

    // For DP
    switch ( input_options.outputFormat )
    {
        case SRA_OUTPUT_FORMAT_SAM_API:
            if ( input_options.isOutputBinary == 1 )
            { samOutputDPFilePtr = samopen ( outputDPFileName, "wb", &samOutputHeader ); }
            else
            { samOutputDPFilePtr = samopen ( outputDPFileName, "wh", &samOutputHeader ); }

            if ( samOutputDPFilePtr == NULL )
            {
                fprintf ( stderr, "Could not open the output file %s\n", outputDPFileName );
                exit ( 1 );
            }

            break;

        default:
            outputDPFile = ( FILE * ) fopen ( outputDPFileName, "w" );

            if ( outputDPFile == NULL ) { fprintf ( stderr, "Could not open the output file %s\n", outputDPFileName ); exit ( 1 );}

            OCCWriteOutputHeader ( hsp, outputDPFile, maxReadLength, 1, input_options.outputFormat ); // will modify the number of reads later
            fclose ( outputDPFile );
            break;
    }

    // For Unpaired reads
#ifdef PERFORM_DEEP_DP_FOR_UNALIGN_READS

    if ( input_options.readType == PAIR_END_READ )
    {
        switch ( input_options.outputFormat )
        {
            case SRA_OUTPUT_FORMAT_SAM_API:
                if ( input_options.isOutputBinary == 1 )
                { samOutputUnpairFilePtr = samopen ( outputUnpairFileName, "wb", &samOutputHeader ); }
                else
                { samOutputUnpairFilePtr = samopen ( outputUnpairFileName, "wh", &samOutputHeader ); }

                if ( samOutputUnpairFilePtr == NULL )
                {
                    fprintf ( stderr, "Could not open the output file %s\n", outputUnpairFileName );
                    exit ( 1 );
                }

                break;
        }
    }

#endif
    // ======================================================================================
    // | LOADING INPUT SHORT READ FILE                                                      |
    // ======================================================================================
    size_t bufferSize;
    uint bufferIndex;
    char queryChar;
    size_t bufferSize2;
    uint bufferIndex2;
    char queryChar2;

    if ( input_options.isReadBAM )
    {
        // the query file is in BAM format
        bamQueryFile = bam_open ( queryFileName, "r" );
        bamHeader = bam_header_init ();
        bamHeader = bam_header_read ( bamQueryFile );
        bam = bam_init1 ();
    }
    else
    {
        //gzQueryFile = ( gzFile * ) gzopen ( queryFileName, "r" );
        gzQueryFile = gzopen ( queryFileName, "r" );

        if ( gzQueryFile == NULL ) { fprintf ( stderr, "Cannot open queryFile\n" ); exit ( 1 );}

        bufferSize = gzread ( gzQueryFile, queryFileBuffer, INPUT_BUFFER_SIZE );

        if ( bufferSize < INPUT_BUFFER_SIZE && ( !gzeof ( gzQueryFile ) ) )
        {
            const char * error_string;
            int err;
            error_string = gzerror ( gzQueryFile, & err );

            if ( err )
            {
                fprintf ( stderr, "Error in reading the read file: %s.\n", error_string );
                exit ( EXIT_FAILURE );
            }
        }

        bufferIndex = 0;
        queryChar = queryFileBuffer[bufferIndex++];

        if ( input_options.readType == PAIR_END_READ )
        {
            // pair-ended reads
            //gzQueryFile2 = ( gzFile * ) gzopen ( queryFileName2, "r" );
            gzQueryFile2 = gzopen ( queryFileName2, "r" );

            if ( gzQueryFile2 == NULL ) { fprintf ( stderr, "Cannot open queryFile2\n" ); exit ( 1 );}

            bufferSize2 = gzread ( gzQueryFile2, queryFileBuffer2, INPUT_BUFFER_SIZE );

            if ( bufferSize2 < INPUT_BUFFER_SIZE && ( !gzeof ( gzQueryFile2 ) ) )
            {
                const char * error_string;
                int err;
                error_string = gzerror ( gzQueryFile2, & err );

                if ( err )
                {
                    fprintf ( stderr, "Error in reading the read file: %s.\n", error_string );
                    exit ( EXIT_FAILURE );
                }
            }

            bufferIndex2 = 0;
            queryChar2 = queryFileBuffer2[bufferIndex2++];
        }
    }

    uint accumReadNum = 0;
    uint numQueries;
    // create buffers
    InputReadsBuffer * buffer0 = InputReadsBufferFullCreate ( maxReadLength,
                                 maxNumQueries, wordPerQuery, quality_constant, queries0,
                                 readLengths0, readIDs0, upkdQualities0, upkdQueryNames0,
                                 isFastq, ini_params.Ini_maxReadNameLen );
    InputReadsBuffer * buffer1 = InputReadsBufferFullCreate ( maxReadLength,
                                 maxNumQueries, wordPerQuery, quality_constant, queries1,
                                 readLengths1, readIDs1, upkdQualities1, upkdQueryNames1,
                                 isFastq, ini_params.Ini_maxReadNameLen );
    AIOInputBuffer * aiob = AIOInputBufferCreate ( buffer0, buffer1 );
    InputFilePointers * ifp = InputFilePointersCreate ();

    if ( input_options.isReadBAM )
    {
        InputFilePointersSetBam ( ifp, bamQueryFile, bamHeader, bam );
    }
    else if ( input_options.readType == SINGLE_READ )
    {
        InputFilePointersSetSingle ( ifp, gzQueryFile );
    }
    else
    {
        InputFilePointersSetPair ( ifp, gzQueryFile, gzQueryFile2 );
    }

    aiob->reads = ifp;
    InputReadsBuffer * readyReadsBuffer;
    //create io thread
    AIOInputThreadCreate ( aiob, bufferSize, bufferIndex, index->charMap, queryChar,
                           queryFileBuffer, bufferSize2, bufferIndex2, queryChar2,
                           queryFileBuffer2 );
    //load reads
    readyReadsBuffer = LoadReadsFromAIOBuffer ( aiob );
    queries = readyReadsBuffer->queries;
    readLengths = readyReadsBuffer->readLengths;
    readIDs = readyReadsBuffer->readIDs;
    upkdQualities = readyReadsBuffer->upkdQualities;
    upkdQueryNames = readyReadsBuffer->upkdQueryNames;
    isFastq = readyReadsBuffer->isFastq;
    hspaux->isFastq = isFastq;
    numQueries = readyReadsBuffer->filledNum;

    while ( numQueries > 0 )
    {
        printf ( "[Main] Loaded %u short reads from the query file.\n", numQueries );
        readLoadTime = getElapsedTime ( startTime );
        printf ( "[Main] Elapsed time on host : %9.4f seconds\n\n", readLoadTime - lastEventTime );
        totalReadLoadTime += readLoadTime - lastEventTime;
        lastEventTime = readLoadTime;

        numOfAnswer = 0;
        numOfAlignedRead = 0;
        numOfUnAlignedPairs = 0;
        uint origNumQueries = numQueries;

        if ( detected_read_length == 0 )
        {
            // printParameters(input_options, ini_params);

            // ==================================================================
            // | DETECT THE READ LENGTH                                         |
            // ==================================================================
            if ( input_options.readType == PAIR_END_READ )
            {
                detected_read_length = GetReadLength ( readLengths, numQueries, 2 );
                detected_read_length2 = GetReadLength ( readLengths + 1, numQueries, 2 );

                // the minimum insert size cannot be smaller than detected_read_length2
                if ( input_options.insert_low < detected_read_length2 )
                { input_options.insert_low = detected_read_length2; }
            }
            else
            {
                detected_read_length = GetReadLength ( readLengths, numQueries, 1 );
            }

            // ==================================================================
            // | FOR DP IS ENABLED                                              |
            // | IF READ LENGTH < MIN_READ_LEN_FOR_DP (i.e. 30)                 |
            // |    THEN DP IS DISABLE.,                                        |
            // | IF READ LENGTH > 150, SKIP SOAP3 MODULE.                       |
            // | IF READ LENGTH <= 50,  ONLY ALLOW 1 MISMATCH IN SOAP3          |
            // ==================================================================
            if ( input_options.readType == SINGLE_READ && detected_read_length < MIN_READ_LEN_FOR_DP && input_options.enableDP == 1 )
            {
                input_options.enableDP = 0;
                printf ( "Dynamic programming is disabled because read length < %i\n", MIN_READ_LEN_FOR_DP );
            }
            else if ( input_options.readType == PAIR_END_READ && ( detected_read_length < MIN_READ_LEN_FOR_DP || detected_read_length2 < MIN_READ_LEN_FOR_DP ) && input_options.enableDP == 1 )
            {
                input_options.enableDP = 0;
                printf ( "Dynamic programming is disabled because read length < %i\n", MIN_READ_LEN_FOR_DP );
            }
            else if ( input_options.readType == PAIR_END_READ && ( detected_read_length > 150 || detected_read_length2 > 150 ) && input_options.enableDP == 1 )
            {
                ini_params.Ini_skipSOAP3Alignment = 1;
                printf ( "All reads are directly processed by DP, because the read length > 150\n" );
            }
            else if ( input_options.readType == PAIR_END_READ && ( detected_read_length <= 50 || detected_read_length2 <= 50 ) && input_options.enableDP == 1 )
            {
                input_options.numMismatch = 1;
            }

            // ==================================================================
            // | IF DP IS DISABLE AND USER DOES NOT SPECIFY # OF MISMATCHES     |
            // | THEN SET THE DEFAULT # OF MISMATCHES AS:                       |
            // |   - IF READ LENGTH < 50, DEFAULT_NUM_MISMATCH_FOR_SHORT_READ   |
            // |   - IF READ LENGTH >= 50, DEFAULT_NUM_MISMATCH_FOR_NORMAL_READ |
            // ==================================================================
            if ( input_options.enableDP == 0 && input_options.numMismatch == -1 )
            {
                // user does not specify # of mismatches
                input_options.numMismatch = getDefaultMismatchNum ( detected_read_length );
                printf ( "Maximum number of mismatches allowed: %i\n",  input_options.numMismatch );
            }

            // get the max hit # for default DP
            if ( input_options.enableDP == 1 && input_options.readType == PAIR_END_READ )
            {
                input_options.maxHitNum = getMaxHitNumForDefaultDP ( detected_read_length );
                input_options.maxHitNum2 = getMaxHitNumForDefaultDP ( detected_read_length2 );
            }
        }

        // Reset the array for mapping quality score calculation
        memset ( hspaux->x0_array, 0, roundUp * sizeof ( int ) );
        memset ( hspaux->x1_array, 0, roundUp * sizeof ( int ) );
        memset ( hspaux->mismatch_array, 0, roundUp * sizeof ( int ) );
        memset ( hspaux->sa_start, 0, roundUp * sizeof ( unsigned int ) );
        memset ( hspaux->occ_start, 0, roundUp * sizeof ( unsigned int ) );
        memset ( hspaux->sa_num, 0, roundUp * sizeof ( unsigned int ) );
        memset ( hspaux->occ_num, 0, roundUp * sizeof ( unsigned int ) );
        memset ( ( ( ReadInputForDP ** ) hspaux->soap3AnsArray ), 0, roundUp * sizeof ( ReadInputForDP * ) );
        // Reset the array for storing the single alignment for those unaligned paired-end reads
#ifdef PERFORM_DEEP_DP_FOR_UNALIGN_READS
        resetAllHits ( ( AllHits * ) hspaux->allHits );
#endif

        // =====================================================================
        // | Reset the arrays for storing alignment results for semi-global DP |
        // =====================================================================

        if ( input_options.enableDP == 1 )
        {
            if ( input_options.readType == PAIR_END_READ )
                for ( int threadId = 0; threadId < ini_params.Ini_NumOfCpuThreads; ++threadId )
                {
                    resetReadInputForDP ( readInputForDPall.inputArrays[threadId] );
                    resetReadInputForDP ( readInputForNewDPall.inputArrays[threadId] );
                    resetReadInputForDP ( readAnsForNonValidInsert.inputArrays[threadId] );
                }

            resetBothUnalignedPairsArrays ( bothUnalignedPairsArrays );
        }

        // =======================================
        // | BATCH PROCESS READ (GPU/GPU/CPU)    |
        // =======================================
        char ** currOutputFileName = outputFileName;
        samfile_t ** currSamOutputFilePtr = samOutputFilePtr;

        // ========================================
        // | IF THE INDEX IS NOT IN DEVICE, THEN  |
        // | COPY INDEX TO DEVICE MEMORY          |
        // ========================================

        if ( indexLoadedToGPU == 0 )
        {
            GPUINDEXUpload ( index, &_bwt, &_occ,
                             &_revBwt, &_revOcc );
            double copyTime = getElapsedTime ( startTime );
            printf ( "[Main] Finished copying index into device (GPU).\n" );
            printf ( "[Main] Loading time : %9.4f seconds\n\n", copyTime - lastEventTime );
            lastEventTime = copyTime;
            indexLoadedToGPU = 1;
        }

        //*******************************//
        // Perform Alignment             //
        //*******************************//

        if ( input_options.readType == PAIR_END_READ )
        {
            soap3_dp_pair_align ( queries, readLengths, input_options.numMismatch, wordPerQuery,
                                  maxBatchSize, numQueries, accumReadNum,
                                  index,
                                  _bwt, _revBwt,
                                  _occ, _revOcc,
                                  ini_params, input_options,
                                  maxReadLength, detected_read_length, detected_read_length2,
                                  upkdQualities,
                                  unAlignedPair, numOfUnAlignedPairs,
                                  readIDs, upkdQueryNames,
                                  currOutputFileName, currSamOutputFilePtr,
                                  outputDPFileName, samOutputDPFilePtr,
                                  samOutputUnpairFilePtr,
                                  numOfAnswer, numOfAlignedRead,
                                  &readInputForDPall, &readInputForNewDPall,
                                  &readAnsForNonValidInsert,
                                  bothUnalignedPairsArrays,
                                  startTime, lastEventTime, totalAlignmentTime,
                                  indexLoadedToGPU );
        }
        else
        {
            soap3_dp_single_align ( queries, readLengths, input_options.numMismatch, wordPerQuery,
                                    maxBatchSize, numQueries, accumReadNum,
                                    index, _bwt, _revBwt,
                                    _occ, _revOcc,
                                    ini_params, input_options,
                                    maxReadLength, detected_read_length,
                                    upkdQualities,
                                    unAlignedPair, numOfUnAlignedPairs,
                                    readIDs, upkdQueryNames,
                                    currOutputFileName, currSamOutputFilePtr,
                                    outputDPFileName, samOutputDPFilePtr,
                                    numOfAnswer, numOfAlignedRead,
                                    &readInputForDPall,
                                    unalignedSinglesArrays,
                                    startTime, lastEventTime, totalAlignmentTime,
                                    indexLoadedToGPU );
        }

        // ========================================
        // | GET NEXT BATCH OF READ               |
        // ========================================
        totalReadsAlignedForInputReads += numOfAlignedRead;
        totalAnsForInputReads += numOfAnswer;
        accumReadNum += origNumQueries;
        ResetBufferStatusToUnfilled ( aiob );
        readyReadsBuffer = LoadReadsFromAIOBuffer ( aiob );
        queries = readyReadsBuffer->queries;
        readLengths = readyReadsBuffer->readLengths;
        readIDs = readyReadsBuffer->readIDs;
        upkdQualities = readyReadsBuffer->upkdQualities;
        upkdQueryNames = readyReadsBuffer->upkdQueryNames;
        isFastq = readyReadsBuffer->isFastq;
        numQueries = readyReadsBuffer->filledNum;

        // If the current opened file still have queries returned
        // skip the code to process the result / open next file; and continue
        if ( numQueries > 0 )
        {
            // Skip
            continue;
        }

        // show the summary of the result and then load another pair of read files
        // ======================================================================================
        // | SHOW THE SUMMARY                                                                   |
        // ======================================================================================
        if ( input_options.readType == PAIR_END_READ )
        {
            // printf("[Main] Overall number of pairs of reads aligned: %llu (number of alignments: %llu)\n", totalReadsAlignedForInputReads/2, totalAnsForInputReads);
            printf ( "[Main] Overall number of pairs of reads aligned: %llu\n", totalReadsAlignedForInputReads / 2 );
        }
        else
        {
            // printf("[Main] Overall number of reads aligned: %llu (number of alignments: %llu)\n", totalReadsAlignedForInputReads, totalAnsForInputReads);
            // printf("[Main] Overall number of unaligned reads: %llu\n", accumReadNum-totalReadsAlignedForInputReads);
            printf ( "[Main] Overall number of reads aligned: %llu\n", totalReadsAlignedForInputReads );
            printf ( "[Main] Overall number of unaligned reads: %llu\n", accumReadNum - totalReadsAlignedForInputReads );
        }

        printf ( "[Main] Overall read load time : %9.4f seconds\n", totalReadLoadTime );
        printf ( "[Main] Overall alignment time (excl. read loading) : %9.4f seconds\n", totalAlignmentTime + totalTrimAlignmentTime );
        // ======================================================================================
        // | SHOW THE COMMAND FOR MERGING THE OUTPUT FILES INTO ONE                             |
        // ======================================================================================

        // update the output files
        switch ( input_options.outputFormat )
        {
            case SRA_OUTPUT_FORMAT_DEFAULT:
                // update the header of the output files
                outputDPFile = ( FILE * ) fopen ( outputDPFileName, "r+" );

                if ( outputDPFile == NULL ) { fprintf ( stderr, "Cannot open outputFile %s\n", outputDPFileName ); exit ( 1 );}

                fseek ( outputDPFile , 0 , SEEK_SET );
                OCCWriteOutputHeader ( hsp, outputDPFile, maxReadLength, accumReadNum, input_options.outputFormat ); // update the number of reads
                fclose ( outputDPFile );
                break;

            case SRA_OUTPUT_FORMAT_SAM_API:
                samclose ( samOutputDPFilePtr );
                break;
        }

        switch ( input_options.outputFormat )
        {
            case SRA_OUTPUT_FORMAT_DEFAULT:

                // update the header of the output files
                for ( int i = 0; i < ini_params.Ini_NumOfCpuThreads; i++ )
                {
                    outputFile = ( FILE * ) fopen ( outputFileName[i], "r+" );

                    if ( outputFile == NULL ) { fprintf ( stderr, "Cannot open outputFile %s\n", outputFileName[i] ); exit ( 1 );}

                    fseek ( outputFile , 0 , SEEK_SET );
                    OCCWriteOutputHeader ( hsp, outputFile, maxReadLength, accumReadNum, input_options.outputFormat ); // update the number of reads
                    fclose ( outputFile );
                }

                break;

            case SRA_OUTPUT_FORMAT_SAM_API:
                for ( int i = 0; i < ini_params.Ini_NumOfCpuThreads; i++ )
                {
                    samclose ( samOutputFilePtr[i] );
                }

                break;
        }

        // For Unpaired reads
        if ( input_options.readType == PAIR_END_READ )
        {
            switch ( input_options.outputFormat )
            {
                case SRA_OUTPUT_FORMAT_SAM_API:
                    samclose ( samOutputUnpairFilePtr );
                    break;
            }
        }

        // create the done file
        outputDoneFile = ( FILE * ) fopen ( outputDoneFileName, "w" );

        if ( outputDoneFile == NULL )
        {
            fprintf ( stderr, "Could not create the output file %s\n",
                      outputDoneFileName );
        }

        fclose ( outputDoneFile );
        free ( outputDoneFileName );

        // release memory and close files
        for ( i = 0; i < MAX_NUM_CPU_THREADS; i++ )
        { free ( outputFileName[i] ); }

        free ( outputDPFileName );
        free ( outputUnpairFileName );

        if ( input_options.isReadBAM )
        {
            bam_close ( bamQueryFile );
            bam_destroy1 ( bam );
        }
        else
        {
            gzclose ( gzQueryFile );

            if ( input_options.readType == PAIR_END_READ )
            {
                gzclose ( gzQueryFile2 );
            }
        }

        // ======================================================================================
        // | Load the next set of files                                                         |
        // ======================================================================================

        if ( input_options.isReadList == 1 && currExp < expNum )
        {
            printf ( "\n[Main] Load the next set of files\n" );
            updateInputOption ( ( &input_options ), multiInput, currExp++ );
            queryFileName = input_options.queryFileName;

            if ( input_options.readType == PAIR_END_READ && input_options.isReadBAM == 0 )
            {
                queryFileName2 = input_options.queryFileName2;
            }

            // Update the HSP information
            hspaux->readGroup = input_options.readGroup;

            if ( strlen ( hspaux->readGroup ) == 0 )
            { hspaux->readGroup = input_options.queryFileName; }

            hspaux->sampleName = input_options.sampleName;

            if ( strlen ( hspaux->sampleName ) == 0 )
            { hspaux->sampleName = DEFAULT_SAMPLE_NAME; }

            hspaux->readGrpOption = input_options.readGrpOption;

            // update the output files
            for ( i = 0; i < MAX_NUM_CPU_THREADS; i++ )
            { outputFileName[i] = ( char * ) malloc ( strlen ( input_options.outputPrefix ) + 10 ); }

            outputDPFileName = ( char * ) malloc ( strlen ( input_options.outputPrefix ) + 9 );
            outputUnpairFileName = ( char * ) malloc ( strlen ( input_options.outputPrefix ) + 8 );

            for ( i = 0; i < MAX_NUM_CPU_THREADS; i++ )
            {
                sprintf ( outputFileName[i], "%s.gout.%d", input_options.outputPrefix, i + 1 );
            }

            sprintf ( outputDPFileName, "%s.dpout.1", input_options.outputPrefix );
            sprintf ( outputUnpairFileName, "%s.unpair", input_options.outputPrefix );
            outputDoneFileName = ( char * ) malloc ( strlen ( input_options.outputPrefix ) + 6 );
            sprintf ( outputDoneFileName, "%s.done", input_options.outputPrefix );

            // initialize the output files
            switch ( input_options.outputFormat )
            {
                case SRA_OUTPUT_FORMAT_SAM_API:
                    SAMOutputHeaderConstruct ( &samOutputHeader, hsp, hspaux, maxReadLength );

                    for ( int i = 0; i < ini_params.Ini_NumOfCpuThreads; i++ )
                    {
                        if ( input_options.isOutputBinary == 1 )
                        { samOutputFilePtr[i] = samopen ( outputFileName[i], "wb", &samOutputHeader ); }
                        else
                        { samOutputFilePtr[i] = samopen ( outputFileName[i], "wh", &samOutputHeader ); }

                        if ( samOutputFilePtr[i] == NULL )
                        {
                            fprintf ( stderr, "Could not open the output file %s\n", outputFileName[i] );
                            exit ( 1 );
                        }
                    }

                    break;

                default:
                    for ( int i = 0; i < ini_params.Ini_NumOfCpuThreads; i++ )
                    {
                        outputFile = ( FILE * ) fopen ( outputFileName[i], "w" );

                        if ( outputFile == NULL ) { fprintf ( stderr, "Cannot open outputFile %s\n", outputFileName[i] ); exit ( 1 );}

                        OCCWriteOutputHeader ( hsp, outputFile, maxReadLength, 1, input_options.outputFormat ); // will modify the number of reads later
                        fclose ( outputFile );
                    }

                    break;
            }

            // For DP
            switch ( input_options.outputFormat )
            {
                case SRA_OUTPUT_FORMAT_SAM_API:
                    if ( input_options.isOutputBinary == 1 )
                    { samOutputDPFilePtr = samopen ( outputDPFileName, "wb", &samOutputHeader ); }
                    else
                    { samOutputDPFilePtr = samopen ( outputDPFileName, "wh", &samOutputHeader ); }

                    if ( samOutputDPFilePtr == NULL )
                    {
                        fprintf ( stderr, "Could not open the output file %s\n", outputDPFileName );
                        exit ( 1 );
                    }

                    break;

                default:
                    outputDPFile = ( FILE * ) fopen ( outputDPFileName, "w" );

                    if ( outputDPFile == NULL ) { fprintf ( stderr, "Cannot open outputFile %s\n", outputDPFileName ); exit ( 1 );}

                    OCCWriteOutputHeader ( hsp, outputDPFile, maxReadLength, 1, input_options.outputFormat ); // will modify the number of reads later
                    fclose ( outputDPFile );
                    break;
            }

            // For Unpaired reads
#ifdef PERFORM_DEEP_DP_FOR_UNALIGN_READS

            if ( input_options.readType == PAIR_END_READ )
            {
                switch ( input_options.outputFormat )
                {
                    case SRA_OUTPUT_FORMAT_SAM_API:
                        if ( input_options.isOutputBinary == 1 )
                        { samOutputUnpairFilePtr = samopen ( outputUnpairFileName, "wb", &samOutputHeader ); }
                        else
                        { samOutputUnpairFilePtr = samopen ( outputUnpairFileName, "wh", &samOutputHeader ); }

                        if ( samOutputUnpairFilePtr == NULL )
                        {
                            fprintf ( stderr, "Could not open the output file %s\n", outputUnpairFileName );
                            exit ( 1 );
                        }

                        break;
                }
            }

#endif
            // reset the variables
            accumReadNum = 0;
            totalReadsAlignedForInputReads = 0;
            totalAnsForInputReads = 0;
            detected_read_length = 0; // the max read length for the first ten reads
            totalAlignmentTime = 0;
            totalReadLoadTime = 0;

            // load reads
            if ( input_options.readType == SINGLE_READ )
            { printf ( "\n\n[Main] Loading read file %s\n", queryFileName ); }
            else if ( input_options.isReadBAM == 0 )
            { printf ( "\n\n[Main] Loading read files %s and %s\n", queryFileName, queryFileName2 ); }

            if ( input_options.isReadBAM )
            {
                // the query file is in BAM format
                bamQueryFile = bam_open ( queryFileName, "r" );
                bamHeader = bam_header_init ();
                bamHeader = bam_header_read ( bamQueryFile );
                bam = bam_init1 ();
            }
            else
            {
                //gzQueryFile = ( gzFile * ) gzopen ( queryFileName, "r" );
                gzQueryFile = gzopen ( queryFileName, "r" );

                if ( gzQueryFile == NULL ) { fprintf ( stderr, "Cannot open queryFile\n" ); exit ( 1 );}

                bufferSize = gzread ( gzQueryFile, queryFileBuffer, INPUT_BUFFER_SIZE );
                bufferIndex = 0;
                queryChar = queryFileBuffer[bufferIndex++];

                if ( input_options.readType == PAIR_END_READ )
                {
                    // pair-ended reads
                    //gzQueryFile2 = ( gzFile * ) gzopen ( queryFileName2, "r" );
                    gzQueryFile2 = gzopen ( queryFileName2, "r" );

                    if ( gzQueryFile2 == NULL ) { fprintf ( stderr, "Cannot open queryFile2\n" ); exit ( 1 );}

                    bufferSize2 = gzread ( gzQueryFile2, queryFileBuffer2, INPUT_BUFFER_SIZE );
                    bufferIndex2 = 0;
                    queryChar2 = queryFileBuffer2[bufferIndex2++];
                }
            }

            if ( input_options.isReadBAM )
            {
                InputFilePointersSetBam ( ifp, bamQueryFile, bamHeader, bam );
            }
            else if ( input_options.readType == SINGLE_READ )
            {
                InputFilePointersSetSingle ( ifp, gzQueryFile );
            }
            else
            {
                InputFilePointersSetPair ( ifp, gzQueryFile, gzQueryFile2 );
            }

            aiob->reads = ifp;
            //clear buffer' status
            AIOInputBufferClear ( aiob );
            //create io thread
            AIOInputThreadCreate ( aiob, bufferSize, bufferIndex, index->charMap, queryChar,
                                   queryFileBuffer, bufferSize2, bufferIndex2, queryChar2,
                                   queryFileBuffer2 );
            //load reads
            readyReadsBuffer = LoadReadsFromAIOBuffer ( aiob );
            queries = readyReadsBuffer->queries;
            readLengths = readyReadsBuffer->readLengths;
            readIDs = readyReadsBuffer->readIDs;
            upkdQualities = readyReadsBuffer->upkdQualities;
            upkdQueryNames = readyReadsBuffer->upkdQueryNames;
            isFastq = readyReadsBuffer->isFastq;
            numQueries = readyReadsBuffer->filledNum;

            // Update the isFastq variable inside HSP
            hspaux->isFastq = isFastq;
        }
    }

    hipDeviceReset ();
    // ======================================================================================
    // | CLEAN UP                                                                           |
    // ======================================================================================
#ifdef PERFORM_DEEP_DP_FOR_UNALIGN_READS
    freeBothUnalignedPairsArrays ( ( BothUnalignedPairsArrays * ) hspaux->readsIDForSingleDP );
    releaseAllHits ( ( AllHits * ) hspaux->allHits );
#endif
    free ( hspaux->x0_array );
    free ( hspaux->x1_array );
    free ( hspaux->mismatch_array );
    free ( hspaux->sa_start );
    free ( hspaux->occ_start );
    free ( hspaux->sa_num );
    free ( hspaux->occ_num );
    ReadInputForDP ** array = ( ReadInputForDP ** ) hspaux->soap3AnsArray;
    free ( array );

    // free device memory
    if ( indexLoadedToGPU == 1 )
    {
        printf ( "[Main] Free device memory..\n" );
        GPUINDEXFree ( _bwt, _occ, _revBwt, _revOcc );
    }

    printf ( "[Main] Free index from host memory..\n" );
    INDEXFree ( index, ini_params.Ini_shareIndex );
    printf ( "[Main] Free host memory..\n" );
    free ( queries0 );
    free ( readLengths0 );
    free ( readIDs0 );
    free ( upkdQualities0 );
    free ( upkdQueryNames0 );
    free ( queries1 );
    free ( readLengths1 );
    free ( readIDs1 );
    free ( upkdQualities1 );
    free ( upkdQueryNames1 );
    AIOInputBufferFree ( aiob );
    // free(queries);
    // free(readLengths);
    // free(readIDs);
    free ( unAlignedPair );
    // free(isBad);
    // free(upkdQueries);
    // free(upkdQualities);
    // free(upkdQueryNames);

    for ( int threadId = 0; threadId < ini_params.Ini_NumOfCpuThreads; ++threadId )
    {
        freeReadInputForDP ( readInputForDPall.inputArrays[threadId] );
        freeReadInputForDP ( readInputForNewDPall.inputArrays[threadId] );
        freeReadInputForDP ( readAnsForNonValidInsert.inputArrays[threadId] );
    }

    free ( readInputForDPall.inputArrays ); // for half-aligned pairs
    free ( readInputForNewDPall.inputArrays ); // for half-aligned pairs
    free ( readAnsForNonValidInsert.inputArrays );
    freeBothUnalignedPairsArrays ( bothUnalignedPairsArrays ); // for both-unaligned pairs OR single-unaligned reads

    if ( multiInput != NULL )
    {
        free ( multiInput );
    }

    if ( input_options.outputFormat == SRA_OUTPUT_FORMAT_SAM_API )
    {
        SAMOutputHeaderDestruct ( &samOutputHeader );
    }

    return 0;
}
