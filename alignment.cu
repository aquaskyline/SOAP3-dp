#include "hip/hip_runtime.h"
/*
 *
 *    alignment.cu
 *    Soap3(gpu)
 *
 *    Copyright (C) 2011, HKU
 *
 *    This program is free software; you can redistribute it and/or
 *    modify it under the terms of the GNU General Public License
 *    as published by the Free Software Foundation; either version 2
 *    of the License, or (at your option) any later version.
 *
 *    This program is distributed in the hope that it will be useful,
 *    but WITHOUT ANY WARRANTY; without even the implied warranty of
 *    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *    GNU General Public License for more details.
 *
 *    You should have received a copy of the GNU General Public License
 *    along with this program; if not, write to the Free Software
 *    Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */

#include "alignment.h"

// COPY INDEX TO DEVICE MEMORY
void GPUINDEXUpload ( Soap3Index * index, uint ** _bwt, uint ** _occ,
                      uint ** _revBwt, uint ** _revOcc )
{
    BWT * bwt = index->sraIndex->bwt;
    BWT * revBwt = index->sraIndex->rev_bwt;
    unsigned int * revOccValue = index->gpu_revOccValue;
    unsigned int * occValue = index->gpu_occValue;
    unsigned int numOfOccValue = index->gpu_numOfOccValue;
    hipError_t gpuErr;
    gpuErr = hipMalloc ( ( void ** ) _bwt, bwt->bwtSizeInWord * sizeof ( uint ) );

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MALLOC FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }

    gpuErr = hipMalloc ( ( void ** ) _occ, numOfOccValue * ALPHABET_SIZE * sizeof ( uint ) );

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MALLOC FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }

    gpuErr = hipMemcpy ( ( *_bwt ), bwt->bwtCode, bwt->bwtSizeInWord * sizeof ( uint ), hipMemcpyHostToDevice );

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MEMCPY FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }

    gpuErr = hipMemcpy ( ( *_occ ), occValue, numOfOccValue * ALPHABET_SIZE * sizeof ( uint ), hipMemcpyHostToDevice );

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MEMCPY FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }

    gpuErr = hipMalloc ( ( void ** ) _revBwt, revBwt->bwtSizeInWord * sizeof ( uint ) );

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MALLOC FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }

    gpuErr = hipMalloc ( ( void ** ) _revOcc, numOfOccValue * ALPHABET_SIZE * sizeof ( uint ) );

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MALLOC FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }

    gpuErr = hipMemcpy ( ( *_revBwt ), revBwt->bwtCode, revBwt->bwtSizeInWord * sizeof ( uint ), hipMemcpyHostToDevice );

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MEMCPY FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }

    gpuErr = hipMemcpy ( ( *_revOcc ), revOccValue, numOfOccValue * ALPHABET_SIZE * sizeof ( uint ), hipMemcpyHostToDevice );

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MEMCPY FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }

    gpuErr = hipMemcpyToSymbol(HIP_SYMBOL( gpuCharMap), index->charMap, sizeof ( unsigned char ) * 256 );

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MEMCPY FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }
}

void GPUINDEXFree ( uint * _bwt, uint * _occ, uint * _revBwt, uint * _revOcc )
{
    hipFree ( _bwt );
    hipFree ( _occ );
    hipFree ( _revBwt );
    hipFree ( _revOcc );
}

// perform round1 alignment in GPU
void perform_round1_alignment ( uint * nextQuery, uint * nextReadLength, uint * answers[][MAX_NUM_CASES],
                                uint numMismatch, uint numCases, uint sa_range_allowed, uint wordPerQuery, uint word_per_ans,
                                bool isExactNumMismatch, int doubleBufferIdx, uint blocksNeeded, ullint batchSize,
                                Soap3Index * index, uint * _bwt, uint * _revBwt, uint * _occ, uint * _revOcc )
{
    hipError_t gpuErr;
    uint * _queries, *_readLengths, *_answers;
    bool * _isBad;
    ullint roundUp = ( batchSize + 31 ) / 32 * 32;
    // allocated device memory for bad read indicator
    bool * isBad;
    isBad = ( bool * ) malloc ( roundUp * sizeof ( bool ) ); // an array to store bad read indicator
    memset ( isBad, 0, roundUp );
    gpuErr = hipMalloc ( ( void ** ) &_isBad, roundUp * sizeof ( bool ) );
    BWT * bwt = index->sraIndex->bwt;
    BWT * revBwt = index->sraIndex->rev_bwt;

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MALLOC FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }

    // to initialize the array _isBad
    gpuErr = hipMemcpy ( _isBad, isBad,
                          roundUp * sizeof ( bool ),
                          hipMemcpyHostToDevice );

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MEMCOPY FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }

    // to initialize the array answers
    // for (uint caseno=0; caseno < numCases; caseno++) {
    //       memset(answers[doubleBufferIdx][caseno], 0, roundUp * word_per_ans * sizeof(uint));
    // }
    // printf("[perform_round1_alignment] sa_range_allowed = %u; word_per_ans = %u\n", sa_range_allowed, word_per_ans);
    // allocate device memory for queries and answers
    hipMalloc ( ( void ** ) &_queries, roundUp * wordPerQuery * sizeof ( uint ) );
    hipMalloc ( ( void ** ) &_readLengths, roundUp * sizeof ( uint ) );
    hipMalloc ( ( void ** ) &_answers, roundUp * word_per_ans * sizeof ( uint ) );
    gpuErr = hipMemcpy ( _queries, nextQuery, roundUp * wordPerQuery * sizeof ( uint ), hipMemcpyHostToDevice );

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MEMCOPY FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }

    gpuErr = hipMemcpy ( _readLengths, nextReadLength, roundUp * sizeof ( uint ), hipMemcpyHostToDevice );

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MEMCOPY FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }

    for ( uint caseno = 0; caseno < numCases; caseno++ )
    {
        // =======================================
        // | GPU-1: FOR EACH CASE                |
        // =======================================
        if ( numMismatch <= 3 )
            kernel <<< blocksNeeded, THREADS_PER_BLOCK>>>
            ( caseno, _queries, _readLengths, batchSize, wordPerQuery,
              _bwt, _occ, bwt->inverseSa0,
              _revBwt, _revOcc, revBwt->inverseSa0, bwt->textLength,
              _answers, _isBad, 0 , numMismatch, sa_range_allowed, word_per_ans, isExactNumMismatch );
        else if ( caseno < 5 ) // 4 mismatch and case 0 - 4
            kernel_4mismatch_1 <<< blocksNeeded, THREADS_PER_BLOCK>>>
            ( caseno, _queries, _readLengths, batchSize, wordPerQuery,
              _bwt, _occ, bwt->inverseSa0,
              _revBwt, _revOcc, revBwt->inverseSa0, bwt->textLength,
              _answers, _isBad, 0 , sa_range_allowed, word_per_ans, isExactNumMismatch );
        else  // 4 mismatch and case 5 - 9
            kernel_4mismatch_2 <<< blocksNeeded, THREADS_PER_BLOCK>>>
            ( caseno, _queries, _readLengths, batchSize, wordPerQuery,
              _bwt, _occ, bwt->inverseSa0,
              _revBwt, _revOcc, revBwt->inverseSa0, bwt->textLength,
              _answers, _isBad, 0 , sa_range_allowed, word_per_ans, isExactNumMismatch );

        gpuErr = hipMemcpy ( answers[doubleBufferIdx][caseno], _answers, roundUp * word_per_ans * sizeof ( uint ), hipMemcpyDeviceToHost );

        if ( gpuErr != hipSuccess )
        {
            printf ( "CUDA MEMCOPY FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
            exit ( 1 );
        }
    }

    // free the memories
    free ( isBad );
    hipFree ( _isBad );
    hipFree ( _queries );
    hipFree ( _readLengths );
    hipFree ( _answers );
}



// perform round2 alignment in GPU
void perform_round2_alignment ( uint * queries, uint * readLengths, uint * answers[][MAX_NUM_CASES],
                                uint numMismatch, uint numCases, uint sa_range_allowed_2, uint wordPerQuery, uint word_per_ans, uint word_per_ans_2,
                                bool isExactNumMismatch, int doubleBufferIdx, uint blocksNeeded, ullint batchSize,
                                Soap3Index * index, uint * _bwt, uint * _revBwt, uint * _occ, uint * _revOcc,
                                uint processedQuery, uint * badReadIndices[][MAX_NUM_CASES],
                                uint * badAnswers[][MAX_NUM_CASES] )
{
    uint * badQueries;
    uint * badReadLengths;
    BWT * bwt = index->sraIndex->bwt;
    BWT * revBwt = index->sraIndex->rev_bwt;

    for ( int whichCase = 0; whichCase < numCases; ++whichCase )
    {
        ullint numBads = 0;

        // Count number of bad reads and
        for ( ullint readId = 0; readId < batchSize; readId++ )
        {
            ullint srcOffset = ( ( readId ) / 32 * 32 * word_per_ans + readId % 32 );
            numBads += ( answers[doubleBufferIdx][whichCase][srcOffset] > 0xFFFFFFFD );
        }

        if ( numBads == 0 )
        { continue; }

        // Allocate memory and copy bad reads to another array
        ullint roundUp = ( numBads + 31 ) / 32 * 32;
        badQueries = ( uint * ) malloc ( roundUp * wordPerQuery * sizeof ( uint ) );
        // printf("size of badQueries = %u\n", roundUp * wordPerQuery);
        badReadLengths = ( uint * ) malloc ( roundUp * sizeof ( uint ) );
        // printf("size of badReadLengths = %u\n", roundUp);
        numBads = 0;

        for ( ullint readId = 0; readId < batchSize; ++readId )
        {
            ullint srcOffset = ( ( readId ) / 32 * 32 * word_per_ans + readId % 32 );
            ullint srcQueryOffset = ( processedQuery + readId ) / 32 * 32 * wordPerQuery + readId % 32;

            if ( answers[doubleBufferIdx][whichCase][srcOffset] > 0xFFFFFFFD ) // is bad
            {
                badReadIndices[doubleBufferIdx][whichCase][numBads] = readId;
                ullint targetOffset = numBads / 32 * 32 * wordPerQuery + numBads % 32;

                for ( ullint i = 0; i < wordPerQuery; ++i ) // copy each word of the read
                {
                    badQueries[targetOffset + i * 32] = * ( queries + ( srcQueryOffset + i * 32 ) );
                }

                badReadLengths[numBads] = readLengths[processedQuery + readId];
                numBads++;
            }
        }

        // Copy reads to device memory, call kernel, and copy results back to host
        uint * _queries, *_readLengths, *_answers;
        bool * _isBad = NULL;
        // bool *isBad = (bool*) malloc((batchSize + 31) / 32 * 32 * sizeof(bool)); // an array to store bad read indicator
        // memset(isBad,0, (batchSize + 31) / 32 * 32 * sizeof(bool));
        // hipMalloc((void**)&_isBad, (batchSize + 31) / 32 * 32 * sizeof(bool));
        // hipMemcpy(_isBad, isBad,(batchSize + 31) / 32 * 32 * sizeof(bool), hipMemcpyHostToDevice);
        // free(isBad);
        hipMalloc ( ( void ** ) &_queries, roundUp * wordPerQuery * sizeof ( uint ) );
        hipMemcpy ( _queries, badQueries,
                     roundUp * wordPerQuery * sizeof ( uint ), hipMemcpyHostToDevice );
        free ( badQueries );
        hipMalloc ( ( void ** ) &_readLengths, roundUp * sizeof ( uint ) );
        hipMemcpy ( _readLengths, badReadLengths,
                     roundUp * sizeof ( uint ), hipMemcpyHostToDevice );
        free ( badReadLengths );
        // reset the values of the array badAnswers
        // memset(badAnswers[doubleBufferIdx][whichCase], 0, roundUp * word_per_ans_2 * sizeof(uint));
        hipMalloc ( ( void ** ) &_answers, roundUp * word_per_ans_2 * sizeof ( uint ) );

        // to reset the values inside the array _answers
        // hipMemcpy(_answers, badAnswers[doubleBufferIdx][whichCase],
        //          roundUp * word_per_ans_2 * sizeof(uint),
        //          hipMemcpyHostToDevice);
        if ( numMismatch <= 3 ) // 3 mismatch
            kernel <<< blocksNeeded, THREADS_PER_BLOCK>>>
            ( whichCase, _queries, _readLengths, numBads, wordPerQuery,
              _bwt, _occ, bwt->inverseSa0,
              _revBwt, _revOcc, revBwt->inverseSa0,
              bwt->textLength, _answers, _isBad, 1, numMismatch, sa_range_allowed_2, word_per_ans_2, isExactNumMismatch );
        else if ( whichCase < 5 ) // 4 mismatch and case no. 0 - 4
            kernel_4mismatch_1 <<< blocksNeeded, THREADS_PER_BLOCK>>>
            ( whichCase, _queries, _readLengths, numBads, wordPerQuery,
              _bwt, _occ, bwt->inverseSa0,
              _revBwt, _revOcc, revBwt->inverseSa0,
              bwt->textLength, _answers, _isBad, 1, sa_range_allowed_2, word_per_ans_2, isExactNumMismatch );
        else   // 4 mismatch and case no. 5 - 9
            kernel_4mismatch_2 <<< blocksNeeded, THREADS_PER_BLOCK>>>
            ( whichCase, _queries, _readLengths, numBads, wordPerQuery,
              _bwt, _occ, bwt->inverseSa0,
              _revBwt, _revOcc, revBwt->inverseSa0,
              bwt->textLength, _answers, _isBad, 1, sa_range_allowed_2, word_per_ans_2, isExactNumMismatch );

        hipMemcpy ( badAnswers[doubleBufferIdx][whichCase], _answers,
                     roundUp * word_per_ans_2 * sizeof ( uint ), hipMemcpyDeviceToHost );
        // free the memory in the device
        // hipFree(_isBad);
        hipFree ( _queries );
        hipFree ( _answers );
        hipFree ( _readLengths );
    }
}

// perform round1 alignment in GPU for 1 mismatch (no pipeline)
void perform_round1_alignment_no_pipeline ( uint * nextQuery, uint * nextReadLength, uint * answers[MAX_NUM_CASES],
        uint numMismatch, uint numCases, uint sa_range_allowed, uint wordPerQuery, uint word_per_ans,
        bool isExactNumMismatch, uint blocksNeeded, ullint batchSize,
        Soap3Index * index, uint * _bwt, uint * _revBwt, uint * _occ, uint * _revOcc )
{
    hipError_t gpuErr;
    uint * _queries, *_readLengths, *_answers;
    bool * _isBad;
    ullint roundUp = ( batchSize + 31 ) / 32 * 32;
    // allocated device memory for bad read indicator
    bool * isBad;
    isBad = ( bool * ) malloc ( roundUp * sizeof ( bool ) ); // an array to store bad read indicator
    memset ( isBad, 0, roundUp );
    gpuErr = hipMalloc ( ( void ** ) &_isBad, roundUp * sizeof ( bool ) );
    BWT * bwt = index->sraIndex->bwt;
    BWT * revBwt = index->sraIndex->rev_bwt;

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MALLOC FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }

    // to initialize the array _isBad
    gpuErr = hipMemcpy ( _isBad, isBad,
                          roundUp * sizeof ( bool ),
                          hipMemcpyHostToDevice );

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MEMCOPY FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }

    // printf("[perform_round1_alignment] sa_range_allowed = %u; word_per_ans = %u\n", sa_range_allowed, word_per_ans);
    // allocate device memory for queries and answers
    hipMalloc ( ( void ** ) &_queries, roundUp * wordPerQuery * sizeof ( uint ) );
    hipMalloc ( ( void ** ) &_readLengths, roundUp * sizeof ( uint ) );
    hipMalloc ( ( void ** ) &_answers, roundUp * word_per_ans * sizeof ( uint ) );
    gpuErr = hipMemcpy ( _queries, nextQuery, roundUp * wordPerQuery * sizeof ( uint ), hipMemcpyHostToDevice );

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MEMCOPY FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }

    gpuErr = hipMemcpy ( _readLengths, nextReadLength, roundUp * sizeof ( uint ), hipMemcpyHostToDevice );

    if ( gpuErr != hipSuccess )
    {
        printf ( "CUDA MEMCOPY FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
        exit ( 1 );
    }

    for ( uint caseno = 0; caseno < numCases; caseno++ )
    {
        // =======================================
        // | GPU-1: FOR EACH CASE                |
        // =======================================
        if ( numMismatch <= 3 )
            kernel <<< blocksNeeded, THREADS_PER_BLOCK>>>
            ( caseno, _queries, _readLengths, batchSize, wordPerQuery,
              _bwt, _occ, bwt->inverseSa0,
              _revBwt, _revOcc, revBwt->inverseSa0, bwt->textLength,
              _answers, _isBad, 0 , numMismatch, sa_range_allowed, word_per_ans, isExactNumMismatch );
        else if ( caseno < 5 ) // 4 mismatch and case 0 - 4
            kernel_4mismatch_1 <<< blocksNeeded, THREADS_PER_BLOCK>>>
            ( caseno, _queries, _readLengths, batchSize, wordPerQuery,
              _bwt, _occ, bwt->inverseSa0,
              _revBwt, _revOcc, revBwt->inverseSa0, bwt->textLength,
              _answers, _isBad, 0 , sa_range_allowed, word_per_ans, isExactNumMismatch );
        else  // 4 mismatch and case 5 - 9
            kernel_4mismatch_2 <<< blocksNeeded, THREADS_PER_BLOCK>>>
            ( caseno, _queries, _readLengths, batchSize, wordPerQuery,
              _bwt, _occ, bwt->inverseSa0,
              _revBwt, _revOcc, revBwt->inverseSa0, bwt->textLength,
              _answers, _isBad, 0 , sa_range_allowed, word_per_ans, isExactNumMismatch );

        gpuErr = hipMemcpy ( answers[caseno], _answers, roundUp * word_per_ans * sizeof ( uint ), hipMemcpyDeviceToHost );

        if ( gpuErr != hipSuccess )
        {
            printf ( "CUDA MEMCOPY FAILED .. %s(%d)\n", hipGetErrorString ( gpuErr ), gpuErr );
            exit ( 1 );
        }
    }

    // free the memories
    free ( isBad );
    hipFree ( _isBad );
    hipFree ( _queries );
    hipFree ( _readLengths );
    hipFree ( _answers );
}

// perform round2 alignment in GPU for 1 mismatch
void perform_round2_alignment_no_pipeline ( uint * queries, uint * readLengths, uint * answers[MAX_NUM_CASES],
        uint numMismatch, uint numCases, uint sa_range_allowed_2, uint wordPerQuery, uint word_per_ans, uint word_per_ans_2,
        bool isExactNumMismatch, uint blocksNeeded, ullint batchSize,
        Soap3Index * index, uint * _bwt, uint * _revBwt, uint * _occ, uint * _revOcc,
        uint processedQuery, uint * badReadIndices[MAX_NUM_CASES],
        uint * badAnswers[MAX_NUM_CASES] )
{
    uint * badQueries;
    uint * badReadLengths;
    BWT * bwt = index->sraIndex->bwt;
    BWT * revBwt = index->sraIndex->rev_bwt;

    for ( int whichCase = 0; whichCase < numCases; ++whichCase )
    {
        ullint numBads = 0;

        // Count number of bad reads and
        for ( ullint readId = 0; readId < batchSize; readId++ )
        {
            ullint srcOffset = ( ( readId ) / 32 * 32 * word_per_ans + readId % 32 );
            numBads += ( answers[whichCase][srcOffset] > 0xFFFFFFFD );
        }

        if ( numBads == 0 )
        { continue; }

        // Allocate memory and copy bad reads to another array
        ullint roundUp = ( numBads + 31 ) / 32 * 32;
        badQueries = ( uint * ) malloc ( roundUp * wordPerQuery * sizeof ( uint ) );
        // printf("size of badQueries = %u\n", roundUp * wordPerQuery);
        badReadLengths = ( uint * ) malloc ( roundUp * sizeof ( uint ) );
        // printf("size of badReadLengths = %u\n", roundUp);
        numBads = 0;

        for ( ullint readId = 0; readId < batchSize; ++readId )
        {
            ullint srcOffset = ( ( readId ) / 32 * 32 * word_per_ans + readId % 32 );
            ullint srcQueryOffset = ( processedQuery + readId ) / 32 * 32 * wordPerQuery + readId % 32;

            if ( answers[whichCase][srcOffset] > 0xFFFFFFFD ) // is bad
            {
                badReadIndices[whichCase][numBads] = readId;
                ullint targetOffset = numBads / 32 * 32 * wordPerQuery + numBads % 32;

                for ( ullint i = 0; i < wordPerQuery; ++i ) // copy each word of the read
                {
                    badQueries[targetOffset + i * 32] = * ( queries + ( srcQueryOffset + i * 32 ) );
                }

                badReadLengths[numBads] = readLengths[processedQuery + readId];
                numBads++;
            }
        }

        // Copy reads to device memory, call kernel, and copy results back to host
        uint * _queries, *_readLengths, *_answers;
        bool * _isBad = NULL;
        // bool *isBad = (bool*) malloc((batchSize + 31) / 32 * 32 * sizeof(bool)); // an array to store bad read indicator
        // memset(isBad,0, (batchSize + 31) / 32 * 32 * sizeof(bool));
        // hipMalloc((void**)&_isBad, (batchSize + 31) / 32 * 32 * sizeof(bool));
        // hipMemcpy(_isBad, isBad,(batchSize + 31) / 32 * 32 * sizeof(bool), hipMemcpyHostToDevice);
        // free(isBad);
        hipMalloc ( ( void ** ) &_queries, roundUp * wordPerQuery * sizeof ( uint ) );
        hipMemcpy ( _queries, badQueries,
                     roundUp * wordPerQuery * sizeof ( uint ), hipMemcpyHostToDevice );
        free ( badQueries );
        hipMalloc ( ( void ** ) &_readLengths, roundUp * sizeof ( uint ) );
        hipMemcpy ( _readLengths, badReadLengths,
                     roundUp * sizeof ( uint ), hipMemcpyHostToDevice );
        free ( badReadLengths );
        // reset the values of the array badAnswers
        // memset(badAnswers[whichCase], 0, roundUp * word_per_ans_2 * sizeof(uint));
        hipMalloc ( ( void ** ) &_answers, roundUp * word_per_ans_2 * sizeof ( uint ) );

        // to reset the values inside the array _answers
        // hipMemcpy(_answers, badAnswers[whichCase],
        //          roundUp * word_per_ans_2 * sizeof(uint),
        //          hipMemcpyHostToDevice);
        if ( numMismatch <= 3 ) // 3 mismatch
            kernel <<< blocksNeeded, THREADS_PER_BLOCK>>>
            ( whichCase, _queries, _readLengths, numBads, wordPerQuery,
              _bwt, _occ, bwt->inverseSa0,
              _revBwt, _revOcc, revBwt->inverseSa0,
              bwt->textLength, _answers, _isBad, 1, numMismatch, sa_range_allowed_2, word_per_ans_2, isExactNumMismatch );
        else if ( whichCase < 5 ) // 4 mismatch and case no. 0 - 4
            kernel_4mismatch_1 <<< blocksNeeded, THREADS_PER_BLOCK>>>
            ( whichCase, _queries, _readLengths, numBads, wordPerQuery,
              _bwt, _occ, bwt->inverseSa0,
              _revBwt, _revOcc, revBwt->inverseSa0,
              bwt->textLength, _answers, _isBad, 1, sa_range_allowed_2, word_per_ans_2, isExactNumMismatch );
        else   // 4 mismatch and case no. 5 - 9
            kernel_4mismatch_2 <<< blocksNeeded, THREADS_PER_BLOCK>>>
            ( whichCase, _queries, _readLengths, numBads, wordPerQuery,
              _bwt, _occ, bwt->inverseSa0,
              _revBwt, _revOcc, revBwt->inverseSa0,
              bwt->textLength, _answers, _isBad, 1, sa_range_allowed_2, word_per_ans_2, isExactNumMismatch );

        hipMemcpy ( badAnswers[whichCase], _answers,
                     roundUp * word_per_ans_2 * sizeof ( uint ), hipMemcpyDeviceToHost );
        // free the memory in the device
        // hipFree(_isBad);
        hipFree ( _queries );
        hipFree ( _answers );
        hipFree ( _readLengths );
    }
}


void all_valid_alignment ( uint * queries, uint * readLengths, uint * seedLengths, uint numMismatch, uint wordPerQuery,
                           ullint maxBatchSize, uint numQueries, uint accumReadNum,
                           Soap3Index * index, uint * _bwt, uint * _revBwt, uint * _occ, uint * _revOcc,
                           IniParams ini_params, InputOptions input_options,
                           char * upkdQualities,
                           uint * unAlignedReads, uint & numOfUnPaired,
                           uint * readIDs, char * upkdQueryNames,
                           char ** currOutputFileName, samfile_t ** currSamOutputFilePtr,
                           unsigned long long & numOfAnswer, uint & numOfAlignedRead,
                           uint8_t isTerminalCase,
                           ReadInputForDP ** readInputForDP,
                           ReadInputForDP ** readInputForNewDP,
                           ReadInputForDP ** otherSoap3Result,
                           BothUnalignedPairs ** bothUnalignedPairs )
{
#ifdef BGS_GPU_CASE_BREAKDOWN_TIME
    double startTime2 = setStartTime ();
    double lastEventTime2 = 0;
    double currEventTime2;
#endif
    //Modified to be double buffered
    uint * answers[2][MAX_NUM_CASES];
    uint * badReadIndices[2][MAX_NUM_CASES]; // IDs of bad reads for each case
    uint * badAnswers[2][MAX_NUM_CASES]; // stores answers (and reads temporarily) for 2nd round
    uint threadBadStarts[2][MAX_NUM_CPU_THREADS][MAX_NUM_CASES];
    uint threadBadCounts[2][MAX_NUM_CPU_THREADS][MAX_NUM_CASES];
    uint * unAlignedId[MAX_NUM_CPU_THREADS];
    int doubleBufferIdx = 0; // expect to be either 0 or 1
    // Host alignment model
    SRAModel * SRAMismatchModel[MAX_READ_LENGTH + 1];
    SRAModel * SRAMismatchModel2[MAX_READ_LENGTH + 1];
    SRAModel * SRAMismatchModel_neg[MAX_READ_LENGTH + 1];
    SRAModel * SRAMismatchModel2_neg[MAX_READ_LENGTH + 1];
    HostKernelArguements hostKernelArguments[MAX_NUM_CPU_THREADS];
    // Host multi-threading variables
    int threadId;
    pthread_t threads[MAX_NUM_CPU_THREADS];
    uint threadBucketSize[MAX_NUM_CPU_THREADS];
    numOfAnswer = 0;
    numOfAlignedRead = 0;
    numOfUnPaired = 0;
    uint blocksNeeded;
    ullint batchSize;
    uint numCases;
    uint sa_range_allowed_1;
    uint sa_range_allowed_2;
    char skip_round_2;
    ullint queriesLeft = numQueries;
    uint maxReadLength = input_options.maxReadLength;
    uint word_per_ans;
    uint word_per_ans_2;
    int i;

    SRASetting _mapqTempSRASetting;

    SRAIndex * sraIndex = index->sraIndex;

    // initialization of arrays
    for ( i = 0; i < 2; i++ )
    {
        for ( int j = 0; j < MAX_NUM_CASES; j++ )
        {
            badReadIndices[i][j] = NULL;
            badAnswers[i][j] = NULL;
            answers[i][j] = NULL;

            for ( int k = 0; k < MAX_NUM_CPU_THREADS; k++ )
            {
                threadBadStarts[i][k][j] = 0;
                threadBadCounts[i][k][j] = 0;
            }
        }
    }

    // initialization of unAlignedId
    for ( int i = 0; i < ini_params.Ini_NumOfCpuThreads; i++ )
    {
        unAlignedId[i] = ( uint * ) malloc ( maxBatchSize * sizeof ( uint ) );
    }

    // obtain the number of cases for this number of mismatch
    // and obtain the number of SA ranges allowed
    getParametersForThisMismatch ( numMismatch, numCases, sa_range_allowed_1,
                                   sa_range_allowed_2, skip_round_2, word_per_ans, word_per_ans_2 );

    // For single-read alignment (and not a long-read mode),
    // unique best and random best requires only 1 answer
    if ( input_options.readType == SINGLE_READ && seedLengths == NULL )
    {
        if ( input_options.alignmentType == OUTPUT_UNIQUE_BEST )
        {
            sa_range_allowed_1 = 1;
            word_per_ans = 2;
            skip_round_2 = 1;
        }

        if ( input_options.alignmentType == OUTPUT_RANDOM_BEST )
        {
            sa_range_allowed_1 = 2;
            word_per_ans = 4;
            skip_round_2 = 1;
        }
    }

    // For 4-mismatch, if all-valid alignment
    // then sa_range_allowed_1 is changed from 1 to 2
    if ( numMismatch == 4 && input_options.alignmentType == OUTPUT_ALL_VALID )
    {
        sa_range_allowed_1 = 2;
        word_per_ans = 4;
    }

    // For paired-end read alignment, (all-best or all-valid) and SAM format
    bool needOutputMAPQ = ( input_options.alignmentType == OUTPUT_ALL_VALID ||
                            input_options.alignmentType == OUTPUT_ALL_BEST ) &&
                          ( input_options.outputFormat == SRA_OUTPUT_FORMAT_SAM_API );

    if ( needOutputMAPQ && ( numMismatch == 2 ) )
    {
        sa_range_allowed_2 = 256;
        word_per_ans_2 = 512;
    }

    // set the multi-threading arguments
    setHostKernelArguments ( hostKernelArguments, threads, ini_params, index, maxReadLength, wordPerQuery, word_per_ans, &input_options );

    for ( threadId = 0; threadId < ini_params.Ini_NumOfCpuThreads; ++threadId )
    {
        hostKernelArguments[threadId].upkdQualities = upkdQualities;
        hostKernelArguments[threadId].upkdQueryNames = upkdQueryNames;
        hostKernelArguments[threadId].SRAMismatchModel = SRAMismatchModel;
        hostKernelArguments[threadId].SRAMismatchModel2 = SRAMismatchModel2;
        hostKernelArguments[threadId].SRAMismatchModel_neg = SRAMismatchModel_neg;
        hostKernelArguments[threadId].SRAMismatchModel2_neg = SRAMismatchModel2_neg;

        if ( currOutputFileName != NULL )
        { hostKernelArguments[threadId].outputFileName = currOutputFileName[threadId]; }
        else
        { hostKernelArguments[threadId].outputFileName = NULL; }

        hostKernelArguments[threadId].readLengths = readLengths;
        hostKernelArguments[threadId].seedLengths = seedLengths;
        hostKernelArguments[threadId].readIDs = readIDs;
        hostKernelArguments[threadId].accumReadNum = accumReadNum;
        hostKernelArguments[threadId].unAlignedIDs = unAlignedId[threadId];
        hostKernelArguments[threadId].unAlignedOcc = 0;
    }

    ullint roundUp = ( maxBatchSize + 31 ) / 32 * 32;

    // allocate host memory for answers
    for ( i = 0; i < 2; i++ )
    {
        for ( int j = 0; j < numCases; j++ )
        {
            answers[i][j] = ( uint * ) malloc ( roundUp * word_per_ans * sizeof ( uint ) );
            //memset(answers[i][j],0xFFFFFFFF, roundUp * word_per_ans * sizeof ( uint ) );
        }
    }

    // if there is only 3G GPU memory, then skip round 2
    if ( ini_params.Ini_GPUMemory == 3 )
    { skip_round_2 = 1; }

    // printParameters(input_options, ini_params);
    // printf("numMismatch = %u; numCases = %u; sa_range_allowed_1 = %u; sa_range_allowed_2 = %u; skip_round_2 = %i; word_per_ans = %u; word_per_ans_2 = %u \n", numMismatch, numCases, sa_range_allowed_1, sa_range_allowed_2, skip_round_2, word_per_ans, word_per_ans_2);

    // update the num of mismatch of the model for host alignment
    for ( threadId = 0; threadId < ini_params.Ini_NumOfCpuThreads; ++threadId )
    {
        hostKernelArguments[threadId].sraQuerySettings.MaxError = numMismatch;
        hostKernelArguments[threadId].sraQuerySettings.MaxNBMismatch = 0;
        hostKernelArguments[threadId].numCases = numCases;
        hostKernelArguments[threadId].numCases2 = GetNumCases ( numMismatch + 1 );
        hostKernelArguments[threadId].sa_range_allowed_1 = sa_range_allowed_1;
        hostKernelArguments[threadId].sa_range_allowed_2 = sa_range_allowed_2;
        hostKernelArguments[threadId].maxNumMismatch = numMismatch;

        if ( readInputForDP != NULL )
        { hostKernelArguments[threadId].readInput = readInputForDP[threadId]; }

        if ( readInputForNewDP != NULL )
        { hostKernelArguments[threadId].readInputForNewDefault = readInputForNewDP[threadId]; }

        if ( otherSoap3Result != NULL )
        { hostKernelArguments[threadId].otherSoap3Result = otherSoap3Result[threadId]; }

        if ( bothUnalignedPairs != NULL )
        { hostKernelArguments[threadId].bothUnalignedPairs = bothUnalignedPairs[threadId]; }

        //Update the SAM output file ptr in QuerySetting
        if ( currSamOutputFilePtr != NULL )
        { hostKernelArguments[threadId].sraQuerySettings.SAMOutFilePtr = currSamOutputFilePtr[threadId]; }
        else
        { hostKernelArguments[threadId].sraQuerySettings.SAMOutFilePtr = NULL; }
    }

    for ( i = 0; i <= MAX_READ_LENGTH; i++ )
    {
        SRAMismatchModel[i] = NULL;
        SRAMismatchModel2[i] = NULL;
        SRAMismatchModel_neg[i] = NULL;
        SRAMismatchModel2_neg[i] = NULL;
    }

    if ( seedLengths == NULL )
    {
        for ( unsigned readId = 0; readId < numQueries; ++readId )
        {
            if ( SRAMismatchModel[readLengths[readId]] == NULL )
            {
                SRAMismatchModel[readLengths[readId]] = SRAModelConstruct ( readLengths[readId], QUERY_POS_STRAND, & ( hostKernelArguments[0].sraQuerySettings ), sraIndex,  ini_params.Ini_HostAlignmentModel );

            }

            if ( SRAMismatchModel_neg[readLengths[readId]] == NULL )
            {
                SRAMismatchModel_neg[readLengths[readId]] = SRAModelConstruct ( readLengths[readId], QUERY_NEG_STRAND, & ( hostKernelArguments[0].sraQuerySettings ), sraIndex,  ini_params.Ini_HostAlignmentModel );

            }
        }
    }
    else
    {
        for ( unsigned readId = 0; readId < numQueries; ++readId )
        {
            if ( SRAMismatchModel[seedLengths[readId]] == NULL )
            {
                SRAMismatchModel[seedLengths[readId]] = SRAModelConstruct ( seedLengths[readId], QUERY_POS_STRAND, & ( hostKernelArguments[0].sraQuerySettings ), sraIndex, ini_params.Ini_HostAlignmentModel );
            }

            if ( SRAMismatchModel_neg[seedLengths[readId]] == NULL )
            {
                SRAMismatchModel_neg[seedLengths[readId]] = SRAModelConstruct ( seedLengths[readId], QUERY_NEG_STRAND, & ( hostKernelArguments[0].sraQuerySettings ), sraIndex, ini_params.Ini_HostAlignmentModel );
            }
        }
    }

    // if (ALL-VALID or ALL-BEST) and SAM format, then need to update the model for mismatch+1
    if ( needOutputMAPQ && numMismatch < 4 )
    {
        memcpy ( &_mapqTempSRASetting, & ( hostKernelArguments[0].sraQuerySettings ), sizeof ( SRASetting ) );
        _mapqTempSRASetting.MaxError = numMismatch + 1;

        if ( seedLengths == NULL )
        {
            for ( unsigned readId = 0; readId < numQueries; ++readId )
            {
                if ( SRAMismatchModel2[readLengths[readId]] == NULL )
                {
                    SRAMismatchModel2[readLengths[readId]] = SRAModelConstruct ( readLengths[readId], QUERY_POS_STRAND, &_mapqTempSRASetting, sraIndex, ini_params.Ini_HostAlignmentModel );
                }

                if ( SRAMismatchModel2_neg[readLengths[readId]] == NULL )
                {
                    SRAMismatchModel2_neg[readLengths[readId]] = SRAModelConstruct ( readLengths[readId], QUERY_NEG_STRAND, &_mapqTempSRASetting, sraIndex, ini_params.Ini_HostAlignmentModel );
                }
            }
        }
        else
        {
            for ( unsigned readId = 0; readId < numQueries; ++readId )
            {
                if ( SRAMismatchModel2[seedLengths[readId]] == NULL )
                {
                    SRAMismatchModel2[seedLengths[readId]] = SRAModelConstruct ( seedLengths[readId], QUERY_POS_STRAND, &_mapqTempSRASetting, sraIndex, ini_params.Ini_HostAlignmentModel );
                }

                if ( SRAMismatchModel2_neg[seedLengths[readId]] == NULL )
                {
                    SRAMismatchModel2_neg[seedLengths[readId]] = SRAModelConstruct ( seedLengths[readId], QUERY_NEG_STRAND, &_mapqTempSRASetting, sraIndex, ini_params.Ini_HostAlignmentModel );
                }
            }
        }
    }

    uint * nextQuery = queries;
    uint * nextReadLength;

    if ( seedLengths != NULL )
    { nextReadLength = seedLengths; }
    else
    { nextReadLength = readLengths; }

    uint * nextUnAlignedReads = unAlignedReads;
#ifdef BGS_GPU_CASE_BREAKDOWN_TIME
    currEventTime2 = getElapsedTime ( startTime2 );
    printf ( "[Main] Time elapsed for initialization: %9.4f seconds.\n", currEventTime2 - lastEventTime2 );
    lastEventTime2 = currEventTime2;
#endif

    while ( queriesLeft > 0 )
    {
        uint processedQuery = numQueries - queriesLeft;

        if ( queriesLeft > maxBatchSize )
        {
            blocksNeeded = NUM_BLOCKS;
            batchSize = maxBatchSize;
        }
        else
        {
            blocksNeeded = ( queriesLeft + THREADS_PER_BLOCK * QUERIES_PER_THREAD - 1 ) /
                           ( THREADS_PER_BLOCK * QUERIES_PER_THREAD );
            batchSize = queriesLeft;
        }

        // allocate the reads to different CPU threads
        uint roughBucketSize = batchSize / ini_params.Ini_NumOfCpuThreads;
        // roughBucketSize has to be divible by 2
        roughBucketSize = roughBucketSize / 2 * 2;
        uint batchSizeUnalloc = batchSize;

        for ( threadId = 0; threadId < ini_params.Ini_NumOfCpuThreads - 1; threadId++ )
        {
            threadBucketSize[threadId] = roughBucketSize;
            batchSizeUnalloc -= roughBucketSize;
        }

        threadBucketSize[ini_params.Ini_NumOfCpuThreads - 1] = batchSizeUnalloc;
        // perform first round alignment in GPU
        perform_round1_alignment ( nextQuery, nextReadLength, answers,
                                   numMismatch, numCases, sa_range_allowed_1, wordPerQuery,
                                   word_per_ans, false, doubleBufferIdx, blocksNeeded, batchSize,
                                   index, _bwt, _revBwt, _occ, _revOcc );
#ifdef BGS_GPU_CASE_BREAKDOWN_TIME
        printf ( "[Main] # of mismatches allowed: %u with # of sa ranges: %u\n", numMismatch, sa_range_allowed_1 );
        currEventTime2 = getElapsedTime ( startTime2 );
        printf ( "[Main] Time elapsed for first round alignment in GPU: %9.4f seconds.\n", currEventTime2 - lastEventTime2 );
        lastEventTime2 = currEventTime2;
#endif

        if ( skip_round_2 == 0 )
        {
            // =======================================
            // | GPU-2                               |
            // =======================================
            // perform second round alignment in GPU
            for ( int whichCase = 0; whichCase < numCases; ++whichCase )
            {
                ullint numBads = 0;

                // Count number of bad reads and
                for ( ullint readId = 0; readId < batchSize; readId++ )
                {
                    ullint srcOffset = ( ( readId ) / 32 * 32 * word_per_ans + readId % 32 );
                    numBads += ( answers[doubleBufferIdx][whichCase][srcOffset] > 0xFFFFFFFD );
                }

                if ( numBads > 0 )
                {
                    // Allocate memory and copy bad reads to another array
                    ullint roundUp = ( numBads + 31 ) / 32 * 32;
                    badReadIndices[doubleBufferIdx][whichCase] = ( uint * ) malloc ( roundUp * sizeof ( uint ) );
                    // printf("size of badReadIndices[%i][%i] = %u\n",doubleBufferIdx, whichCase, roundUp);
                    badAnswers[doubleBufferIdx][whichCase] = ( uint * ) malloc ( roundUp * word_per_ans_2 * sizeof ( uint ) );
                    // printf("size of badAnswers[%i][%i] = %u\n",doubleBufferIdx, whichCase, roundUp * word_per_ans_2);
                }

                // printf("numBads == %u in case %i of mismatch %u\n", numBads, whichCase, numMismatch);
            }

            if ( seedLengths == NULL )
            {
                perform_round2_alignment ( queries, readLengths, answers,
                                           numMismatch, numCases, sa_range_allowed_2, wordPerQuery, word_per_ans,
                                           word_per_ans_2, false, doubleBufferIdx, blocksNeeded, batchSize,
                                           index, _bwt, _revBwt, _occ, _revOcc,
                                           processedQuery, badReadIndices, badAnswers );
            }
            else
            {
                perform_round2_alignment ( queries, seedLengths, answers,
                                           numMismatch, numCases, sa_range_allowed_2, wordPerQuery, word_per_ans,
                                           word_per_ans_2, false, doubleBufferIdx, blocksNeeded, batchSize,
                                           index, _bwt, _revBwt, _occ, _revOcc,
                                           processedQuery, badReadIndices, badAnswers );
            }

            for ( int whichCase = 0; whichCase < numCases; ++whichCase )
            {
                ullint readId = 0;

                for ( threadId = 0; threadId < ini_params.Ini_NumOfCpuThreads; threadId ++ )
                {
                    uint threadNumBad = 0;

                    // printf("whichCase = %i; threadId = %i; threadBucketSize[threadId] = %u\n",
                    //        whichCase, threadId, threadBucketSize[threadId]);
                    for ( uint n = 0 ; n < threadBucketSize[threadId]; ++n )
                    {
                        ullint srcOffset = ( readId / 32 * 32 * word_per_ans + readId % 32 );
                        threadNumBad += ( answers[doubleBufferIdx][whichCase][srcOffset] > 0xFFFFFFFD );
                        readId++;
                    }

                    threadBadCounts[doubleBufferIdx][threadId][whichCase] = threadNumBad;
                    // printf("threadBadCounts[%i][%i][%i] = %u\n",
                    //    doubleBufferIdx, threadId, whichCase, threadBadCounts[doubleBufferIdx][threadId][whichCase]);
                }
            }

            for ( int whichCase = 0; whichCase < numCases; ++whichCase )
            {
                threadBadStarts[doubleBufferIdx][0][whichCase] = 0;

                for ( threadId = 1; threadId < ini_params.Ini_NumOfCpuThreads; threadId ++ )
                {
                    threadBadStarts[doubleBufferIdx][threadId][whichCase] = threadBadStarts[doubleBufferIdx][threadId - 1][whichCase] + threadBadCounts[doubleBufferIdx][threadId - 1][whichCase];
                    // printf("threadBadStarts[%i][%i][%i] = %u\n",
                    //  doubleBufferIdx, threadId, whichCase, threadBadStarts[doubleBufferIdx][threadId][whichCase]);
                }
            }

#ifdef BGS_GPU_CASE_BREAKDOWN_TIME
            currEventTime2 = getElapsedTime ( startTime2 );
            printf ( "[Main] Time elapsed for second round alignment in GPU: %9.4f seconds.\n", currEventTime2 - lastEventTime2 );
            printf ( "[Main] # of sa ranges allowed: %u \n", sa_range_allowed_2 );
            lastEventTime2 = currEventTime2;
#endif
        } // skip the round 2

        for ( threadId = 0; threadId < ini_params.Ini_NumOfCpuThreads ; threadId++ )
        {
            if ( threads[threadId] != 0 )
            {
                if ( pthread_join ( threads[threadId], NULL ) )
                { fprintf ( stderr, "[Main:Thread%u] Crash!\n", threadId ), exit ( 1 ); }

                // pthread_detach(threads[threadId]);
                numOfAnswer += hostKernelArguments[threadId].alignedOcc;
                numOfAlignedRead += hostKernelArguments[threadId].alignedReads;
                threads[threadId] = 0;

                // consolidate the unAlignedId;
                if ( hostKernelArguments[threadId].unAlignedOcc > 0 )
                {
                    memcpy ( nextUnAlignedReads, unAlignedId[threadId],
                             ( ullint ) hostKernelArguments[threadId].unAlignedOcc * sizeof ( uint ) );
                    nextUnAlignedReads += hostKernelArguments[threadId].unAlignedOcc;
                    numOfUnPaired += hostKernelArguments[threadId].unAlignedOcc;
                }
            }
        }

#ifdef BGS_GPU_CASE_BREAKDOWN_TIME
        currEventTime2 = getElapsedTime ( startTime2 );
        printf ( "[Main] Time elapsed for waiting for CPU threads: %9.4f seconds.\n", currEventTime2 - lastEventTime2 );
        lastEventTime2 = currEventTime2;
#endif

        for ( int whichCase = 0; whichCase < numCases; ++whichCase )
        {
            if ( badReadIndices[1 - doubleBufferIdx][whichCase] != NULL )
            {
                free ( badReadIndices[1 - doubleBufferIdx][whichCase] );
                badReadIndices[1 - doubleBufferIdx][whichCase] = NULL;
            }

            if ( badAnswers[1 - doubleBufferIdx][whichCase] != NULL )
            {
                free ( badAnswers[1 - doubleBufferIdx][whichCase] );
                badAnswers[1 - doubleBufferIdx][whichCase] = NULL;
            }
        }

        // =======================================
        // | CPU: Thread #0,1,2,3,...            |
        // =======================================
        unsigned int threadProcessedQuery = 0;

        for ( threadId = 0; threadId < ini_params.Ini_NumOfCpuThreads ; threadId++ )
        {
            hostKernelArguments[threadId].batchFirstReadId = processedQuery;
            hostKernelArguments[threadId].skipFirst = threadProcessedQuery;
            hostKernelArguments[threadId].numQueries = threadBucketSize[threadId];
            hostKernelArguments[threadId].word_per_query = wordPerQuery;
            hostKernelArguments[threadId].queries = queries;
            hostKernelArguments[threadId].answers = answers[doubleBufferIdx];
            hostKernelArguments[threadId].alignedOcc = 0;
            hostKernelArguments[threadId].alignedReads = 0;
            hostKernelArguments[threadId].badReadIndices = badReadIndices[doubleBufferIdx];
            hostKernelArguments[threadId].badAnswers = badAnswers[doubleBufferIdx];
            hostKernelArguments[threadId].badStartOffset = threadBadStarts[doubleBufferIdx][threadId];
            hostKernelArguments[threadId].badCountOffset = threadBadCounts[doubleBufferIdx][threadId];
            hostKernelArguments[threadId].outputGoodReads = TRUE;
            hostKernelArguments[threadId].skip_round_2 = skip_round_2;
            hostKernelArguments[threadId].isTerminalCase = isTerminalCase;

            if ( pthread_create ( & ( threads[threadId] ), NULL, hostKernelThreadWrapper, ( void * ) & ( hostKernelArguments[threadId] ) ) )
            { fprintf ( stderr, "[Main:Threads%u] Can't create hostKernelThreadWrapper\n", threadId ), exit ( 1 ); }

            threadProcessedQuery += threadBucketSize[threadId];
        }

        // Swap the double buffer
        doubleBufferIdx = 1 - doubleBufferIdx;
        queriesLeft -= batchSize;
        nextQuery += batchSize * wordPerQuery;
        nextReadLength += batchSize;
    }

    for ( threadId = 0; threadId < ini_params.Ini_NumOfCpuThreads ; threadId++ )
    {
        if ( threads[threadId] != 0 )
        {
            if ( pthread_join ( threads[threadId], NULL ) )
            { fprintf ( stderr, "[Main:Thread%u] Crash!\n", threadId ), exit ( 1 ); }

            // pthread_detach(threads[threadId]);
            numOfAnswer += hostKernelArguments[threadId].alignedOcc;
            numOfAlignedRead += hostKernelArguments[threadId].alignedReads;
            threads[threadId] = 0;

            // consolidate the unAlignedId;
            if ( hostKernelArguments[threadId].unAlignedOcc > 0 )
            {
                memcpy ( nextUnAlignedReads, unAlignedId[threadId],
                         ( ullint ) hostKernelArguments[threadId].unAlignedOcc * sizeof ( uint ) );
                nextUnAlignedReads += hostKernelArguments[threadId].unAlignedOcc;
                numOfUnPaired += hostKernelArguments[threadId].unAlignedOcc;
            }
        }
    }

    for ( int whichCase = 0; whichCase < numCases; ++whichCase )
    {
        if ( badReadIndices[1 - doubleBufferIdx][whichCase] != NULL )
        {
            free ( badReadIndices[1 - doubleBufferIdx][whichCase] );
            badReadIndices[1 - doubleBufferIdx][whichCase] = NULL;
        }

        if ( badAnswers[1 - doubleBufferIdx][whichCase] != NULL )
        {
            free ( badAnswers[1 - doubleBufferIdx][whichCase] );
            badAnswers[1 - doubleBufferIdx][whichCase] = NULL;
        }
    }

    // CLEAN UP for each MISMATCH iteration                                                                          |
    for ( i = 0; i <= MAX_READ_LENGTH; i++ )
    {
        if ( SRAMismatchModel[i] != NULL )
        {
            SRAModelFree ( SRAMismatchModel[i] );
            SRAMismatchModel[i] = NULL;
        }

        if ( SRAMismatchModel2[i] != NULL )
        {
            SRAModelFree ( SRAMismatchModel2[i] );
            SRAMismatchModel2[i] = NULL;
        }

        if ( SRAMismatchModel_neg[i] != NULL )
        {
            SRAModelFree ( SRAMismatchModel_neg[i] );
            SRAMismatchModel_neg[i] = NULL;
        }

        if ( SRAMismatchModel2_neg[i] != NULL )
        {
            SRAModelFree ( SRAMismatchModel2_neg[i] );
            SRAMismatchModel2_neg[i] = NULL;
        }
    }

    for ( threadId = 0; threadId < ini_params.Ini_NumOfCpuThreads; ++threadId )
    {
        OCCFree ( hostKernelArguments[threadId].occ );
        free ( unAlignedId[threadId] );
    }

    for ( i = 0; i < 2; i++ )
    {
        for ( int j = 0; j < numCases; j++ )
        {
            free ( answers[i][j] );
        }
    }
}

// pair-end alignment: for random-best
// 4-phases [0,1,2,4]
void four_phases_alignment ( uint * queries, uint * readLengths, uint numMismatch, uint wordPerQuery,
                             ullint maxBatchSize, uint numQueries, uint accumReadNum,
                             Soap3Index * index, uint * _bwt, uint * _revBwt, uint * _occ, uint * _revOcc,
                             IniParams ini_params, InputOptions input_options,
                             char * upkdQualities,
                             uint * unAlignedReads, uint & numOfUnPaired,
                             uint * readIDs, char * upkdQueryNames,
                             char ** currOutputFileName, samfile_t ** currSamOutputFilePtr,
                             unsigned long long & numOfAnswer, uint & numOfAlignedRead,
                             ReadInputForDP ** readInputForDP,
                             ReadInputForDP ** readInputForNewDP,
                             ReadInputForDP ** otherSoap3Result,
                             BothUnalignedPairs ** bothUnalignedPairs )
{
    // for performing pair-end alignment with random-best output
    unsigned long long numOfAnswer0 = 0;
    uint numOfAlignedRead0 = 0;
    unsigned long long numOfAnswer1 = 0;
    uint numOfAlignedRead1 = 0;
    unsigned long long numOfAnswer2 = 0;
    uint numOfAlignedRead2 = 0;
    unsigned long long numOfAnswer3 = 0;
    uint numOfAlignedRead3 = 0;
    //********************************//
    // Phase 0: Perform 0 Alignment //
    //********************************//
    uint numMismatch_phase0 = 0;
    all_valid_alignment ( queries, readLengths, NULL, numMismatch_phase0, wordPerQuery,
                          maxBatchSize, numQueries, accumReadNum,
                          index, _bwt, _revBwt, _occ, _revOcc,
                          ini_params, input_options,
                          upkdQualities,
                          unAlignedReads, numOfUnPaired,
                          readIDs, upkdQueryNames,
                          currOutputFileName,  currSamOutputFilePtr,
                          numOfAnswer0, numOfAlignedRead0, numMismatch == 0,
                          readInputForDP, readInputForNewDP, otherSoap3Result, bothUnalignedPairs );

    if ( numMismatch > 0 && numOfAlignedRead0 < numQueries )
    {
        //********************************//
        // Phase 1: Perform 0/1 Alignment //
        //********************************//
        uint numMismatch_phase1 = 1;
        // pack the reads which are not paired
        packUnPairedReads ( queries, readIDs, readLengths, unAlignedReads,
                            wordPerQuery, numOfUnPaired, maxBatchSize );
        numQueries = numOfUnPaired;
        all_valid_alignment ( queries, readLengths, NULL, numMismatch_phase1, wordPerQuery,
                              maxBatchSize, numQueries, accumReadNum,
                              index, _bwt, _revBwt, _occ, _revOcc,
                              ini_params, input_options,
                              upkdQualities,
                              unAlignedReads, numOfUnPaired,
                              readIDs, upkdQueryNames,
                              currOutputFileName,  currSamOutputFilePtr,
                              numOfAnswer1, numOfAlignedRead1, numMismatch == 1,
                              readInputForDP, readInputForNewDP, otherSoap3Result, bothUnalignedPairs );

        // printf("numOfAlignedRead1 = %u\n", numOfAlignedRead1);
        // printf("numOfUnPaired = %u\n", numOfUnPaired);

        if ( numMismatch > 1 && numOfAlignedRead1 < numQueries )
        {
            //****************************************//
            // Phase 2: Perform 0/1/2 alignment     //
            //****************************************//
            uint numMismatch_phase2 = 2;
            // pack the reads which are not paired
            packUnPairedReads ( queries, readIDs, readLengths, unAlignedReads,
                                wordPerQuery, numOfUnPaired, maxBatchSize );
            numQueries = numOfUnPaired;
            all_valid_alignment ( queries, readLengths, NULL, numMismatch_phase2, wordPerQuery,
                                  maxBatchSize, numQueries, accumReadNum,
                                  index, _bwt, _revBwt, _occ, _revOcc,
                                  ini_params, input_options,
                                  upkdQualities,
                                  unAlignedReads, numOfUnPaired,
                                  readIDs, upkdQueryNames,
                                  currOutputFileName, currSamOutputFilePtr,
                                  numOfAnswer2, numOfAlignedRead2, numMismatch == 2,
                                  readInputForDP, readInputForNewDP, otherSoap3Result, bothUnalignedPairs );

            // printf("numOfAlignedRead2 = %u\n", numOfAlignedRead2);
            // printf("numOfUnPaired = %u\n", numOfUnPaired, 0);

            if ( numMismatch > 2 && numOfAlignedRead2 < numQueries )
            {
                //******************************************//
                // Phase 3: Perform 0/1/2/3/4 alignment     //
                //******************************************//
                // pack the reads which are not paired
                packUnPairedReads ( queries, readIDs, readLengths, unAlignedReads,
                                    wordPerQuery, numOfUnPaired, maxBatchSize );
                numQueries = numOfUnPaired;
                all_valid_alignment ( queries, readLengths, NULL, numMismatch, wordPerQuery,
                                      maxBatchSize, numQueries, accumReadNum,
                                      index, _bwt, _revBwt, _occ, _revOcc,
                                      ini_params, input_options,
                                      upkdQualities,
                                      unAlignedReads, numOfUnPaired,
                                      readIDs, upkdQueryNames,
                                      currOutputFileName, currSamOutputFilePtr,
                                      numOfAnswer3, numOfAlignedRead3, 1,
                                      readInputForDP, readInputForNewDP, otherSoap3Result, bothUnalignedPairs );
                // printf("numOfAlignedRead3 = %u\n", numOfAlignedRead3);
                // printf("numOfUnPaired = %u\n", numOfUnPaired)read;
            }
        }
    }

    numOfAnswer = numOfAnswer0 + numOfAnswer1 + numOfAnswer2 + numOfAnswer3;
    numOfAlignedRead = numOfAlignedRead0 + numOfAlignedRead1 + numOfAlignedRead2 + numOfAlignedRead3;
}

// pair-end all-best alignment
// 3-phases [1,2,4]
void all_best_alignment ( uint * queries, uint * readLengths, uint numMismatch, uint wordPerQuery,
                          ullint maxBatchSize, uint numQueries, uint accumReadNum,
                          Soap3Index * index, uint * _bwt, uint * _revBwt, uint * _occ, uint * _revOcc,
                          IniParams ini_params, InputOptions input_options,
                          char * upkdQualities,
                          uint * unAlignedReads, uint & numOfUnPaired,
                          uint * readIDs, char * upkdQueryNames,
                          char ** currOutputFileName, samfile_t ** currSamOutputFilePtr,
                          unsigned long long & numOfAnswer, uint & numOfAlignedRead,
                          ReadInputForDPArrays * readInputForDPall,
                          ReadInputForDPArrays * readInputForNewDPall,
                          ReadInputForDPArrays * otherSoap3Resultall,
                          BothUnalignedPairsArrays * bothUnalignedPairsArrays )
{
    // for performing pair-end alignment with the-best and the-second-best output
    unsigned long long numOfAnswer1 = 0;
    uint numOfAlignedRead1 = 0;
    unsigned long long numOfAnswer2 = 0;
    uint numOfAlignedRead2 = 0;
    unsigned long long numOfAnswer3 = 0;
    uint numOfAlignedRead3 = 0;
    unsigned long long numOfAnswer4 = 0;
    uint numOfAlignedRead4 = 0;
    uint numMismatchPerform = 1;
    //********************************//
    // Phase 1: Perform 0/1 Alignment //
    //********************************//
    all_valid_alignment ( queries, readLengths, NULL, numMismatchPerform, wordPerQuery,
                          maxBatchSize, numQueries, accumReadNum,
                          index, _bwt, _revBwt, _occ, _revOcc,
                          ini_params, input_options,
                          upkdQualities,
                          unAlignedReads, numOfUnPaired,
                          readIDs, upkdQueryNames,
                          currOutputFileName,  currSamOutputFilePtr,
                          numOfAnswer1, numOfAlignedRead1, numMismatch <= 1,
                          readInputForDPall->inputArrays, readInputForNewDPall->inputArrays,
                          otherSoap3Resultall->inputArrays,
                          bothUnalignedPairsArrays->array );

    // printf("numOfAlignedRead1 = %u\n", numOfAlignedRead1);
    // printf("numOfUnPaired = %u\n", numOfUnPaired);

    if ( numMismatch > 1 && numOfAlignedRead1 < numQueries )
    {
        //****************************************//
        // Phase 2: Perform 0/1/2 alignment     //
        //****************************************//
        numMismatchPerform = 2;
        // pack the reads which are not paired
        packUnPairedReads ( queries, readIDs, readLengths, unAlignedReads,
                            wordPerQuery, numOfUnPaired, maxBatchSize );
        numQueries = numOfUnPaired;
        all_valid_alignment ( queries, readLengths, NULL, numMismatchPerform, wordPerQuery,
                              maxBatchSize, numQueries, accumReadNum,
                              index, _bwt, _revBwt, _occ, _revOcc,
                              ini_params, input_options,
                              upkdQualities,
                              unAlignedReads, numOfUnPaired,
                              readIDs, upkdQueryNames,
                              currOutputFileName, currSamOutputFilePtr,
                              numOfAnswer2, numOfAlignedRead2, numMismatch == 2,
                              readInputForDPall->inputArrays, readInputForNewDPall->inputArrays,
                              otherSoap3Resultall->inputArrays,
                              bothUnalignedPairsArrays->array );

        // printf("numOfAlignedRead2 = %u\n", numOfAlignedRead2);
        // printf("numOfUnPaired = %u\n", numOfUnPaired, 0);

        if ( numMismatch > 2 && numOfAlignedRead2 < numQueries )
        {
            //******************************************//
            // Phase 3: Perform 0/1/2/3/4 alignment     //
            //******************************************//
            numMismatchPerform = numMismatch;
            // pack the reads which are not paired
            packUnPairedReads ( queries, readIDs, readLengths, unAlignedReads,
                                wordPerQuery, numOfUnPaired, maxBatchSize );
            numQueries = numOfUnPaired;
            all_valid_alignment ( queries, readLengths, NULL, numMismatchPerform, wordPerQuery,
                                  maxBatchSize, numQueries, accumReadNum,
                                  index, _bwt, _revBwt, _occ, _revOcc,
                                  ini_params, input_options,
                                  upkdQualities,
                                  unAlignedReads, numOfUnPaired,
                                  readIDs, upkdQueryNames,
                                  currOutputFileName, currSamOutputFilePtr,
                                  numOfAnswer3, numOfAlignedRead3, 1,
                                  readInputForDPall->inputArrays, readInputForNewDPall->inputArrays,
                                  otherSoap3Resultall->inputArrays,
                                  bothUnalignedPairsArrays->array );
            // printf("numOfAlignedRead3 = %u\n", numOfAlignedRead3);
            // printf("numOfUnPaired = %u\n", numOfUnPaired);
        }
    }

    /*
    // for those reads which needs to further process to get the second-best hits
    if (numOfUnPaired > 0 && numMismatchPerform < 4) {

          numMismatchPerform++;
          uint numOfReadToProcess = numOfUnPaired;

          // repack the reads
          // no read will be removed, but
          // the reads which need to be processed in next-round by soap3 will be duplicated
          // to the front of the list. The readIDs are stored inside the array called "needProcessPair"
          // the corresponding readIDs inside "readInputForDP", "readInputForNewDP" and
          // "bothUnalignedPairs" need to be updated correspondingly.

          printf("Start repacking the reads....\n");
          printf("numOfReadToProcess = %u\n", numOfReadToProcess);
          repackUnPairedReads(&queries, &readIDs, &readLengths, unAlignedReads,
                              wordPerQuery, numOfReadToProcess, numQueries,
                              readInputForDPall, readInputForNewDPall,
                              bothUnalignedPairsArrays);

          printf("Finish repacking the reads.\n");

          all_valid_alignment(queries, readLengths, NULL, numMismatchPerform, wordPerQuery,
                 maxBatchSize, numOfReadToProcess, accumReadNum,
                 index, _bwt, _revBwt, _occ, _revOcc,
                 ini_params, input_options,
                 upkdQualities,
                 unAlignedReads, numOfUnPaired,
                 readIDs, upkdQueryNames,
                 currOutputFileName, currSamOutputFilePtr,
                 numOfAnswer4, numOfAlignedRead4, 1,
                 readInputForDPall->inputArrays, readInputForNewDPall->inputArrays,
                 otherSoap3Resultall->inputArrays,
                 bothUnalignedPairsArrays->array);

          printf("numOfAlignedRead4 = %u\n", numOfAlignedRead4);
          printf("numOfUnPaired = %u\n", numOfUnPaired);
    }
    */
    numOfAnswer = numOfAnswer1 + numOfAnswer2 + numOfAnswer3 + numOfAnswer4;
    numOfAlignedRead = numOfAlignedRead1 + numOfAlignedRead2 + numOfAlignedRead3 + numOfAlignedRead4;
}


void best_single_alignment ( uint * queries, uint * readLengths, uint * seedLengths, uint numMismatch, uint wordPerQuery,
                             ullint maxBatchSize, uint numQueries, uint accumReadNum,
                             Soap3Index * index, uint * _bwt, uint * _revBwt, uint * _occ, uint * _revOcc,
                             IniParams ini_params, InputOptions input_options,
                             char * upkdQualities,
                             uint * unAlignedReads, uint & numOfUnAligned,
                             uint * readIDs, char * upkdQueryNames,
                             char ** currOutputFileName, samfile_t ** currSamOutputFilePtr,
                             unsigned long long & numOfAnswer, uint & numOfAlignedRead,
                             ReadInputForDP ** readInputForDP,
                             BothUnalignedPairs ** bothUnalignedPairs )
{
    // for performing single-end alignment
    // random-best, unique-best
    numOfAnswer = 0;
    numOfAlignedRead = 0;

    for ( uint currNumMismatch = 0; currNumMismatch <= numMismatch; currNumMismatch++ )
    {
        unsigned long long currNumOfAnswer = 0;
        uint currNumOfAlignedRead = 0;
        all_valid_alignment ( queries, readLengths, seedLengths, currNumMismatch, wordPerQuery,
                              maxBatchSize, numQueries, accumReadNum,
                              index, _bwt, _revBwt, _occ, _revOcc,
                              ini_params, input_options,
                              upkdQualities,
                              unAlignedReads, numOfUnAligned,
                              readIDs, upkdQueryNames,
                              currOutputFileName,  currSamOutputFilePtr,
                              currNumOfAnswer, currNumOfAlignedRead, currNumMismatch == numMismatch,
                              readInputForDP, NULL, NULL, bothUnalignedPairs );
        numOfAnswer += currNumOfAnswer;
        numOfAlignedRead += currNumOfAlignedRead;

        if ( currNumMismatch < numMismatch )
        {
            // need to proceed the next round
            // pack the reads which have no hits
            packUnPairedReads ( queries, readIDs, readLengths, unAlignedReads,
                                wordPerQuery, numOfUnAligned, maxBatchSize );
            numQueries = numOfUnAligned;
        }
    }
}

void all_best_single_alignment ( uint * queries, uint * readLengths, uint * seedLengths, uint numMismatch, uint wordPerQuery,
                                 ullint maxBatchSize, uint numQueries, uint accumReadNum,
                                 Soap3Index * index, uint * _bwt, uint * _revBwt, uint * _occ, uint * _revOcc,
                                 IniParams ini_params, InputOptions input_options,
                                 char * upkdQualities,
                                 uint * unAlignedReads, uint & numOfUnAligned,
                                 uint * readIDs, char * upkdQueryNames,
                                 char ** currOutputFileName, samfile_t ** currSamOutputFilePtr,
                                 unsigned long long & numOfAnswer, uint & numOfAlignedRead,
                                 ReadInputForDP ** readInputForDP,
                                 BothUnalignedPairs ** bothUnalignedPairs )
{
    // for performing single-end alignment
    // all-best
    numOfAnswer = 0;
    numOfAlignedRead = 0;

    for ( uint currNumMismatch = 1; currNumMismatch <= ( numMismatch > 0 ? numMismatch : 1 ); currNumMismatch++ )
    {
        unsigned long long currNumOfAnswer = 0;
        uint currNumOfAlignedRead = 0;
        all_valid_alignment ( queries, readLengths, seedLengths, currNumMismatch, wordPerQuery,
                              maxBatchSize, numQueries, accumReadNum,
                              index, _bwt, _revBwt, _occ, _revOcc,
                              ini_params, input_options,
                              upkdQualities,
                              unAlignedReads, numOfUnAligned,
                              readIDs, upkdQueryNames,
                              currOutputFileName,  currSamOutputFilePtr,
                              currNumOfAnswer, currNumOfAlignedRead, currNumMismatch >= numMismatch,
                              readInputForDP, NULL, NULL, bothUnalignedPairs );
        numOfAnswer += currNumOfAnswer;
        numOfAlignedRead += currNumOfAlignedRead;

        if ( currNumMismatch < numMismatch )
        {
            // need to proceed the next round
            // pack the reads which have no hits
            packUnPairedReads ( queries, readIDs, readLengths, unAlignedReads,
                                wordPerQuery, numOfUnAligned, maxBatchSize );
            numQueries = numOfUnAligned;
        }
    }
}


// Perform single-read all-valid alignment for seeds
// This function would NOT reset the SingleAlgnResultArray
void single_all_valid_seed_alignment (
    unsigned int * queries, unsigned int * readLengths, uint numMismatch,
    unsigned int maxReadLength, unsigned int wordPerQuery, unsigned int maxBatchSize,
    unsigned int numQueries,
    Soap3Index * index,
    uint * _bwt, unsigned int * _revBwt,
    unsigned int * _occ, unsigned int * _revOcc,
    unsigned long long & numOfAnswer,
    unsigned int & numOfAlignedRead, unsigned int cpuNumThreads,
    SingleAlgnResultArray * alignResultArray, unsigned int maxHitNum,
    unsigned char outNoAlign, unsigned char * noAlignment,
    unsigned int * readIDs )
{
#ifdef BGS_GPU_CASE_BREAKDOWN_TIME
    double startTime2 = setStartTime ();
    double lastEventTime2 = 0;
    double currEventTime2;
#endif
    uint * answers[MAX_NUM_CASES];
    uint * badReadIndices[MAX_NUM_CASES]; // IDs of bad reads for each case
    uint * badAnswers[MAX_NUM_CASES]; // stores answers (and reads temporarily) for 2nd round
    uint threadBadStarts[MAX_NUM_CPU_THREADS][MAX_NUM_CASES];
    uint threadBadCounts[MAX_NUM_CPU_THREADS][MAX_NUM_CASES];
    // Host alignment model
    SRAModel * SRAMismatchModel[MAX_READ_LENGTH + 1];
    SRAModel * SRAMismatchModel_neg[MAX_READ_LENGTH + 1];
    HostKernelArguements hostKernelArguments[MAX_NUM_CPU_THREADS];
    // Host multi-threading variables
    int threadId;
    pthread_t threads[MAX_NUM_CPU_THREADS];
    uint threadBucketSize[MAX_NUM_CPU_THREADS];
    numOfAnswer = 0;
    numOfAlignedRead = 0;
    uint blocksNeeded;
    ullint batchSize;
    uint numCases;
    uint sa_range_allowed_1;
    uint sa_range_allowed_2;
    char skip_round_2;
    ullint queriesLeft = numQueries;
    uint word_per_ans;
    uint word_per_ans_2;
    int i;

    if ( numQueries == 0 )
    { return; }

    SRAIndex * sraIndex = index->sraIndex;

    // initialization of arrays
    for ( int j = 0; j < MAX_NUM_CASES; j++ )
    {
        badReadIndices[j] = NULL;
        badAnswers[j] = NULL;
        answers[j] = NULL;

        for ( int k = 0; k < MAX_NUM_CPU_THREADS; k++ )
        {
            threadBadStarts[k][j] = 0;
            threadBadCounts[k][j] = 0;
        }
    }

    // obtain the number of cases for this number of mismatch
    // and obtain the number of SA ranges allowed
    getParametersForThisMismatch2 ( numMismatch, numCases, sa_range_allowed_1,
                                    sa_range_allowed_2, skip_round_2, word_per_ans, word_per_ans_2 );
    // set the multi-threading arguments
    setHostKernelArguments2 ( hostKernelArguments, threads, index, maxReadLength, wordPerQuery, word_per_ans, cpuNumThreads );

    for ( threadId = 0; threadId < cpuNumThreads; ++threadId )
    {
        hostKernelArguments[threadId].SRAMismatchModel = SRAMismatchModel;
        hostKernelArguments[threadId].SRAMismatchModel_neg = SRAMismatchModel_neg;
        hostKernelArguments[threadId].readLengths = readLengths;
        hostKernelArguments[threadId].maxHitNum = maxHitNum;
        hostKernelArguments[threadId].readIDs = readIDs;
    }

    ullint roundUp = ( maxBatchSize + 31 ) / 32 * 32;

    // allocate host memory for answers
    for ( int j = 0; j < numCases; j++ )
    {
        answers[j] = ( uint * ) malloc ( roundUp * word_per_ans * sizeof ( uint ) );
    }

    // printParameters(input_options, ini_params);
    // printf("numMismatch = %u; numCases = %u; sa_range_allowed_1 = %u; sa_range_allowed_2 = %u; skip_round_2 = %i; word_per_ans = %u; word_per_ans_2 = %u \n\n\n\n\n", 1, numCases, sa_range_allowed_1, sa_range_allowed_2, skip_round_2, word_per_ans, word_per_ans_2);

    // update the num of mismatch of the model for host alignment
    for ( threadId = 0; threadId < cpuNumThreads; ++threadId )
    {
        hostKernelArguments[threadId].sraQuerySettings.MaxError = numMismatch;
        hostKernelArguments[threadId].sraQuerySettings.MaxNBMismatch = 0;
        hostKernelArguments[threadId].numCases = numCases;
        hostKernelArguments[threadId].sa_range_allowed_1 = sa_range_allowed_1;
        hostKernelArguments[threadId].sa_range_allowed_2 = sa_range_allowed_2;
        hostKernelArguments[threadId].maxNumMismatch = numMismatch;
        hostKernelArguments[threadId].alignResult = alignResultArray->array[threadId];
    }

    for ( i = 0; i <= MAX_READ_LENGTH; i++ )
    {
        SRAMismatchModel[i] = NULL;
        SRAMismatchModel_neg[i] = NULL;
    }

    for ( unsigned readId = 0; readId < numQueries; ++readId )
    {
        if ( SRAMismatchModel[readLengths[readId]] == NULL )
        {
            SRAMismatchModel[readLengths[readId]] = SRAModelConstruct ( readLengths[readId], QUERY_POS_STRAND, & ( hostKernelArguments[0].sraQuerySettings ), sraIndex, SRA_MODEL_16G );
        }

        if ( SRAMismatchModel_neg[readLengths[readId]] == NULL )
        {
            SRAMismatchModel_neg[readLengths[readId]] = SRAModelConstruct ( readLengths[readId], QUERY_NEG_STRAND, & ( hostKernelArguments[0].sraQuerySettings ), sraIndex, SRA_MODEL_16G );
        }
    }

    uint * nextQuery = queries;
    uint * nextReadLength = readLengths;
#ifdef BGS_GPU_CASE_BREAKDOWN_TIME
    currEventTime2 = getElapsedTime ( startTime2 );
    printf ( "[Main] Time elapsed for initialization: %9.4f seconds.\n", currEventTime2 - lastEventTime2 );
    lastEventTime2 = currEventTime2;
#endif

    while ( queriesLeft > 0 )
    {
        uint processedQuery = numQueries - queriesLeft;

        if ( queriesLeft > maxBatchSize )
        {
            blocksNeeded = NUM_BLOCKS;
            batchSize = maxBatchSize;
        }
        else
        {
            blocksNeeded = ( queriesLeft + THREADS_PER_BLOCK * QUERIES_PER_THREAD - 1 ) /
                           ( THREADS_PER_BLOCK * QUERIES_PER_THREAD );
            batchSize = queriesLeft;
        }

        // allocate the reads to different CPU threads
        uint roughBucketSize = batchSize / cpuNumThreads;
        uint batchSizeUnalloc = batchSize;

        for ( threadId = 0; threadId < cpuNumThreads - 1; threadId++ )
        {
            threadBucketSize[threadId] = roughBucketSize;
            batchSizeUnalloc -= roughBucketSize;
        }

        threadBucketSize[cpuNumThreads - 1] = batchSizeUnalloc;
        // perform first round alignment in GPU
        perform_round1_alignment_no_pipeline ( nextQuery, nextReadLength, answers,
                                               numMismatch, numCases, sa_range_allowed_1, wordPerQuery,
                                               word_per_ans, false, blocksNeeded, batchSize,
                                               index, _bwt, _revBwt, _occ, _revOcc );
#ifdef BGS_GPU_CASE_BREAKDOWN_TIME
        printf ( "[Main] # of mismatches allowed: %u with # of sa ranges: %u\n", 1, sa_range_allowed_1 );
        currEventTime2 = getElapsedTime ( startTime2 );
        printf ( "[Main] Time elapsed for first round alignment in GPU: %9.4f seconds.\n", currEventTime2 - lastEventTime2 );
        lastEventTime2 = currEventTime2;
#endif

        // identify which read has no hit
        if ( outNoAlign == 1 )
        {
            uint * ans;

            for ( ullint j = 0; j < batchSize; ++j )
            {
                ullint batchReadId = j;
                ullint readId = processedQuery + j;
                char isNoAlign = 1;

                for ( int whichCase = 0; whichCase < numCases && isNoAlign; whichCase++ )
                {
                    ans = answers[whichCase] + ( ( batchReadId ) / 32 * 32 * word_per_ans + ( batchReadId ) % 32 );

                    if ( * ( ans ) != 0xFFFFFFFD )
                    { isNoAlign = 0; }
                }

                if ( isNoAlign == 1 )
                {
                    noAlignment[readId] = 1;
                }
            }
        }

        if ( skip_round_2 == 0 )
        {
            // =======================================
            // | GPU-2                               |
            // =======================================
            // perform second round alignment in GPU
            for ( int whichCase = 0; whichCase < numCases; ++whichCase )
            {
                ullint numBads = 0;

                // Count number of bad reads and
                for ( ullint readId = 0; readId < batchSize; readId++ )
                {
                    ullint srcOffset = ( ( readId ) / 32 * 32 * word_per_ans + readId % 32 );
                    numBads += ( answers[whichCase][srcOffset] > 0xFFFFFFFD );
                }

                if ( numBads > 0 )
                {
                    // Allocate memory and copy bad reads to another array
                    ullint roundUp = ( numBads + 31 ) / 32 * 32;
                    badReadIndices[whichCase] = ( uint * ) malloc ( roundUp * sizeof ( uint ) );
                    // printf("size of badReadIndices[%i] = %u\n",whichCase, roundUp);
                    badAnswers[whichCase] = ( uint * ) malloc ( roundUp * word_per_ans_2 * sizeof ( uint ) );
                    // printf("size of badAnswers[%i] = %u\n", whichCase, roundUp * word_per_ans_2);
                }

                // printf("numBads == %u in case %i of mismatch %u\n", numBads, whichCase, numMismatch);
            }

            perform_round2_alignment_no_pipeline ( queries, readLengths, answers,
                                                   numMismatch, numCases, sa_range_allowed_2, wordPerQuery, word_per_ans,
                                                   word_per_ans_2, false, blocksNeeded, batchSize,
                                                   index, _bwt, _revBwt, _occ, _revOcc,
                                                   processedQuery, badReadIndices, badAnswers );

            for ( int whichCase = 0; whichCase < numCases; ++whichCase )
            {
                ullint readId = 0;

                for ( threadId = 0; threadId < cpuNumThreads; threadId ++ )
                {
                    uint threadNumBad = 0;

                    // printf("whichCase = %i; threadId = %i; threadBucketSize[threadId] = %u\n",
                    //        whichCase, threadId, threadBucketSize[threadId]);
                    for ( uint n = 0 ; n < threadBucketSize[threadId]; ++n )
                    {
                        ullint srcOffset = ( readId / 32 * 32 * word_per_ans + readId % 32 );
                        threadNumBad += ( answers[whichCase][srcOffset] > 0xFFFFFFFD );
                        readId++;
                    }

                    threadBadCounts[threadId][whichCase] = threadNumBad;
                    // printf("threadBadCounts[%i][%i] = %u\n",
                    //    threadId, whichCase, threadBadCounts[threadId][whichCase]);
                }
            }

            for ( int whichCase = 0; whichCase < numCases; ++whichCase )
            {
                threadBadStarts[0][whichCase] = 0;

                for ( threadId = 1; threadId < cpuNumThreads; threadId ++ )
                {
                    threadBadStarts[threadId][whichCase] = threadBadStarts[threadId - 1][whichCase] + threadBadCounts[threadId - 1][whichCase];
                    // printf("threadBadStarts[%i][%i] = %u\n",
                    //  threadId, whichCase, threadBadStarts[threadId][whichCase]);
                }
            }

#ifdef BGS_GPU_CASE_BREAKDOWN_TIME
            currEventTime2 = getElapsedTime ( startTime2 );
            printf ( "[Main] Time elapsed for second round alignment in GPU: %9.4f seconds.\n", currEventTime2 - lastEventTime2 );
            printf ( "[Main] # of sa ranges allowed: %u \n", sa_range_allowed_2 );
            lastEventTime2 = currEventTime2;
#endif
        } // skip the round 2

#ifdef BGS_GPU_CASE_BREAKDOWN_TIME
        currEventTime2 = getElapsedTime ( startTime2 );
        printf ( "[Main] Time elapsed for waiting for CPU threads: %9.4f seconds.\n", currEventTime2 - lastEventTime2 );
        lastEventTime2 = currEventTime2;
#endif
        // ================ PIPE SECTION FINAL========================
        // =======================================
        // | CPU: Thread #0,1,2,3,...            |
        // =======================================
        unsigned int threadProcessedQuery = 0;

        for ( threadId = 0; threadId < cpuNumThreads ; threadId++ )
        {
            hostKernelArguments[threadId].batchFirstReadId = processedQuery;
            hostKernelArguments[threadId].skipFirst = threadProcessedQuery;
            hostKernelArguments[threadId].numQueries = threadBucketSize[threadId];
            hostKernelArguments[threadId].queries = queries;
            hostKernelArguments[threadId].word_per_query = wordPerQuery;
            hostKernelArguments[threadId].answers = answers;
            hostKernelArguments[threadId].alignedOcc = 0;
            hostKernelArguments[threadId].alignedReads = 0;
            hostKernelArguments[threadId].badReadIndices = badReadIndices;
            hostKernelArguments[threadId].badAnswers = badAnswers;
            hostKernelArguments[threadId].badStartOffset = threadBadStarts[threadId];
            hostKernelArguments[threadId].badCountOffset = threadBadCounts[threadId];
            hostKernelArguments[threadId].skip_round_2 = skip_round_2;

            if ( pthread_create ( & ( threads[threadId] ), NULL, hostKernelThreadWrapperSingle, ( void * ) & ( hostKernelArguments[threadId] ) ) )
            { fprintf ( stderr, "[Main:Threads%u] Can't create hostKernelThreadWrapper\n", threadId ), exit ( 1 ); }

            threadProcessedQuery += threadBucketSize[threadId];
        }

        for ( threadId = 0; threadId < cpuNumThreads ; threadId++ )
        {
            if ( threads[threadId] != 0 )
            {
                if ( pthread_join ( threads[threadId], NULL ) )
                { fprintf ( stderr, "[Main:Thread%u] Crash!\n", threadId ), exit ( 1 ); }

                numOfAnswer += hostKernelArguments[threadId].alignedOcc;
                numOfAlignedRead += hostKernelArguments[threadId].alignedReads;
                threads[threadId] = 0;
            }
        }

        for ( int whichCase = 0; whichCase < numCases; ++whichCase )
        {
            if ( badReadIndices[whichCase] != NULL )
            {
                free ( badReadIndices[whichCase] );
                badReadIndices[whichCase] = NULL;
            }

            if ( badAnswers[whichCase] != NULL )
            {
                free ( badAnswers[whichCase] );
                badAnswers[whichCase] = NULL;
            }
        }

        // ================ PIPE SECTION FINAL========================
        queriesLeft -= batchSize;
        nextQuery += batchSize * wordPerQuery;
        nextReadLength += batchSize;
    }

    // CLEAN UP for each MISMATCH iteration                                                                          |
    for ( i = 0; i <= MAX_READ_LENGTH; i++ )
    {
        if ( SRAMismatchModel[i] != NULL )
        {
            SRAModelFree ( SRAMismatchModel[i] );
            SRAMismatchModel[i] = NULL;
        }

        if ( SRAMismatchModel_neg[i] != NULL )
        {
            SRAModelFree ( SRAMismatchModel_neg[i] );
            SRAMismatchModel_neg[i] = NULL;
        }
    }

    for ( threadId = 0; threadId < cpuNumThreads; ++threadId )
    {
        OCCFree ( hostKernelArguments[threadId].occ );
    }

    for ( int j = 0; j < numCases; j++ )
    {
        free ( answers[j] );
    }
}

// Perform single-read all-best 1-mismatch alignment for seeds
// This function would reset the SingleAlgnResultArray
void single_1_mismatch_alignment2 ( unsigned int * queries, unsigned int * readLengths,
                                    unsigned int maxReadLength, unsigned int wordPerQuery, unsigned int maxBatchSize,
                                    unsigned int numQueries, Soap3Index * index, uint * _bwt, unsigned int * _revBwt,
                                    unsigned int * _occ, unsigned int * _revOcc,
                                    unsigned long long & numOfAnswer,
                                    unsigned int & numOfAlignedRead, unsigned int cpuNumThreads,
                                    SingleAlgnResultArray * alignResultArray, unsigned int maxHitNum )
{
    // reset the array alignResult
    for ( unsigned int threadId = 0; threadId < cpuNumThreads; ++threadId )
    {
        resetSingleAlgnResult ( alignResultArray->array[threadId] );
    }

    // create the arrays for processing
    unsigned char * noAlignment = ( unsigned char * ) malloc ( sizeof ( unsigned char ) * numQueries );
    memset ( noAlignment, 0, numQueries );
    single_all_valid_seed_alignment (
        queries, readLengths, 0,
        maxReadLength, wordPerQuery, maxBatchSize,
        numQueries, index, _bwt, _revBwt,
        _occ, _revOcc,
        numOfAnswer,
        numOfAlignedRead, cpuNumThreads,
        alignResultArray, maxHitNum,
        1, noAlignment, NULL );
    // create the arrays for processing
    unsigned long long roundUp = ( numQueries + 31 ) / 32 * 32;
    unsigned int * queries2 = ( unsigned int * ) malloc ( sizeof ( unsigned int ) * roundUp * wordPerQuery );
    unsigned int * readLengths2 = ( unsigned int * ) malloc ( sizeof ( unsigned int ) * roundUp );
    memcpy ( queries2, queries, sizeof ( unsigned int ) * roundUp * wordPerQuery );
    memcpy ( readLengths2, readLengths, sizeof ( unsigned int ) * numQueries );
    // pack the unaligned reads
    unsigned int numUnAligned;
    unsigned int * readIDs2 = packReads2 ( queries2, readLengths2, noAlignment,
                                           wordPerQuery, numQueries, numUnAligned );
    unsigned int numOfAlignedRead2;
    unsigned long long numOfAnswer2;
    single_all_valid_seed_alignment (
        queries2, readLengths2, 1,
        maxReadLength, wordPerQuery, maxBatchSize,
        numUnAligned, index, _bwt, _revBwt,
        _occ, _revOcc,
        numOfAnswer2,
        numOfAlignedRead2, cpuNumThreads,
        alignResultArray, maxHitNum,
        0, noAlignment, readIDs2 );
    numOfAnswer += numOfAnswer2;
    numOfAlignedRead += numOfAlignedRead2;
    free ( queries2 );
    free ( readLengths2 );
    free ( readIDs2 );
    free ( noAlignment );
}


// Perform SOAP3-DP Paired-End Alignment
void soap3_dp_pair_align ( uint * queries, uint * readLengths, uint numMismatch, uint wordPerQuery,
                           ullint maxBatchSize, uint numQueries, uint accumReadNum,
                           Soap3Index * index,
                           uint * _bwt, uint * _revBwt,
                           uint * _occ, uint * _revOcc,
                           IniParams ini_params, InputOptions input_options,
                           uint maxReadLength, uint detected_read_length, uint detected_read_length2,
                           char * upkdQualities,
                           uint * unAlignedReads, uint & numOfUnPaired,
                           uint * readIDs, char * upkdQueryNames,
                           char ** currOutputFileName, samfile_t ** currSamOutputFilePtr,
                           char * outputDPFileName, samfile_t * samOutputDPFilePtr,
                           samfile_t * samOutputUnpairFilePtr,
                           unsigned long long & numOfAnswer, uint & numOfAlignedRead,
                           ReadInputForDPArrays * readInputForDPall,
                           ReadInputForDPArrays * readInputForNewDPall,
                           ReadInputForDPArrays * otherSoap3Resultall,
                           BothUnalignedPairsArrays * bothUnalignedPairsArrays,
                           double startTime, double & lastEventTime, double & totalAlignmentTime,
                           uint & indexLoadedToGPU )
{
    double alignmentTime, copyTime;
    // ======================================================================================
    // | IF THE INDEX IS NOT IN DEVICE, THEN                                                |
    // | COPY INDEX TO DEVICE MEMORY                                                        |
    // ======================================================================================
    unsigned int numDPAlignedPair = 0;
    unsigned int numDPAlignment = 0;
    DPParameters dpParameters;
    int orig_align_type;
    HSPAux * hspaux = index->sraIndex->hspaux;

    if ( ini_params.Ini_skipSOAP3Alignment == 1 )
    {
        // Add all first reads of the pairs to BothUnalignedPairs (i.e. 0, 2, 4, ..., totalReadNum-2)
        addAllFirstReadIDToBothUnalignedPairs ( bothUnalignedPairsArrays->array[0], numQueries );
        // For DP module, if SAM format and all-best are both selected, then
        // output format is needed to set to all-valid.
        orig_align_type = input_options.alignmentType;

        if ( input_options.alignmentType == OUTPUT_ALL_BEST &&
                input_options.outputFormat == SRA_OUTPUT_FORMAT_SAM_API )
        {
            input_options.alignmentType = OUTPUT_ALL_VALID;
        }

        // Parameters for DP
        getParameterForAllDP ( dpParameters, input_options, ini_params );
    }
    else
    {
        if ( indexLoadedToGPU == 0 )
        {
            GPUINDEXUpload ( index, &_bwt, &_occ,
                             &_revBwt, &_revOcc );
            copyTime = getElapsedTime ( startTime );
            printf ( "[Main] Finished copying index into device (GPU).\n" );
            printf ( "[Main] Loading time : %9.4f seconds\n\n", copyTime - lastEventTime );
            lastEventTime = copyTime;
            indexLoadedToGPU = 1;
        }

        //*******************************//
        // Perform Alignment             //
        //*******************************//

        if ( input_options.alignmentType == OUTPUT_ALL_BEST &&
                input_options.outputFormat == SRA_OUTPUT_FORMAT_SAM_API )
        {
            all_best_alignment ( queries, readLengths, input_options.numMismatch, wordPerQuery,
                                 maxBatchSize, numQueries, accumReadNum,
                                 index,
                                 _bwt, _revBwt, _occ, _revOcc,
                                 ini_params, input_options,
                                 upkdQualities,
                                 unAlignedReads, numOfUnPaired,
                                 readIDs, upkdQueryNames,
                                 currOutputFileName, currSamOutputFilePtr,
                                 numOfAnswer, numOfAlignedRead,
                                 readInputForDPall, readInputForNewDPall,
                                 otherSoap3Resultall,
                                 bothUnalignedPairsArrays );
        }
        else if ( input_options.alignmentType == OUTPUT_UNIQUE_BEST ||
                  input_options.alignmentType == OUTPUT_RANDOM_BEST ||
                  input_options.alignmentType == OUTPUT_ALL_BEST )
        {
            four_phases_alignment ( queries, readLengths, input_options.numMismatch, wordPerQuery,
                                    maxBatchSize, numQueries, accumReadNum,
                                    index,
                                    _bwt, _revBwt, _occ, _revOcc,
                                    ini_params, input_options,
                                    upkdQualities,
                                    unAlignedReads, numOfUnPaired,
                                    readIDs, upkdQueryNames,
                                    currOutputFileName, currSamOutputFilePtr,
                                    numOfAnswer, numOfAlignedRead,
                                    readInputForDPall->inputArrays, readInputForNewDPall->inputArrays,
                                    otherSoap3Resultall->inputArrays,
                                    bothUnalignedPairsArrays->array );
        }
        else
        {
            all_valid_alignment ( queries, readLengths, NULL, input_options.numMismatch, wordPerQuery,
                                  maxBatchSize, numQueries, accumReadNum,
                                  index,
                                  _bwt, _revBwt, _occ, _revOcc,
                                  ini_params, input_options,
                                  upkdQualities,
                                  unAlignedReads, numOfUnPaired,
                                  readIDs, upkdQueryNames,
                                  currOutputFileName, currSamOutputFilePtr,
                                  numOfAnswer, numOfAlignedRead, 1,
                                  readInputForDPall->inputArrays, readInputForNewDPall->inputArrays,
                                  otherSoap3Resultall->inputArrays,
                                  bothUnalignedPairsArrays->array );
        }

        printf ( "[Main] Finished alignment with <= %i mismatches\n", input_options.numMismatch );
        // printf("[Main] Number of pairs aligned: %u (number of alignments: %llu)\n", numOfAlignedRead/2, numOfAnswer);
        printf ( "[Main] Number of pairs aligned: %u\n", numOfAlignedRead / 2 );
        alignmentTime = getElapsedTime ( startTime );
        printf ( "[Main] Elapsed time : %9.4f seconds\n\n", alignmentTime - lastEventTime );
        totalAlignmentTime += alignmentTime - lastEventTime;
        lastEventTime = alignmentTime;
        // For DP module, if SAM format and all-best are both selected, then
        // output format is needed to set to all-valid.
        orig_align_type = input_options.alignmentType;

        if ( input_options.alignmentType == OUTPUT_ALL_BEST &&
                input_options.outputFormat == SRA_OUTPUT_FORMAT_SAM_API )
        {
            input_options.alignmentType = OUTPUT_ALL_VALID;
        }

        numDPAlignedPair = 0;
        numDPAlignment = 0;
        // Parameters for DP
        getParameterForAllDP ( dpParameters, input_options, ini_params );
        
        numDPAlignedPair = 0;
        numDPAlignment = 0;

        ////////////////////////////////////////////////////////////
        // PERFORM NEW SEMI-GLOBAL DP IF NECESSARY                    //
        ////////////////////////////////////////////////////////////

#ifdef PERFORM_NEW_DEFAULT_DP_FOR_SEMI_ALIGNED_PAIR
        if ( input_options.enableDP == 1 && input_options.readType == PAIR_END_READ )
        {
            // DP Parameters for half-aligned reads
            getParameterForNewDefaultDP ( dpParameters, input_options, ini_params, detected_read_length, detected_read_length2 );
            printDPParameters ( dpParameters );
            /*
            if (indexLoadedToGPU == 1) {
                  // free device memory
                  // printf("[Main] Free index from device memory..\n");
                  hipFree(_bwt);
                  hipFree(_occ);
                  hipFree(_revBwt);
                  hipFree(_revOcc);


                  indexLoadedToGPU=0;
            }
            */
#ifdef BGS_OUTPUT_DP_MESSAGE
            printf ( "*********************************************************\n" );
            printf ( "NEW SEMI-GLOBAL MESSAGE:\n" );
#endif
            unsigned int totalReadsProceedToDP = 0;
            unsigned int totalSARanges = 0;
            unsigned int totalOccs = 0;
            unsigned int totalHits = 0;

            for ( int threadId = 0; threadId < ini_params.Ini_NumOfCpuThreads; ++threadId )
            {
                totalReadsProceedToDP += readInputForNewDPall->inputArrays[threadId]->readNum;
                totalSARanges += readInputForNewDPall->inputArrays[threadId]->saRangeTotalNum;
                totalOccs += readInputForNewDPall->inputArrays[threadId]->occTotalNum;

                for ( int h = 0; h < readInputForNewDPall->inputArrays[threadId]->saRangeTotalNum; h++ )
                { totalHits += readInputForNewDPall->inputArrays[threadId]->sa_list[h].saIndexRight - readInputForNewDPall->inputArrays[threadId]->sa_list[h].saIndexLeft + 1; }

                totalHits += readInputForNewDPall->inputArrays[threadId]->occTotalNum;
            }

            if ( totalReadsProceedToDP > 0 )
            {
                printf ( "[Main] %u half-aligned pairs of reads are proceeded to new default DP.\n", totalReadsProceedToDP );
#ifdef BGS_OUTPUT_HALF_ALIGNED_DP_MESSAGE
                printf ( "%u half-aligned pairs of reads (with %u SA Ranges and \n", totalReadsProceedToDP, totalSARanges );
                printf ( "%u occurrences) are proceeded to new default DP.\n", totalOccs );
                printf ( "total %u number of hits) are proceeded to new default DP.\n", totalHits );
#endif
                // prepare the file pointer for output
                FILE * outputDPFile = NULL;

                //Output file for SAM output is handled by SAM API
                //=> OutFilePtr is not used for SAM API output format.
                switch ( input_options.outputFormat )
                {
                    case SRA_OUTPUT_FORMAT_SAM_API:
                        break;

                    default:
                        outputDPFile = ( FILE * ) fopen ( outputDPFileName, "a" );
                }

                newSemiGlobalDP ( readInputForNewDPall, input_options.insert_high, input_options.insert_low,
                                  queries, readLengths, readIDs, upkdQueryNames, upkdQualities,
                                  maxReadLength, ini_params.Ini_PEStrandLeftLeg, ini_params.Ini_PEStrandRightLeg,
                                  index,
                                  _bwt,  _revBwt,
                                  _occ,  _revOcc,
                                  input_options.alignmentType, &dpParameters,
                                  numDPAlignedPair, numDPAlignment,
                                  accumReadNum, input_options.outputFormat,
                                  outputDPFile, samOutputDPFilePtr, bothUnalignedPairsArrays->array[ini_params.Ini_NumOfCpuThreads] );
                // the index is released in the function "newSemiGlobalDP"
                indexLoadedToGPU = 0;

                switch ( input_options.outputFormat )
                {
                    case SRA_OUTPUT_FORMAT_SAM_API:
                        break;

                    default:
                        fclose ( outputDPFile );
                }

                // printf("Finished semi-global DP\n");
                // printf("[Main] Number of pairs aligned by new default DP: %u (number of alignments: %u)\n", numDPAlignedPair, numDPAlignment);
                printf ( "[Main] Number of pairs aligned by new default DP: %u\n", numDPAlignedPair );
                alignmentTime = getElapsedTime ( startTime );
                printf ( "[Main] Elapsed time : %9.4f seconds\n", alignmentTime - lastEventTime );
                totalAlignmentTime += alignmentTime - lastEventTime;
                lastEventTime = alignmentTime;
                numOfAlignedRead +=  numDPAlignedPair * 2;
                numOfAnswer += numDPAlignment;
                // printf("[Main] Total Number of pairs aligned: %u (number of alignments: %llu)\n", numOfAlignedRead/2, numOfAnswer);
                printf ( "[Main] Total Number of pairs aligned: %u\n", numOfAlignedRead / 2 );
#ifdef BGS_OUTPUT_DP_MESSAGE
                printf ( "*********************************************************\n" );
#endif
                printf ( "\n" );
            }
        }
#endif

        numDPAlignedPair = 0;
        numDPAlignment = 0;
    }

    ////////////////////////////////////////////////////////////
    // PERFORM SEMI-GLOBAL DP IF NECESSARY                    //
    ////////////////////////////////////////////////////////////

#ifdef PERFORM_DEFAULT_DP_FOR_SEMI_ALIGNED_PAIR
    if ( input_options.enableDP == 1 && input_options.readType == PAIR_END_READ )
    {
        // DP Parameters for half-aligned reads
        getParameterForDefaultDP ( dpParameters, input_options, ini_params, detected_read_length, detected_read_length2 );
        printDPParameters ( dpParameters );

        if ( indexLoadedToGPU == 1 )
        {
            // free device memory
            // printf("[Main] Free index from device memory..\n");
            hipFree ( _bwt );
            hipFree ( _occ );
            hipFree ( _revBwt );
            hipFree ( _revOcc );
            indexLoadedToGPU = 0;
        }

#ifdef BGS_OUTPUT_DP_MESSAGE
        printf ( "*********************************************************\n" );
        printf ( "SEMI-GLOBAL MESSAGE:\n" );
#endif
        unsigned int totalReadsProceedToDP = 0;
        unsigned int totalSARanges = 0;
        unsigned int totalOccs = 0;

        for ( int threadId = 0; threadId < ini_params.Ini_NumOfCpuThreads; ++threadId )
        {
            totalReadsProceedToDP += readInputForDPall->inputArrays[threadId]->readNum;
            totalSARanges += readInputForDPall->inputArrays[threadId]->saRangeTotalNum;
            totalOccs += readInputForDPall->inputArrays[threadId]->occTotalNum;
        }

        if ( totalReadsProceedToDP > 0 )
        {
            printf ( "[Main] %u half-aligned pairs of reads are proceeded to DP.\n", totalReadsProceedToDP );
#ifdef BGS_OUTPUT_HALF_ALIGNED_DP_MESSAGE
            printf ( "%u half-aligned pairs of reads (with %u SA Ranges and \n", totalReadsProceedToDP, totalSARanges );
            printf ( "%u occurrences) are proceeded to DP.\n", totalOccs );
#endif
            // prepare the file pointer for output
            FILE * outputDPFile = NULL;

            //Output file for SAM output is handled by SAM API
            //=> OutFilePtr is not used for SAM API output format.
            switch ( input_options.outputFormat )
            {
                case SRA_OUTPUT_FORMAT_SAM_API:
                    break;

                default:
                    outputDPFile = ( FILE * ) fopen ( outputDPFileName, "a" );
            }

            semiGlobalDP2 ( readInputForDPall, input_options.insert_high, input_options.insert_low,
                            queries, readLengths, readIDs, upkdQueryNames, upkdQualities,
                            maxReadLength, ini_params.Ini_PEStrandLeftLeg, ini_params.Ini_PEStrandRightLeg,
                            index,
                            _bwt,  _revBwt,
                            _occ,  _revOcc,
                            input_options.alignmentType, &dpParameters,
                            numDPAlignedPair, numDPAlignment,
                            accumReadNum, input_options.outputFormat,
                            outputDPFile, samOutputDPFilePtr, bothUnalignedPairsArrays->array[ini_params.Ini_NumOfCpuThreads] );

            switch ( input_options.outputFormat )
            {
                case SRA_OUTPUT_FORMAT_SAM_API:
                    break;

                default:
                    fclose ( outputDPFile );
            }

            // printf("Finished semi-global DP\n");
            // printf("[Main] Number of pairs aligned by DP: %u (number of alignments: %u)\n", numDPAlignedPair, numDPAlignment);
            printf ( "[Main] Number of pairs aligned by DP: %u\n", numDPAlignedPair );
            // printf(" Number of reads passed to deep-dp: %u\n", bothUnalignedPairsArrays->array[ini_params.Ini_NumOfCpuThreads]->totalNum);
            alignmentTime = getElapsedTime ( startTime );
            printf ( "[Main] Elapsed time : %9.4f seconds\n", alignmentTime - lastEventTime );
            totalAlignmentTime += alignmentTime - lastEventTime;
            lastEventTime = alignmentTime;
            numOfAlignedRead +=  numDPAlignedPair * 2;
            numOfAnswer += numDPAlignment;
            // printf("[Main] Total Number of pairs aligned: %u (number of alignments: %llu)\n", numOfAlignedRead/2, numOfAnswer);
            printf ( "[Main] Total Number of pairs aligned: %u\n", numOfAlignedRead / 2 );
#ifdef BGS_OUTPUT_DP_MESSAGE
            printf ( "*********************************************************\n" );
#endif
            printf ( "\n" );
        }
    }

#endif
    ////////////////////////////////////////////////////////////
    // PERFORM DP FOR BOTH ENDS UNALIGNED                     //
    ////////////////////////////////////////////////////////////
    numDPAlignedPair = 0;
    numDPAlignment = 0;

#ifdef PERFORM_DEEP_DP_FOR_UNALIGN_PAIR
    if ( input_options.enableDP == 1 && input_options.readType == PAIR_END_READ )
    {
        // DP Parameters for Deep DP
        getParameterForDeepDP ( dpParameters, input_options, ini_params, detected_read_length, detected_read_length2 );
        printDPParameters ( dpParameters );

        // ======================================================================================
        // | IF THE INDEX IS NOT IN DEVICE, THEN                                                |
        // | COPY INDEX TO DEVICE MEMORY                                                        |
        // ======================================================================================

        if ( indexLoadedToGPU == 0 )
        {
            GPUINDEXUpload ( index, &_bwt, &_occ,
                             &_revBwt, &_revOcc );
            copyTime = getElapsedTime ( startTime );
            printf ( "[Main] Finished copying index into device (GPU).\n" );
            printf ( "[Main] Loading time : %9.4f seconds\n\n", copyTime - lastEventTime );
            lastEventTime = copyTime;
            indexLoadedToGPU = 1;
        }

#ifdef BGS_OUTPUT_DP_MESSAGE
        printf ( "*********************************************************\n" );
        printf ( "DP FOR BOTH UNALIGNED READS\n" );
#endif
        unsigned int totalReadsProceedToDP = 0;

        for ( int threadId = 0; threadId <= ini_params.Ini_NumOfCpuThreads; ++threadId )
        {
            totalReadsProceedToDP += bothUnalignedPairsArrays->array[threadId]->totalNum;
        }

        if ( totalReadsProceedToDP > 0 )
        {
            printf ( "[Main] %u pairs of reads are proceeded to deep DP.\n", totalReadsProceedToDP );
            // print out the read IDs
            // printAllReadIDs(bothUnalignedPairsArrays, accumReadNum);
            // prepare the file pointer for output
            FILE * outputDPFile = NULL;

            //Output file for SAM output is handled by SAM API
            //=> OutFilePtr is not used for SAM API output format.
            switch ( input_options.outputFormat )
            {
                case SRA_OUTPUT_FORMAT_SAM_API:
                    break;

                default:
                    outputDPFile = ( FILE * ) fopen ( outputDPFileName, "a" );
            }

            DPForUnalignPairs2 ( bothUnalignedPairsArrays, input_options.insert_high, input_options.insert_low,
                                 queries, readLengths, readIDs, upkdQueryNames, upkdQualities,
                                 maxReadLength, ini_params.Ini_PEStrandLeftLeg, ini_params.Ini_PEStrandRightLeg,
                                 index,
                                 _bwt, _revBwt,
                                 _occ, _revOcc,
                                 input_options.alignmentType, &dpParameters,
                                 numDPAlignedPair, numDPAlignment,
                                 accumReadNum, input_options.outputFormat,
                                 outputDPFile, samOutputDPFilePtr );
            indexLoadedToGPU = 0;

            switch ( input_options.outputFormat )
            {
                case SRA_OUTPUT_FORMAT_SAM_API:
                    break;

                default:
                    fclose ( outputDPFile );
            }

            // printf("Finished DP for both-end unaligned reads\n");
            // printf("[Main] Number of pairs aligned by DP: %u (number of alignments: %u)\n", numDPAlignedPair, numDPAlignment);
            printf ( "[Main] Number of pairs aligned by DP: %u\n", numDPAlignedPair );
            alignmentTime = getElapsedTime ( startTime );
            printf ( "[Main] Elapsed time : %9.4f seconds\n", alignmentTime - lastEventTime );
            totalAlignmentTime += alignmentTime - lastEventTime;
            lastEventTime = alignmentTime;
            numOfAlignedRead +=  numDPAlignedPair * 2;
            numOfAnswer += numDPAlignment;
            // printf("[Main] Total Number of pairs aligned: %u (number of alignments: %llu)\n", numOfAlignedRead/2, numOfAnswer);
            printf ( "[Main] Total Number of pairs aligned: %u\n", numOfAlignedRead / 2 );
#ifdef BGS_OUTPUT_DP_MESSAGE
            printf ( "*********************************************************\n" );
#endif
            printf ( "\n" );
        }
    }
#endif

    ////////////////////////////////////////////////////////////
    // TO PERFORM SINGLE DEEP-DP FOR UNALIGNED READS          //
    ////////////////////////////////////////////////////////////
#ifdef PERFORM_DEEP_DP_FOR_UNALIGN_READS
    unsigned int numSingleDPAligned = 0;
    unsigned int numSingleDPAlignment = 0;

    if ( input_options.enableDP == 1 && input_options.readType == PAIR_END_READ && input_options.outputFormat == SRA_OUTPUT_FORMAT_SAM_API )
    {
        // reset the readType to single read
        input_options.readType = SINGLE_READ;
        // DP Parameters for single-end alignment
        getParameterForSingleDP ( dpParameters, input_options, ini_params, detected_read_length );
        hspaux->singleDPcutoffThreshold = dpParameters.paramRead[0].cutoffThreshold;
        // printDPParameters(dpParameters);
#ifdef BGS_OUTPUT_DP_MESSAGE
        printf ( "*********************************************************\n" );
        printf ( "DP FOR UNALIGNED SINGLE READS\n" );
#endif
        unsigned int totalReadsProceedToDP = 0;
        UnalignedSinglesArrays * unalignedSingleEndArrays = ( UnalignedSinglesArrays * ) ( hspaux->readsIDForSingleDP );
        totalReadsProceedToDP += unalignedSingleEndArrays->array[0]->totalNum;
        printf ( "[Main] %u unaligned reads are proceeded to DP.\n", totalReadsProceedToDP );

        if ( totalReadsProceedToDP > 0 )
        {
            // ======================================================================================
            // | IF THE INDEX IS NOT IN DEVICE, THEN                                                |
            // | COPY INDEX TO DEVICE MEMORY                                                        |
            // ======================================================================================
            if ( indexLoadedToGPU == 0 )
            {
                GPUINDEXUpload ( index, &_bwt, &_occ,
                                 &_revBwt, &_revOcc );
                copyTime = getElapsedTime ( startTime );
                printf ( "[Main] Finished copying index into device (GPU).\n" );
                printf ( "[Main] Loading time : %9.4f seconds\n\n", copyTime - lastEventTime );
                lastEventTime = copyTime;
                indexLoadedToGPU = 1;
            }

            DPForUnalignSingle2 ( unalignedSingleEndArrays,
                                  queries, readLengths, readIDs, upkdQueryNames, upkdQualities,
                                  maxReadLength,
                                  index,
                                  _bwt, _revBwt,
                                  _occ, _revOcc,
                                  input_options.alignmentType, &dpParameters,
                                  numSingleDPAligned, numSingleDPAlignment,
                                  accumReadNum, input_options.outputFormat,
                                  NULL, samOutputUnpairFilePtr );
            indexLoadedToGPU = 0;
            // sort the alignment results
            sortReadPtrs ( ( AllHits * ) hspaux->allHits );
            // output the alignment results
            outputSingleResultForPairEnds ( ( AllHits * ) hspaux->allHits,
                                            queries, readLengths, readIDs, upkdQueryNames, upkdQualities,
                                            maxReadLength, accumReadNum, input_options.outputFormat,
                                            samOutputUnpairFilePtr, index );
        }

        // printf("Finished DP for single unaligned reads\n");
        // printf("[Main] Number of reads aligned by DP: %u (number of alignments: %u)\n", numDPAlignedSingle, numDPAlignment);
        printf ( "[Main] Number of reads aligned by single-end DP: %u\n", numSingleDPAligned );
        alignmentTime = getElapsedTime ( startTime );
        printf ( "[Main] Elapsed time : %9.4f seconds\n", alignmentTime - lastEventTime );
        totalAlignmentTime += alignmentTime - lastEventTime;
        lastEventTime = alignmentTime;
        // numOfAlignedRead +=  numDPAlignedSingle;
        // numOfAnswer += numDPAlignment;
        // printf("[Main] Total Number of reads aligned: %u (number of alignments: %llu)\n", numOfAlignedRead, numOfAnswer);
        // printf("[Main] Total Number of reads aligned: %u\n", numOfAlignedRead);
#ifdef BGS_OUTPUT_DP_MESSAGE
        printf ( "*********************************************************\n" );
#endif
        printf ( "\n" );
        // reset the unaligned-singles array
        unalignedSingleEndArrays->array[0]->totalNum = 0;
        input_options.readType = PAIR_END_READ;
    }

#endif
    input_options.alignmentType = orig_align_type;
}

// Perform SOAP3-DP Single Alignment
void soap3_dp_single_align ( uint * queries, uint * readLengths, uint numMismatch, uint wordPerQuery,
                             ullint maxBatchSize, uint numQueries, uint accumReadNum,
                             Soap3Index * index,
                             uint * _bwt, uint * _revBwt,
                             uint * _occ, uint * _revOcc,
                             IniParams ini_params, InputOptions input_options,
                             uint maxReadLength, uint detected_read_length,
                             char * upkdQualities,
                             uint * unAlignedReads, uint & numOfUnPaired,
                             uint * readIDs, char * upkdQueryNames,
                             char ** currOutputFileName, samfile_t ** currSamOutputFilePtr,
                             char * outputDPFileName, samfile_t * samOutputDPFilePtr,
                             unsigned long long & numOfAnswer, uint & numOfAlignedRead,
                             ReadInputForDPArrays * readInputForDPall,
                             UnalignedSinglesArrays * unalignedSinglesArrays,
                             double startTime, double & lastEventTime, double & totalAlignmentTime,
                             uint & indexLoadedToGPU )
{
    double alignmentTime, copyTime;
    HSPAux * hspaux = index->sraIndex->hspaux;

    // ======================================================================================
    // | IF THE INDEX IS NOT IN DEVICE, THEN                                                |
    // | COPY INDEX TO DEVICE MEMORY                                                        |
    // ======================================================================================

    if ( indexLoadedToGPU == 0 )
    {
        GPUINDEXUpload ( index, &_bwt, &_occ,
                         &_revBwt, &_revOcc );
        copyTime = getElapsedTime ( startTime );
        printf ( "[Main] Finished copying index into device (GPU).\n" );
        printf ( "[Main] Loading time : %9.4f seconds\n\n", copyTime - lastEventTime );
        lastEventTime = copyTime;
        indexLoadedToGPU = 1;
    }

    // ======================================================================================
    // | FOR LONG READS (with length > 120), only align the first 100 bases
    // ======================================================================================
    uint * seedLengths = NULL;

    if ( maxReadLength > LONG_READ_LEN )
    {
        seedLengths = ( uint * ) malloc ( numQueries * sizeof ( uint ) );
        uint i;

        for ( i = 0; i < numQueries; i++ )
        {
            if ( readLengths[i] > LONG_READ_LEN )
            {
                seedLengths[i] = SOAP3_SEED_LEN;
            }
            else
            {
                seedLengths[i] = readLengths[i];
            }
        }
    }

    //*******************************//
    // Perform Alignment             //
    //*******************************//

    if ( input_options.alignmentType == OUTPUT_ALL_BEST )
    {
        all_best_single_alignment ( queries, readLengths, seedLengths, input_options.numMismatch, wordPerQuery,
                                    maxBatchSize, numQueries, accumReadNum,
                                    index,
                                    _bwt, _revBwt, _occ, _revOcc,
                                    ini_params, input_options,
                                    upkdQualities,
                                    unAlignedReads, numOfUnPaired,
                                    readIDs, upkdQueryNames,
                                    currOutputFileName, currSamOutputFilePtr,
                                    numOfAnswer, numOfAlignedRead, readInputForDPall->inputArrays,
                                    unalignedSinglesArrays->array );
    }
    else if ( input_options.alignmentType == OUTPUT_UNIQUE_BEST ||
              input_options.alignmentType == OUTPUT_RANDOM_BEST )
    {
        best_single_alignment ( queries, readLengths, seedLengths, input_options.numMismatch, wordPerQuery,
                                maxBatchSize, numQueries, accumReadNum,
                                index,
                                _bwt, _revBwt, _occ, _revOcc,
                                ini_params, input_options,
                                upkdQualities,
                                unAlignedReads, numOfUnPaired,
                                readIDs, upkdQueryNames,
                                currOutputFileName, currSamOutputFilePtr,
                                numOfAnswer, numOfAlignedRead, readInputForDPall->inputArrays,
                                unalignedSinglesArrays->array );
    }
    else
    {
        all_valid_alignment ( queries, readLengths, seedLengths, input_options.numMismatch, wordPerQuery,
                              maxBatchSize, numQueries, accumReadNum,
                              index,
                              _bwt, _revBwt, _occ, _revOcc,
                              ini_params, input_options,
                              upkdQualities,
                              unAlignedReads, numOfUnPaired,
                              readIDs, upkdQueryNames,
                              currOutputFileName, currSamOutputFilePtr,
                              numOfAnswer, numOfAlignedRead, 1, readInputForDPall->inputArrays,
                              readInputForDPall->inputArrays, readInputForDPall->inputArrays,
                              unalignedSinglesArrays->array );
    }

    printf ( "[Main] Finished alignment with <= %i mismatches\n", input_options.numMismatch );
    // printf("[Main] Number of reads aligned: %u (number of alignments: %llu)\n", numOfAlignedRead, numOfAnswer);
    printf ( "[Main] Number of reads aligned: %u\n", numOfAlignedRead );
    alignmentTime = getElapsedTime ( startTime );
    printf ( "[Main] Elapsed time : %9.4f seconds\n\n", alignmentTime - lastEventTime );
    totalAlignmentTime += alignmentTime - lastEventTime;
    lastEventTime = alignmentTime;

    // ======================================================================================
    // | FOR LONG READS (with length > 120), release the seedLengths
    // ======================================================================================

    if ( maxReadLength > LONG_READ_LEN )
    {
        delete ( seedLengths );
    }

    /////////////////////////////////////////////////
    // FOR SINGLE READS TO PERFORM DP IF NECESSARY //
    /////////////////////////////////////////////////
    // For DP module, if SAM format and all-best are both selected, then
    // output format is needed to set to all-valid.
    int orig_align_type = input_options.alignmentType;

    if ( input_options.alignmentType == OUTPUT_ALL_BEST &&
            input_options.outputFormat == SRA_OUTPUT_FORMAT_SAM_API )
    {
        input_options.alignmentType = OUTPUT_ALL_VALID;
    }

    unsigned int numDPAlignedSingle = 0;
    unsigned int numDPAlignment = 0;
    // Parameters for DP
    DPParameters dpParameters;
    getParameterForAllDP ( dpParameters, input_options, ini_params );

    if ( input_options.enableDP == 1 )
    {
        // DP Parameters for single-end alignment
        getParameterForSingleDP ( dpParameters, input_options, ini_params, detected_read_length );
        hspaux->singleDPcutoffThreshold = dpParameters.paramRead[0].cutoffThreshold;
        // printDPParameters(dpParameters);
#ifdef BGS_OUTPUT_DP_MESSAGE
        printf ( "*********************************************************\n" );
        printf ( "DP FOR UNALIGNED SINGLE READS\n" );
#endif
        unsigned int totalReadsProceedToDP = 0;

        for ( int threadId = 0; threadId < ini_params.Ini_NumOfCpuThreads; ++threadId )
        {
            totalReadsProceedToDP += unalignedSinglesArrays->array[threadId]->totalNum;
        }

        printf ( "[Main] %u unaligned reads are proceeded to DP.\n", totalReadsProceedToDP );

        // print out the read IDs
        // printAllReadIDs(unalignedSinglesArrays, accumReadNum);

        if ( totalReadsProceedToDP > 0 )
        {
            // ======================================================================================
            // | IF THE INDEX IS NOT IN DEVICE, THEN                                                |
            // | COPY INDEX TO DEVICE MEMORY                                                        |
            // ======================================================================================
            if ( indexLoadedToGPU == 0 )
            {
                GPUINDEXUpload ( index, &_bwt, &_occ,
                                 &_revBwt, &_revOcc );
                copyTime = getElapsedTime ( startTime );
                printf ( "[Main] Finished copying index into device (GPU).\n" );
                printf ( "[Main] Loading time : %9.4f seconds\n\n", copyTime - lastEventTime );
                lastEventTime = copyTime;
                indexLoadedToGPU = 1;
            }

            // prepare the file pointer for output
            FILE * outputDPFile = NULL;

            //Output file for SAM output is handled by SAM API
            //=> OutFilePtr is not used for SAM API output format.
            if ( outputDPFileName != NULL )
            {
                switch ( input_options.outputFormat )
                {
                    case SRA_OUTPUT_FORMAT_SAM_API:
                        break;

                    default:
                        outputDPFile = ( FILE * ) fopen ( outputDPFileName, "a" );
                }
            }

            // printAllReadIDs(unalignedSinglesArrays, accumReadNum, readIDs);
            DPForUnalignSingle2 ( unalignedSinglesArrays,
                                  queries, readLengths, readIDs, upkdQueryNames, upkdQualities,
                                  maxReadLength,
                                  index,
                                  _bwt, _revBwt,
                                  _occ, _revOcc,
                                  input_options.alignmentType, &dpParameters,
                                  numDPAlignedSingle, numDPAlignment,
                                  accumReadNum, input_options.outputFormat,
                                  outputDPFile, samOutputDPFilePtr );
            indexLoadedToGPU = 0;

            if ( outputDPFileName != NULL )
            {
                switch ( input_options.outputFormat )
                {
                    case SRA_OUTPUT_FORMAT_SAM_API:
                        break;

                    default:
                        fclose ( outputDPFile );
                }
            }
        }

        // printf("Finished DP for single unaligned reads\n");
        // printf("[Main] Number of reads aligned by DP: %u (number of alignments: %u)\n", numDPAlignedSingle, numDPAlignment);
        printf ( "[Main] Number of reads aligned by DP: %u\n", numDPAlignedSingle );
        alignmentTime = getElapsedTime ( startTime );
        printf ( "[Main] Elapsed time : %9.4f seconds\n", alignmentTime - lastEventTime );
        totalAlignmentTime += alignmentTime - lastEventTime;
        lastEventTime = alignmentTime;
        numOfAlignedRead +=  numDPAlignedSingle;
        numOfAnswer += numDPAlignment;
        //printf("[Main] Total Number of reads aligned: %u (number of alignments: %llu)\n", numOfAlignedRead, numOfAnswer);
        printf ( "[Main] Total Number of reads aligned: %u\n", numOfAlignedRead );
#ifdef BGS_OUTPUT_DP_MESSAGE
        printf ( "*********************************************************\n" );
#endif
        printf ( "\n" );
    }

    input_options.alignmentType = orig_align_type;
}
