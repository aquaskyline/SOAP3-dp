#include "hip/hip_runtime.h"
/*
 *  soap3-dp-module.cpp
 *  soap3_ext_r5
 *
 *  Created by kfwong on 14/3/12.
 *  Copyright 2012 HKU. All rights reserved.
 *
 */


#include "soap3-dp-module.h"

// set Parameters
void setParam ( SingleAlignParam * param, IniParams & ini_params, InputOptions & input_options )
{
    ini_params.Ini_NumOfCpuThreads = param->cpuNumThreads; // number of CPU threads
    // ini_params.Ini_HostAlignmentModelStr = "16G";
    ini_params.Ini_HostAlignmentModel = 1; // 16G
    ini_params.Ini_GPUMemory = 6; // 6G
    ini_params.Ini_PEStrandLeftLeg = 1; // pos strand
    ini_params.Ini_PEStrandRightLeg = 2; // neg strand
    ini_params.Ini_MaxOutputPerRead = param->maxHitNum;
    // if there are too many hits, then only outputs the first "maxHitNum" hits
    ini_params.Ini_PEMaxOutputPerPair = 1000; // not applicable

    ini_params.Ini_shareIndex = 0; // index to be shared among multiple copies of soap3-dp
    ini_params.Ini_maxReadNameLen = 64; // max length allowed for read name
    // max length from the front of the read for clipping
    ini_params.Ini_maxFrontLenClipped = 3;
    // max length from the end of the read for clipping
    ini_params.Ini_maxEndLenClipped = 8;
    // whether the seed will proceed to perform DP if there are too many hits
    ini_params.Ini_proceedDPForTooManyHits = 0;
    // whether the read will perform SOAP3 module
    ini_params.Ini_skipSOAP3Alignment = 0;

    if ( param->enableDP == 1 )
    {
        ini_params.Ini_MatchScore = param->scoring.matchScore;
        ini_params.Ini_MismatchScore = param->scoring.mismatchScore;
        ini_params.Ini_GapOpenScore = param->scoring.openGapScore;
        ini_params.Ini_GapExtendScore = param->scoring.extendGapScore;
        ini_params.Ini_DPScoreThreshold = param->scoring.cutoffThreshold;
        ini_params.Ini_isDefaultThreshold = 0;
    }

    input_options.maxReadLength = param->maxReadLength;
    input_options.outputFormat = 1; // plain format (fixed)
    input_options.enableDP = param->enableDP;
    input_options.numMismatch = param->numMismatch;
    input_options.alignmentType = param->outputOption;
    input_options.insert_low = -1; // not applicable
    input_options.insert_high = -1; // not applicable
    input_options.readType = 1; // single-end
    input_options.isReadList = 0; // not applicable
}


// To perform the single alignment
// The resulting alignments are stored inside "algnResultArrays"
// "algnResultArrays" has to be constructed before calling this function
void alignSingleR ( unsigned int * queries, unsigned int * readLengths, unsigned int * readIDs,
                    unsigned int wordPerQuery,
                    unsigned int numQueries, Soap3Index * index,
                    SingleAlignParam * param,
                    unsigned long long & numOfAnswer,
                    unsigned int & numOfAlignedRead,
                    AlgnResultArrays * algnResultArrays )
{
    uint * _bwt, *_occ;
    uint * _revBwt, *_revOcc;
    double startTime, copyTime;
    double lastEventTime;
    double totalAlignmentTime = 0.0;
    //Start measuring runtime..
    startTime = setStartTime ();
    lastEventTime = 0;
    // Indicate whether the index has been loaded to GPU
    uint indexLoadedToGPU = 0;

    if ( indexLoadedToGPU == 0 )
    {
        GPUINDEXUpload ( index, &_bwt, &_occ,
                         &_revBwt, &_revOcc );
        copyTime = getElapsedTime ( startTime );
        printf ( "[Main] Finished copying index into device (GPU).\n" );
        printf ( "[Main] Loading time : %9.4f seconds\n\n", copyTime - lastEventTime );
        lastEventTime = copyTime;
        indexLoadedToGPU = 1;
    }

    // ======================================================================================
    // | ALLOCATE MEMORY FOR THE ARRAYS                                                     |
    // ======================================================================================
    ullint roundUp = ( numQueries + 31 ) / 32 * 32;
    uint maxReadLength = param->maxReadLength;
    char * upkdQualities = ( char * ) malloc ( roundUp * maxReadLength * sizeof ( char ) ); // may not need
    memset ( upkdQualities, 0, roundUp * maxReadLength ); // may not need
    unsigned int * unAlignedPair = ( unsigned int * ) malloc ( roundUp * sizeof ( unsigned int ) );
    char * upkdQueryNames = ( char * ) malloc ( roundUp * MAX_READ_NAME_LENGTH * sizeof ( char ) );
    uint maxBatchSize = NUM_BLOCKS * THREADS_PER_BLOCK * QUERIES_PER_THREAD; // queries processed in one kernel call
    // maxBatchSize has to be divible by 2
    maxBatchSize = maxBatchSize / 2 * 2;
    // set parameters
    InputOptions input_options;
    IniParams ini_params;
    setParam ( param, ini_params, input_options );
    index->sraIndex->hspaux->algnResultArrays = algnResultArrays;
    // Declare the structure for storing the alignment results
    // for the pairs of reads with one end has no hit but another has.
    // The structure will be used for proceeding semi-global DP
    ReadInputForDPArrays readInputForDPall;
    readInputForDPall.inputArrays = ( ReadInputForDP ** ) malloc ( ini_params.Ini_NumOfCpuThreads * sizeof ( ReadInputForDP * ) );
    readInputForDPall.numArrays = ini_params.Ini_NumOfCpuThreads;

    for ( int threadId = 0; threadId < ini_params.Ini_NumOfCpuThreads; ++threadId )
    {
        readInputForDPall.inputArrays[threadId] = constructReadInputForDP ( ini_params.Ini_NumOfCpuThreads );
    }

    // Declare the structure for storing the read IDs
    // for those read cannot be aligned
    UnalignedSinglesArrays * unalignedSinglesArrays = constructBothUnalignedPairsArrays ( ini_params.Ini_NumOfCpuThreads + 1 );
    // for single-end reads
    // get the max read length for the first ten reads (i.e. 0, 1, ..., 9)
    // ==================================================================
    // | DETECT THE READ LENGTH                                         |
    // ==================================================================
    uint detected_read_length = GetReadLength ( readLengths, numQueries, 1 );
    numOfAnswer = 0;
    numOfAlignedRead = 0;
    unsigned int numOfUnAlignedPairs = 0;
    unsigned int accumReadNum = 0;
    // ======================================================================================
    // | Configuration on GPU functions                                                     |
    // ======================================================================================
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferShared);
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_4mismatch_1), hipFuncCachePreferShared);
    // hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_4mismatch_2), hipFuncCachePreferShared);
    // ======================================================================================
    // | Perform alignment                                                                  |
    // ======================================================================================
    soap3_dp_single_align ( queries, readLengths, param->numMismatch, wordPerQuery,
                            maxBatchSize, numQueries, accumReadNum,
                            index, _bwt, _revBwt,
                            _occ, _revOcc,
                            ini_params, input_options,
                            param->maxReadLength, detected_read_length,
                            upkdQualities,
                            unAlignedPair, numOfUnAlignedPairs,
                            readIDs, upkdQueryNames,
                            NULL, NULL,
                            NULL, NULL,
                            numOfAnswer, numOfAlignedRead,
                            &readInputForDPall,
                            unalignedSinglesArrays,
                            startTime, lastEventTime, totalAlignmentTime,
                            indexLoadedToGPU );
    // ======================================================================================
    // | CLEAN UP                                                                           |
    // ======================================================================================
    free ( upkdQualities );
    free ( upkdQueryNames );
    free ( unAlignedPair );

    // free device memory
    if ( indexLoadedToGPU == 1 )
    {
        printf ( "[Main] Free device memory..\n" );
        GPUINDEXFree ( _bwt, _occ, _revBwt, _revOcc );
    }

    for ( int threadId = 0; threadId < ini_params.Ini_NumOfCpuThreads; ++threadId )
    {
        freeReadInputForDP ( readInputForDPall.inputArrays[threadId] );
    }

    free ( readInputForDPall.inputArrays ); // for half-aligned pairs
    freeBothUnalignedPairsArrays ( unalignedSinglesArrays ); // for single-unaligned reads
}


// pending.... NOT YET IMPLEMENTED
// To perform the paired alignment
// The resulting alignments are stored inside "algnResultArrays"
void alignPairR ( unsigned int * queries, unsigned int * readLengths, unsigned int * readIDs,
                  unsigned int wordPerQuery,
                  unsigned int numQueries, Soap3Index * index,
                  PairAlignParam * param,
                  unsigned long long & numOfAnswer,
                  unsigned int & numOfAlignedRead )
{
}



