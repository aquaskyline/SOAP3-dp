#include "hip/hip_runtime.h"
/*
 *
 *    DV-DPfunctions.cu
 *    Soap3(gpu)
 *
 *    Copyright (C) 2011, HKU
 *
 *    This program is free software; you can redistribute it and/or
 *    modify it under the terms of the GNU General Public License
 *    as published by the Free Software Foundation; either version 2
 *    of the License, or (at your option) any later version.
 *
 *    This program is distributed in the hope that it will be useful,
 *    but WITHOUT ANY WARRANTY; without even the implied warranty of
 *    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *    GNU General Public License for more details.
 *
 *    You should have received a copy of the GNU General Public License
 *    along with this program; if not, write to the Free Software
 *    Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */

#include "DV-DPfunctions.h"
#include "OutputDPResult.h"
#include <assert.h>

#include <algorithm>
using namespace std;

///////////////////////////////////////////////////////////////////////////////
//////////////////////////// GPU function definition //////////////////////////
///////////////////////////////////////////////////////////////////////////////

__forceinline__ __device__ int _MAX ( int x, int y )
{
    //return (x > y ? x : y);
    return max ( x, y );
}
__forceinline__ __device__ int _MIN ( int x, int y )
{
    //return (x < y ? x : y);
    return min ( x, y );
}
__forceinline__ __device__ short _LOW_THRESHOLD ( int x )
{
    return ( short ) max ( x, -32000 );
}

texture <uint> texPatterns;
texture <uint> texSequences;
#define DP_SCORE_NEG_INFINITY -32000
#define dist(x,y) (x==y ? MatchScore : MismatchScore)
#define GapInit (GapOpenScore - GapExtendScore)
#define PatternLength() (maxReadLength + maxDPTableLength)

#define MC_ScoreAddr(X,j,i) *(X + ((j)*LPARA + (((i)>>1)<<6) + TPARA + ((i)&0x1)))
#define MC_DnaUnpack(X,i) ((X[dnaTPARA + (((i)>>4)<<5)] >> ((15-((i)&0xF))<<1)) & 3)
#define MC_ReadUnpack(X,i) ((tex1Dfetch(texPatterns, readTPARA + (((i)>>4)<<5)) >> ((15-((i)&0xF))<<1)) & 3)

__device__ void DPScoreNHitPos ( uint * packedDNASequence, uint DNALength, uint maxDNALength, uint maxDPTableLength,
                                 uint * readSequence, uint readLength, uint maxReadLength,
                                 int MatchScore, int MismatchScore,
                                 int GapOpenScore, int GapExtendScore,
                                 int clipLtCheckLoc, int clipRtCheckLoc,
                                 int anchorLeftLoc, int anchorRightLoc,
                                 int & maxScore, uint & hitPos,
                                 void * DPTable, uint threadId )
{
    short * score = ( short * ) DPTable + ( threadId / 32 ) * ( maxDPTableLength * maxReadLength * 32 * 2 );
    short * scoreOpen = ( short * ) score + maxDPTableLength * maxReadLength * 32;
    uint LPARA = maxReadLength << 5;
    uint TPARA = ( threadId & 0x1F ) << 1;
    uint readTPARA = ( threadId >> 5 ) * ( MC_CeilDivide16 ( maxReadLength ) << 5 ) + ( threadId & 0x1F );
    uint dnaTPARA = ( threadId >> 5 ) * ( MC_CeilDivide16 ( maxDNALength ) << 5 ) + ( threadId & 0x1F );
    int i, j;
    maxScore = DP_SCORE_NEG_INFINITY;
    //Initialize the first column
    MC_ScoreAddr ( score, 0, 0 ) = _LOW_THRESHOLD ( 0 );
    MC_ScoreAddr ( scoreOpen, 0, 0 ) = _LOW_THRESHOLD ( GapInit );
    int upScore = GapInit;

    for ( i = 1; i <= readLength; i++ )
    {
        if ( i <= clipLtCheckLoc )
        {
            MC_ScoreAddr ( score, 0, i ) = _LOW_THRESHOLD ( GapOpenScore );
            MC_ScoreAddr ( scoreOpen, 0, i ) = _LOW_THRESHOLD ( GapOpenScore + GapInit );
        }
        else
        {
            upScore += GapExtendScore;
            MC_ScoreAddr ( score, 0, i ) = _LOW_THRESHOLD ( upScore );
            MC_ScoreAddr ( scoreOpen, 0, i ) = _LOW_THRESHOLD ( upScore + GapInit );
        }
    }

    //prepare for filling table
    //start!
    int prevInitScore = 0;

    for ( j = 1; j <= DNALength; j++ )
    {
        uchar refChar = MC_DnaUnpack ( packedDNASequence, j );
        int initScore = ( ( j >= anchorLeftLoc ) ? DP_SCORE_NEG_INFINITY : 0 );
        int upScore = initScore;
        int scoreOpenUp = initScore + GapInit;
        int prevScoreUp = prevInitScore;
        int prevScoreR;
        int gappedScore;
#define Mc_innerDPSO_UpdateScoreTable() { \
        prevScoreR = MC_ScoreAddr(score,0,i); \
        gappedScore = _MAX(GapOpenScore + prevScoreR, GapExtendScore + MC_ScoreAddr(scoreOpen,0,i)); \
        MC_ScoreAddr(scoreOpen,0,i) = _LOW_THRESHOLD( gappedScore ); \
        scoreOpenUp = _MAX(GapExtendScore + scoreOpenUp, GapOpenScore + upScore); \
        gappedScore = _MAX(scoreOpenUp, gappedScore); \
        upScore = _MAX(gappedScore, prevScoreUp + dist(refChar, MC_ReadUnpack(readSequence, i))); \
        MC_ScoreAddr(score,0,i) = _LOW_THRESHOLD( upScore ); \
        prevScoreUp = prevScoreR; \
    }

        for ( i = 1; i <= readLength; i++ )
        {
            Mc_innerDPSO_UpdateScoreTable ();

            if ( i <= clipLtCheckLoc )
            {
                scoreOpenUp = _MAX ( initScore + GapInit, scoreOpenUp );
                prevScoreUp = _MAX ( prevInitScore, prevScoreUp );
            }

            if ( i >= clipRtCheckLoc &&
                    j >= anchorRightLoc &&
                    upScore > maxScore )
            {
                // update max score
                maxScore = upScore;
                hitPos = j;
            }
        }

        prevInitScore = initScore;
    }
}

__device__ void GenerateDPTable ( uint * packedDNASequence, uint DNALength, uint maxDNALength, uint maxDPTableLength,
                                  uint * readSequence, uint readLength, uint maxReadLength,
                                  int MatchScore, int MismatchScore,
                                  int GapOpenScore, int GapExtendScore,
                                  int clipLtCheckLoc, int clipRtCheckLoc,
                                  int anchorLeftLoc, int anchorRightLoc,
                                  uint refOffset, int & maxScore, uint & hitPos, uint & scRight,
                                  uint & maxScoreCount,
                                  void * DPTable, uint threadId )
{
    short * score = ( short * ) DPTable + ( threadId / 32 ) * ( maxDPTableLength * maxReadLength * 32 * 2 );
    short * scoreOpen = ( short * ) score + maxDPTableLength * maxReadLength * 32;
    uint LPARA = maxReadLength << 5;
    uint TPARA = ( threadId & 0x1F ) << 1;
    uint readTPARA = ( threadId >> 5 ) * ( MC_CeilDivide16 ( maxReadLength ) << 5 ) + ( threadId & 0x1F );
    uint dnaTPARA = ( threadId >> 5 ) * ( MC_CeilDivide16 ( maxDNALength ) << 5 ) + ( threadId & 0x1F );
    int i, j;
    maxScore = DP_SCORE_NEG_INFINITY;
    maxScoreCount = 0;
    //Initialize the first column
    MC_ScoreAddr ( score, 0, 0 ) = _LOW_THRESHOLD ( 0 );
    MC_ScoreAddr ( scoreOpen, 0, 0 ) = _LOW_THRESHOLD ( GapInit );
    int upScore = GapInit;

    for ( i = 1; i <= readLength; i++ )
    {
        if ( i <= clipLtCheckLoc )
        {
            MC_ScoreAddr ( score, 0, i ) = _LOW_THRESHOLD ( GapOpenScore );
            MC_ScoreAddr ( scoreOpen, 0, i ) = _LOW_THRESHOLD ( GapOpenScore + GapInit );
        }
        else
        {
            upScore += GapExtendScore;
            MC_ScoreAddr ( score, 0, i ) = _LOW_THRESHOLD ( upScore );
            MC_ScoreAddr ( scoreOpen, 0, i ) = _LOW_THRESHOLD ( upScore + GapInit );
        }
    }

    //prepare for filling table
    //start!
    int prevInitScore = 0;

    for ( j = 1; j <= DNALength; j++ )
    {
        uchar refChar = MC_DnaUnpack ( packedDNASequence, j + refOffset );
        int initScore = ( ( j + refOffset >= anchorLeftLoc ) ? DP_SCORE_NEG_INFINITY : 0 );
        int upScore = initScore;
        int scoreOpenUp = initScore + GapInit;
        int prevScoreUp = prevInitScore;
        int prevScoreR;
        int gappedScore;
        MC_ScoreAddr ( score, j, 0 ) = _LOW_THRESHOLD ( upScore );
        MC_ScoreAddr ( scoreOpen, j, 0 ) = _LOW_THRESHOLD ( scoreOpenUp );
#define Mc_UpdateScoreTable() { \
        prevScoreR = MC_ScoreAddr(score,j-1,i); \
        gappedScore = _MAX(GapOpenScore + prevScoreR, GapExtendScore + MC_ScoreAddr(scoreOpen,j-1,i)); \
        MC_ScoreAddr(scoreOpen,j,i) = _LOW_THRESHOLD( gappedScore ); \
        scoreOpenUp = _MAX(GapExtendScore + scoreOpenUp, GapOpenScore + upScore); \
        gappedScore = _MAX(scoreOpenUp, gappedScore); \
        upScore = _MAX(gappedScore, prevScoreUp + dist(refChar, MC_ReadUnpack(readSequence, i))); \
        MC_ScoreAddr(score,j,i) = _LOW_THRESHOLD( upScore ); \
        prevScoreUp = prevScoreR; \
    }

        for ( i = 1; i <= readLength; i++ )
        {
            Mc_UpdateScoreTable ();

            if ( i <= clipLtCheckLoc )
            {
                scoreOpenUp = _MAX ( initScore + GapInit, scoreOpenUp );
                prevScoreUp = _MAX ( prevInitScore, prevScoreUp );
            }

            if ( i >= clipRtCheckLoc &&
                    j + refOffset >= anchorRightLoc )
            {
                // update max score
                if ( upScore > maxScore )
                {
                    maxScore = upScore;
                    hitPos = j;
                    scRight = readLength - i;
                    maxScoreCount = 1;
                }
                else if ( upScore == maxScore )
                {
                    ++maxScoreCount;
                }
            }
        }

        prevInitScore = initScore;
    }
}

__global__ void SemiGlobalAligntment ( uint * packedDNASequence, uint * DNALengths, uint maxDNALength, uint maxDPTableLength,
                                       uint * readSequence, uint * readLengths, uint maxReadLength,
                                       int * maxScores, uint * hitLocs, uint * startOffsets,
                                       uint * clipLtSizes, uint * clipRtSizes,
                                       uint * anchorLeftLocs, uint * anchorRightLocs, uint numOfThreads,
                                       int MatchScore, int MismatchScore,
                                       int GapOpenScore, int GapExtendScore,
                                       void * DPTable, uint * maxScoreCounts,
                                       int alignmentScheme = 1 )
{
    // precondition: MAX_READ_LENGTH should be multiple of 4
    uint threadId = blockIdx.x * DP_THREADS_PER_BLOCK + threadIdx.x;

    if ( threadId < numOfThreads )
    {
        uint clipLtSize = ( clipLtSizes == NULL ) ? 0 : clipLtSizes[threadId];
        uint clipRtSize = ( clipRtSizes == NULL ) ? 0 : clipRtSizes[threadId];
        uint anchorLeftLoc = ( anchorLeftLocs == NULL ) ?
                             maxDNALength : anchorLeftLocs[threadId];
        uint anchorRightLoc = ( anchorRightLocs == NULL ) ?
                              0 : anchorRightLocs[threadId];
        uint readLength = readLengths[threadId];
        uint DNALength = DNALengths[threadId];
        uint refOffset = 0, hitPos = 0, scRight = 0;
        uint maxScoreCount = 0;
        int maxScore;

        if ( alignmentScheme == 2 )
        {
            // get maxScore & hitPos
            DPScoreNHitPos ( packedDNASequence, DNALength, maxDNALength, maxDPTableLength,
                             readSequence, readLength, maxReadLength,
                             MatchScore, MismatchScore,
                             GapOpenScore, GapExtendScore,
                             clipLtSize, readLength - clipRtSize,
                             anchorLeftLoc, anchorRightLoc,
                             maxScore, hitPos,
                             DPTable, threadId );
            // decide offset
            DNALength = _MIN ( DNALength, maxDPTableLength - 1 );

            if ( hitPos > DNALength )
            {
                refOffset = hitPos - DNALength;

                if ( refOffset >= anchorLeftLoc )
                {
                    refOffset = ( anchorLeftLoc > 0 ? anchorLeftLoc - 1 : 0 );
                }
            }
        }

        GenerateDPTable ( packedDNASequence, DNALength, maxDNALength, maxDPTableLength,
                          readSequence, readLength, maxReadLength,
                          MatchScore, MismatchScore,
                          GapOpenScore, GapExtendScore,
                          clipLtSize, readLength - clipRtSize,
                          anchorLeftLoc, anchorRightLoc,
                          refOffset, maxScore, hitPos, scRight,
                          maxScoreCount,
                          DPTable, threadId );
        startOffsets[threadId] = refOffset;
        hitLocs[threadId] = hitPos;
        hitLocs[threadId] = hitLocs[threadId];

        if ( clipRtSizes != NULL )
        { clipRtSizes[threadId] = scRight; }

        maxScores[threadId] = maxScore;
        maxScoreCounts[threadId] = maxScoreCount;
    }
}

__global__ void GPUBacktrack ( uint * packedDNASequence, uint * DNALengths, uint maxDNALength, uint maxDPTableLength,
                               uint * readSequence, uint * readLengths, uint maxReadLength,
                               int * maxScores, uint * hitLocs, uint * startOffsets,
                               uint * clipLtSizes, uint * clipRtSizes, uint * anchorLeftLocs, uint numOfThreads,
                               int MatchScore, int MismatchScore,
                               int GapOpenScore, int GapExtendScore, int * cutoffThresholds,
                               void * DPTable, uchar * pattern )
{
    uint threadId = blockIdx.x * DP_THREADS_PER_BLOCK + threadIdx.x;

    if ( threadId < numOfThreads )
    {
        if ( maxScores[threadId] >= cutoffThresholds[threadId] )
        {
            short * score = ( short * ) DPTable + ( threadId / 32 ) * ( maxDPTableLength * maxReadLength * 32 * 2 );
            short * scoreOpen = ( short * ) score + maxDPTableLength * maxReadLength * 32;
            uint LPARA = maxReadLength << 5;
            uint TPARA = ( threadId & 0x1F ) << 1;
            uint readTPARA = ( threadId >> 5 ) * ( MC_CeilDivide16 ( maxReadLength ) << 5 ) + ( threadId & 0x1F );
            uint dnaTPARA = ( threadId >> 5 ) * ( MC_CeilDivide16 ( maxDNALength ) << 5 ) + ( threadId & 0x1F );
            uchar * curPattern = pattern + threadId * PatternLength ();
            uint pIndex = 0;
            uint readLength = readLengths[threadId];
            uint refOffset = startOffsets[threadId];
            uint anchorLeftLoc = ( anchorLeftLocs == NULL ) ?
                                 maxDNALength : anchorLeftLocs[threadId];
#define MC_PatternAppend(x) { curPattern[pIndex] = (x); ++pIndex; }
            uint clipLtCheckLoc = ( ( clipLtSizes == NULL ) ? 0 : clipLtSizes[threadId] );
            uint clipRtLength = ( clipRtSizes == NULL ) ? 0 : clipRtSizes[threadId];

            if ( clipRtLength > 0 )
            {
                MC_PatternAppend ( 'S' );
                MC_PatternAppend ( 'V' );
                MC_PatternAppend ( clipRtLength );
            }

            uint readPos = readLength - clipRtLength;
            uint refIndex = hitLocs[threadId];
            uchar readChar = MC_ReadUnpack ( readSequence, readPos );
            uchar refChar = MC_DnaUnpack ( packedDNASequence, refOffset + refIndex );
#define MC_NextRefCharNInitScore() { \
        --refIndex; \
        refChar = MC_DnaUnpack(packedDNASequence, refOffset+refIndex); \
        initScore = prevInitScore; \
        prevInitScore = ((refOffset+refIndex > anchorLeftLoc) ? \
                         DP_SCORE_NEG_INFINITY : 0); \
    }
#define MC_NextReadChar() { --readPos; readChar = MC_ReadUnpack(readSequence, readPos); }
            short curScore = MC_ScoreAddr ( score, refIndex, readPos );
            short nextScore;
            short initScore = ( ( refOffset + refIndex >= anchorLeftLoc ) ?
                                DP_SCORE_NEG_INFINITY : 0 );
            short prevInitScore = ( ( refOffset + refIndex > anchorLeftLoc ) ?
                                    DP_SCORE_NEG_INFINITY : 0 );
            enum DP_BacktrackState { DP_BT_NORMAL, DP_BT_I_EXT, DP_BT_D_EXT, \
                                     DP_BT_SM_EXIT, DP_BT_SI_EXIT
                                   };
            DP_BacktrackState state = DP_BT_NORMAL;

            while ( readPos > 0 && refIndex > 0 )
            {
                // check match
                if ( state == DP_BT_NORMAL )
                {
                    if ( curScore == dist ( refChar, readChar ) +
                            ( nextScore = MC_ScoreAddr ( score, refIndex - 1, readPos - 1 ) ) )
                    {
                        // Match/Mismatch --> reference X : read X
                        MC_PatternAppend ( 'm' - ( ( refChar == readChar ) << 5 ) );
                        MC_NextRefCharNInitScore ();
                        MC_NextReadChar ();
                        curScore = nextScore;
                    }
                    else if ( curScore == GapOpenScore +
                              ( nextScore = MC_ScoreAddr ( score, refIndex - 1, readPos ) ) )
                    {
                        // Deletion --> reference X : read -
                        MC_PatternAppend ( 'D' );
                        MC_NextRefCharNInitScore ();
                        curScore = nextScore;
                    }
                    else if ( curScore == GapExtendScore +
                              MC_ScoreAddr ( scoreOpen, refIndex - 1, readPos ) )
                    {
                        // (start extension) Deletion --> reference X : read -
                        MC_PatternAppend ( 'D' );
                        MC_NextRefCharNInitScore ();
                        curScore -= GapExtendScore;
                        state = DP_BT_D_EXT;
                    }
                    else
                    {
                        // check for left soft clip
                        if ( readPos <= clipLtCheckLoc + 1 )
                        {
                            if ( curScore == prevInitScore + dist ( refChar, readChar ) )
                            {
                                state = DP_BT_SM_EXIT;
                                break;
                            }
                            else if ( curScore == initScore + GapOpenScore )
                            {
                                state = DP_BT_SI_EXIT;
                                break;
                            }
                        }

                        if ( curScore == GapOpenScore +
                                ( nextScore = MC_ScoreAddr ( score, refIndex, readPos - 1 ) ) )
                        {
                            //  Insertion --> reference - : read X
                            MC_PatternAppend ( 'I' );
                            MC_NextReadChar ();
                            curScore = nextScore;
                        }
                        else
                        {
                            // (start extension) Insertion --> reference - : read X
                            MC_PatternAppend ( 'I' );
                            MC_NextReadChar ();
                            curScore -= GapExtendScore;
                            state = DP_BT_I_EXT;
                        }
                    }
                }
                else
                {
                    // extension state
                    if ( state == DP_BT_D_EXT )
                    {
                        // (extension) Deletion --> reference X : read -
                        MC_PatternAppend ( 'D' );
                        MC_NextRefCharNInitScore ();
                    }
                    else
                    {
                        // (extension) check for left soft clip
                        if ( readPos <= clipLtCheckLoc + 1 &&
                                curScore == initScore + GapOpenScore )
                        {
                            state = DP_BT_SI_EXIT;
                            break;
                        }

                        // (extension) Insertion --> reference - : read X
                        MC_PatternAppend ( 'I' );
                        MC_NextReadChar ();
                    }

                    if ( curScore == GapOpenScore + ( nextScore = MC_ScoreAddr ( score, refIndex, readPos ) ) )
                    {
                        state = DP_BT_NORMAL;
                        curScore = nextScore;
                    }
                    else
                    { curScore -= GapExtendScore; }
                }
            }

            //last proc
            if ( refIndex == 0 )
            {
                uint scNum = min ( clipLtCheckLoc, readPos );

                if ( scNum < readPos )
                {
                    MC_PatternAppend ( 'I' );
                    MC_PatternAppend ( 'V' );
                    MC_PatternAppend ( readPos - scNum );
                }

                MC_PatternAppend ( 'S' );
                MC_PatternAppend ( 'V' );
                MC_PatternAppend ( scNum );
            }
            else if ( state == DP_BT_SI_EXIT )
            {
                MC_PatternAppend ( 'I' );
                MC_PatternAppend ( 'S' );
                MC_PatternAppend ( 'V' );
                MC_PatternAppend ( readPos - 1 );
            }
            else if ( state == DP_BT_SM_EXIT )
            {
                MC_PatternAppend ( 'm' - ( ( refChar == readChar ) << 5 ) );
                MC_PatternAppend ( 'S' );
                MC_PatternAppend ( 'V' );
                MC_PatternAppend ( readPos - 1 );
                refIndex -= 1;
            }

            MC_PatternAppend ( 0 );
            hitLocs[threadId] = refOffset + refIndex;
        }
    }
}



///////////////////////////////////////////////////////////////////////////////
/////////////////////////////// SemiGlobalAligner  ////////////////////////////
///////////////////////////////////////////////////////////////////////////////

uint SemiGlobalAligner::estimateThreadSize ( int maxReadLength, int maxDNALength )
{
    uint tableSizeOfThread = 2 * maxDNALength * maxReadLength * sizeof ( short );
    uint otherSizeOfThread = MC_CeilDivide16 ( maxDNALength ) * sizeof ( uint ) + //_packedDNASequence
                             MC_CeilDivide16 ( maxReadLength ) * sizeof ( uint ) + //_packedReadSequence
                             sizeof ( uint ) + //_DNALengths
                             sizeof ( uint ) + //_readLengths
                             sizeof ( uint ) + //_startLocs
                             sizeof ( int ) * 2 + //_scores, _cutoffThresholds
                             sizeof ( uint ) + //_startOffsets
                             sizeof ( uint ) * 2 + //_clipLtSizes, Rt
                             sizeof ( uint ) * 2 + //_anchorLeftLocs, Right
                             sizeof ( uint ) + //_hitLocs
                             sizeof ( uint ) + //_maxScoreCounts
                             ( maxReadLength + maxDNALength ) * sizeof ( uchar ); //_pattern
    return tableSizeOfThread + otherSizeOfThread + 16; // 16 is padding size
}

SemiGlobalAligner::SemiGlobalAligner ()
{
    n_conf = 6;

    for ( int i = 0; i < 4; i++ )
    {
        blockConf[i] = 64 - 16 * i;
        coefConf[i] = 3;
    }

    blockConf[4] = 8;
    coefConf[4] = 2;
    blockConf[5] = 2;
    coefConf[5] = 1.5; // <- desperate
}

int SemiGlobalAligner::tryAlloc ( size_t estimatedThreadSize, size_t numOfBlocks )
{
    void * _testMalloc;
    hipError_t err = hipMalloc ( ( void ** ) &_testMalloc, estimatedThreadSize * numOfBlocks * DP_THREADS_PER_BLOCK );

    if ( err == hipSuccess )
    {
        hipFree ( _testMalloc );
        return 0;
    }

    return -1;
}

void SemiGlobalAligner::decideConfiguration (
    int maxReadLength, int maxDNALength,
    int & maxDPTableLength, int & numOfBlocks,
    int & patternLength, DPParameters & dpPara
)
{
#define DP_Effective_Region(l)  (l + l/2 + 8)
    size_t avail, total;
    hipMemGetInfo ( &avail, &total );
    size_t availableMemory = avail * 88 / 100; // 88%
    size_t estimatedThreadSize      = estimateThreadSize ( maxReadLength, maxDNALength );
    size_t availableBlocks          = availableMemory / ( estimatedThreadSize * DP_THREADS_PER_BLOCK );
    size_t estimatedThreadSize_2    = estimateThreadSize ( maxReadLength, DP_Effective_Region ( maxReadLength ) );
    size_t availableBlocks_2        = availableMemory / ( estimatedThreadSize_2 * DP_THREADS_PER_BLOCK );
    int successFlag = 0;

    for ( int i = 0; i < n_conf; i++ )
    {
        if ( availableBlocks >= blockConf[i] )
        {
            //              printf("[%d] scheme 1, try allocate\n", i);
            if ( 0 == tryAlloc ( estimatedThreadSize, blockConf[i] ) )
            {
                successFlag = 1;
                alignmentScheme = 1;
                numOfBlocks = blockConf[i];
                maxDPTableLength = maxDNALength;
                break;
            }
        }

        if ( availableBlocks_2 >= blockConf[i] &&
                maxDNALength >= ( int ) ( maxReadLength * coefConf[i] ) )
        {
            //              printf("[%d] scheme 2, try allocate\n", i);
            if ( 0 == tryAlloc ( estimatedThreadSize_2, blockConf[i] ) )
            {
                successFlag = 1;
                alignmentScheme = 2;
                numOfBlocks = blockConf[i];
                maxDPTableLength = DP_Effective_Region ( maxReadLength );
                break;
            }
        }
    }

    if ( !successFlag )
    {
        // configuration failed
        printf ( "[DPfunc] error: insufficient GPU memory, cannot perform DP\n" );
        exit ( -1 );
    }

    // check invalid configuration
    if ( numOfBlocks < 32 )
    {
        printf ( "[DPfunc] warning: insufficient GPU memory, performance might degrade\n" );
    }

    if ( dpPara.matchScore > 30 )
    {
        printf ( "[DPfunc] warning: MatchScore (set to %d) should not exceed 30\n", dpPara.matchScore );
    }

    fflush ( stdout );
    patternLength = PatternLength ();
}

void SemiGlobalAligner::init (
    int batchSize,
    int maxReadLength, int maxDNALength, int maxDPTableLength,
    DPParameters & dpPara
)
{
    MC_MemberCopy5 ( this->, , batchSize, maxReadLength, maxDNALength, maxDPTableLength, dpPara );
    hipFuncSetCacheConfig(reinterpret_cast<const void*>( SemiGlobalAligntment), hipFuncCachePreferL1 );
    //      hipFuncSetCacheConfig(reinterpret_cast<const void*>( GPUBacktrack), hipFuncCachePreferL1 );
    //      showGPUMemInfo("before");
    DP_HANDLE_ERROR ( hipMalloc ( ( void ** ) &_DPTable, ( size_t ) 2 * maxDPTableLength * maxReadLength * batchSize * sizeof ( short ) ) );
    DP_HANDLE_ERROR ( hipMalloc ( ( void ** ) &_packedDNASequence, batchSize * MC_CeilDivide16 ( maxDNALength ) * sizeof ( uint ) ) );
    DP_HANDLE_ERROR ( hipMalloc ( ( void ** ) &_packedReadSequence, batchSize * MC_CeilDivide16 ( maxReadLength ) * sizeof ( uint ) ) );
    DP_HANDLE_ERROR ( hipMalloc ( ( void ** ) &_DNALengths, batchSize * sizeof ( uint ) ) );
    DP_HANDLE_ERROR ( hipMalloc ( ( void ** ) &_readLengths, batchSize * sizeof ( uint ) ) );
    DP_HANDLE_ERROR ( hipMalloc ( ( void ** ) &_startLocs, batchSize * sizeof ( uint ) ) );
    DP_HANDLE_ERROR ( hipMalloc ( ( void ** ) &_startOffsets, batchSize * sizeof ( uint ) ) );
    DP_HANDLE_ERROR ( hipMalloc ( ( void ** ) &_hitLocs, batchSize * sizeof ( uint ) ) );
    DP_HANDLE_ERROR ( hipMalloc ( ( void ** ) &_scores, batchSize * sizeof ( int ) ) );
    DP_HANDLE_ERROR ( hipMalloc ( ( void ** ) &_cutoffThresholds, batchSize * sizeof ( int ) ) );
    DP_HANDLE_ERROR ( hipMalloc ( ( void ** ) &_clipLtSizes, batchSize * sizeof ( uint ) ) );
    DP_HANDLE_ERROR ( hipMalloc ( ( void ** ) &_clipRtSizes, batchSize * sizeof ( uint ) ) );
    DP_HANDLE_ERROR ( hipMalloc ( ( void ** ) &_anchorLeftLocs, batchSize * sizeof ( uint ) ) );
    DP_HANDLE_ERROR ( hipMalloc ( ( void ** ) &_anchorRightLocs, batchSize * sizeof ( uint ) ) );
    DP_HANDLE_ERROR ( hipMalloc ( ( void ** ) &_pattern, batchSize * PatternLength () * sizeof ( uchar ) ) );
    DP_HANDLE_ERROR ( hipMalloc ( ( void ** ) &_maxScoreCounts, batchSize * sizeof ( uint ) ) );
    //      showGPUMemInfo("alloc");
    hipBindTexture(NULL, texPatterns, _packedReadSequence,
            batchSize * MC_CeilDivide16 ( maxReadLength ) * sizeof ( uint ));
    hipBindTexture(NULL, texSequences, _packedDNASequence,
            batchSize * MC_CeilDivide16 ( maxDNALength ) * sizeof ( uint ));
}

void SemiGlobalAligner::performAlignment (
    uint * packedDNASequence, uint * DNALengths,
    uint * packedReadSequence, uint * readLengths,
    int * cutoffThresholds, int * scores, uint * hitLocs,
    uint * maxScoreCounts,
    uchar * pattern, int numOfThreads,
    uint * clipLtSizes, uint * clipRtSizes,
    uint * anchorLeftLocs, uint * anchorRightLocs )
{
    DP_HANDLE_ERROR ( hipMemcpy ( _packedDNASequence, packedDNASequence, batchSize * MC_CeilDivide16 ( maxDNALength ) * sizeof ( uint ), hipMemcpyHostToDevice ) );
    DP_HANDLE_ERROR ( hipMemcpy ( _packedReadSequence, packedReadSequence, batchSize * MC_CeilDivide16 ( maxReadLength ) * sizeof ( uint ), hipMemcpyHostToDevice ) );
    DP_HANDLE_ERROR ( hipMemcpy ( _DNALengths, DNALengths, batchSize * sizeof ( uint ), hipMemcpyHostToDevice ) );
    DP_HANDLE_ERROR ( hipMemcpy ( _readLengths, readLengths, batchSize * sizeof ( uint ), hipMemcpyHostToDevice ) );
    DP_HANDLE_ERROR ( hipMemcpy ( _cutoffThresholds, cutoffThresholds, batchSize * sizeof ( int ), hipMemcpyHostToDevice ) );
#define MC_CheckCopy_TypeUINT(CPU_para, GPU_para, tmp_para) { \
        if (CPU_para != NULL) { \
            DP_HANDLE_ERROR( hipMemcpy(GPU_para, CPU_para, batchSize * sizeof(uint), hipMemcpyHostToDevice) ); \
            tmp_para = GPU_para; \
        } \
    }
    uint * _clipLt = NULL, *_clipRt = NULL;
    MC_CheckCopy_TypeUINT ( clipLtSizes, _clipLtSizes, _clipLt );
    MC_CheckCopy_TypeUINT ( clipRtSizes, _clipRtSizes, _clipRt );
    uint * _anchorLtLocs = NULL, *_anchorRtLocs = NULL;
    MC_CheckCopy_TypeUINT ( anchorLeftLocs, _anchorLeftLocs, _anchorLtLocs );
    MC_CheckCopy_TypeUINT ( anchorRightLocs, _anchorRightLocs, _anchorRtLocs );
    int blocksNeeded = ( numOfThreads + DP_THREADS_PER_BLOCK - 1 ) / DP_THREADS_PER_BLOCK;
    SemiGlobalAligntment <<< blocksNeeded, DP_THREADS_PER_BLOCK>>> (
        _packedDNASequence, _DNALengths, maxDNALength, maxDPTableLength,
        _packedReadSequence, _readLengths, maxReadLength,
        _scores, _hitLocs, _startOffsets,
        _clipLt, _clipRt, _anchorLtLocs, _anchorRtLocs, numOfThreads,
        dpPara.matchScore, dpPara.mismatchScore,
        dpPara.openGapScore, dpPara.extendGapScore,
        _DPTable, _maxScoreCounts,
        alignmentScheme
    );
    GPUBacktrack <<< blocksNeeded, DP_THREADS_PER_BLOCK>>> (
        _packedDNASequence, _DNALengths, maxDNALength, maxDPTableLength,
        _packedReadSequence, _readLengths, maxReadLength,
        _scores, _hitLocs, _startOffsets,
        _clipLt, _clipRt, _anchorLtLocs, numOfThreads,
        dpPara.matchScore, dpPara.mismatchScore,
        dpPara.openGapScore, dpPara.extendGapScore, _cutoffThresholds,
        _DPTable, _pattern
    );
    //Fetch results
    DP_HANDLE_ERROR ( hipMemcpy ( scores, _scores, batchSize * sizeof ( int ), hipMemcpyDeviceToHost ) );
    DP_HANDLE_ERROR ( hipMemcpy ( hitLocs, _hitLocs, batchSize * sizeof ( uint ), hipMemcpyDeviceToHost ) );
    DP_HANDLE_ERROR ( hipMemcpy ( pattern, _pattern, batchSize * PatternLength () * sizeof ( uchar ), hipMemcpyDeviceToHost ) );
    DP_HANDLE_ERROR ( hipMemcpy ( maxScoreCounts, _maxScoreCounts, batchSize * sizeof ( uint ), hipMemcpyDeviceToHost ) );

}

void SemiGlobalAligner::freeMemory ()
{
    hipFree ( _DPTable );
    hipFree ( _packedDNASequence );
    hipFree ( _packedReadSequence );
    hipFree ( _DNALengths );
    hipFree ( _readLengths );
    hipFree ( _hitLocs );
    hipFree ( _startLocs );
    hipFree ( _startOffsets );
    hipFree ( _scores );
    hipFree ( _cutoffThresholds );
    hipFree ( _clipLtSizes );
    hipFree ( _clipRtSizes );
    hipFree ( _anchorLeftLocs );
    hipFree ( _anchorRightLocs );
    hipFree ( _pattern );
    hipFree ( _maxScoreCounts );
}



//////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////// For output ////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////

AlgnmtFlags::AlgnmtFlags ( uint range )
{
    size = ( range + 31 ) / 32;
    MC_CheckMalloc ( flags,   uint,   size );

    for ( int i = 0; i < 32; i++ )
    {
        MASK[i] = 1 << i;
    }

    pthread_mutex_init ( &occupy_mutex, NULL );
    clear ();
}

void AlgnmtFlags::clear ()
{
    memset ( flags, 0, size * sizeof ( uint ) );
}

void AlgnmtFlags::increaseSize ( uint newSize )
{
    uint * oldFlags = flags;
    uint oldSize = size;
    size  = newSize;
    MC_CheckMalloc ( flags,   uint,   size );
    memcpy ( flags, oldFlags, oldSize * sizeof ( uint ) );
    memset ( flags + oldSize, 0, ( newSize - oldSize ) * sizeof ( uint ) );
    free ( oldFlags );
}

void AlgnmtFlags::set ( int readID )
{
    pthread_mutex_lock ( &occupy_mutex );
    uint offset = readID >> 5;

    if ( offset >= size )
    {
        uint newSize = size * 2;

        while ( offset >= newSize )
        { newSize *= 2; }

        increaseSize ( newSize );
    }

    flags[offset] |= MASK[readID & 0x1F];
    pthread_mutex_unlock ( &occupy_mutex );
}

#define AlgnmtFlags_Get(in_flag, int32Offset, diff) { \
        uint flag = in_flag; \
        if (flag != 0) { \
            int offset = int32Offset << 5; \
            for (int j = 0; flag != 0; j++) { \
                if (flag & 1) { \
                    diff->push_back(offset + j); \
                } \
                flag >>= 1; \
            } \
        } \
    }

void AlgnmtFlags::get ( vector<int> * diff )
{
    for ( int i = 0; i < size; i++ )
    {
        AlgnmtFlags_Get ( flags[i], i, diff );
    }
}

inline void AlgnmtFlags::reserveSize ( AlgnmtFlags * algnFlags )
{
    if ( size < algnFlags->size )
    {
        this->increaseSize ( algnFlags->size );
    }
    else if ( size > algnFlags->size )
    {
        algnFlags->increaseSize ( size );
    }
}

void AlgnmtFlags::getXOR ( AlgnmtFlags * algnFlags, vector<int> * diff )
{
    reserveSize ( algnFlags );

    for ( int i = 0; i < size; i++ )
    {
        AlgnmtFlags_Get ( flags[i] ^ algnFlags->flags[i], i, diff );
    }
}

void AlgnmtFlags::AND ( AlgnmtFlags * algnFlags )
{
    reserveSize ( algnFlags );

    for ( int i = 0; i < size; i++ )
    {
        flags[i] &= algnFlags->flags[i];
    }
}

void AlgnmtFlags::XOR ( AlgnmtFlags * algnFlags )
{
    reserveSize ( algnFlags );

    for ( int i = 0; i < size; i++ )
    {
        flags[i] ^= algnFlags->flags[i];
    }
}

AlgnmtFlags::~AlgnmtFlags ()
{
    free ( flags );
}

//////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////// Alignment modules //////////////////////////////////////
/////////////////// The following code better be placed in a seperate file ///////////////////
//////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////


//////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////// standard space ///////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////


template <>
int isValid ( AlgnmtDPResult & a )
{
    return ( a.whichFromDP < 2 );
}

template <>
int ScoreCompare ( AlgnmtDPResult & a, AlgnmtDPResult & b )
{
#define MC_DPScoreCompare_SetValue(result, aligned, mismatch, score) { \
        if (result.whichFromDP == 0) { \
            aligned = 1; \
            mismatch = result.score_2; \
            score = result.score_1; \
        } \
        else \
            if (result.whichFromDP == 1) { \
                aligned = 1; \
                mismatch = result.score_1; \
                score = result.score_2; \
            } \
            else { \
                aligned = 0; \
                if (result.algnmt_1 != 0xFFFFFFFF) \
                    mismatch = result.score_1; \
                else \
                    mismatch = result.score_2; \
                score = 0; \
            } \
    }
    uint aligned_a, mismatch_a, score_a;
    uint aligned_b, mismatch_b, score_b;
    MC_DPScoreCompare_SetValue ( a, aligned_a, mismatch_a, score_a );
    MC_DPScoreCompare_SetValue ( b, aligned_b, mismatch_b, score_b );
    uint64 value_a = ( ( uint64 ) aligned_a << 63 ) | ( ( uint64 ) ( 0x1FFFFFFF - mismatch_a ) << 32 ) | ( score_a + 0x1FFFFFFF );
    uint64 value_b = ( ( uint64 ) aligned_b << 63 ) | ( ( uint64 ) ( 0x1FFFFFFF - mismatch_b ) << 32 ) | ( score_b + 0x1FFFFFFF );

    if ( value_a > value_b )
    { return 1; }
    else if ( value_a < value_b )
    { return -1; }
    else
    { return 0; }
}

template <>
int ScoreCompare ( SingleAlgnmtResult & a, SingleAlgnmtResult & b )
{
    if ( a.score > b.score )
    { return 1; }
    else if ( a.score < b.score )
    { return -1; }
    else
    { return 0; }
}

template <>
int ScoreCompare ( DeepDPAlignResult & a, DeepDPAlignResult & b )
{
    int score_a = a.score_1 + a.score_2;
    int score_b = b.score_1 + b.score_2;

    if ( score_a > score_b )
    { return 1; }
    else if ( score_a < score_b )
    { return -1; }
    else
    { return 0; }
}

template <>
bool ResultCompare ( const AlgnmtDPResult & a, const AlgnmtDPResult & b )
{
    return ( a.algnmt_1 < b.algnmt_1 );
}
template <>
bool ResultCompare ( const SingleAlgnmtResult & a, const SingleAlgnmtResult & b )
{
    return ( a.algnmt < b.algnmt );
}
template <>
bool ResultCompare ( const DeepDPAlignResult & a, const DeepDPAlignResult & b )
{
    return ( a.algnmt_1 < b.algnmt_1 );
}

QueryIDStream::QueryIDStream ()
{
    data = new vector<int>;
}
QueryIDStream::QueryIDStream ( BothUnalignedPairsArrays * input )
{
    data = new vector<int>;

    for ( int arrIndex = 0; arrIndex < input->arrayNum; arrIndex++ )
    {
        BothUnalignedPairs * array = input->array[arrIndex];

        for ( int pairIndex = 0; pairIndex < array->totalNum; pairIndex++ )
        {
            data->push_back ( array->readIDs[pairIndex] );
        }
    }
}
QueryIDStream::~QueryIDStream ()
{
    delete data;
}
void QueryIDStream::append ( QueryIDStream * stream )
{
    data->insert ( data->end (), stream->data->begin (), stream->data->end () );
}
void QueryIDStream::setBuffer ( vector<int> * input )
{
    delete data;
    data = input;
}



//////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////// single-dp space //////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////

using namespace SingleDP_Space;
#define DPS_SEEDING_BATCH_SIZE 256 * 1024
#define DPS_MARGIN(l) ((l>100) ? (l>>2) : 25)

CandidateStream::CandidateStream ()
{
    pthread_mutex_init ( &occupy_mutex, NULL );
}
void CandidateStream::append ( vector<CandidateInfo> * canInfo, AlgnmtFlags * alignFlags )
{
    pthread_mutex_lock ( &occupy_mutex );

    for ( vector<CandidateInfo>::iterator it = canInfo->begin ();
            it < canInfo->end (); ++it )
    {
        data.push_back ( *it );
        alignFlags->set ( it->readID );
    }

    pthread_mutex_unlock ( &occupy_mutex );
}

// ****
SingleEndSeedingEngine::SingleEndSeedingBatch::SingleEndSeedingBatch (
    uint batchSize, DPParameters * dpPara,
    uint * queries, uint * queryLengths, uint inputMaxReadLength
)
{
    MC_MemberCopy3 ( this->, , batchSize, queries, queryLengths );
    this->numOfCPUForSeeding    = dpPara->numOfCPUForSeeding;
    this->maxHitNum             = dpPara->paramRead[0].maxHitNum;
    this->maxSeedLength     = inputMaxReadLength;
    this->wordPerSeed       = MC_CeilDivide16 ( maxSeedLength );
    this->wordPerQuery      = getWordPerQuery ( inputMaxReadLength );
    this->algnResultArray = constructSingleAlgnResultArray ( numOfCPUForSeeding );
    MC_CheckMalloc ( readIDs, uint,   batchSize );
    MC_CheckMalloc ( lengths, uint,   batchSize );
    MC_CheckMalloc ( offsets, uint,   batchSize );
    MC_CheckMalloc ( seeds,   uint,   batchSize * wordPerSeed );
    MC_CheckMalloc ( seedPositions,   int,    inputMaxReadLength );
    clear ();
}
SingleEndSeedingEngine::SingleEndSeedingBatch::~SingleEndSeedingBatch ()
{
    freeSingleAlgnResultArray ( algnResultArray );
    free ( readIDs );
    free ( lengths );
    free ( offsets );
    free ( seeds );
    free ( seedPositions );
}
void SingleEndSeedingEngine::SingleEndSeedingBatch::clear ()
{
    numQueries = 0;
}
inline void SingleEndSeedingEngine::SingleEndSeedingBatch::pack ( uint readID, int off, int seedLength )
{
    readIDs[numQueries] = readID;
    lengths[numQueries] = seedLength;
    offsets[numQueries] = off;
#define MC_OldReadUnpackIn(X,i) ((X[oldReadTPARA + ((i>>4)<<5)] >> ((i & 0xF) << 1)) & 0x3)
    uint oldReadTPARA = ( readID / 32 ) * 32 * wordPerQuery + ( readID % 32 );
    uint seedTPARA = ( numQueries / 32 ) * 32 * wordPerSeed + ( numQueries % 32 );

    for ( int i = 0; i < wordPerSeed; i++ )
    {
        seeds[seedTPARA + ( i << 5 )] = 0;
    }

    for ( int i = 0; i < seedLength; i++ )
    {
        int pos = off + i;
        seeds[seedTPARA + ( ( i >> 4 ) << 5 )] |= ( uint ) MC_OldReadUnpackIn ( queries, pos ) << ( ( i & 0xF ) << 1 ) ;
    }

    ++numQueries;
}

int SingleEndSeedingEngine::SingleEndSeedingBatch::packSeeds (
    uint readID, int stage
)
{
    int seedNum, seedLength;
    int readLength = queryLengths[readID];
    getSeedPositions ( stage, readLength, &seedLength, seedPositions, &seedNum );

    if ( numQueries + seedNum > batchSize )
    {
        return 0;
    }

    for ( int i = 0; i < seedNum; i++ )
    {
        pack ( readID, seedPositions[i], seedLength );
    }

    return seedNum;
}

vector<CandidateInfo> * SingleEndSeedingEngine::SingleEndSeedingBatch::singleMerge (
    SeedPos * readPos
)
{
    vector<CandidateInfo> * canInfo = new vector<CandidateInfo> ();
    SeedPos * readIter = readPos;
    uint seedReadCnt = 0;

    while ( true )
    {
        uint readID = readIter->readID;

        if ( readID == 0x7FFFFFFF )
        { break; }

        SeedPos * readStart = readIter;

        while ( readIter->readID == readID )
        { ++readIter; }

        SeedPos * readEnd = readIter;
        canInfo->push_back ( *readStart );
        register uint prevLoc = readStart->pos;

        for ( SeedPos * p = readStart + 1; p < readEnd; p++ )
        {
            register uint curLoc = p->pos;

            if ( prevLoc + DPS_DIVIDE_GAP < curLoc )
            {
                canInfo->push_back ( *p );
                prevLoc = curLoc;
            }
        }

        ++seedReadCnt;
    }

    return canInfo;
}

SeedPos * SingleEndSeedingEngine::SingleEndSeedingBatch::decodePositions (
    BWT * bwt
)
{
#define MC_AppendPos(posIter, id, seedStrand, ePos, off) { \
        posIter->readID = id; \
        posIter->pos = ePos; \
        posIter->strand = seedStrand; \
        ++posIter; \
    }
    // printf("num of raw positions = %llu\n", numOfAnswer);
    SeedPos * pos, *auxPos;
    MC_CheckMalloc ( pos,     SeedPos,    numOfAnswer + 1 );
    MC_CheckMalloc ( auxPos,  SeedPos,    numOfAnswer + 1 );
    SeedPos * iter_pos = pos;

    for ( uint cpuThread = 0; cpuThread < algnResultArray->arrayNum; cpuThread++ )
    {
#define MC_EstimatedPos(x) ( strand == 1 ? \
                             x - offset : \
                             x + seedLength + offset - readLength )
        SingleAlgnResult * result = algnResultArray->array[cpuThread];
        SARecord * iter_sa = result->sa_list;
        OccRecord * iter_occ = result->occ_list;

        for ( uint i = 0; i < result->readNum; i++ )
        {
            uint seedID = result->readIDs[i];
            uint readID = readIDs[seedID];
            uint offset = offsets[seedID];
            uint seedLength = lengths[seedID];
            uint readLength = queryLengths[readID];

            if ( result->saEnds[i] < 0xFFFFFFFF )
            {
                SARecord * end_sa = result->sa_list + result->saEnds[i];

                while ( iter_sa <= end_sa )
                {
                    uint strand = iter_sa->strand;

                    for ( uint k = iter_sa->saLeft; k <= iter_sa->saRight; k++ )
                    {
                        uint estimatedPos = MC_EstimatedPos ( ( *bwt->_bwtSaValue ) ( bwt, k ) );
                        MC_AppendPos ( iter_pos, readID, strand,
                                       estimatedPos, offset );
                    }

                    ++iter_sa;
                }
            }

            if ( result->occEnds[i] < 0xFFFFFFFF )
            {
                OccRecord * end_occ = result->occ_list + result->occEnds[i];

                while ( iter_occ <= end_occ )
                {
                    uint strand = iter_occ->strand;
                    uint estimatedPos = MC_EstimatedPos ( iter_occ->pos );
                    MC_AppendPos ( iter_pos, readID, strand,
                                   estimatedPos, offset );
                    ++iter_occ;
                }
            }
        }
    }

    // array guard
    MC_AppendPos ( iter_pos, 0x7FFFFFFF, 0, 0xFFFFFFFF, 0 );
    uint len = iter_pos - pos;
    MC_RadixSort_32_16 ( pos, pos, auxPos, len );
    MC_RadixSort_32_16 ( pos, readID, auxPos, len );
    MC_RadixSort_8_8 ( pos, strand, auxPos, len );
    free ( auxPos );
    return pos;
}

vector<CandidateInfo> * SingleEndSeedingEngine::SingleEndSeedingBatch::decodeMergePositions (
    BWT * bwt
)
{
    SeedPos * pos = decodePositions ( bwt );
    vector<CandidateInfo> * canInfo = singleMerge ( pos );
    free ( pos );
    return canInfo;
}

// ****
void SingleEndSeedingEngine::SingleEndSeedingThreadContext::init ( SingleEndSeedingBatch * batch )
{
    sem_init ( &ACKSem, 0, 0 );
    sem_init ( &GPUFinishSem, 0, 0 );
    this->batch = batch;
    this->batch->clear ();
}

void SingleEndSeedingEngine::SingleEndSeedingThreadContext::freeMemory ()
{
    delete batch;
}

// ****
SingleEndSeedingEngine::SingleEndSeedingEngine () {}

void SingleEndSeedingEngine::performSeeding ()
{
    hipCtxPopCurrent ( & ( ctx ) );
    seedingSwapBatch =
        new SingleEndSeedingBatch ( DPS_SEEDING_BATCH_SIZE, dpPara,
                                    queries, queryLengths, inputMaxReadLength );
    seedingThreadContext =
        new SingleEndSeedingThreadContext[dpPara->numOfCPUForSeeding];

    for ( int i = 0; i < dpPara->numOfCPUForSeeding; i++ )
    {
        SingleEndSeedingBatch * batch =
            new SingleEndSeedingBatch ( DPS_SEEDING_BATCH_SIZE, dpPara,
                                        queries, queryLengths, inputMaxReadLength );
        seedingThreadContext[i].init ( batch );
    }

    seedingGPUThreadDelegator.init ( 1, SeedingGPUThread,
                                     SeedingGPUThreadInit, SeedingGPUThreadFinalize );
    seedingCPUThreadDelegator.init ( dpPara->numOfCPUForSeeding,
                                     SeedingCPUThread );
    inputFlags = new AlgnmtFlags;
    alignFlags = new AlgnmtFlags;
    int threadId;
    void * empty;

    for ( uint i = 0; i < queryIDStream->data->size (); i++ )
    {
        int readID = ( * ( queryIDStream->data ) ) [i];
        inputFlags->set ( readID );

        if ( !seedingSwapBatch->packSeeds ( readID, STAGE_SINGLE_DP ) )
        {
            // launch one batch
            threadId = seedingCPUThreadDelegator.schedule ( empty );
            sem_wait ( & ( seedingThreadContext[threadId].ACKSem ) );
            seedingSwapBatch->clear ();
            seedingSwapBatch->packSeeds ( readID, STAGE_SINGLE_DP );
        }
    }

    // last batch
    if ( seedingSwapBatch->numQueries > 0 )
    {
        threadId = seedingCPUThreadDelegator.schedule ( empty );
        sem_wait ( & ( seedingThreadContext[threadId].ACKSem ) );
    }

    seedingCPUThreadDelegator.finalize ();
    seedingGPUThreadDelegator.finalize ();
    alignFlags->getXOR ( inputFlags, unseededIDStream->data );
    delete inputFlags;
    delete alignFlags;
    delete seedingSwapBatch;

    for ( int i = 0; i < dpPara->numOfCPUForSeeding; i++ )
    {
        seedingThreadContext[i].freeMemory ();
    }

    delete[] seedingThreadContext;
    hipCtxPushCurrent ( ctx );
}

void SingleEndSeedingEngine::performSeeding (
    /* input */
    QueryIDStream    *    queryIDStream,
    DPParameters     *    dpPara,
    uint * queries, uint * queryLengths, int inputMaxReadLength,
    /* soap3 seeding related */
    SOAP3Wrapper<void>  * soap3Wrapper,
    Soap3Index      *     index,
    /* output */
    CandidateStream   *   canStream,
    QueryIDStream    *    unseededIDStream
)
{
    engine = new SingleEndSeedingEngine ();
    MC_MemberCopy5 ( engine->, , queryIDStream, dpPara, queries, queryLengths, inputMaxReadLength );
    MC_MemberCopy4 ( engine->, , soap3Wrapper, index, canStream, unseededIDStream );
    engine->performSeeding ();
    delete engine;
}
SingleEndSeedingEngine * SingleEndSeedingEngine::engine;

void SingleDP_Space::SeedingCPUThread ( int threadId, void *& empty )
{
    SingleEndSeedingEngine * engine = SingleEndSeedingEngine::engine;
    SingleEndSeedingEngine::SingleEndSeedingBatch * batch = engine->seedingSwapBatch;
    engine->seedingSwapBatch = engine->seedingThreadContext[threadId].batch;
    sem_post ( & ( engine->seedingThreadContext[threadId].ACKSem ) );
    engine->seedingThreadContext[threadId].batch = batch;
    int * pThreadId = &threadId;
    engine->seedingGPUThreadDelegator.schedule ( pThreadId );
    sem_wait ( & ( engine->seedingThreadContext[threadId].GPUFinishSem ) );
    vector<CandidateInfo> * canInfo = batch->decodeMergePositions ( engine->index->sraIndex->bwt );
    engine->canStream->append ( canInfo, engine->alignFlags );
    delete canInfo;
}

void SingleDP_Space::SeedingGPUThreadInit ()
{
    hipCtxPushCurrent ( SingleEndSeedingEngine::engine->ctx );
}

void SingleDP_Space::SeedingGPUThread ( int threadId, int *& pCallThreadId )
{
    SingleEndSeedingEngine * engine = SingleEndSeedingEngine::engine;
    SingleEndSeedingEngine::SingleEndSeedingBatch * batch = engine->seedingThreadContext[*pCallThreadId].batch;
    engine->soap3Wrapper->seeding (
        batch->seeds, batch->lengths,
        batch->maxSeedLength, batch->wordPerSeed, batch->batchSize,
        batch->numQueries, batch->numOfAnswer, batch->numOfAlignedRead,
        batch->numOfCPUForSeeding,
        batch->algnResultArray, batch->maxHitNum
    );
    sem_post ( & ( engine->seedingThreadContext[*pCallThreadId].GPUFinishSem ) );
}

void SingleDP_Space::SeedingGPUThreadFinalize ()
{
    hipCtxPopCurrent ( & ( SingleEndSeedingEngine::engine->ctx ) );
}

// ****
SingleEndAlignmentEngine::SingleEndAlgnBatch::SingleEndAlgnBatch (
    int batchSize, DPParameters * dpPara,
    int maxReadLength, int maxDNALength, int maxDPTableLength, int patternLength,
    Soap3Index * index, uint * queries, uint inputMaxReadLength, uint * upkdLengths
)
{
    MC_MemberCopy5 ( this->, , batchSize, maxReadLength, maxDNALength, maxDPTableLength, patternLength );
    MC_MemberCopy3 ( this->, , queries, inputMaxReadLength, upkdLengths );
    MC_MemberCopy2 ( this->, dpPara->, softClipLeft, softClipRight );
    this->cutoffThreshold   = dpPara->paramRead[0].cutoffThreshold;
    this->wordPerOldQuery   = getWordPerQuery ( inputMaxReadLength );
    this->wordPerQuery      = MC_CeilDivide16 ( maxReadLength );
    this->wordPerDNA        = MC_CeilDivide16 ( maxDNALength );
    this->packedDNA         = index->sraIndex->hsp->packedDNA;
    this->fullDNALength     = index->sraIndex->hsp->dnaLength;
    this->index             = index;
    MC_CheckMalloc ( canInfos,            CandidateInfo,  batchSize );
    MC_CheckMalloc ( DNALengths,          uint,           batchSize );
    MC_CheckMalloc ( lengths,             uint,           batchSize );
    MC_CheckMalloc ( packedDNASeq,        uint,           batchSize * MC_CeilDivide16 ( maxDNALength ) );
    MC_CheckMalloc ( packedReadSeq,       uint,           batchSize * MC_CeilDivide16 ( maxReadLength ) );
    MC_CheckMalloc ( scores,              int,            batchSize );
    MC_CheckMalloc ( cutoffThresholds,    int,            batchSize );
    MC_CheckMalloc ( softClipLtSizes,     uint,           batchSize );
    MC_CheckMalloc ( softClipRtSizes,     uint,           batchSize );
    MC_CheckMalloc ( hitLocs,             uint,           batchSize );
    MC_CheckMalloc ( pattern,             uchar,          batchSize * patternLength );
    MC_CheckMalloc ( maxScoreCounts,      uint,           batchSize );
    clear ();
}

SingleEndAlignmentEngine::SingleEndAlgnBatch::~SingleEndAlgnBatch ()
{
    free ( canInfos );
    free ( DNALengths );
    free ( lengths );
    free ( packedDNASeq );
    free ( packedReadSeq );
    free ( scores );
    free ( cutoffThresholds );
    free ( softClipLtSizes );
    free ( softClipRtSizes );
    free ( hitLocs );
    free ( pattern );
    free ( maxScoreCounts );
}

void SingleEndAlignmentEngine::SingleEndAlgnBatch::clear ()
{
    numOfThreads = 0;
}

int SingleEndAlignmentEngine::SingleEndAlgnBatch::pack (
    CandidateInfo & canInfo
)
{
    if ( numOfThreads >= batchSize )
    {
        return 0;
    }

    uint readID = canInfo.readID;
    uint readLength = upkdLengths[readID];
    int margin = DPS_MARGIN ( readLength );
    uint DNAStart = canInfo.pos - margin;

    if ( DNAStart >= fullDNALength )
    {
        DNAStart = 0;
    }

    uint DNALength = readLength + margin * 2;

    if ( DNAStart + DNALength > fullDNALength )
    {
        DNALength = fullDNALength - DNAStart;
    }

    packRead ( packedReadSeq, numOfThreads,
               readID, readLength,
               canInfo.strand );
    repackDNA ( packedDNASeq, numOfThreads,
                packedDNA, DNAStart, DNALength );
    softClipLtSizes[numOfThreads] = ( canInfo.strand == 1 ) ?
                                    softClipLeft : softClipRight;
    softClipRtSizes[numOfThreads] = ( canInfo.strand == 1 ) ?
                                    softClipRight : softClipLeft;
    DNALengths[numOfThreads] = DNALength;
    lengths[numOfThreads] = readLength;
    cutoffThresholds[numOfThreads] = cutoffThreshold;
    canInfo.pos = DNAStart;
    canInfos[numOfThreads] = canInfo;
    ++numOfThreads;
    return 1;
}

inline void SingleEndAlignmentEngine::SingleEndAlgnBatch::packRead (
    uint * packedSeq, uint threadId,
    uint readID, uint length, int strand
)
{
#define MC_OldReadUnpack(X,i) ((X[oldReadTPARA + (((i)>>4)<<5)] >> (((i) & 0xF) << 1)) & 0x3)
    uint oldReadTPARA = ( readID / 32 ) * 32 * wordPerOldQuery + ( readID % 32 );
    uint readTPARA = ( threadId / 32 ) * 32 * wordPerQuery + ( threadId % 32 );

    for ( uint i = 0; i <= ( length / CHAR_PER_WORD ); i++ )
    {
        packedSeq[readTPARA + ( i << 5 )] = 0;
    }

    if ( strand == 1 )
    {
        for ( int i = 1; i <= length; i++ )
        {
            int fwd_i = i - 1;
            register uint c_nucleotide = ( uint ) MC_OldReadUnpack ( queries, fwd_i );
#ifdef BS_MOD
            c_nucleotide = c_nucleotide ^ ( ( c_nucleotide == index->sraIndex->hsp->flag ) << 1 );
#endif
            packedSeq[readTPARA + ( ( i >> 4 ) << 5 )] |= c_nucleotide << ( ( 15 - ( i & 0xF ) ) << 1 );
        }
    }
    else   // strand == 2
    {
        for ( int i = 1; i <= length; i++ )
        {
            int rev_i = length - i;
            register uint c_nucleotide = soap3DnaComplement[ ( uint ) MC_OldReadUnpack ( queries, rev_i )];
#ifdef BS_MOD
            c_nucleotide = c_nucleotide ^ ( ( c_nucleotide == index->sraIndex->hsp->flag ) << 1 );
#endif
            packedSeq[readTPARA + ( ( i >> 4 ) << 5 )] |= c_nucleotide << ( ( 15 - ( i & 0xF ) ) << 1 );
        }
    }
}

inline void SingleEndAlignmentEngine::SingleEndAlgnBatch::repackDNA (
    uint * packedSeq, uint threadId,
    uint * seq, uint start, uint length
)
{
#define MC_OldDnaUnpack(X,i) ((X[(i)>>4] >> ((15-((i)&0xF))<<1)) & 3)
    uint dnaTPARA = ( threadId / 32 ) * 32 * wordPerDNA + ( threadId & 0x1F );

    for ( uint i = 0; i <= ( length / CHAR_PER_WORD ); i++ )
    {
        packedSeq[dnaTPARA + ( i << 5 )] = 0;
    }

    for ( int i = 1; i <= length; i++ )
    { packedSeq[dnaTPARA + ( ( i >> 4 ) << 5 )] |= ( uint ) ( MC_OldDnaUnpack ( seq, start + i - 1 ) ) << ( ( 15 - ( i & 0xF ) ) << 1 ); }
}

// ****
void SingleEndAlignmentEngine::SingleEndAlgnThreadContext::init ( SingleEndAlgnBatch * batch )
{
    sem_init ( &ACKSem, 0, 0 );
    sem_init ( &GPUFinishSem, 0, 0 );
    sem_init ( &outputACKSem, 0, 0 );
    this->batch = batch;
}

void SingleEndAlignmentEngine::SingleEndAlgnThreadContext::freeMemory ()
{
    delete batch;
}

// ****
SingleEndAlignmentEngine::AlgnmtResultStream::AlgnmtResultStream ()
{
    numOut = 0;
    pthread_mutex_init ( &occupy_mutex, NULL );
}

SingleEndAlignmentEngine::AlgnmtResultStream::~AlgnmtResultStream ()
{
    for ( int i = 0; i < dpSResult.size (); i++ )
    {
        SingleDPResultBatch & resultBatch = * ( dpSResult[i] );

        for ( int j = 0; j < resultBatch.size (); j++ )
        {
            free ( resultBatch[j].cigarString );
        }

        delete dpSResult[i];
    }

    dpSResult.clear ();
}

// ****
void SingleEndAlignmentEngine::performAlignment (
    uint & numDPAlignedRead, uint & numDPAlignment
)
{
    /* initialize */
    hipCtxPopCurrent ( & ( ctx ) );
    algnBatchCount = 0;
    dpSAlignedRead = 0;
    dpSAlignment = 0;
    lastReadID = -1;
    inputFlags = new AlgnmtFlags;
    alignFlags = new AlgnmtFlags;
    resultStream = new AlgnmtResultStream;
    outputBuf = new OutputBuffer<SingleAlgnmtResult> ();
    outputBuf->setAlignmentType ( alignmentType );
    maxReadLength = ( inputMaxReadLength / 4 + 1 ) * 4;
    maxDNALength = maxReadLength + 2 * DPS_MARGIN ( inputMaxReadLength ) + 8;
    semiGlobalAligner.decideConfiguration ( maxReadLength, maxDNALength,
                                            maxDPTableLength, DPS_ALGN_NUM_OF_BLOCKS,
                                            patternLength, *dpPara );
    algnSwapBatch =
        new SingleEndAlgnBatch ( DPS_ALGN_NUM_OF_BLOCKS * DP_THREADS_PER_BLOCK, dpPara,
                                 maxReadLength, maxDNALength, maxDPTableLength, patternLength,
                                 index, queries, inputMaxReadLength, upkdReadLengths );
    algnThreadContext = new SingleEndAlgnThreadContext[dpPara->numOfCPUThreads];

    for ( int i = 0; i < dpPara->numOfCPUThreads; i++ )
    {
        SingleEndAlgnBatch * batch =
            new SingleEndAlgnBatch ( DPS_ALGN_NUM_OF_BLOCKS * DP_THREADS_PER_BLOCK, dpPara,
                                     maxReadLength, maxDNALength, maxDPTableLength, patternLength,
                                     index, queries, inputMaxReadLength, upkdReadLengths );
        algnThreadContext[i].init ( batch );
    }

    algnmtGPUThreadDelegator.init ( 1, algnmtGPUThread,
                                    algnmtGPUThreadInit, algnmtGPUThreadFinalize );
    outputThreadDelegator.init ( 1, DPSOutputThread,
                                 NULL, DPSOutputThreadFinalize );
    algnmtCPUThreadDelegator.init ( dpPara->numOfCPUThreads, algnmtCPUThread );
    /* perform alignment */
    int threadId;
    void * empty;

    for ( uint i = 0; i < canStream->data.size (); i++ )
    {
        CandidateInfo & info = canStream->data[i];
        inputFlags->set ( info.readID );

        if ( !algnSwapBatch->pack ( info ) )
        {
            // launch one batch
            threadId = algnmtCPUThreadDelegator.schedule ( empty );
            sem_wait ( & ( algnThreadContext[threadId].ACKSem ) );
            algnSwapBatch->clear ();
            algnSwapBatch->pack ( info );
        }
    }

    // last batch
    if ( algnSwapBatch->numOfThreads > 0 )
    {
        threadId = algnmtCPUThreadDelegator.schedule ( empty );
        sem_wait ( & ( algnThreadContext[threadId].ACKSem ) );
    }

    /* finalize */
    algnmtCPUThreadDelegator.finalize ();
    algnmtGPUThreadDelegator.finalize ();
    outputThreadDelegator.finalize ();
    alignFlags->getXOR ( inputFlags, unalignedIDStream->data );
    delete inputFlags;
    delete alignFlags;
    delete algnSwapBatch;

    for ( int i = 0; i < dpPara->numOfCPUThreads; i++ )
    {
        algnThreadContext[i].freeMemory ();
    }

    delete[] algnThreadContext;
    delete outputBuf;
    delete resultStream;
    numDPAlignedRead = this->dpSAlignedRead;
    numDPAlignment = this->dpSAlignment;
    hipCtxPushCurrent ( ctx );
}

void SingleEndAlignmentEngine::performAlignment (
    /* input */
    CandidateStream   *   canStream,
    DPParameters     *    dpPara,
    uint * queries, uint * upkdReadLengths, int inputMaxReadLength,
    char * upkdQueryNames, uint * origReadIDs, char * upkdQualities,
    Soap3Index * index,
    int alignmentType,
    uint accumReadNum, int outputFormat,
    FILE * outputFile, samfile_t * samOutputDPFilePtr,
    /* output */
    QueryIDStream    *    unalignedIDStream,
    uint         &        numDPAlignedRead,
    uint         &        numDPAlignment
)
{
    engine = new SingleEndAlignmentEngine ();
    MC_MemberCopy2 ( engine->, , canStream, dpPara );
    MC_MemberCopy4 ( engine->, , queries, upkdQueryNames, upkdReadLengths, inputMaxReadLength );
    MC_MemberCopy2 ( engine->, , origReadIDs, upkdQualities );
    MC_MemberCopy ( engine->, , index );
    MC_MemberCopy4 ( engine->, , accumReadNum, outputFormat, outputFile, samOutputDPFilePtr );
    MC_MemberCopy2 ( engine->, , alignmentType, unalignedIDStream );
    engine->performAlignment ( numDPAlignedRead, numDPAlignment );
    delete engine;
}

SingleEndAlignmentEngine * SingleEndAlignmentEngine::engine;

void SingleDP_Space::algnmtCPUThread ( int threadId, void *& empty )
{
    SingleEndAlignmentEngine * engine = SingleEndAlignmentEngine::engine;
    SingleEndAlignmentEngine::SingleEndAlgnBatch * batch = engine->algnSwapBatch;
    engine->algnSwapBatch = engine->algnThreadContext[threadId].batch;
    engine->algnThreadContext[threadId].batchID = engine->algnBatchCount++;
    sem_post ( & ( engine->algnThreadContext[threadId].ACKSem ) );
    engine->algnThreadContext[threadId].batch = batch;
    int * pThreadId = &threadId;
    engine->algnmtGPUThreadDelegator.schedule ( pThreadId );
    sem_wait ( & ( engine->algnThreadContext[threadId].GPUFinishSem ) );
    MC_MemberCopy2 ( int, engine->dpPara->, matchScore, mismatchScore );
    MC_MemberCopy2 ( int, engine->dpPara->, openGapScore, extendGapScore );
    int cutoffThreshold = engine->dpPara->paramRead[0].cutoffThreshold;
    // Rearrange result and Output
    SingleDPResultBatch * resultBatch = new SingleDPResultBatch;

    for ( int i = 0; i < batch->numOfThreads; i++ )
    {
        if ( batch->scores[i] >= cutoffThreshold )
        {
            CigarStringEncoder<void> encoder;
            uchar lastType = 'N';

            for ( uchar * p = batch->pattern + i * engine->patternLength; *p != 0; p++ )
            {
                if ( *p == 'V' )
                {
                    encoder.append ( lastType, ( int ) ( * ( ++p ) ) - 1 );
                }
                else
                {
                    encoder.append ( *p, 1 );
                    lastType = *p;
                }
            }

            SingleAlgnmtResult result;
            result.readID = batch->canInfos[i].readID;
            result.strand = batch->canInfos[i].strand;
            result.algnmt = batch->canInfos[i].pos + batch->hitLocs[i];
            result.score = batch->scores[i];
            encoder.encodeCigarString ( openGapScore, extendGapScore );
            result.cigarString = encoder.cigarString;
            int L = batch->lengths[i] - encoder.charCount['I'] - encoder.charCount['S'];
            int numOfMismatch = ( L * matchScore + encoder.gapPenalty - batch->scores[i] ) /
                                ( matchScore - mismatchScore );
            result.editdist = encoder.charCount['I'] + encoder.charCount['D'] + numOfMismatch;
            result.num_sameScore = batch->maxScoreCounts[i]; // TODO
            resultBatch->push_back ( result );
        }
    }

    // Output
    engine->algnThreadContext[threadId].resultBatch = resultBatch;
    int * pid = &threadId;
    engine->outputThreadDelegator.schedule ( pid );
    // printf("ALgn CPU Thread done.\n");
    sem_wait ( & ( engine->algnThreadContext[threadId].outputACKSem ) );
}

void SingleDP_Space::algnmtGPUThreadInit ()
{
    SingleEndAlignmentEngine * engine = SingleEndAlignmentEngine::engine;
    hipCtxPushCurrent ( engine->ctx );
    //  showGPUMemInfo("algn enter");
    int batchSize = engine->DPS_ALGN_NUM_OF_BLOCKS * DP_THREADS_PER_BLOCK;
    engine->semiGlobalAligner.init ( batchSize, engine->maxReadLength,
                                     engine->maxDNALength, engine->maxDPTableLength, * ( engine->dpPara ) );
}

void SingleDP_Space::algnmtGPUThread ( int threadId, int *& pCallThreadId )
{
    SingleEndAlignmentEngine * engine = SingleEndAlignmentEngine::engine;
    SingleEndAlignmentEngine::SingleEndAlgnBatch * batch =
        engine->algnThreadContext[*pCallThreadId].batch;
    engine->semiGlobalAligner.performAlignment (
        batch->packedDNASeq, batch->DNALengths,
        batch->packedReadSeq, batch->lengths,
        batch->cutoffThresholds, batch->scores, batch->hitLocs,
        batch->maxScoreCounts,
        batch->pattern, batch->numOfThreads,
        batch->softClipLtSizes, batch->softClipRtSizes );
    sem_post ( & ( engine->algnThreadContext[*pCallThreadId].GPUFinishSem ) );
}

void SingleDP_Space::algnmtGPUThreadFinalize ()
{
    SingleEndAlignmentEngine * engine = SingleEndAlignmentEngine::engine;
    engine->semiGlobalAligner.freeMemory ();
    //  showGPUMemInfo("algn exit");
    hipCtxPopCurrent ( & ( engine->ctx ) );
}

void SingleDP_Space::DPSOutputThread ( int threadId, int *& pCallThreadId )
{
    int callThreadId = *pCallThreadId;
    SingleEndAlignmentEngine * engine = SingleEndAlignmentEngine::engine;
    int batchID = engine->algnThreadContext[callThreadId].batchID;
    SingleDPResultBatch * resultBatch = engine->algnThreadContext[callThreadId].resultBatch;
    sem_post ( & ( engine->algnThreadContext[callThreadId].outputACKSem ) );
    vector<SingleDPResultBatch *> & dpResult = engine->resultStream->dpSResult;

    while ( dpResult.size () <= batchID )
    {
        dpResult.push_back ( NULL );
    }

    dpResult[batchID] = resultBatch;
#define MC_DPSOutputRead() { \
        engine->outputBuf->ready(); \
        if (engine->outputBuf->size > 0) { \
            outputDPSingleResult2( \
                                   engine->outputBuf->elements, engine->outputBuf->size, \
                                   engine->queries, engine->upkdReadLengths, engine->origReadIDs, \
                                   engine->upkdQueryNames, engine->upkdQualities, \
                                   engine->inputMaxReadLength, engine->accumReadNum, engine->outputFormat, \
                                   engine->outputFile, engine->samOutputDPFilePtr, engine->index); \
            engine->dpSAlignedRead += 1; \
            engine->dpSAlignment += engine->outputBuf->size; \
            engine->alignFlags->set(engine->lastReadID); \
        } \
    }
    uint numOut = engine->resultStream->numOut;

    while ( numOut < dpResult.size () && dpResult[numOut] != NULL )
    {
        //OUTPUT HERE
        SingleDPResultBatch & batch = *dpResult[numOut];

        for ( int i = 0; i < batch.size (); i++ )
        {
            SingleAlgnmtResult & result = batch[i];
            int readID = result.readID;

            if ( readID != engine->lastReadID )
            {
                MC_DPSOutputRead ();
                engine->outputBuf->clear ();
                engine->lastReadID = readID;
            }

            engine->outputBuf->add ( result );
        }

        ++numOut;
    }

    engine->resultStream->numOut = numOut;
}

void SingleDP_Space::DPSOutputThreadFinalize ()
{
    SingleEndAlignmentEngine * engine = SingleEndAlignmentEngine::engine;
    MC_DPSOutputRead ();
    engine->outputBuf->clear ();
}

void SingleDP_Space::DPSOutputUnalignedReads (
    QueryIDStream * unalignedIDStream,
    uint * queries, uint * upkdReadLengths, int inputMaxReadLength,
    Soap3Index * index,
    char * upkdQueryNames, uint * origReadIDs, char * upkdQualities,
    uint accumReadNum, int outputFormat,
    FILE * outputFile, samfile_t * samOutputDPFilePtr
)
{
    // output unaligned result
#define MC_DPSOutputUnalgnRead() { \
        outputDPSingleResult2(buf, idx, \
                              queries, upkdReadLengths, origReadIDs, \
                              upkdQueryNames, upkdQualities, \
                              inputMaxReadLength, accumReadNum, outputFormat, \
                              outputFile, samOutputDPFilePtr, index); }
    SingleAlgnmtResult * buf;
    MC_CheckMalloc ( buf, SingleAlgnmtResult, 1024 );
    int idx = 0;

    for ( uint i = 0; i < unalignedIDStream->data->size (); i++ )
    {
        buf[idx].readID = ( * ( unalignedIDStream->data ) ) [i];
        buf[idx].algnmt = 0xFFFFFFFF;
        buf[idx].cigarString = NULL;
        ++idx;

        if ( idx >= 1024 )
        {
            MC_DPSOutputUnalgnRead ();
            idx = 0;
        }
    }

    if ( idx > 0 )
    { MC_DPSOutputUnalgnRead (); }

    free ( buf );
}



//////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////// default-dp space /////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////
using namespace DP_Space;

// ****
HalfEndOccStream::HalfEndOccStream ( ReadInputForDPArrays * input, BWT * bwt )
{
    this->data = input;
    this->bwt = bwt;
    arrayIndex = 0;
    iter_readInput = data->inputArrays[arrayIndex];
    iter_occ = iter_readInput->occ_list;
    end_occ = iter_occ + iter_readInput->occTotalNum;
    iter_sa = iter_readInput->sa_list;
    end_sa = iter_sa + iter_readInput->saRangeTotalNum;
    nextSAIndex = -1;
}

int HalfEndOccStream::fetchNextOcc ( SRAOccurrence & occ )
{
    while ( true )
    {
#define SA2OCC() { \
        occ.readID = iter_sa->readID; \
        occ.mismatchCount = iter_sa->mismatchCount; \
        occ.strand = iter_sa->strand; \
        occ.ambPosition = (*bwt->_bwtSaValue)(bwt, nextSAIndex++); \
        if (nextSAIndex > iter_sa->saIndexRight) { \
            nextSAIndex = -1; \
            ++iter_sa; \
        } \
    }

        if ( nextSAIndex != -1 )
        {
            SA2OCC ();
        }
        else if ( iter_occ == end_occ )
        {
            if ( iter_sa == end_sa )
            {
                ++arrayIndex;

                if ( arrayIndex >= data->numArrays )
                { return 0; }  // finished
                else
                {
                    iter_readInput = data->inputArrays[arrayIndex];
                    iter_occ = iter_readInput->occ_list;
                    end_occ = iter_occ + iter_readInput->occTotalNum;
                    iter_sa = iter_readInput->sa_list;
                    end_sa = iter_sa + iter_readInput->saRangeTotalNum;
                    continue;
                }
            }
            else
            {
                nextSAIndex = iter_sa->saIndexLeft;
                SA2OCC ();
            }
        }
        else
        {
            if ( iter_sa == end_sa )
            {
                occ = * ( iter_occ++ );
            }
            else
            {
                if ( ( iter_occ->readID >> 1 ) < ( iter_sa->readID >> 1 ) )
                { occ = * ( iter_occ++ ); }
                else
                {
                    nextSAIndex = iter_sa->saIndexLeft;
                    SA2OCC ();
                }
            }
        }

        return 1;
    }
}

// ****
HalfEndAlignmentEngine::HalfEndAlgnBatch::HalfEndAlgnBatch (
    int batchSize, DPParameters * dpPara,
    int peStrandLeftLeg, int peStrandRightLeg, int insert_high, int insert_low,
    int maxReadLength, int maxDNALength, int maxDPTableLength, int patternLength,
    Soap3Index * index, uint * queries, int inputMaxReadLength, uint * upkdReadLengths
)
{
    MC_MemberCopy5 ( this->, , batchSize, peStrandLeftLeg, peStrandRightLeg, insert_high, insert_low );
    MC_MemberCopy4 ( this->, , maxReadLength, maxDNALength, maxDPTableLength, patternLength );
    MC_MemberCopy4 ( this->, , index, queries, inputMaxReadLength, upkdReadLengths );
    MC_MemberCopy2 ( this->, dpPara->, softClipLeft, softClipRight );
    this->cutoffThreshold[0]    = dpPara->paramRead[0].cutoffThreshold;
    this->cutoffThreshold[1]    = dpPara->paramRead[1].cutoffThreshold;
    this->isDoubleStrand        = ( peStrandLeftLeg == peStrandRightLeg );
    this->fullDNALength         = index->sraIndex->hsp->dnaLength;
    this->wordPerOldQuery       = getWordPerQuery ( inputMaxReadLength );
    this->wordPerQuery          = MC_CeilDivide16 ( maxReadLength );
    this->wordPerDNA            = MC_CeilDivide16 ( maxDNALength );
    MC_CheckMalloc ( canInfo,             CandidateInfo,  batchSize );
    MC_CheckMalloc ( DNALengths,          uint,           batchSize );
    MC_CheckMalloc ( lengths,             uint,           batchSize );
    MC_CheckMalloc ( packedDNASequence,   uint,           batchSize * wordPerDNA );
    MC_CheckMalloc ( packedReadSequence,  uint,           batchSize * wordPerQuery );
    MC_CheckMalloc ( startLocs,           uint,           batchSize );
    MC_CheckMalloc ( hitLocs,             uint,           batchSize );
    MC_CheckMalloc ( scores,              int,            batchSize );
    MC_CheckMalloc ( cutoffThresholds,    int,            batchSize );
    MC_CheckMalloc ( softClipLtSizes,     uint,           batchSize );
    MC_CheckMalloc ( softClipRtSizes,     uint,           batchSize );
    MC_CheckMalloc ( peLeftAnchorLocs,    uint,           batchSize );
    MC_CheckMalloc ( peRightAnchorLocs,   uint,           batchSize );
    MC_CheckMalloc ( pattern,             uchar,          batchSize * patternLength );
    MC_CheckMalloc ( maxScoreCounts,      uint,           batchSize );
    clear ();
}

HalfEndAlignmentEngine::HalfEndAlgnBatch::~HalfEndAlgnBatch ()
{
    free ( canInfo );
    free ( DNALengths );
    free ( packedDNASequence );
    free ( lengths );
    free ( packedReadSequence );
    free ( startLocs );
    free ( hitLocs );
    free ( scores );
    free ( cutoffThresholds );
    free ( softClipLtSizes );
    free ( softClipRtSizes );
    free ( peLeftAnchorLocs );
    free ( peRightAnchorLocs );
    free ( pattern );
    free ( maxScoreCounts );
}

void HalfEndAlignmentEngine::HalfEndAlgnBatch::clear ()
{
    numOfThreads = 0;
}

int HalfEndAlignmentEngine::HalfEndAlgnBatch::pack (
    SRAOccurrence & curOcc
)
{
    uint alignedStrand = curOcc.strand;

    if ( alignedStrand != peStrandLeftLeg && alignedStrand != peStrandRightLeg )
    { return numOfThreads; }
    else
    {
        if ( numOfThreads + 1 + isDoubleStrand > batchSize )
        { return -1; }
    }

    uint alignedReadID = curOcc.readID;
    uint alignedPos = curOcc.ambPosition;
    uint alignedReadLength = upkdReadLengths[alignedReadID];
    int  unalignedIsReadOrMate = 1 - ( alignedReadID & 1 );
    uint unalignedReadID = ( unalignedIsReadOrMate == 0 ?
                             alignedReadID - 1 : alignedReadID + 1 );
    uint unalignedReadLength = upkdReadLengths[unalignedReadID];
#define MC_SetRead(strand) { \
        packRead(packedReadSequence, numOfThreads, \
                 unalignedReadID, unalignedReadLength, strand); \
        cutoffThresholds[numOfThreads] = cutoffThreshold[unalignedIsReadOrMate]; \
        softClipLtSizes[numOfThreads] = (strand == 1) ? softClipLeft : softClipRight;  \
        softClipRtSizes[numOfThreads] = (strand == 1) ? softClipRight : softClipLeft; \
    }

    if ( peStrandLeftLeg == alignedStrand )
    {
        //aligned read: at left, unaligned read: at right
        uint rightEnd = alignedPos + insert_high;
        uint rightStart = alignedPos + insert_low - unalignedReadLength;

        // rightStart has to be >= alignedPos
        if ( rightStart < alignedPos )
        { rightStart = alignedPos; }

        if ( rightStart < fullDNALength && rightEnd <= fullDNALength )
        {
            canInfo[numOfThreads].refer = curOcc;
            canInfo[numOfThreads].leftOrRight = 1;
            lengths[numOfThreads] = unalignedReadLength;
            startLocs[numOfThreads] = rightStart;
            DNALengths[numOfThreads] = rightEnd - rightStart;
            peLeftAnchorLocs[numOfThreads] = maxDNALength;
            peRightAnchorLocs[numOfThreads] = unalignedReadLength;
            repackDNA ( packedDNASequence, numOfThreads,
                        ( uint * ) index->sraIndex->hsp->packedDNA, rightStart, DNALengths[numOfThreads] );
            MC_SetRead ( peStrandRightLeg );
            ++numOfThreads;
        }
    }

    if ( peStrandRightLeg == alignedStrand )
    {
        //aligned read: at right, unaligned read: at left
        uint leftStart = alignedPos + alignedReadLength - insert_high;
        uint leftEnd = alignedPos + alignedReadLength - insert_low + unalignedReadLength;

        // leftEnd has to be < alignedPos + alignedReadLength
        if ( leftEnd >= alignedPos + alignedReadLength )
        { leftEnd = alignedPos + alignedReadLength - 1; }

        if ( leftStart < fullDNALength && leftEnd <= fullDNALength )
        {
            canInfo[numOfThreads].refer = curOcc;
            canInfo[numOfThreads].leftOrRight = 0;
            lengths[numOfThreads] = unalignedReadLength;
            startLocs[numOfThreads] = leftStart;
            DNALengths[numOfThreads] = leftEnd - leftStart;
            peLeftAnchorLocs[numOfThreads] = insert_high - insert_low + 1;
            peRightAnchorLocs[numOfThreads] = 0;
            repackDNA ( packedDNASequence, numOfThreads,
                        ( uint * ) index->sraIndex->hsp->packedDNA, leftStart, DNALengths[numOfThreads] );
            MC_SetRead ( peStrandLeftLeg );
            ++numOfThreads;
        }
    }

    return numOfThreads;
}

inline void HalfEndAlignmentEngine::HalfEndAlgnBatch::packRead (
    uint * packedSeq, uint threadId,
    uint readID, uint length, int strand
)
{
#define MC_OldReadUnpack(X,i) ((X[oldReadTPARA + (((i)>>4)<<5)] >> (((i) & 0xF) << 1)) & 0x3)
    uint oldReadTPARA = ( readID / 32 ) * 32 * wordPerOldQuery + ( readID % 32 );
    uint readTPARA = ( threadId / 32 ) * 32 * wordPerQuery + ( threadId % 32 );

    for ( uint i = 0; i <= ( length / CHAR_PER_WORD ); i++ )
    {
        packedSeq[readTPARA + ( i << 5 )] = 0;
    }

    if ( strand == 1 )
    {
        for ( int i = 1; i <= length; i++ )
        {
            int fwd_i = i - 1;
            register uint c_nucleotide = ( uint ) MC_OldReadUnpack ( queries, fwd_i );
#ifdef BS_MOD
            c_nucleotide = c_nucleotide ^ ( ( c_nucleotide == index->sraIndex->hsp->flag ) << 1 );
#endif
            packedSeq[readTPARA + ( ( i >> 4 ) << 5 )] |= c_nucleotide << ( ( 15 - ( i & 0xF ) ) << 1 );
        }
    }
    else   // strand == 2
    {
        for ( int i = 1; i <= length; i++ )
        {
            int rev_i = length - i;
            register uint c_nucleotide = soap3DnaComplement[ ( uint ) MC_OldReadUnpack ( queries, rev_i )];
#ifdef BS_MOD
            c_nucleotide = c_nucleotide ^ ( ( c_nucleotide == index->sraIndex->hsp->flag ) << 1 );
#endif
            packedSeq[readTPARA + ( ( i >> 4 ) << 5 )] |= c_nucleotide << ( ( 15 - ( i & 0xF ) ) << 1 );
        }
    }
}
inline void HalfEndAlignmentEngine::HalfEndAlgnBatch::repackDNA (
    uint * packedSeq, uint threadId,
    uint * seq, uint start, uint length
)
{
#define MC_OldDnaUnpack(X,i) ((X[(i)>>4] >> ((15-((i)&0xF))<<1)) & 3)
    uint dnaTPARA = ( threadId / 32 ) * 32 * wordPerDNA + ( threadId & 0x1F );

    for ( uint i = 0; i <= ( length / CHAR_PER_WORD ); i++ )
    {
        packedSeq[dnaTPARA + ( i << 5 )] = 0;
    }

    for ( int i = 1; i <= length; i++ )
    {
        packedSeq[dnaTPARA + ( ( i >> 4 ) << 5 )] |= ( uint ) ( MC_OldDnaUnpack ( seq, start + i - 1 ) ) << ( ( 15 - ( i & 0xF ) ) << 1 );
    }
}

// ****
void HalfEndAlignmentEngine::HalfEndAlgnThreadContext::init (
    HalfEndAlgnBatch * batch
)
{
    sem_init ( &dispatchACKSem, 0, 0 );
    sem_init ( &GPUFinishSem, 0, 0 );
    sem_init ( &outputACKSem, 0, 0 );
    this->batch = batch;
}

void HalfEndAlignmentEngine::HalfEndAlgnThreadContext::freeMemory ()
{
    resultBatch = NULL;
    delete batch;
}

// ****
HalfEndAlignmentEngine::AlgnmtResultStream::AlgnmtResultStream ()
{
    numOut = 0;
    pthread_mutex_init ( &occupy_mutex, NULL );
}
HalfEndAlignmentEngine::AlgnmtResultStream::~AlgnmtResultStream ()
{
    for ( int i = 0; i < dpResult.size (); i++ )
    {
        DPResultBatch & resultBatch = * ( dpResult[i] );

        for ( int j = 0; j < resultBatch.size (); j++ )
        {
            free ( resultBatch[j].cigarString );
        }

        delete dpResult[i];
    }

    dpResult.clear ();
}

// ****
void HalfEndAlignmentEngine::performAlignment ( uint & numDPAlignedRead, uint & numDPAlignment )
{
    /* initialize */
    hipCtxPopCurrent ( & ( ctx ) );
    algnBatchCount = 0;
    dpAlignedRead = 0;
    dpAlignment = 0;
    lastReadID = -1;
    inputFlags = new AlgnmtFlags;
    alignFlags = new AlgnmtFlags;
    resultStream = new AlgnmtResultStream;
    outputBuf = new OutputBuffer<AlgnmtDPResult> ();
    outputBuf->setAlignmentType ( alignmentType );
    maxReadLength = ( inputMaxReadLength / 4 + 1 ) * 4;
    maxDNALength = insert_high - insert_low + inputMaxReadLength + 1;
    semiGlobalAligner.decideConfiguration ( maxReadLength, maxDNALength,
                                            maxDPTableLength, DP_ALGN_NUM_OF_BLOCKS,
                                            patternLength, *dpPara );
    algnSwapBatch =
        new HalfEndAlgnBatch ( DP_ALGN_NUM_OF_BLOCKS * DP_THREADS_PER_BLOCK, dpPara,
                               peStrandLeftLeg, peStrandRightLeg, insert_high, insert_low,
                               maxReadLength, maxDNALength, maxDPTableLength, patternLength,
                               index, queries, inputMaxReadLength, upkdReadLengths );
    algnThreadContext = new HalfEndAlgnThreadContext[dpPara->numOfCPUThreads];

    for ( int i = 0; i < dpPara->numOfCPUThreads; i++ )
    {
        HalfEndAlgnBatch * batch =
            new HalfEndAlgnBatch ( DP_ALGN_NUM_OF_BLOCKS * DP_THREADS_PER_BLOCK, dpPara,
                                   peStrandLeftLeg, peStrandRightLeg, insert_high, insert_low,
                                   maxReadLength, maxDNALength, maxDPTableLength, patternLength,
                                   index, queries, inputMaxReadLength, upkdReadLengths );
        algnThreadContext[i].init ( batch );
    }

    algnmtGPUThreadDelegator.init ( 1, algnmtGPUThread,
                                    algnmtGPUThreadInit, algnmtGPUThreadFinalize );
    outputThreadDelegator.init ( 1, DPOutputThread,
                                 NULL, DPOutputThreadFinalize );
    algnmtCPUThreadDelegator.init ( dpPara->numOfCPUThreads, algnmtCPUThread );
    /* perform alignment */
    int threadId;
    void * empty;
    SRAOccurrence occ;

    while ( canStream->fetchNextOcc ( occ ) )
    {
        inputFlags->set ( ( occ.readID >> 1 ) << 1 );

        if ( algnSwapBatch->pack ( occ ) == -1 )
        {
            threadId = algnmtCPUThreadDelegator.schedule ( empty );
            sem_wait ( & ( algnThreadContext[threadId].dispatchACKSem ) );
            algnSwapBatch->clear ();
            algnSwapBatch->pack ( occ );
        }
    }

    // last batch
    if ( algnSwapBatch->numOfThreads > 0 )
    {
        threadId = algnmtCPUThreadDelegator.schedule ( empty );
        sem_wait ( & ( algnThreadContext[threadId].dispatchACKSem ) );
    }

    /* finalize */
    algnmtCPUThreadDelegator.finalize ();
    algnmtGPUThreadDelegator.finalize ();
    outputThreadDelegator.finalize ();
    alignFlags->getXOR ( inputFlags, unalignedIDStream->data );
    delete inputFlags;
    delete alignFlags;
    delete algnSwapBatch;

    for ( int i = 0; i < dpPara->numOfCPUThreads; i++ )
    {
        algnThreadContext[i].freeMemory ();
    }

    delete[] algnThreadContext;
    delete outputBuf;
    delete resultStream;
    numDPAlignedRead = this->dpAlignedRead;
    numDPAlignment = this->dpAlignment;
    hipCtxPushCurrent ( ctx );
}

void HalfEndAlignmentEngine::performAlignment (
    /* input */
    HalfEndOccStream   *  canStream,
    DPParameters     *    dpPara,
    uint * queries, uint * upkdReadLengths, int inputMaxReadLength,
    int insert_high, int insert_low,
    int peStrandLeftLeg, int peStrandRightLeg,
    char * upkdQueryNames, uint * origReadIDs, char * upkdQualities,
    Soap3Index * index,
    int alignmentType,
    uint accumReadNum, int outputFormat,
    FILE * outputFile, samfile_t * samOutputDPFilePtr,
    /* output */
    QueryIDStream    *    unalignedIDStream,
    uint         &        numDPAlignedRead,
    uint         &        numDPAlignment
)
{
    engine = new HalfEndAlignmentEngine ();
    MC_MemberCopy2 ( engine->, , canStream, dpPara );
    MC_MemberCopy4 ( engine->, , queries, upkdQueryNames, upkdReadLengths, inputMaxReadLength );
    MC_MemberCopy4 ( engine->, , insert_high, insert_low, peStrandLeftLeg, peStrandRightLeg );
    MC_MemberCopy2 ( engine->, , origReadIDs, upkdQualities );
    MC_MemberCopy ( engine->, , index );
    MC_MemberCopy4 ( engine->, , accumReadNum, outputFormat, outputFile, samOutputDPFilePtr );
    MC_MemberCopy2 ( engine->, , alignmentType, unalignedIDStream );
    engine->performAlignment ( numDPAlignedRead, numDPAlignment );
    delete engine;
}
HalfEndAlignmentEngine::HalfEndAlignmentEngine * HalfEndAlignmentEngine::engine;

// ****
void DP_Space::algnmtCPUThread ( int threadId, void *& empty )
{
    // Copy data, then ACK to dispatching thread
    HalfEndAlignmentEngine * engine = HalfEndAlignmentEngine::engine;
    engine->algnThreadContext[threadId].batchID = engine->algnBatchCount++;
    HalfEndAlignmentEngine::HalfEndAlgnBatch * batch = engine->algnSwapBatch;
    engine->algnSwapBatch = engine->algnThreadContext[threadId].batch;
    sem_post ( & ( engine->algnThreadContext[threadId].dispatchACKSem ) );
    engine->algnThreadContext[threadId].batch = batch;
    // launch kernel
    int * pThreadId = &threadId;
    engine->algnmtGPUThreadDelegator.schedule ( pThreadId );
    sem_wait ( & ( engine->algnThreadContext[threadId].GPUFinishSem ) );
    // rearrange result and Output
    MC_MemberCopy2 ( int, engine->, peStrandLeftLeg, peStrandRightLeg );
    MC_MemberCopy2 ( uint *, batch->, startLocs, hitLocs );
    MC_MemberCopy2 ( int, engine->dpPara->, matchScore, mismatchScore );
    MC_MemberCopy2 ( int, engine->dpPara->, openGapScore, extendGapScore );
    uchar * pattern = batch->pattern;
    CandidateInfo * canInfo =  batch->canInfo;
    DPResultBatch * resultBatch = new DPResultBatch;

    for ( int id = 0; id < batch->numOfThreads; id++ )
    {
        //Create record for AlgnmtDPResult;
        AlgnmtDPResult result;
        int alignedID = canInfo[id].refer.readID;
        int alignedIsReadOrMate = alignedID & 1;
        result.readID = alignedID - alignedIsReadOrMate;
        uint dpAlgnmtPos;

        if ( batch->scores[id] >= engine->dpPara->paramRead[1 - alignedIsReadOrMate].cutoffThreshold )
        {
//fprintf ( stderr, "%u %u %u\n", alignedID, batch->scores[id], canInfo[id].refer.strand );
            CigarStringEncoder<void> encoder;
            uchar lastType = 'N';

            for ( uchar * p = pattern + id * engine->patternLength; *p != 0; p++ )
            {
                if ( *p == 'V' )
                {
                    encoder.append ( lastType, ( int ) ( * ( ++p ) ) - 1 );
                }
                else
                {
                    encoder.append ( *p, 1 );
                    lastType = *p;
                }
            }

            encoder.encodeCigarString ( openGapScore, extendGapScore );
            result.cigarString = encoder.cigarString;
            // To get edit distance
            int L = batch->lengths[id] - encoder.charCount['I'] - encoder.charCount['S'];
            int numOfMismatch = ( L * matchScore + encoder.gapPenalty - batch->scores[id] ) /
                                ( matchScore - mismatchScore );
            result.editdist = encoder.charCount['I'] + encoder.charCount['D'] + numOfMismatch;
            result.whichFromDP = 1 - alignedIsReadOrMate;
            dpAlgnmtPos = startLocs[id] + hitLocs[id];

            if ( dpAlgnmtPos < canInfo[id].refer.ambPosition )
            {
                // dp is on left
                result.insertSize = canInfo[id].refer.ambPosition - dpAlgnmtPos +
                                    engine->upkdReadLengths[alignedID];
            }
            else
            {
                // dp is on right
                result.insertSize = dpAlgnmtPos - canInfo[id].refer.ambPosition +
                                    batch->lengths[id] + encoder.charCount['D'] -
                                    encoder.charCount['I'] - encoder.charCount['S'];
            }

            result.num_sameScore = batch->maxScoreCounts[id]; //TODO
        }
        else
        {
            result.cigarString = NULL;
            result.whichFromDP = 2;
            dpAlgnmtPos = 0xFFFFFFFF;
        }

        if ( alignedIsReadOrMate == 0 )
        {
            // aligned is read, unaligned is mate
            result.algnmt_1 = canInfo[id].refer.ambPosition;
            result.algnmt_2 = dpAlgnmtPos;
            result.score_1 = canInfo[id].refer.mismatchCount;
            result.score_2 = batch->scores[id];
            result.strand_1 = canInfo[id].refer.strand;
            result.strand_2 = ( canInfo[id].leftOrRight == 0 ? peStrandLeftLeg : peStrandRightLeg );
        }
        else
        {
            // aligned is mate, unaligned is read
            result.algnmt_1 = dpAlgnmtPos;
            result.algnmt_2 = canInfo[id].refer.ambPosition;
            result.score_1 = batch->scores[id];
            result.score_2 = canInfo[id].refer.mismatchCount;
            result.strand_1 = ( canInfo[id].leftOrRight == 0 ? peStrandLeftLeg : peStrandRightLeg );
            result.strand_2 = canInfo[id].refer.strand;
        }

        resultBatch->push_back ( result );
    }

    // output thread
    engine->algnThreadContext[threadId].resultBatch = resultBatch;
    pThreadId = &threadId;
    engine->outputThreadDelegator.schedule ( pThreadId );
    sem_wait ( & ( engine->algnThreadContext[threadId].outputACKSem ) );
}

void DP_Space::algnmtGPUThreadInit ()
{
    HalfEndAlignmentEngine * engine = HalfEndAlignmentEngine::engine;
    hipCtxPushCurrent ( engine->ctx );
    int batchSize = engine->DP_ALGN_NUM_OF_BLOCKS * DP_THREADS_PER_BLOCK;
    engine->semiGlobalAligner.init ( batchSize, engine->maxReadLength,
                                     engine->maxDNALength, engine->maxDPTableLength, * ( engine->dpPara ) );
}

void DP_Space::algnmtGPUThreadFinalize ()
{
    HalfEndAlignmentEngine * engine = HalfEndAlignmentEngine::engine;
    engine->semiGlobalAligner.freeMemory ();
    hipCtxPopCurrent ( & ( engine->ctx ) );
}

void DP_Space::algnmtGPUThread ( int gpuThreadId, int *& pCallThreadId )
{
    int threadId = *pCallThreadId;
    HalfEndAlignmentEngine * engine = HalfEndAlignmentEngine::engine;
    HalfEndAlignmentEngine::HalfEndAlgnBatch
    *batch = engine->algnThreadContext[threadId].batch;
    //  timeRecorder.appendStart("GPUTime");
    engine->semiGlobalAligner.performAlignment ( batch->packedDNASequence, batch->DNALengths,
            batch->packedReadSequence, batch->lengths,
            batch->cutoffThresholds, batch->scores, batch->hitLocs,
            batch->maxScoreCounts,
            batch->pattern, batch->numOfThreads,
            batch->softClipLtSizes, batch->softClipRtSizes,
            batch->peLeftAnchorLocs, batch->peRightAnchorLocs );
    //  timeRecorder.appendEnd("GPUTime");
    sem_post ( & ( engine->algnThreadContext[threadId].GPUFinishSem ) );
}

void DP_Space::DPOutputThread ( int outputThreadId, int *& pCallThreadId )
{
    int threadId = *pCallThreadId;
    HalfEndAlignmentEngine * engine = HalfEndAlignmentEngine::engine;
    DPResultBatch * resultBatch = engine->algnThreadContext[threadId].resultBatch;
    int batchID = engine->algnThreadContext[threadId].batchID;
    sem_post ( & ( engine->algnThreadContext[threadId].outputACKSem ) );
    vector<DPResultBatch *> & dpResult = engine->resultStream->dpResult;

    while ( dpResult.size () <= batchID )
    {
        dpResult.push_back ( NULL );
    }

    dpResult[batchID] = resultBatch;
#define MC_OutputRead() { \
        engine->outputBuf->ready(3); \
        if (engine->outputBuf->size > 0 && engine->outputBuf->elements[0].whichFromDP < 2) { \
            outputRead2(engine->outputBuf->elements, engine->outputBuf->size, \
                        engine->queries, engine->upkdReadLengths, \
                        engine->origReadIDs, engine->upkdQueryNames, \
                        engine->upkdQualities, engine->inputMaxReadLength, \
                        engine->accumReadNum, engine->outputFormat, \
                        engine->outputFile, engine->samOutputDPFilePtr, engine->index, \
                        engine->peStrandLeftLeg, engine->peStrandRightLeg); \
            engine->dpAlignedRead += 1; \
            engine->dpAlignment += engine->outputBuf->size; \
            engine->alignFlags->set(engine->lastReadID); \
        } \
    }
    uint numOut = engine->resultStream->numOut;

    while ( numOut < dpResult.size () && dpResult[numOut] != NULL )
    {
        //OUTPUT HERE
        DPResultBatch & batch = *dpResult[numOut];

        for ( int i = 0; i < batch.size (); i++ )
        {
            AlgnmtDPResult & result = batch[i];

            if ( result.readID != engine->lastReadID )
            {
                MC_OutputRead ();
                engine->outputBuf->clear ();
                engine->lastReadID = result.readID;
            }

            engine->outputBuf->add ( result );
        }

        ++numOut;
    }

    engine->resultStream->numOut = numOut;
}

void DP_Space::DPOutputThreadFinalize ()
{
    // last read
    HalfEndAlignmentEngine * engine = HalfEndAlignmentEngine::engine;
    MC_OutputRead ();
    engine->outputBuf->clear ();
}



//////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////// deep-dp space ///////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////
using namespace DeepDP_Space;
#define DP2_SEEDING_BATCH_SIZE 128 * 1024
#define DP2_MARGIN(l) ((l>100) ? (l>>2) : 25)

// ****
DeepDP_Space::CandidateStream::CandidateStream ()
{
    pthread_mutex_init ( &occupy_mutex, NULL );
}
void DeepDP_Space::CandidateStream::append ( vector<CandidateInfo> * canInfo, AlgnmtFlags * alignFlags )
{
    pthread_mutex_lock ( &occupy_mutex );

    for ( vector<CandidateInfo>::iterator it = canInfo->begin ();
            it < canInfo->end (); ++it )
    {
        data.push_back ( *it );
        alignFlags->set ( ( it->readIDLeft >> 1 ) << 1 );
    }

    pthread_mutex_unlock ( &occupy_mutex );
}

// ****
PairEndSeedingEngine::PairEndSeedingBatch::PairEndSeedingBatch (
    uint batchSize, DPParameters * dpPara,
    uint * queries, uint * readLengths, uint inputMaxReadLength,
    int insert_high, int insert_low,
    int peStrandLeftLeg, int peStrandRightLeg, BWT * bwt
)
{
    MC_MemberCopy4 ( this->, , batchSize, queries, readLengths, bwt );
    MC_MemberCopy4 ( this->, , insert_high, insert_low, peStrandLeftLeg, peStrandRightLeg );
    this->numOfCPUForSeeding = dpPara->numOfCPUForSeeding;

    for ( int i = 0; i < 2; i++ )
    {
        this->maxHitNum[i]  = dpPara->paramRead[i].maxHitNum;
    }

    this->maxSeedLength = inputMaxReadLength;
    this->wordPerSeed   = MC_CeilDivide16 ( maxSeedLength );
    this->wordPerQuery  = getWordPerQuery ( inputMaxReadLength );

    for ( int lOr = 0; lOr < 2; lOr++ )
    {
        algnResultArray[lOr] = constructSingleAlgnResultArray ( numOfCPUForSeeding );
        MC_CheckMalloc ( readIDs[lOr],    uint,   batchSize );
        MC_CheckMalloc ( lengths[lOr],    uint,   batchSize );
        MC_CheckMalloc ( offsets[lOr],    uint,   batchSize );
        MC_CheckMalloc ( seeds[lOr],      uint,   batchSize * wordPerSeed );
    }

    MC_CheckMalloc ( seedPositions,       int,    inputMaxReadLength );
    clear ();
}

PairEndSeedingEngine::PairEndSeedingBatch::~PairEndSeedingBatch ()
{
    for ( int lOr = 0; lOr < 2; lOr++ )
    {
        freeSingleAlgnResultArray ( algnResultArray[lOr] );
        free ( readIDs[lOr] );
        free ( lengths[lOr] );
        free ( offsets[lOr] );
        free ( seeds[lOr] );
    }
}

void PairEndSeedingEngine::PairEndSeedingBatch::clear ()
{
    for ( int i = 0; i < 2; i++ )
    {
        numQueries[i] = 0;
        inPosArr[i].clear ();
    }

    lastPairID = -1;
}
uint PairEndSeedingEngine::PairEndSeedingBatch::findRevStart (
    SeedPos * arr, uint len
)
{
    if ( len == 0 || ! ( arr[len - 1].strand_readID >> 31 ) )
    { return len; }

    uint start = 0;
    uint end = len - 1;

    while ( start < end )
    {
        uint mid = ( start + end ) / 2;

        if ( ( arr[mid].strand_readID >> 31 ) )
        {
            // reverse
            end = mid;
        }
        else
        {
            // forward
            start = mid + 1;
        }
    }

    return start;
}

inline void PairEndSeedingEngine::PairEndSeedingBatch::pack (
    uint evenReadID, uint readID, int off, int seedLength, int readOrMate
)
{
    int seedID = numQueries[readOrMate];
    readIDs[readOrMate][seedID] = evenReadID;
    lengths[readOrMate][seedID] = seedLength;
    offsets[readOrMate][seedID] = off;
#define MC_OldReadUnpackIn(X,i) ((X[oldReadTPARA + ((i>>4)<<5)] >> ((i & 0xF) << 1)) & 0x3)
    uint oldReadTPARA = ( readID / 32 ) * 32 * wordPerQuery + ( readID % 32 );
    uint seedTPARA = ( seedID / 32 ) * 32 * wordPerSeed + ( seedID % 32 );

    for ( int i = 0; i < wordPerSeed; i++ )
    {
        seeds[readOrMate][seedTPARA + ( i << 5 )] = 0;
    }

    for ( int i = 0; i < seedLength; i++ )
    {
        int pos = off + i;
        seeds[readOrMate][seedTPARA + ( ( i >> 4 ) << 5 )] |= ( uint ) MC_OldReadUnpackIn ( queries, pos ) << ( ( i & 0xF ) << 1 ) ;
    }

    ++numQueries[readOrMate];
}

int PairEndSeedingEngine::PairEndSeedingBatch::packSeeds (
    uint evenReadID, int stage
)
{
    for ( int i = 0; i < 2; i++ )
    {
        uint readID = evenReadID + i;
        uint readLength = readLengths[readID];
        int seedNum, seedLength;
        getSeedPositions ( stage, readLength, &seedLength, seedPositions, &seedNum );

        if ( numQueries[i] + seedNum > batchSize )
        {
            return 0;
        }

        for ( int j = 0; j < seedNum; j++ )
        {
            pack ( evenReadID, readID, seedPositions[j], seedLength, i );
        }

        // pack(evenReadID, readID, 0, 40, i);
    }

    lastPairID = evenReadID >> 1;
    return 1;
}

int PairEndSeedingEngine::PairEndSeedingBatch::packSeeds (
    SRAOccurrence & occ, int stage
)
{
    uint readID = occ.readID;
    uint pairID = readID >> 1;
    int readOrMate = readID & 1;
    uint evenReadID = readID - readOrMate;

    if ( pairID != lastPairID )
    {
        if ( !packSeeds ( evenReadID, stage ) )
        {
            return 0;
        }
    }

    SeedPos pos;
    pos.pos = occ.ambPosition;
    pos.strand_readID = ( ( occ.strand - 1 ) << 31 ) | evenReadID;
    inPosArr[readOrMate].push_back ( pos );
    return 1;
}

inline int PairEndSeedingEngine::PairEndSeedingBatch::packSeedsOneSide (
    uint evenReadID, int readOrMate, int stage
)
{
    uint readID = evenReadID + readOrMate;
    uint readLength = readLengths[readID];
    int seedNum, seedLength;
    getSeedPositions ( stage, readLength, &seedLength, seedPositions, &seedNum );

    if ( numQueries[readOrMate] + seedNum > batchSize )
    {
        return 0;
    }

    for ( int j = 0; j < seedNum; j++ )
    {
        pack ( evenReadID, readID, seedPositions[j], seedLength, readOrMate );
    }

    lastPairID = evenReadID >> 1;
    return 1;
}

int PairEndSeedingEngine::PairEndSeedingBatch::packSeedsOneSide (
    SRAOccurrence & occ, int stage
)
{
    uint readID = occ.readID;
    uint pairID = readID >> 1;
    int readOrMate = readID & 1;
    uint evenReadID = readID - readOrMate;

    if ( pairID != lastPairID )
    {
        if ( !packSeedsOneSide ( evenReadID, 1 - readOrMate, stage ) )
        {
            return 0;
        }
    }

    SeedPos pos;
    pos.pos = occ.ambPosition;
    pos.strand_readID = ( ( occ.strand - 1 ) << 31 ) | evenReadID;
    inPosArr[readOrMate].push_back ( pos );
    return 1;
}

void PairEndSeedingEngine::PairEndSeedingBatch::pairEndMerge (
    vector<CandidateInfo> * pairEndPos,
    SeedPos * readPos, SeedPos * matePos,
    int leftReadOrMate
)
{
#define MC_DecodePos(x) ((x)->pos)
#define MC_DecodeID(x) ((x)->strand_readID & 0x7FFFFFFF)
#define MC_ReadID() MC_DecodeID(readIter)
#define MC_MateID() MC_DecodeID(mateIter)
    SeedPos * readIter = readPos;
    SeedPos * mateIter = matePos;
    uint readID;
    uint mateID;

    while ( true )
    {
        mateID = MC_MateID ();

        while ( MC_ReadID () < mateID )
        { ++readIter; }

        readID = MC_ReadID ();

        while ( MC_MateID () < readID )
        { ++mateIter; }

        mateID = MC_MateID ();

        if ( mateID == 0x7FFFFFFF )
        { break; }
        else if ( readID < mateID )
        { continue; }

        SeedPos * readStart = readIter;
        SeedPos * mateStart = mateIter;

        // assert : readID == mateID
        while ( MC_ReadID () == readID )
        { ++readIter; }

        while ( MC_MateID () == mateID )
        { ++mateIter; }

        SeedPos * readEnd = readIter;
        SeedPos * mateEnd = mateIter;
#define MC_Compress(start, end, divideGap) { \
        SeedPos *cmprReadIter = start; \
        register uint prevLoc = MC_DecodePos(cmprReadIter); \
        for (SeedPos* p = start+1; p < end; p++) { \
            register uint curLoc = MC_DecodePos(p); \
            if (prevLoc + divideGap < curLoc) { \
                *(++cmprReadIter) = *p; \
                prevLoc = curLoc; \
            } \
        } \
        end = cmprReadIter + 1; \
    }
        MC_Compress ( readStart, readEnd, DP2_DIVIDE_GAP );
        //          MC_Compress(mateStart, mateEnd, DP2_DIVIDE_GAP);
        int readLength = readLengths[readID];
        int margin = DP2_MARGIN ( readLength );
        int length_low = insert_low - readLength - margin;

        if ( length_low < 0 )
        { length_low = 0; }

        int length_high = insert_high - readLength + margin;
        SeedPos * readP = readStart;
        SeedPos * mateP = mateStart;
        register uint readLoc = MC_DecodePos ( readP );
        register uint mateLoc = MC_DecodePos ( mateP );

        while ( readP < readEnd && mateP < mateEnd )
        {
            if ( readLoc + length_low > mateLoc )
            {
                ++mateP;
                mateLoc = MC_DecodePos ( mateP );
            }
            else if ( readLoc + length_high < mateLoc )
            {
                ++readP;
                readLoc = MC_DecodePos ( readP );
            }
            else
            {
                CandidateInfo ci;
                ci.pos[0] = readLoc;
                ci.pos[1] = mateLoc;
                ci.readIDLeft = MC_DecodeID ( readP ) + leftReadOrMate;
                pairEndPos->push_back ( ci );
                //                  ++mateP;
                //                  mateLoc = MC_DecodePos(mateP);
                // TODO
                ++readP;
                readLoc = MC_DecodePos ( readP );
            }
        }
    }
}

int PairEndSeedingEngine::PairEndSeedingBatch::decodePositions (
    int readOrMate, SeedPos *& pos, AlgnmtFlags * tooManyHitFlags
)
{
#define MC_Inc2(x) (x+2)
#define MC_SingleDP_AppendPos(posIter, readID, strandIndex, ePos, off) { \
        posIter->strand_readID = readID | (strandIndex << 31); \
        posIter->pos = ePos; \
        ++posIter; \
    }
    int inPosSize = inPosArr[readOrMate].size ();
    int arrSize = inPosSize + MC_Inc2 ( numOfAnswer[readOrMate] );
    SeedPos * auxPos;
    MC_CheckMalloc ( pos,     SeedPos,    arrSize );
    MC_CheckMalloc ( auxPos,  SeedPos,    arrSize );
    // pre-input value
    copy ( inPosArr[readOrMate].begin (), inPosArr[readOrMate].end (), pos );
    inPosArr[readOrMate].clear ();
    SeedPos * iter_pos = pos + inPosSize;

    for ( uint cpuThread = 0; cpuThread < algnResultArray[readOrMate]->arrayNum; cpuThread++ )
    {
#define MC_SingleDP_EstimatedPos(x) ( \
                                      strandIndex == 0 ? \
                                      x - offset : \
                                      x + seedLength + offset - readLength \
                                    )
        SingleAlgnResult * result = algnResultArray[readOrMate]->array[cpuThread];
        SARecord * iter_sa = result->sa_list;
        OccRecord * iter_occ = result->occ_list;

        for ( uint i = 0; i < result->readNum; i++ )
        {
            uint seedID = result->readIDs[i];
            uint readID = readIDs[readOrMate][seedID];
            uint offset = offsets[readOrMate][seedID];
            uint seedLength = lengths[readOrMate][seedID];
            uint readLength = readLengths[readID + readOrMate];

            if ( result->saEnds[i] < 0xFFFFFFFF )
            {
                SARecord * end_sa = result->sa_list + result->saEnds[i];

                while ( iter_sa <= end_sa )
                {
                    uint strandIndex = iter_sa->strand - 1;

                    for ( uint k = iter_sa->saLeft; k <= iter_sa->saRight; k++ )
                    {
                        uint estimatedPos = MC_SingleDP_EstimatedPos ( ( *bwt->_bwtSaValue ) ( bwt, k ) );
                        MC_SingleDP_AppendPos ( iter_pos, readID, strandIndex,
                                                estimatedPos, offset );
                    }

                    ++iter_sa;
                }
            }

            if ( result->occEnds[i] < 0xFFFFFFFF )
            {
                OccRecord * end_occ = result->occ_list + result->occEnds[i];

                while ( iter_occ <= end_occ )
                {
                    uint strandIndex = iter_occ->strand - 1;
                    uint estimatedPos = MC_SingleDP_EstimatedPos ( iter_occ->pos );
                    MC_SingleDP_AppendPos ( iter_pos, readID, strandIndex,
                                            estimatedPos, offset );
                    ++iter_occ;
                }
            }

            // set tooManyHitflags
            if ( result->isTooManyHit[i] )
            {
                tooManyHitFlags->set ( readID );
            }
        }
    }

    // array guard
    MC_SingleDP_AppendPos ( iter_pos, 0x7FFFFFFF, 0, 0xFFFFFFFF, 0 );
    MC_SingleDP_AppendPos ( iter_pos, 0x7FFFFFFF, 1, 0xFFFFFFFF, 0 );
    uint len = iter_pos - pos;
    MC_RadixSort_32_16 ( pos, pos, auxPos, len );
    MC_RadixSort_32_16 ( pos, strand_readID, auxPos, len );
    free ( auxPos );
    return len;
}

vector<DeepDP_Space::CandidateInfo> * PairEndSeedingEngine::PairEndSeedingBatch::decodeMergePositions (
    AlgnmtFlags * tooManyHitFlags
)
{
    SeedPos * readPos, *matePos;
    uint readPosLen = decodePositions ( 0, readPos, tooManyHitFlags );
    uint matePosLen = decodePositions ( 1, matePos, tooManyHitFlags );
    SeedPos * readArr[2], *mateArr[2];
    // 0 -- forward, 1 -- reverse
    readArr[0] = readPos;
    mateArr[0] = matePos;
    readArr[1] = readPos + findRevStart ( readPos, readPosLen );
    mateArr[1] = matePos + findRevStart ( matePos, matePosLen );
    vector<CandidateInfo> * canInfo = new vector<CandidateInfo>;
    // read left, mate right
    pairEndMerge ( canInfo, readArr[peStrandLeftLeg - 1], mateArr[peStrandRightLeg - 1], 0 );
    pairEndMerge ( canInfo, mateArr[peStrandLeftLeg - 1], readArr[peStrandRightLeg - 1], 1 );
    free ( readPos );
    free ( matePos );
    // Sort the candidates so that readID will be in order
    // To be revised
    vector<CandidateInfo> & candArr = *canInfo;
    uint arrLength = candArr.size ();
    CandidateInfo * auxCandArr;
    MC_CheckMalloc ( auxCandArr, CandidateInfo, arrLength );
    MC_RadixSort_32_16 ( candArr, readIDLeft, auxCandArr, arrLength );
    free ( auxCandArr );
    return canInfo;
}

// ****
void PairEndSeedingEngine::PairEndSeedingThreadContext::init (
    PairEndSeedingBatch * batch
)
{
    sem_init ( &ACKSem, 0, 0 );
    sem_init ( &GPUFinishSem, 0, 0 );
    this->batch = batch;
    this->batch->clear ();
}
void PairEndSeedingEngine::PairEndSeedingThreadContext::freeMemory ()
{
    delete batch;
}

// ****
PairEndSeedingEngine::PairEndSeedingEngine ()
{
    queryIDStream = NULL;
    halfEndOccStream = NULL;
    tooManyHitIDStream = NULL;
}

void PairEndSeedingEngine::performSeeding ()
{
    hipCtxPopCurrent ( & ( ctx ) );
    seedingSwapBatch =
        new PairEndSeedingBatch ( DP2_SEEDING_BATCH_SIZE, dpPara,
                                  queries, queryLengths, inputMaxReadLength,
                                  insert_high, insert_low,
                                  peStrandLeftLeg, peStrandRightLeg, index->sraIndex->bwt );
    seedingThreadContext =
        new PairEndSeedingThreadContext[dpPara->numOfCPUForSeeding];

    for ( int i = 0; i < dpPara->numOfCPUForSeeding; i++ )
    {
        PairEndSeedingBatch * batch =
            new PairEndSeedingBatch ( DP2_SEEDING_BATCH_SIZE, dpPara,
                                      queries, queryLengths, inputMaxReadLength,
                                      insert_high, insert_low,
                                      peStrandLeftLeg, peStrandRightLeg, index->sraIndex->bwt );
        seedingThreadContext[i].init ( batch );
    }

    seedingGPUThreadDelegator.init ( 1, SeedingGPUThread,
                                     SeedingGPUThreadInit, SeedingGPUThreadFinalize );
    seedingCPUThreadDelegator.init ( dpPara->numOfCPUForSeeding,
                                     SeedingCPUThread );
    inputFlags = new AlgnmtFlags;
    alignFlags = new AlgnmtFlags;
    tooManyHitFlags = new AlgnmtFlags;
    int threadId;
    void * empty;
    int lastPairID = -1;

    if ( halfEndOccStream != NULL )
    {
        SRAOccurrence occ;

        while ( halfEndOccStream->fetchNextOcc ( occ ) )
        {
            int pairID = occ.readID >> 1;

            if ( lastPairID != pairID )
            {
                inputFlags->set ( pairID << 1 );
                lastPairID = pairID;
            }

            if ( !seedingSwapBatch->packSeeds ( occ, seedingStage ) )
            {
                // launch one batch
                threadId = seedingCPUThreadDelegator.schedule ( empty );
                sem_wait ( & ( seedingThreadContext[threadId].ACKSem ) );
                seedingSwapBatch->clear ();
                seedingSwapBatch->packSeeds ( occ, seedingStage );
            }
        }
    }

    if ( queryIDStream != NULL )
    {
        for ( uint i = 0; i < queryIDStream->data->size (); i++ )
        {
            int readID = ( * ( queryIDStream->data ) ) [i];
            inputFlags->set ( readID );

            if ( !seedingSwapBatch->packSeeds ( readID, seedingStage ) )
            {
                // launch one batch
                threadId = seedingCPUThreadDelegator.schedule ( empty );
                sem_wait ( & ( seedingThreadContext[threadId].ACKSem ) );
                seedingSwapBatch->clear ();
                seedingSwapBatch->packSeeds ( readID, seedingStage );
            }
        }
    }

    // last batch
    threadId = seedingCPUThreadDelegator.schedule ( empty );
    sem_wait ( & ( seedingThreadContext[threadId].ACKSem ) );
    seedingCPUThreadDelegator.finalize ();
    seedingGPUThreadDelegator.finalize ();

    if ( tooManyHitIDStream == NULL )
    {
        // put together
        alignFlags->getXOR ( inputFlags, unseededIDStream->data );
    }
    else
    {
        // should separate
        // step 1, get unaligned reads
        alignFlags->XOR ( inputFlags );
        // step 2, get unaligned & tooManyHitReads
        tooManyHitFlags->AND ( alignFlags );
        // step 3
        alignFlags->getXOR ( tooManyHitFlags, unseededIDStream->data );
        tooManyHitFlags->get ( tooManyHitIDStream->data );
    }

    delete inputFlags;
    delete alignFlags;
    delete tooManyHitFlags;
    delete seedingSwapBatch;

    for ( int i = 0; i < dpPara->numOfCPUForSeeding; i++ )
    {
        seedingThreadContext[i].freeMemory ();
    }

    delete[] seedingThreadContext;
    hipCtxPushCurrent ( ctx );
}

void PairEndSeedingEngine::performSeeding (
    /* input */
    QueryIDStream    *    queryIDStream,
    DPParameters     *    dpPara,
    uint * queries, uint * queryLengths, int inputMaxReadLength,
    int insert_high, int insert_low,
    int peStrandLeftLeg, int peStrandRightLeg,
    int seedingStage,
    /* soap3 seeding related */
    SOAP3Wrapper<void>  * soap3Wrapper,
    Soap3Index      *     index,
    /* output */
    CandidateStream   *   canStream,
    QueryIDStream    *    unseededIDStream
)
{
    engine = new PairEndSeedingEngine ();
    MC_MemberCopy5 ( engine->, , queryIDStream, dpPara, queries, queryLengths, inputMaxReadLength );
    MC_MemberCopy4 ( engine->, , insert_high, insert_low, peStrandLeftLeg, peStrandRightLeg );
    MC_MemberCopy4 ( engine->, , soap3Wrapper, index, canStream, unseededIDStream );
    engine->seedingStage = seedingStage;
    engine->performSeeding ();
    delete engine;
}

void PairEndSeedingEngine::performSeeding (
    /* input */
    QueryIDStream    *    queryIDStream,
    DPParameters     *    dpPara,
    uint * queries, uint * queryLengths, int inputMaxReadLength,
    int insert_high, int insert_low,
    int peStrandLeftLeg, int peStrandRightLeg,
    int seedingStage,
    /* soap3 seeding related */
    SOAP3Wrapper<void>  * soap3Wrapper,
    Soap3Index      *     index,
    /* output */
    CandidateStream   *   canStream,
    QueryIDStream    *    tooManyHitIDStream,
    QueryIDStream    *    unseededIDStream
)
{
    engine = new PairEndSeedingEngine ();
    MC_MemberCopy5 ( engine->, , queryIDStream, dpPara, queries, queryLengths, inputMaxReadLength );
    MC_MemberCopy4 ( engine->, , insert_high, insert_low, peStrandLeftLeg, peStrandRightLeg );
    MC_MemberCopy5 ( engine->, , soap3Wrapper, index, canStream, tooManyHitIDStream, unseededIDStream );
    engine->seedingStage = seedingStage;
    engine->performSeeding ();
    delete engine;
}


void PairEndSeedingEngine::performSeeding (
    /* input */
    DP_Space::HalfEndOccStream
    * halfEndOccStream,
    DPParameters     *    dpPara,
    uint * queries, uint * queryLengths, int inputMaxReadLength,
    int insert_high, int insert_low,
    int peStrandLeftLeg, int peStrandRightLeg,
    int seedingStage,
    /* soap3 seeding related */
    SOAP3Wrapper<void>  * soap3Wrapper,
    Soap3Index      *     index,
    /* output */
    CandidateStream   *   canStream,
    QueryIDStream    *    unseededIDStream
)
{
    engine = new PairEndSeedingEngine ();
    MC_MemberCopy5 ( engine->, , halfEndOccStream, dpPara, queries, queryLengths, inputMaxReadLength );
    MC_MemberCopy4 ( engine->, , insert_high, insert_low, peStrandLeftLeg, peStrandRightLeg );
    MC_MemberCopy4 ( engine->, , soap3Wrapper, index, canStream, unseededIDStream );
    engine->seedingStage = seedingStage;
    engine->performSeeding ();
    delete engine;
}

PairEndSeedingEngine * PairEndSeedingEngine::engine;

void DeepDP_Space::SeedingGPUThreadInit ()
{
    hipCtxPushCurrent ( PairEndSeedingEngine::engine->ctx );
    //  showGPUMemInfo("seeding enter");
}
void DeepDP_Space::SeedingGPUThread ( int threadId, int *& pCallThreadId )
{
    PairEndSeedingEngine * engine = PairEndSeedingEngine::engine;
    PairEndSeedingEngine::PairEndSeedingBatch * batch =
        engine->seedingThreadContext[*pCallThreadId].batch;

    for ( int r = 0; r < 2; r++ )
    {
        engine->soap3Wrapper->seeding (
            batch->seeds[r], batch->lengths[r],
            batch->maxSeedLength, batch->wordPerSeed, batch->batchSize,
            batch->numQueries[r], batch->numOfAnswer[r], batch->numOfAlignedRead[r],
            batch->numOfCPUForSeeding,
            batch->algnResultArray[r], batch->maxHitNum[r]
        );
    }

    sem_post ( & ( engine->seedingThreadContext[*pCallThreadId].GPUFinishSem ) );
}
void DeepDP_Space::SeedingGPUThreadFinalize ()
{
    //  showGPUMemInfo("seeding exit");
    hipCtxPopCurrent ( & ( PairEndSeedingEngine::engine->ctx ) );
}
void DeepDP_Space::SeedingCPUThread ( int threadId, void *& empty )
{
    PairEndSeedingEngine * engine = PairEndSeedingEngine::engine;
    PairEndSeedingEngine::PairEndSeedingBatch * batch = engine->seedingSwapBatch;
    engine->seedingSwapBatch = engine->seedingThreadContext[threadId].batch;
    // printf("[%u][%d] Launching a seeding batch... ", threadId, engine->seedingBatchCount++); fflush(stdout);
    sem_post ( & ( engine->seedingThreadContext[threadId].ACKSem ) );
    engine->seedingThreadContext[threadId].batch = batch;
    int * pThreadId = &threadId;
    engine->seedingGPUThreadDelegator.schedule ( pThreadId );
    sem_wait ( & ( engine->seedingThreadContext[threadId].GPUFinishSem ) );
    // printf("[%u] Seeding thread done.\n", threadId); fflush(stdout);
    vector<CandidateInfo> * candidates = batch->decodeMergePositions ( engine->tooManyHitFlags );
    engine->canStream->append ( candidates, engine->alignFlags );
    delete candidates;
}

void DeepDP_Space::DP2OutputUnalignedReads (
    QueryIDStream * unalignedIDStream,
    uint * queries, uint * upkdReadLengths, int inputMaxReadLength,
    Soap3Index * index, int peStrandLeftLeg, int peStrandRightLeg,
    char * upkdQueryNames, uint * origReadIDs, char * upkdQualities,
    uint accumReadNum, int outputFormat,
    FILE * outputFile, samfile_t * samOutputDPFilePtr
)
{
    // output unaligned result
#define MC_DP2OutputUnalgnRead() { \
        outputDeepDPResult2(buf, idx, \
                            queries, upkdReadLengths, \
                            origReadIDs, upkdQueryNames, upkdQualities, \
                            inputMaxReadLength, accumReadNum, outputFormat, \
                            outputFile, samOutputDPFilePtr, index, \
                            peStrandLeftLeg, peStrandRightLeg); }
    DeepDPAlignResult * buf;
    MC_CheckMalloc ( buf, DeepDPAlignResult, 1024 );
    int idx = 0;

    for ( uint i = 0; i < unalignedIDStream->data->size (); i++ )
    {
        buf[idx].readID = ( * ( unalignedIDStream->data ) ) [i];
        buf[idx].algnmt_1 = 0xFFFFFFFF;
        buf[idx].algnmt_2 = 0xFFFFFFFF;
        buf[idx].cigarString_1 = NULL;
        buf[idx].cigarString_2 = NULL;
        ++idx;

        if ( idx >= 1024 )
        {
            MC_DP2OutputUnalgnRead ();
            idx = 0;
        }
    }

    if ( idx > 0 )
    { MC_DP2OutputUnalgnRead (); }

    free ( buf );
}

// ****
PairEndAlignmentEngine::PairEndAlgnBatch::PairEndAlgnBatch (
    int batchSize, DPParameters * dpPara,
    int peStrandLeftLeg, int peStrandRightLeg, int insert_high, int insert_low,
    int maxReadLength, int maxDNALength, int maxDPTableLength, int patternLength,
    Soap3Index * index, uint * queries, uint inputMaxReadLength, uint * upkdLengths
)
{
    MC_MemberCopy5 ( this->, , batchSize, maxReadLength, maxDNALength, maxDPTableLength, patternLength );
    MC_MemberCopy4 ( this->, , peStrandLeftLeg, peStrandRightLeg, insert_high, insert_low );
    MC_MemberCopy3 ( this->, , queries, inputMaxReadLength, upkdLengths );
    MC_MemberCopy2 ( this->, dpPara->, softClipLeft, softClipRight );
    this->cutoffThreshold[0]    = dpPara->paramRead[0].cutoffThreshold;
    this->cutoffThreshold[1]    = dpPara->paramRead[1].cutoffThreshold;
    this->wordPerOldQuery   = getWordPerQuery ( inputMaxReadLength );
    this->wordPerQuery      = MC_CeilDivide16 ( maxReadLength );
    this->wordPerDNA        = MC_CeilDivide16 ( maxDNALength );
    this->packedDNA         = index->sraIndex->hsp->packedDNA;
    this->fullDNALength     = index->sraIndex->hsp->dnaLength;
    this->index             = index;
    MC_CheckMalloc ( packedDNASeq,        uint,           batchSize * wordPerDNA );
    MC_CheckMalloc ( packedReadSeq,       uint,           batchSize * wordPerQuery );
    MC_CheckMalloc ( canInfos,            CandidateInfo,  batchSize );
    MC_CheckMalloc ( DNALengths,          uint,           batchSize );
    MC_CheckMalloc ( lengths,             uint,           batchSize );
    MC_CheckMalloc ( cutoffThresholds,    int,            batchSize );

    for ( int lOr = 0; lOr < 2; lOr++ )
    {
        MC_CheckMalloc ( scores[lOr],     int,            batchSize );
        MC_CheckMalloc ( hitLocs[lOr],    uint,           batchSize );
        MC_CheckMalloc ( pattern[lOr],    uchar,          batchSize * patternLength );
        MC_CheckMalloc ( maxScoreCounts[lOr],     uint,           batchSize );
    }

    MC_CheckMalloc ( softClipLtSizes,     uint,           batchSize );
    MC_CheckMalloc ( softClipRtSizes,     uint,           batchSize );
    MC_CheckMalloc ( peLeftAnchorLocs,    uint,           batchSize );
    MC_CheckMalloc ( peRightAnchorLocs,   uint,           batchSize );
    clear ();
}
PairEndAlignmentEngine::PairEndAlgnBatch::~PairEndAlgnBatch ()
{
    free ( packedDNASeq );
    free ( packedReadSeq );
    free ( canInfos );
    free ( DNALengths );
    free ( lengths );
    free ( cutoffThresholds );

    for ( int lOr = 0; lOr < 2; lOr++ )
    {
        free ( scores[lOr] );
        free ( hitLocs[lOr] );
        free ( pattern[lOr] );
        free ( maxScoreCounts[lOr] );
    }

    free ( softClipLtSizes );
    free ( softClipRtSizes );
    free ( peLeftAnchorLocs );
    free ( peRightAnchorLocs );
}
void PairEndAlignmentEngine::PairEndAlgnBatch::clear ()
{
    numOfThreads = 0;
}

int PairEndAlignmentEngine::PairEndAlgnBatch::packLeft (
    CandidateInfo & canInfo
)
{
    if ( numOfThreads >= batchSize )
    {
        return 0;
    }

    uint readIDLeft = canInfo.readIDLeft;
    uint readLength = upkdLengths[readIDLeft];
    int margin = DP2_MARGIN ( readLength );
    uint DNAStartLeft = canInfo.pos[0] - margin;

    if ( DNAStartLeft >= fullDNALength )
    {
        DNAStartLeft = 0;
    }

    uint DNALength = readLength + margin * 2;

    if ( DNAStartLeft + DNALength > fullDNALength )
    {
        DNALength = fullDNALength - DNAStartLeft;
    }

    // no anchor requirement
    peLeftAnchorLocs[numOfThreads] = maxDNALength;
    peRightAnchorLocs[numOfThreads] = 0;
    packRead ( packedReadSeq, numOfThreads,
               readIDLeft, readLength, peStrandLeftLeg );
    repackDNA ( packedDNASeq, numOfThreads,
                packedDNA, DNAStartLeft, DNALength );
    softClipLtSizes[numOfThreads] = ( peStrandLeftLeg == 1 ) ?
                                    softClipLeft : softClipRight;
    softClipRtSizes[numOfThreads] = ( peStrandLeftLeg == 1 ) ?
                                    softClipRight : softClipLeft;
    DNALengths[numOfThreads] = DNALength;
    lengths[numOfThreads] = readLength;
    cutoffThresholds[numOfThreads] = cutoffThreshold[readIDLeft & 1];
    canInfo.pos[0] = DNAStartLeft;
    canInfos[numOfThreads] = canInfo;
    ++numOfThreads;
    return 1;
}

void PairEndAlignmentEngine::PairEndAlgnBatch::packRight ()
{
    for ( int i = 0; i < numOfThreads; i++ )
    {
        uint readIDLeft = canInfos[i].readIDLeft;
        uint leftIsOdd = readIDLeft & 1;

        if ( scores[0][i] >= cutoffThreshold[leftIsOdd] )
        {
            uint readIDLeft = canInfos[i].readIDLeft;
            uint readIDRight = ( leftIsOdd ) ? ( readIDLeft - 1 ) : ( readIDLeft + 1 );
            uint readLength = upkdLengths[readIDRight];
            uint margin = DP2_MARGIN ( readLength );
            uint DNAStartRight = canInfos[i].pos[1] - margin;

            if ( DNAStartRight >= fullDNALength )
            {
                DNAStartRight = 0;
            }

            uint DNALength = readLength + margin * 2;

            if ( DNAStartRight + DNALength > fullDNALength )
            {
                DNALength = fullDNALength - DNAStartRight;
            }

            uint hitPosLeft = canInfos[i].pos[0] + hitLocs[0][i];
            // restrict maximum insert size
            uint boundedLength = hitPosLeft + insert_high - DNAStartRight;

            if ( boundedLength < DNALength ) \
                DNALength = boundedLength;

            // set pair-end anchor boundary, restrict minimum insert size
            peLeftAnchorLocs[i] = maxDNALength;
            int rightAnchor = hitPosLeft + insert_low - DNAStartRight;
            peRightAnchorLocs[i] = rightAnchor > 0 ? rightAnchor : 0;
            packRead ( packedReadSeq, i,
                       readIDRight, readLength, peStrandRightLeg );
            repackDNA ( packedDNASeq, i,
                        packedDNA, DNAStartRight, DNALength );
            softClipLtSizes[i] = ( peStrandRightLeg == 1 ) ?
                                 softClipLeft : softClipRight;
            softClipRtSizes[i] = ( peStrandRightLeg == 1 ) ?
                                 softClipRight : softClipLeft;
            DNALengths[i] = DNALength;
            lengths[i] = readLength;
            cutoffThresholds[i] = cutoffThreshold[readIDRight & 1];
            canInfos[i].pos[1] = DNAStartRight;
        }
    }
}

inline void PairEndAlignmentEngine::PairEndAlgnBatch::packRead (
    uint * packedSeq, uint threadId,
    uint readID, uint length, int strand
)
{
#define MC_OldReadUnpack(X,i) ((X[oldReadTPARA + (((i)>>4)<<5)] >> (((i) & 0xF) << 1)) & 0x3)
    uint oldReadTPARA = ( readID / 32 ) * 32 * wordPerOldQuery + ( readID % 32 );
    uint readTPARA = ( threadId / 32 ) * 32 * wordPerQuery + ( threadId % 32 );

    for ( uint i = 0; i <= ( length / CHAR_PER_WORD ); i++ )
    {
        packedSeq[readTPARA + ( i << 5 )] = 0;
    }

    if ( strand == 1 )
    {
        for ( int i = 1; i <= length; i++ )
        {
            int fwd_i = i - 1;
            register uint c_nucleotide = ( uint ) MC_OldReadUnpack ( queries, fwd_i );
#ifdef BS_MOD
            c_nucleotide = c_nucleotide ^ ( ( c_nucleotide == index->sraIndex->hsp->flag ) << 1 );
#endif
            packedSeq[readTPARA + ( ( i >> 4 ) << 5 )] |= c_nucleotide << ( ( 15 - ( i & 0xF ) ) << 1 );
        }
    }
    else   // strand == 2
    {
        for ( int i = 1; i <= length; i++ )
        {
            int rev_i = length - i;
            register uint c_nucleotide = soap3DnaComplement[ ( uint ) MC_OldReadUnpack ( queries, rev_i )];
#ifdef BS_MOD
            c_nucleotide = c_nucleotide ^ ( ( c_nucleotide == index->sraIndex->hsp->flag ) << 1 );
#endif
            packedSeq[readTPARA + ( ( i >> 4 ) << 5 )] |= c_nucleotide << ( ( 15 - ( i & 0xF ) ) << 1 );
        }
    }
}

inline void PairEndAlignmentEngine::PairEndAlgnBatch::repackDNA (
    uint * packedSeq, uint threadId,
    uint * seq, uint start, uint length
)
{
#define MC_OldDnaUnpack(X,i) ((X[(i)>>4] >> ((15-((i)&0xF))<<1)) & 3)
    uint dnaTPARA = ( threadId / 32 ) * 32 * wordPerDNA + ( threadId & 0x1F );

    for ( uint i = 0; i <= ( length / CHAR_PER_WORD ); i++ )
    {
        packedSeq[dnaTPARA + ( i << 5 )] = 0;
    }

    for ( int i = 1; i <= length; i++ )
    { packedSeq[dnaTPARA + ( ( i >> 4 ) << 5 )] |= ( uint ) ( MC_OldDnaUnpack ( seq, start + i - 1 ) ) << ( ( 15 - ( i & 0xF ) ) << 1 ); }
}

// ****
void PairEndAlignmentEngine::PairEndAlgnThreadContext::init (
    PairEndAlgnBatch * batch
)
{
    sem_init ( &ACKSem, 0, 0 );
    sem_init ( &GPUFinishSem, 0, 0 );
    sem_init ( &outputACKSem, 0, 0 );
    this->batch = batch;
}
void PairEndAlignmentEngine::PairEndAlgnThreadContext::freeMemory ()
{
    delete batch;
}

// ****
PairEndAlignmentEngine::AlgnmtResultStream::AlgnmtResultStream ()
{
    numOut = 0;
    pthread_mutex_init ( &occupy_mutex, NULL );
}

PairEndAlignmentEngine::AlgnmtResultStream::~AlgnmtResultStream ()
{
    for ( int i = 0; i < dp2Result.size (); i++ )
    {
        DP2ResultBatch & resultBatch = * ( dp2Result[i] );

        for ( int j = 0; j < resultBatch.size (); j++ )
        {
            free ( resultBatch[j].cigarString_1 );
            free ( resultBatch[j].cigarString_2 );
        }

        delete dp2Result[i];
    }

    dp2Result.clear ();
}

// ****
PairEndAlignmentEngine::PairEndAlignmentEngine () {}

void PairEndAlignmentEngine::performAlignment ( uint & numDPAlignedRead, uint & numDPAlignment )
{
    /* initialize */
    hipCtxPopCurrent ( & ( ctx ) );
    algnBatchCount = 0;
    dp2AlignedRead = 0;
    dp2Alignment = 0;
    lastReadID = -1;
    inputFlags = new AlgnmtFlags;
    alignFlags = new AlgnmtFlags;
    resultStream = new AlgnmtResultStream;
    outputBuf = new OutputBuffer<DeepDPAlignResult> ();
    outputBuf->setAlignmentType ( alignmentType );
    maxReadLength = ( inputMaxReadLength / 4 + 1 ) * 4;
    maxDNALength = maxReadLength + 2 * DP2_MARGIN ( inputMaxReadLength ) + 8;
    semiGlobalAligner.decideConfiguration ( maxReadLength, maxDNALength,
                                            maxDPTableLength, DP2_ALGN_NUM_OF_BLOCKS,
                                            patternLength, *dpPara );
    algnSwapBatch =
        new PairEndAlgnBatch ( DP2_ALGN_NUM_OF_BLOCKS * DP_THREADS_PER_BLOCK, dpPara,
                               peStrandLeftLeg, peStrandRightLeg, insert_high, insert_low,
                               maxReadLength, maxDNALength, maxDPTableLength, patternLength,
                               index, queries, inputMaxReadLength, upkdReadLengths );
    algnThreadContext = new PairEndAlgnThreadContext[dpPara->numOfCPUThreads];

    for ( int i = 0; i < dpPara->numOfCPUThreads; i++ )
    {
        PairEndAlgnBatch * batch =
            new PairEndAlgnBatch ( DP2_ALGN_NUM_OF_BLOCKS * DP_THREADS_PER_BLOCK, dpPara,
                                   peStrandLeftLeg, peStrandRightLeg, insert_high, insert_low,
                                   maxReadLength, maxDNALength, maxDPTableLength, patternLength,
                                   index, queries, inputMaxReadLength, upkdReadLengths );
        algnThreadContext[i].init ( batch );
    }

    algnmtGPUThreadDelegator.init ( 1, DP2GPUAlgnThread,
                                    DP2GPUAlgnThreadInit, DP2GPUAlgnThreadFinalize );
    outputThreadDelegator.init ( 1, DP2OutputThread,
                                 NULL, DP2OutputThreadFinalize );
    algnmtCPUThreadDelegator.init ( dpPara->numOfCPUThreads, DP2CPUAlgnThread );
    /* perform alignment */
    int threadId;
    void * empty;

    for ( uint i = 0; i < canStream->data.size (); i++ )
    {
        CandidateInfo & info = canStream->data[i];
        inputFlags->set ( ( info.readIDLeft >> 1 ) << 1 );

        if ( !algnSwapBatch->packLeft ( info ) )
        {
            // launch one batch
            threadId = algnmtCPUThreadDelegator.schedule ( empty );
            sem_wait ( & ( algnThreadContext[threadId].ACKSem ) );
            algnSwapBatch->clear ();
            algnSwapBatch->packLeft ( info );
        }
    }

    // last batch
    if ( algnSwapBatch->numOfThreads > 0 )
    {
        threadId = algnmtCPUThreadDelegator.schedule ( empty );
        sem_wait ( & ( algnThreadContext[threadId].ACKSem ) );
    }

    /* finalize */
    algnmtCPUThreadDelegator.finalize ();
    algnmtGPUThreadDelegator.finalize ();
    outputThreadDelegator.finalize ();
    alignFlags->getXOR ( inputFlags, unalignedIDStream->data );
    delete inputFlags;
    delete alignFlags;
    delete algnSwapBatch;

    for ( int i = 0; i < dpPara->numOfCPUThreads; i++ )
    {
        algnThreadContext[i].freeMemory ();
    }

    delete[] algnThreadContext;
    delete outputBuf;
    delete resultStream;
    numDPAlignedRead = this->dp2AlignedRead;
    numDPAlignment = this->dp2Alignment;
    hipCtxPushCurrent ( ctx );
}

void PairEndAlignmentEngine::performAlignment (
    /* input */
    CandidateStream   *   canStream,
    DPParameters     *    dpPara,
    uint * queries, uint * upkdReadLengths, int inputMaxReadLength,
    int insert_high, int insert_low,
    int peStrandLeftLeg, int peStrandRightLeg,
    char * upkdQueryNames, uint * origReadIDs, char * upkdQualities,
    Soap3Index * index,
    int alignmentType,
    uint accumReadNum, int outputFormat,
    FILE * outputFile, samfile_t * samOutputDPFilePtr,
    /* output */
    QueryIDStream    *    unalignedIDStream,
    uint         &        numDPAlignedRead,
    uint         &        numDPAlignment
)
{
    engine = new PairEndAlignmentEngine ();
    MC_MemberCopy2 ( engine->, , canStream, dpPara );
    MC_MemberCopy4 ( engine->, , queries, upkdQueryNames, upkdReadLengths, inputMaxReadLength );
    MC_MemberCopy4 ( engine->, , insert_high, insert_low, peStrandLeftLeg, peStrandRightLeg );
    MC_MemberCopy2 ( engine->, , origReadIDs, upkdQualities );
    MC_MemberCopy ( engine->, , index );
    MC_MemberCopy4 ( engine->, , accumReadNum, outputFormat, outputFile, samOutputDPFilePtr );
    MC_MemberCopy2 ( engine->, , alignmentType, unalignedIDStream );
    engine->performAlignment ( numDPAlignedRead, numDPAlignment );
    delete engine;
}
PairEndAlignmentEngine * PairEndAlignmentEngine::engine;

// ****
void DeepDP_Space::DP2GPUAlgnThreadInit ()
{
    PairEndAlignmentEngine * engine = PairEndAlignmentEngine::engine;
    hipCtxPushCurrent ( engine->ctx );
    //showGPUMemInfo("algn enter");
    int batchSize = engine->DP2_ALGN_NUM_OF_BLOCKS * DP_THREADS_PER_BLOCK;
    engine->semiGlobalAligner.init ( batchSize, engine->maxReadLength,
                                     engine->maxDNALength, engine->maxDPTableLength, * ( engine->dpPara ) );
}

void DeepDP_Space::DP2GPUAlgnThread ( int threadId, int *& pCallThreadId )
{
    PairEndAlignmentEngine * engine = PairEndAlignmentEngine::engine;
    PairEndAlignmentEngine::PairEndAlgnBatch * batch =
        engine->algnThreadContext[*pCallThreadId].batch;
    //  timeRecorder.appendStart("DP2");
    int lOr = batch->leftOrRight;
    engine->semiGlobalAligner.performAlignment (
        batch->packedDNASeq, batch->DNALengths,
        batch->packedReadSeq, batch->lengths,
        batch->cutoffThresholds, batch->scores[lOr], batch->hitLocs[lOr],
        batch->maxScoreCounts[lOr],
        batch->pattern[lOr], batch->numOfThreads,
        batch->softClipLtSizes, batch->softClipRtSizes,
        batch->peLeftAnchorLocs, batch->peRightAnchorLocs
    );
    //  timeRecorder.appendEnd("DP2");
    sem_post ( & ( engine->algnThreadContext[*pCallThreadId].GPUFinishSem ) );
}

void DeepDP_Space::DP2GPUAlgnThreadFinalize ()
{
    PairEndAlignmentEngine * engine = PairEndAlignmentEngine::engine;
    engine->semiGlobalAligner.freeMemory ();
    hipCtxPopCurrent ( & ( engine->ctx ) );
}

void DeepDP_Space::DP2CPUAlgnThread ( int threadId, void *& empty )
{
    PairEndAlignmentEngine * engine = PairEndAlignmentEngine::engine;
    PairEndAlignmentEngine::PairEndAlgnBatch * batch = engine->algnSwapBatch;
    engine->algnSwapBatch = engine->algnThreadContext[threadId].batch;
    engine->algnThreadContext[threadId].batchID = engine->algnBatchCount++;
    sem_post ( & ( engine->algnThreadContext[threadId].ACKSem ) );
    engine->algnThreadContext[threadId].batch = batch;
    int * pThreadId = &threadId;
    // align left side
    batch->leftOrRight = 0;
    engine->algnmtGPUThreadDelegator.schedule ( pThreadId );
    sem_wait ( & ( engine->algnThreadContext[threadId].GPUFinishSem ) );
    // align right side
    batch->packRight ();
    batch->leftOrRight = 1;
    engine->algnmtGPUThreadDelegator.schedule ( pThreadId );
    sem_wait ( & ( engine->algnThreadContext[threadId].GPUFinishSem ) );
    MC_MemberCopy2 ( int, engine->dpPara->, matchScore, mismatchScore );
    MC_MemberCopy2 ( int, engine->dpPara->, openGapScore, extendGapScore );
    int cutoffThreshold[2];
    cutoffThreshold[0] = engine->dpPara->paramRead[0].cutoffThreshold;
    cutoffThreshold[1] = engine->dpPara->paramRead[1].cutoffThreshold;
    // rearrange result and Output
    vector<DeepDPAlignResult> * resultBatch = new vector<DeepDPAlignResult> ();

    for ( int i = 0; i < batch->numOfThreads; i++ )
    {
        int readSide = batch->canInfos[i].readIDLeft & 1;
        int mateSide = 1 - readSide;

        if ( batch->scores[0][i] >= cutoffThreshold[readSide] &&
                batch->scores[1][i] >= cutoffThreshold[mateSide] )
        {
            char * cigarString[2];
            int editdist[2], DIS[2];

            for ( int lOr = 0; lOr < 2; lOr++ )
            {
                CigarStringEncoder<void> encoder;
                uchar lastType = 'N';

                for ( uchar * p = batch->pattern[lOr] + i * engine->patternLength; *p != 0; p++ )
                {
                    if ( *p == 'V' )
                    {
                        encoder.append ( lastType, ( int ) ( * ( ++p ) ) - 1 );
                    }
                    else
                    {
                        encoder.append ( *p, 1 );
                        lastType = *p;
                    }
                }

                encoder.encodeCigarString ( openGapScore, extendGapScore );
                cigarString[lOr] = encoder.cigarString;
                // To get edit distance
                int L = batch->lengths[i] - encoder.charCount['I'] - encoder.charCount['S'];
                int numOfMismatch = ( L * matchScore + encoder.gapPenalty - batch->scores[lOr][i] ) /
                                    ( matchScore - mismatchScore );
                editdist[lOr] = encoder.charCount['I'] + encoder.charCount['D'] + numOfMismatch;
                DIS[lOr] = encoder.charCount['D'] - encoder.charCount['I'] - encoder.charCount['S'];
            }

            //#define MC_GetMateID(x) (((x)&1)?((x)-1):((x)+1))
            DeepDPAlignResult result;
            result.readID = batch->canInfos[i].readIDLeft - readSide;
            result.strand_1 = ( ( readSide == 0 ) ? engine->peStrandLeftLeg : engine->peStrandRightLeg );
            result.strand_2 = ( ( mateSide == 0 ) ? engine->peStrandLeftLeg : engine->peStrandRightLeg );
            result.algnmt_1 = batch->canInfos[i].pos[readSide] + batch->hitLocs[readSide][i];
            result.algnmt_2 = batch->canInfos[i].pos[mateSide] + batch->hitLocs[mateSide][i];
            result.score_1 = batch->scores[readSide][i];
            result.score_2 = batch->scores[mateSide][i];
            result.cigarString_1 = cigarString[readSide];
            result.cigarString_2 = cigarString[mateSide];
            result.editdist_1 = editdist[readSide];
            result.editdist_2 = editdist[mateSide];

            if ( result.algnmt_1 < result.algnmt_2 )
                result.insertSize = result.algnmt_2 - result.algnmt_1 +
                                    batch->lengths[i] + DIS[mateSide];
            else
                result.insertSize = result.algnmt_1 - result.algnmt_2 +
                                    batch->lengths[i] + DIS[readSide];

            result.num_sameScore_1 = batch->maxScoreCounts[readSide][i]; //TODO
            result.num_sameScore_2 = batch->maxScoreCounts[mateSide][i];
            resultBatch->push_back ( result );
        }
    }

    // Output
    engine->algnThreadContext[threadId].resultBatch = resultBatch;
    int * pid = &threadId;
    engine->outputThreadDelegator.schedule ( pid );
    sem_wait ( & ( engine->algnThreadContext[threadId].outputACKSem ) );
}

void DeepDP_Space::DP2OutputThread ( int threadId, int *& pCallThreadId )
{
    PairEndAlignmentEngine * engine = PairEndAlignmentEngine::engine;
    int callThreadId = *pCallThreadId;
    int batchID = engine->algnThreadContext[callThreadId].batchID;
    DP2ResultBatch * resultBatch = engine->algnThreadContext[callThreadId].resultBatch;
    sem_post ( & ( engine->algnThreadContext[callThreadId].outputACKSem ) );
    vector<DP2ResultBatch *> & dpResult = engine->resultStream->dp2Result;

    while ( dpResult.size () <= batchID )
    {
        dpResult.push_back ( NULL );
    }

    dpResult[batchID] = resultBatch;
#define MC_DP2OutputRead() { \
        engine->outputBuf->ready(); \
        if (engine->outputBuf->size > 0) { \
            outputDeepDPResult2(engine->outputBuf->elements, engine->outputBuf->size, \
                                engine->queries, engine->upkdReadLengths, \
                                engine->origReadIDs, engine->upkdQueryNames, engine->upkdQualities, \
                                engine->inputMaxReadLength, engine->accumReadNum, engine->outputFormat, \
                                engine->outputFile, engine->samOutputDPFilePtr, engine->index, \
                                engine->peStrandLeftLeg, engine->peStrandRightLeg); \
            engine->dp2AlignedRead += 1; \
            engine->dp2Alignment += engine->outputBuf->size; \
            engine->alignFlags->set(engine->lastReadID << 1); \
        } \
    }
    uint numOut = engine->resultStream->numOut;

    while ( numOut < dpResult.size () && dpResult[numOut] != NULL )
    {
        //OUTPUT HERE
        DP2ResultBatch & batch = *dpResult[numOut];

        for ( int i = 0; i < batch.size (); i++ )
        {
            DeepDPAlignResult & result = batch[i];
            int pairID = result.readID >> 1;

            if ( pairID != engine->lastReadID )
            {
                MC_DP2OutputRead ();
                engine->outputBuf->clear ();
                engine->lastReadID = pairID;
            }

            engine->outputBuf->add ( result );
        }

        ++numOut;
    }

    engine->resultStream->numOut = numOut;
}

void DeepDP_Space::DP2OutputThreadFinalize ()
{
    PairEndAlignmentEngine * engine = PairEndAlignmentEngine::engine;
    // last read
    MC_DP2OutputRead ();
    engine->outputBuf->clear ();
}




// Temporary data for testing
// DPParameters Constants::deepDPPara_Len100;

Constants::Constants ()
{
    /*  deepDPPara_Len100.paramRead[0].cutoffThreshold = 30;
        deepDPPara_Len100.paramRead[0].maxHitNum = 100;
        deepDPPara_Len100.paramRead[0].seedLength = 26;
        deepDPPara_Len100.paramRead[0].sampleDist = 13;
    */
}

Constants constants;

