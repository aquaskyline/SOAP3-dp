#include "hip/hip_runtime.h"
/*
 *
 *    DV-Kernel.cu
 *    Soap3(gpu)
 *
 *    Copyright (C) 2011, HKU
 *
 *    This program is free software; you can redistribute it and/or
 *    modify it under the terms of the GNU General Public License
 *    as published by the Free Software Foundation; either version 2
 *    of the License, or (at your option) any later version.
 *
 *    This program is distributed in the hope that it will be useful,
 *    but WITHOUT ANY WARRANTY; without even the implied warranty of
 *    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *    GNU General Public License for more details.
 *
 *    You should have received a copy of the GNU General Public License
 *    along with this program; if not, write to the Free Software
 *    Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */

#include "DV-Kernel.h"

// copied from DNACount.c
__forceinline__ __device__ uint GPUDNAOccCount ( uint * dna, uint index, char character, uint backward )
{
    uint wordToCount, charToCount;
    uint i;
    uint sum = 0;
    wordToCount = index / 32;
    charToCount = index - wordToCount * 32;
    dna -= backward * 4;
    ulonglong2 dd;
    //ulonglong2 is a 16Byte = 128bit CUDA vector
    //.x is the first 8Byte while .y is the last.
    dd = * ( ( ulonglong2 * ) dna );
    i = 0;

    if ( wordToCount > 0 ) // i = 0
    {
        unsigned long long d = backward ? dd.y : dd.x;
        unsigned long long b = ( d >> 1 );
        b ^= ( ( character & 0x2 ) ? 0 : 0xFFFFFFFFFFFFFFFF );
        unsigned long long a = d ^ ( ( character & 0x1 ) ? 0 : 0xFFFFFFFFFFFFFFFF );
        a &= b;
        a &= 0x5555555555555555;
        sum += __popcll ( a );
        i = 1;
    }

    if ( wordToCount == 2 || charToCount > 0 )
    {
        if ( backward ) { i = 3 - i; }

        if ( wordToCount == 2 ) { charToCount = 32; }

        unsigned long long d = ( i & 1 ) ? dd.y : dd.x;
        unsigned long long b = ( d >> 1 );
        b ^= ( ( character & 0x2 ) ? 0 : 0xFFFFFFFFFFFFFFFF );
        unsigned long long a = d ^ ( ( character & 0x1 ) ? 0 : 0xFFFFFFFFFFFFFFFF );
        a &= b;
        unsigned long long mask;

        if ( charToCount < 16 )
        {
            mask = ( 0x00000000FFFFFFFF << ( 32 - charToCount * 2 ) ) & 0x00000000FFFFFFFF;
        }
        else
        {
            mask = ( 0xFFFFFFFF00000000 << ( 64 - charToCount * 2 ) ) | 0x00000000FFFFFFFF;
        }

        if ( backward )
        { mask = __brevll ( mask ); }

        a &= ( mask & 0x5555555555555555 );
        sum += __popcll ( a );
    }

    return sum;
}

__forceinline__ __device__ void GPUDNAAllOccCount ( uint * dna, uint index, uint backward, uint occCount[] )
{
    uint wordToCount, charToCount;
    uint i;
    wordToCount = index / 32;
    charToCount = index - wordToCount * 32;
    dna -= backward * 4;
    ulonglong2 dd;
    dd = * ( ( ulonglong2 * ) dna );
    i = 0;

    if ( wordToCount > 0 ) // i = 0
    {
        unsigned long long d = backward ? dd.y : dd.x;

        for ( int j = 0; j < ALPHABET_SIZE; ++j )
        {
            unsigned long long b = ( d >> 1 );
            b ^= ( ( j & 0x2 ) ? 0 : 0xFFFFFFFFFFFFFFFF );
            unsigned long long a = d ^ ( ( j & 0x1 ) ? 0 : 0xFFFFFFFFFFFFFFFF );
            a &= b;
            a &= 0x5555555555555555;

            if ( backward )
            { occCount[j] -= __popcll ( a ); }
            else
            { occCount[j] += __popcll ( a ); }
        }

        i = 1;
    }

    if ( wordToCount == 2 || charToCount > 0 )
    {
        if ( backward ) { i = 3 - i; }

        if ( wordToCount == 2 ) { charToCount = 32; }

        unsigned long long d = ( i & 1 ) ? dd.y : dd.x;

        for ( int j = 0; j < ALPHABET_SIZE; ++j )
        {
            unsigned long long b = ( d >> 1 );
            b ^= ( ( j & 0x2 ) ? 0 : 0xFFFFFFFFFFFFFFFF );
            unsigned long long a = d ^ ( ( j & 0x1 ) ? 0 : 0xFFFFFFFFFFFFFFFF );
            a &= b;
            unsigned long long mask;

            if ( charToCount < 16 )
            {
                mask = ( 0x00000000FFFFFFFF << ( 32 - charToCount * 2 ) ) & 0x00000000FFFFFFFF;
            }
            else
            {
                mask = ( 0xFFFFFFFF00000000 << ( 64 - charToCount * 2 ) ) | 0x00000000FFFFFFFF;
            }

            if ( backward )
            { mask = __brevll ( mask ); }

            a &= ( mask & 0x5555555555555555 );

            if ( backward )
            { occCount[j] -= __popcll ( a ); }
            else
            { occCount[j] += __popcll ( a ); }
        }
    }
}

__forceinline__ __device__ uint GPUDNAOccCountWithCumu ( uint * dna, uint index, char c, uint backward, uint & cumu )
{
    uint wordToCount, charToCount;
    uint i;
    uint sum = 0;
    wordToCount = index / 32;
    charToCount = index - wordToCount * 32;
    dna -= backward * 4;
    ulonglong2 dd;
    dd = * ( ( ulonglong2 * ) dna );
    i = 0;

    if ( wordToCount > 0 ) // i = 0
    {
        unsigned long long d = backward ? dd.y : dd.x;

        for ( int j = 3; j > c; --j ) // TODO hardcode
        {
            unsigned long long b = ( d >> 1 );
            b ^= ( ( j & 0x2 ) ? 0 : 0xFFFFFFFFFFFFFFFF );
            unsigned long long a = d ^ ( ( j & 0x1 ) ? 0 : 0xFFFFFFFFFFFFFFFF );
            a &= b;
            a &= 0x5555555555555555;
            cumu += __popcll ( a );
        }

        {
            // copy&paste for c
            unsigned long long b = ( d >> 1 );
            b ^= ( ( c & 0x2 ) ? 0 : 0xFFFFFFFFFFFFFFFF );
            unsigned long long a = d ^ ( ( c & 0x1 ) ? 0 : 0xFFFFFFFFFFFFFFFF );
            a &= b;
            a &= 0x5555555555555555;
            sum += __popcll ( a );
        }

        i = 1;
    }

    if ( wordToCount == 2 || charToCount > 0 )
    {
        if ( backward ) { i = 3 - i; }

        if ( wordToCount == 2 ) { charToCount = 32; }

        unsigned long long d = ( i & 1 ) ? dd.y : dd.x;

        for ( int j = 3; j > c; --j ) // hardcode
        {
            unsigned long long b = ( d >> 1 );
            b ^= ( ( j & 0x2 ) ? 0 : 0xFFFFFFFFFFFFFFFF );
            unsigned long long a = d ^ ( ( j & 0x1 ) ? 0 : 0xFFFFFFFFFFFFFFFF );
            a &= b;
            unsigned long long mask;

            if ( charToCount < 16 )
            {
                mask = ( 0x00000000FFFFFFFF << ( 32 - charToCount * 2 ) ) & 0x00000000FFFFFFFF;
            }
            else
            {
                mask = ( 0xFFFFFFFF00000000 << ( 64 - charToCount * 2 ) ) | 0x00000000FFFFFFFF;
            }

            if ( backward )
            { mask = __brevll ( mask ); }

            a &= ( mask & 0x5555555555555555 );
            cumu += __popcll ( a );
        }

        {
            // copy&paste for c
            unsigned long long b = ( d >> 1 );
            b ^= ( ( c & 0x2 ) ? 0 : 0xFFFFFFFFFFFFFFFF );
            unsigned long long a = d ^ ( ( c & 0x1 ) ? 0 : 0xFFFFFFFFFFFFFFFF );
            a &= b;
            unsigned long long mask;

            if ( charToCount < 16 )
            {
                mask = ( 0x00000000FFFFFFFF << ( 32 - charToCount * 2 ) ) & 0x00000000FFFFFFFF;
            }
            else
            {
                mask = ( 0xFFFFFFFF00000000 << ( 64 - charToCount * 2 ) ) | 0x00000000FFFFFFFF;
            }

            if ( backward )
            { mask = __brevll ( mask ); }

            a &= ( mask & 0x5555555555555555 );
            sum += __popcll ( a );
        }
    }

    return sum;
}


// copied from BWT.c
__forceinline__ __device__ uint GPUBWTOccValue ( uint * bwt, uint * occ, uint index, char c, uint inverseSa0 )
{
    // $ is supposed to be positioned at inverseSa0 but it is not encoded
    // therefore index is subtracted by 1 for adjustment
    index -= ( index > inverseSa0 );
    uint occExplicitIndex = ( index + GPU_OCC_INTERVAL / 2 - 1 ) / GPU_OCC_INTERVAL; // Bidirectional encoding
    uint occIndex = occExplicitIndex * GPU_OCC_INTERVAL;
    uint occValue = occ[occExplicitIndex * ALPHABET_SIZE + c];

    if ( occIndex != index )
    {
        // __usad(x,y,z) = |x-y| + z
        // GPUDNAOccCount(explicitIndex, len, character, isBackwardDirectionCount)
        //   - pointer to position of explicitIndex on BWT: Reference point on sampled occ
        //   - len: number of characters on BWT to count
        //   - character: the character to count
        //   - isBackwardDirectionCount: a boolean flag to identify backward counting
        uint cnt = GPUDNAOccCount ( bwt + occIndex / CHAR_PER_WORD, __usad ( index, occIndex, 0 ), c, occIndex > index );
        return occIndex > index ? occValue - cnt : occValue + cnt;
    }
    else
    {
        return occValue;
    }
}

__forceinline__ __device__ void GPUBWTAllOccValue ( uint * bwt,
        uint * occ,
        uint inverseSa0,
        uint index,
        uint occCount[] )
{
    // $ is supposed to be positioned at inverseSa0 but it is not encoded
    // therefore index is subtracted by 1 for adjustment
    index -= ( index > inverseSa0 );
    uint occExplicitIndex = ( index + GPU_OCC_INTERVAL / 2 - 1 ) / GPU_OCC_INTERVAL; // Bidirectional encoding
    uint occIndex = occExplicitIndex * GPU_OCC_INTERVAL;
    * ( ( uint4 * ) occCount ) = * ( ( uint4 * ) ( occ + occExplicitIndex * ALPHABET_SIZE ) );

    if ( occIndex != index )
    {
        GPUDNAAllOccCount ( bwt + occIndex / CHAR_PER_WORD, __usad ( index, occIndex, 0 ), occIndex > index, occCount );
    }
}

__forceinline__ __device__ uint GPUBWTOccValueWithCumu ( uint * bwt, uint * occ, uint index, char c, uint inverseSa0, uint & cumu )
{
    // $ is supposed to be positioned at inverseSa0 but it is not encoded
    // therefore index is subtracted by 1 for adjustment
    index -= ( index > inverseSa0 );
    uint occExplicitIndex = ( index + GPU_OCC_INTERVAL / 2 - 1 ) / GPU_OCC_INTERVAL; // Bidirectional encoding
    uint occIndex = occExplicitIndex * GPU_OCC_INTERVAL;
    uint occValues[ALPHABET_SIZE];
    * ( ( uint4 * ) occValues ) = * ( ( uint4 * ) ( occ + occExplicitIndex * ALPHABET_SIZE ) );
    uint occValue = occValues[c];
    cumu = 0;

    for ( int j = 3; j > c; --j ) // TODO hardcode
    {
        cumu += occValues[j];
    }

    uint cum = 0;

    if ( occIndex != index )
    {
        uint cnt = GPUDNAOccCountWithCumu ( bwt + occIndex / CHAR_PER_WORD, __usad ( index, occIndex, 0 ), c, occIndex > index, cum );

        if ( occIndex > index )
        {
            cumu -= cum;
            return occValue - cnt;
        }

        cumu += cum;
        return occValue + cnt;
    }

    return occValue;
}

__forceinline__ __device__
void contBackwardSearch ( uint * query, uint start, uint len,
                          uint * bwt, uint * occ, uint inverseSa0,
                          uint saL, uint saR,
                          uint & saCount, uint * output,
                          uint maxSARangesAllowed,
                          uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;

    for ( i = start + len; i > start && saL <= saR; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        saL = GPUBWTOccValue ( bwt, occ, saL, c, inverseSa0 ) + 1;
        saR = GPUBWTOccValue ( bwt, occ, saR + 1, c, inverseSa0 );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
        printf ( "[Global] Address %u\n", output );
        printf ( "[contBackwardSearch] Reporting of SA ranges %u %u ...", saL, saR );
#endif

        if ( saCount < maxSARangesAllowed )
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "DONE\n" );
#endif
            * ( output + 32 * 2 * saCount ) = saL;
            * ( output + 32 * ( 2 * saCount + 1 ) ) = ( saR - saL ) + ( strand << ( BGS_GPU_ANSWER_OFFSET_LENGTH + 3 ) ) +
                    ( accumMismatches << BGS_GPU_ANSWER_OFFSET_LENGTH );
        }
        else
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "FAILED\n" );
#endif
        }

        ++saCount;
    }
}

__forceinline__ __device__
void revContForwardSearch ( uint * query, uint start, uint len,
                            uint * revBwt, uint * revOcc, uint revInverseSa0,
                            uint saL, uint saR,
                            uint revSaL, uint revSaR,
                            uint & saCount, uint * output,
                            uint maxSARangesAllowed,
                            uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;

    for ( i = start; i < start + len && saL <= saR; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        uint start, end;
        revSaL =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaL, c, revInverseSa0, start ) + 1;
        revSaR =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaR + 1, c, revInverseSa0, end );
        saR = saR + start - end;
        saL = saR - ( revSaR - revSaL );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
        printf ( "[Global] Address %u\n", output );
        printf ( "[revContForwardSearch] Reporting of SA ranges %u %u ...", saL, saR );
#endif

        if ( saCount < maxSARangesAllowed )
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "DONE\n" );
#endif
            * ( output + 32 * 2 * saCount ) = saL;
            * ( output + 32 * ( 2 * saCount + 1 ) ) = ( saR - saL ) + ( strand << ( BGS_GPU_ANSWER_OFFSET_LENGTH + 3 ) ) +
                    ( accumMismatches << BGS_GPU_ANSWER_OFFSET_LENGTH );
        }
        else
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "FAILED\n" );
#endif
        }

        ++saCount;
    }
}


__forceinline__ __device__
void backward1Mismatch ( uint * query, uint start, uint len,
                         uint * bwt, uint * occ, uint inverseSa0,
                         uint pl, uint pr,
                         uint & saCount, uint * output,
                         uint maxSARangesAllowed,
                         uint strand, uint accumMismatches )
{
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr  && saCount <= maxSARangesAllowed; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            uint mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            uint mkR = occCountPEnd[ec];
            contBackwardSearch ( query, start, i - start - 1,
                                 bwt, occ, inverseSa0,
                                 mkL, mkR,
                                 saCount, output,
                                 maxSARangesAllowed,
                                 strand, accumMismatches + 1 );

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
    }
}

__forceinline__ __device__
void backward1MismatchAndExact ( uint * query, uint start, uint len,
                                 uint * bwt, uint * occ, uint inverseSa0,
                                 uint pl, uint pr,
                                 uint & saCount, uint * output,
                                 uint maxSARangesAllowed,
                                 uint strand, uint accumMismatches )
{
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr && saCount <= maxSARangesAllowed; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            uint mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            uint mkR = occCountPEnd[ec];

            if ( mkL <= mkR )
            {
                contBackwardSearch ( query, start, i - start - 1,
                                     bwt, occ, inverseSa0,
                                     mkL, mkR,
                                     saCount, output,
                                     maxSARangesAllowed, strand, accumMismatches + 1 );
            }

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
    }

    if ( pl <= pr && i == start && saCount <= maxSARangesAllowed )
    {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
        printf ( "[Global] Address %u\n", output );
        printf ( "[backward1MismatchOrExact] Reporting of SA ranges %u %u ...", pl, pr );
#endif

        if ( saCount < maxSARangesAllowed )
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "DONE\n" );
#endif
            * ( output + 32 * 2 * saCount ) = pl;
            * ( output + 32 * ( 2 * saCount + 1 ) ) = ( pr - pl ) + ( strand << ( BGS_GPU_ANSWER_OFFSET_LENGTH + 3 ) ) +
                    ( accumMismatches << BGS_GPU_ANSWER_OFFSET_LENGTH );
        }
        else
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "FAILED\n" );
#endif
        }

        ++saCount;
    }
}

__forceinline__ __device__
void backward2Mismatch ( uint * query, uint start, uint len,
                         uint * bwt, uint * occ, uint inverseSa0,
                         uint pl, uint pr,
                         uint & saCount, uint * output,
                         uint maxSARangesAllowed,
                         uint strand, uint accumMismatches )
{
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            uint mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            uint mkR = occCountPEnd[ec];
            backward1Mismatch ( query, start, i - start - 1,
                                bwt, occ, inverseSa0,
                                mkL, mkR,
                                saCount, output,
                                maxSARangesAllowed,
                                strand, accumMismatches + 1 );

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
    }
}

__forceinline__ __device__
void backward2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
                                     uint * bwt, uint * occ, uint inverseSa0,
                                     uint pl, uint pr,
                                     uint & saCount, uint * output,
                                     uint maxSARangesAllowed,
                                     uint strand, uint accumMismatches )
{
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr && saCount <= maxSARangesAllowed; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            uint mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            uint mkR = occCountPEnd[ec];

            if ( mkL <= mkR )
            {
                backward1MismatchAndExact ( query, start, i - start - 1,
                                            bwt, occ, inverseSa0,
                                            mkL, mkR,
                                            saCount, output,
                                            maxSARangesAllowed,
                                            strand, accumMismatches + 1 );
            }

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
    }
}

__forceinline__ __device__
void backward2MismatchAnd1MismatchAndExact ( uint * query, uint start, uint len,
        uint * bwt, uint * occ, uint inverseSa0,
        uint pl, uint pr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed,
        uint strand, uint accumMismatches )
{
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr && saCount <= maxSARangesAllowed; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            uint mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            uint mkR = occCountPEnd[ec];

            if ( mkL <= mkR )
            {
                backward1MismatchAndExact ( query, start, i - start - 1,
                                            bwt, occ, inverseSa0,
                                            mkL, mkR,
                                            saCount, output,
                                            maxSARangesAllowed,
                                            strand, accumMismatches + 1 );
            }

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
    }

    if ( pl <= pr && i == start && saCount <= maxSARangesAllowed )
    {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
        printf ( "[Global] Address %u\n", output );
        printf ( "[backward1MismatchOrExact] Reporting of SA ranges %u %u ...", pl, pr );
#endif

        if ( saCount < maxSARangesAllowed )
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "DONE\n" );
#endif
            * ( output + 32 * 2 * saCount ) = pl;
            * ( output + 32 * ( 2 * saCount + 1 ) ) = ( pr - pl ) + ( strand << ( BGS_GPU_ANSWER_OFFSET_LENGTH + 3 ) ) +
                    ( accumMismatches << BGS_GPU_ANSWER_OFFSET_LENGTH );
        }
        else
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "FAILED\n" );
#endif
        }

        ++saCount;
    }
}

__forceinline__ __device__
void backward3Mismatch ( uint * query, uint start, uint len,
                         uint * bwt, uint * occ, uint inverseSa0,
                         uint pl, uint pr,
                         uint & saCount, uint * output,
                         uint maxSARangesAllowed,
                         uint strand, uint accumMismatches )
{
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            uint mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            uint mkR = occCountPEnd[ec];
            backward2Mismatch ( query, start, i - start - 1,
                                bwt, occ, inverseSa0,
                                mkL, mkR,
                                saCount, output,
                                maxSARangesAllowed,
                                strand, accumMismatches + 1 );

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
    }
}

__forceinline__ __device__
void backward3MismatchAnd2Mismatch ( uint * query, uint start, uint len,
                                     uint * bwt, uint * occ, uint inverseSa0,
                                     uint pl, uint pr,
                                     uint & saCount, uint * output,
                                     uint maxSARangesAllowed,
                                     uint strand, uint accumMismatches )
{
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr && saCount <= maxSARangesAllowed; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            uint mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            uint mkR = occCountPEnd[ec];

            if ( mkL <= mkR )
            {
                backward2MismatchAnd1Mismatch ( query, start, i - start - 1,
                                                bwt, occ, inverseSa0,
                                                mkL, mkR,
                                                saCount, output,
                                                maxSARangesAllowed,
                                                strand, accumMismatches + 1 );
            }

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
    }
}

__forceinline__ __device__
void backward3MismatchAnd2MismatchAnd1MismatchAndExact ( uint * query, uint start, uint len,
        uint * bwt, uint * occ, uint inverseSa0,
        uint pl, uint pr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed,
        uint strand, uint accumMismatches )
{
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr && saCount <= maxSARangesAllowed; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            uint mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            uint mkR = occCountPEnd[ec];

            if ( mkL <= mkR )
            {
                backward2MismatchAnd1MismatchAndExact ( query, start, i - start - 1,
                                                        bwt, occ, inverseSa0,
                                                        mkL, mkR,
                                                        saCount, output,
                                                        maxSARangesAllowed,
                                                        strand, accumMismatches + 1 );
            }

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
    }

    if ( pl <= pr && i == start && saCount <= maxSARangesAllowed )
    {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
        printf ( "[Global] Address %u\n", output );
        printf ( "[backward3MismatchAnd2MismatchAnd1MismatchAndExact] Reporting of SA ranges %u %u ...", pl, pr );
#endif

        if ( saCount < maxSARangesAllowed )
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "DONE\n" );
#endif
            * ( output + 32 * 2 * saCount ) = pl;
            * ( output + 32 * ( 2 * saCount + 1 ) ) = ( pr - pl ) + ( strand << ( BGS_GPU_ANSWER_OFFSET_LENGTH + 3 ) ) +
                    ( accumMismatches << BGS_GPU_ANSWER_OFFSET_LENGTH );
        }
        else
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "FAILED\n" );
#endif
        }

        ++saCount;
    }
}


__forceinline__ __device__
void backward4MismatchAnd3MismatchAnd2MismatchAnd1MismatchAndExact ( uint * query, uint start, uint len,
        uint * bwt, uint * occ, uint inverseSa0,
        uint pl, uint pr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed,
        uint strand, uint accumMismatches )
{
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            uint mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            uint mkR = occCountPEnd[ec];
            backward3MismatchAnd2MismatchAnd1MismatchAndExact ( query, start, i - start - 1,
                    bwt, occ, inverseSa0,
                    mkL, mkR,
                    saCount, output,
                    maxSARangesAllowed,
                    strand, accumMismatches + 1 );

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
    }

    if ( pl <= pr && i == start && saCount <= maxSARangesAllowed )
    {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
        printf ( "[Global] Address %u\n", output );
        printf ( "[backward4MismatchAnd3MismatchAnd2MismatchAnd1MismatchAndExact] Reporting of SA ranges %u %u ...", pl, pr );
#endif

        if ( saCount < maxSARangesAllowed )
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "DONE\n" );
#endif
            * ( output + 32 * 2 * saCount ) = pl;
            * ( output + 32 * ( 2 * saCount + 1 ) ) = ( pr - pl ) + ( strand << ( BGS_GPU_ANSWER_OFFSET_LENGTH + 3 ) ) +
                    ( accumMismatches << BGS_GPU_ANSWER_OFFSET_LENGTH );
        }
        else
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "FAILED\n" );
#endif
        }

        ++saCount;
    }
}


__forceinline__ __device__
void revForward1Mismatch ( uint * query, uint start, uint len,
                           uint * revBwt, uint * revOcc, uint revInverseSa0,
                           uint pl, uint pr,
                           uint revPl, uint revPr,
                           uint & saCount, uint * output,
                           uint maxSARangesAllowed,
                           uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr && saCount <= maxSARangesAllowed; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revContForwardSearch ( query, i + 1, start + len - i - 1,
                                       // search range = query[i+1, start+len)
                                       revBwt, revOcc, revInverseSa0,
                                       mkL, mkR,
                                       revMkL, revMkR,
                                       saCount, output,
                                       maxSARangesAllowed,
                                       strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }
}

__forceinline__ __device__
void revForward1MismatchAndExact ( uint * query, uint start, uint len,
                                   uint * revBwt, uint * revOcc, uint revInverseSa0,
                                   uint pl, uint pr,
                                   uint revPl, uint revPr,
                                   uint & saCount, uint * output,
                                   uint maxSARangesAllowed,
                                   uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr && saCount <= maxSARangesAllowed; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revContForwardSearch ( query, i + 1, start + len - i - 1,
                                       // search range = query[i+1, start+len)
                                       revBwt, revOcc, revInverseSa0,
                                       mkL, mkR,
                                       revMkL, revMkR,
                                       saCount, output,
                                       maxSARangesAllowed,
                                       strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }

    if ( pl <= pr && saCount <= maxSARangesAllowed )
    {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
        printf ( "[Global] Address %u\n", output );
        printf ( "[backward3MismatchAndExact] Reporting of SA ranges %u %u ...", pl, pr );
#endif

        if ( saCount < maxSARangesAllowed )
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "DONE\n" );
#endif
            * ( output + 32 * 2 * saCount ) = pl;
            * ( output + 32 * ( 2 * saCount + 1 ) ) = ( pr - pl ) + ( strand << ( BGS_GPU_ANSWER_OFFSET_LENGTH + 3 ) ) +
                    ( accumMismatches << BGS_GPU_ANSWER_OFFSET_LENGTH );
        }
        else
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "FAILED\n" );
#endif
        }

        ++saCount;
    }
}

__forceinline__ __device__
void revForward2Mismatch ( uint * query, uint start, uint len,
                           uint * revBwt, uint * revOcc, uint revInverseSa0,
                           uint pl, uint pr,
                           uint revPl, uint revPr,
                           uint & saCount, uint * output,
                           uint maxSARangesAllowed,
                           uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr && saCount <= maxSARangesAllowed; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revForward1Mismatch ( query, i + 1, start + len - i - 1,
                                      // search range = query[i+1, start+len)
                                      revBwt, revOcc, revInverseSa0,
                                      mkL, mkR,
                                      revMkL, revMkR,
                                      saCount, output,
                                      maxSARangesAllowed,
                                      strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }
}

__forceinline__ __device__
void revForward2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
                                       uint * revBwt, uint * revOcc, uint revInverseSa0,
                                       uint pl, uint pr,
                                       uint revPl, uint revPr,
                                       uint & saCount, uint * output,
                                       uint maxSARangesAllowed,
                                       uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr && saCount <= maxSARangesAllowed; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revForward1MismatchAndExact ( query, i + 1, start + len - i - 1,
                                              // search range = query[i+1, start+len)
                                              revBwt, revOcc, revInverseSa0,
                                              mkL, mkR,
                                              revMkL, revMkR,
                                              saCount, output,
                                              maxSARangesAllowed,
                                              strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }
}

__forceinline__ __device__
void revForward2MismatchAnd1MismatchAndExact ( uint * query, uint start, uint len,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed,
        uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr && saCount <= maxSARangesAllowed; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revForward1MismatchAndExact ( query, i + 1, start + len - i - 1,
                                              // search range = query[i+1, start+len)
                                              revBwt, revOcc, revInverseSa0,
                                              mkL, mkR,
                                              revMkL, revMkR,
                                              saCount, output,
                                              maxSARangesAllowed, strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }

    if ( pl <= pr && saCount <= maxSARangesAllowed )
    {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
        printf ( "[Global] Address %u\n", output );
        printf ( "[backward3MismatchAndExact] Reporting of SA ranges %u %u ...", pl, pr );
#endif

        if ( saCount < maxSARangesAllowed )
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "DONE\n" );
#endif
            * ( output + 32 * 2 * saCount ) = pl;
            * ( output + 32 * ( 2 * saCount + 1 ) ) = ( pr - pl ) + ( strand << ( BGS_GPU_ANSWER_OFFSET_LENGTH + 3 ) ) +
                    ( accumMismatches << BGS_GPU_ANSWER_OFFSET_LENGTH );
        }
        else
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "FAILED\n" );
#endif
        }

        ++saCount;
    }
}

__forceinline__ __device__
void revForward3MismatchAnd2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr && saCount <= maxSARangesAllowed; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revForward2MismatchAnd1MismatchAndExact ( query, i + 1, start + len - i - 1,
                        // search range = query[i+1, start+len)
                        revBwt, revOcc, revInverseSa0,
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed, strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }
}

__forceinline__ __device__
void revForward3MismatchAnd2MismatchAnd1MismatchAndExact ( uint * query, uint start, uint len,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr && saCount <= maxSARangesAllowed; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revForward2MismatchAnd1MismatchAndExact ( query, i + 1, start + len - i - 1,
                        // search range = query[i+1, start+len)
                        revBwt, revOcc, revInverseSa0,
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed,
                        strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }

    if ( pl <= pr && saCount <= maxSARangesAllowed )
    {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
        printf ( "[Global] Address %u\n", output );
        printf ( "[revForward3MismatchAnd2MismatchAnd1MismatchAndExact] Reporting of SA ranges %u %u ...", pl, pr );
#endif

        if ( saCount < maxSARangesAllowed )
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "DONE\n" );
#endif
            * ( output + 32 * 2 * saCount ) = pl;
            * ( output + 32 * ( 2 * saCount + 1 ) ) = ( pr - pl ) + ( strand << ( BGS_GPU_ANSWER_OFFSET_LENGTH + 3 ) ) +
                    ( accumMismatches << BGS_GPU_ANSWER_OFFSET_LENGTH );
        }
        else
        {
#ifdef BGS_OUTPUT_GENERAL_DEBUG_MESSAGE
            printf ( "FAILED\n" );
#endif
        }

        ++saCount;
    }
}

__forceinline__ __device__
void revForward4MismatchAnd3MismatchAnd2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revForward3MismatchAnd2MismatchAnd1MismatchAndExact ( query, i + 1, start + len - i - 1,
                        // search range = query[i+1, start+len)
                        revBwt, revOcc, revInverseSa0,
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed,
                        strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }
}


__forceinline__ __device__
void revContForwardSearchAndBackward1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2, // for backward
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint * bwt, uint * occ, uint inverseSa0, // for backward
        uint saL, uint saR,
        uint revSaL, uint revSaR,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;
    uint cum_start, cum_end;

    for ( i = start; i < start + len && saL <= saR; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        revSaL =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaL, c, revInverseSa0, cum_start ) + 1;
        revSaR =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaR + 1, c, revInverseSa0, cum_end );
        saR = saR + cum_start - cum_end;
        saL = saR - ( revSaR - revSaL );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
        backward1Mismatch ( query, start2, len2,
                            bwt, occ, inverseSa0,
                            saL, saR,
                            saCount, output,
                            maxSARangesAllowed, strand, accumMismatches );
    }
}

__forceinline__ __device__
void revContForwardSearchAndForward1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint saL, uint saR,
        uint revSaL, uint revSaR,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;
    uint cum_start, cum_end;

    for ( i = start; i < start + len && saL <= saR; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        revSaL =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaL, c, revInverseSa0, cum_start ) + 1;
        revSaR =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaR + 1, c, revInverseSa0, cum_end );
        saR = saR + cum_start - cum_end;
        saL = saR - ( revSaR - revSaL );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
        revForward1Mismatch ( query, start2, len2,
                              revBwt, revOcc, revInverseSa0,
                              saL, saR,
                              revSaL, revSaR,
                              saCount, output,
                              maxSARangesAllowed,
                              strand, accumMismatches );
    }
}


__forceinline__ __device__
void revContForwardSearchAndBackward2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2, // for backward
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint * bwt, uint * occ, uint inverseSa0, // for backward
        uint saL, uint saR,
        uint revSaL, uint revSaR,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;
    uint cum_start, cum_end;

    for ( i = start; i < start + len && saL <= saR; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        revSaL =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaL, c, revInverseSa0, cum_start ) + 1;
        revSaR =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaR + 1, c, revInverseSa0, cum_end );
        saR = saR + cum_start - cum_end;
        saL = saR - ( revSaR - revSaL );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
        backward2MismatchAnd1Mismatch ( query, start2, len2,
                                        bwt, occ, inverseSa0,
                                        saL, saR,
                                        saCount, output,
                                        maxSARangesAllowed,
                                        strand, accumMismatches );
    }
}


__forceinline__ __device__
void revContForwardSearchAndBackward3Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2, // for backward
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint * bwt, uint * occ, uint inverseSa0, // for backward
        uint saL, uint saR,
        uint revSaL, uint revSaR,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;
    uint cum_start, cum_end;

    for ( i = start; i < start + len && saL <= saR; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        revSaL =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaL, c, revInverseSa0, cum_start ) + 1;
        revSaR =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaR + 1, c, revInverseSa0, cum_end );
        saR = saR + cum_start - cum_end;
        saL = saR - ( revSaR - revSaL );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
        backward3Mismatch ( query, start2, len2,
                            bwt, occ, inverseSa0,
                            saL, saR,
                            saCount, output,
                            maxSARangesAllowed,
                            strand, accumMismatches );
    }
}


__forceinline__ __device__
void revForward1MismatchAndBackward1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2, // for backward
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint * bwt, uint * occ, uint inverseSa0, // for backward
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr && saCount <= maxSARangesAllowed; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revContForwardSearchAndBackward1Mismatch ( query, i + 1, start + len - i - 1,
                        start2, len2, // for backward
                        revBwt, revOcc, revInverseSa0,
                        bwt, occ, inverseSa0, // for backward
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed,
                        strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }
}

__forceinline__ __device__
void revForward1MismatchAndForward1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr && saCount <= maxSARangesAllowed; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revContForwardSearchAndForward1Mismatch ( query, i + 1, start + len - i - 1,
                        start2, len2,
                        revBwt, revOcc, revInverseSa0,
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed, strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }
}


__forceinline__ __device__
void revContForwardSearchAndBackward3MismatchAnd2Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2, // for backward
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint * bwt, uint * occ, uint inverseSa0, // for backward
        uint saL, uint saR,
        uint revSaL, uint revSaR,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;
    uint cum_start, cum_end;

    for ( i = start; i < start + len && saL <= saR; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        revSaL =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaL, c, revInverseSa0, cum_start ) + 1;
        revSaR =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaR + 1, c, revInverseSa0, cum_end );
        saR = saR + cum_start - cum_end;
        saL = saR - ( revSaR - revSaL );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
        backward3MismatchAnd2Mismatch ( query, start2, len2,
                                        bwt, occ, inverseSa0,
                                        saL, saR,
                                        saCount, output,
                                        maxSARangesAllowed,
                                        strand, accumMismatches );
    }
}

__forceinline__ __device__
void revForward1MismatchAndBackward2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2, // for backward
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint * bwt, uint * occ, uint inverseSa0, // for backward
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr && saCount <= maxSARangesAllowed; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revContForwardSearchAndBackward2MismatchAnd1Mismatch ( query, i + 1, start + len - i - 1,
                        start2, len2, // for backward
                        revBwt, revOcc, revInverseSa0,
                        bwt, occ, inverseSa0, // for backward
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed,
                        strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }
}

__forceinline__ __device__
void contBackwardSearchAndBackward2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint * bwt, uint * occ, uint inverseSa0,
        uint saL, uint saR,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;

    for ( i = start + len; i > start && saL <= saR; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        saL = GPUBWTOccValue ( bwt, occ, saL, c, inverseSa0 ) + 1;
        saR = GPUBWTOccValue ( bwt, occ, saR + 1, c, inverseSa0 );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
        backward2MismatchAnd1Mismatch ( query, start2, len2,
                                        bwt, occ, inverseSa0,
                                        saL, saR,
                                        saCount, output,
                                        maxSARangesAllowed,
                                        strand, accumMismatches );
    }
}


__forceinline__ __device__
void backward1MismatchAndBackward2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint * bwt, uint * occ, uint inverseSa0,
        uint pl, uint pr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr && saCount <= maxSARangesAllowed; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            uint mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            uint mkR = occCountPEnd[ec];

            if ( mkL <= mkR )
            {
                contBackwardSearchAndBackward2MismatchAnd1Mismatch ( query, start, i - start - 1,
                        start2, len2,
                        bwt, occ, inverseSa0,
                        mkL, mkR,
                        saCount, output,
                        maxSARangesAllowed,
                        strand, accumMismatches + 1 );
            }

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
    }
}


__forceinline__ __device__
void contBackwardSearchAndForward2Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint * bwt, uint * occ, uint inverseSa0,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint saL, uint saR,
        uint revSaL, uint revSaR,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;
    uint cum_start, cum_end;

    for ( i = start + len; i > start && saL <= saR; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        saL =
            GPUBWTOccValueWithCumu ( bwt, occ, saL, c, inverseSa0, cum_start ) + 1;
        saR =
            GPUBWTOccValueWithCumu ( bwt, occ, saR + 1, c, inverseSa0, cum_end );
        revSaR = revSaR + cum_start - cum_end;
        revSaL = revSaR - ( saR - saL );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
        revForward2Mismatch ( query, start2, len2,
                              revBwt, revOcc, revInverseSa0,
                              saL, saR,
                              revSaL, revSaR,
                              saCount, output,
                              maxSARangesAllowed,
                              strand, accumMismatches );
    }
}

__forceinline__ __device__
void backward1MismatchAndForward2Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint * bwt, uint * occ, uint inverseSa0,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr && saCount <= maxSARangesAllowed; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            mkR = occCountPEnd[ec];
            revMkR = revPr - occCountP[ec];
            revMkL = revMkR - ( mkR - mkL );

            if ( mkL <= mkR )
            {
                contBackwardSearchAndForward2Mismatch ( query, start, i - start - 1,
                                                        start2, len2,
                                                        bwt, occ, inverseSa0,
                                                        revBwt, revOcc, revInverseSa0,
                                                        mkL, mkR,
                                                        revMkL, revMkR,
                                                        saCount, output,
                                                        maxSARangesAllowed,
                                                        strand, accumMismatches + 1 );
            }

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
        revPr = revPr - occCountP[c];
    }
}

__forceinline__ __device__
void revContForwardSearchAndForward2Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2, // for second-step-forward
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint saL, uint saR,
        uint revSaL, uint revSaR,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;
    uint cum_start, cum_end;

    for ( i = start; i < start + len && saL <= saR; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        revSaL =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaL, c, revInverseSa0, cum_start ) + 1;
        revSaR =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaR + 1, c, revInverseSa0, cum_end );
        saR = saR + cum_start - cum_end;
        saL = saR - ( revSaR - revSaL );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
        revForward2Mismatch ( query, start2, len2,
                              revBwt, revOcc, revInverseSa0,
                              saL, saR,
                              revSaL, revSaR,
                              saCount, output,
                              maxSARangesAllowed, strand, accumMismatches );
    }
}

__forceinline__ __device__
void revContForwardSearchAndForward2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2, // for second-step-forward
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint saL, uint saR,
        uint revSaL, uint revSaR,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;
    uint cum_start, cum_end;

    for ( i = start; i < start + len && saL <= saR; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        revSaL =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaL, c, revInverseSa0, cum_start ) + 1;
        revSaR =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaR + 1, c, revInverseSa0, cum_end );
        saR = saR + cum_start - cum_end;
        saL = saR - ( revSaR - revSaL );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
        revForward2MismatchAnd1Mismatch ( query, start2, len2,
                                          revBwt, revOcc, revInverseSa0,
                                          saL, saR,
                                          revSaL, revSaR,
                                          saCount, output,
                                          maxSARangesAllowed, strand, accumMismatches );
    }
}

__forceinline__ __device__
void revContForwardSearchAndForward3MismatchAnd2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2, // for second-step-forward
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint saL, uint saR,
        uint revSaL, uint revSaR,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;
    uint cum_start, cum_end;

    for ( i = start; i < start + len && saL <= saR; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        revSaL =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaL, c, revInverseSa0, cum_start ) + 1;
        revSaR =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revSaR + 1, c, revInverseSa0, cum_end );
        saR = saR + cum_start - cum_end;
        saL = saR - ( revSaR - revSaL );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
        revForward3MismatchAnd2MismatchAnd1Mismatch ( query, start2, len2,
                revBwt, revOcc, revInverseSa0,
                saL, saR,
                revSaL, revSaR,
                saCount, output,
                maxSARangesAllowed, strand, accumMismatches );
    }
}

__forceinline__ __device__
void revForward1MismatchAndBackward3Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2, // for backward
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint * bwt, uint * occ, uint inverseSa0, // for backward
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr && saCount <= maxSARangesAllowed; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revContForwardSearchAndBackward3Mismatch ( query, i + 1, start + len - i - 1,
                        start2, len2, // for backward
                        revBwt, revOcc, revInverseSa0,
                        bwt, occ, inverseSa0, // for backward
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed, strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }
}


__forceinline__ __device__
void revForward1MismatchAndForward2Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2, // for second-step-forward
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr && saCount <= maxSARangesAllowed; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revContForwardSearchAndForward2Mismatch ( query, i + 1, start + len - i - 1,
                        start2, len2, // for second-step-forward
                        revBwt, revOcc, revInverseSa0,
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed, strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }
}


__forceinline__ __device__
void revForward1MismatchAndForward2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2, // for second-step-forward
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr && saCount <= maxSARangesAllowed; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revContForwardSearchAndForward2MismatchAnd1Mismatch ( query, i + 1, start + len - i - 1,
                        start2, len2, // for second-step-forward
                        revBwt, revOcc, revInverseSa0,
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed, strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }
}

__forceinline__ __device__
void revForward1MismatchAndForward3MismatchAnd2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2, // for second-step-forward
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr && saCount <= maxSARangesAllowed; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revContForwardSearchAndForward3MismatchAnd2MismatchAnd1Mismatch ( query, i + 1, start + len - i - 1,
                        start2, len2, // for second-step-forward
                        revBwt, revOcc, revInverseSa0,
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed, strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }
}

__forceinline__ __device__
void revForward2MismatchAndForward2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2, // for second-step-forward
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr && saCount <= maxSARangesAllowed; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revForward1MismatchAndForward2MismatchAnd1Mismatch ( query, i + 1, start + len - i - 1,
                        start2, len2, // for second-step-forward
                        revBwt, revOcc, revInverseSa0,
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed, strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }
}

__forceinline__ __device__
void contBackwardSearchAndForward2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint * bwt, uint * occ, uint inverseSa0,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint saL, uint saR,
        uint revSaL, uint revSaR,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;
    uint cum_start, cum_end;

    for ( i = start + len; i > start && saL <= saR; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        saL =
            GPUBWTOccValueWithCumu ( bwt, occ, saL, c, inverseSa0, cum_start ) + 1;
        saR =
            GPUBWTOccValueWithCumu ( bwt, occ, saR + 1, c, inverseSa0, cum_end );
        revSaR = revSaR + cum_start - cum_end;
        revSaL = revSaR - ( saR - saL );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
        revForward2MismatchAnd1Mismatch ( query, start2, len2,
                                          revBwt, revOcc, revInverseSa0,
                                          saL, saR,
                                          revSaL, revSaR,
                                          saCount, output,
                                          maxSARangesAllowed, strand, accumMismatches );
    }
}

__forceinline__ __device__
void backward1MismatchAndForward2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint * bwt, uint * occ, uint inverseSa0,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr && saCount <= maxSARangesAllowed; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            mkR = occCountPEnd[ec];
            revMkR = revPr - occCountP[ec];
            revMkL = revMkR - ( mkR - mkL );

            if ( mkL <= mkR )
            {
                contBackwardSearchAndForward2MismatchAnd1Mismatch ( query, start, i - start - 1,
                        start2, len2,
                        bwt, occ, inverseSa0,
                        revBwt, revOcc, revInverseSa0,
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed, strand, accumMismatches + 1 );
            }

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
        revPr = revPr - occCountP[c];
    }
}

__forceinline__ __device__
void contBackwardSearchAndForward1MismatchAndForward2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint start3, uint len3,
        uint * bwt, uint * occ, uint inverseSa0,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint saL, uint saR,
        uint revSaL, uint revSaR,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;
    uint cum_start, cum_end;

    for ( i = start + len; i > start && saL <= saR; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        saL =
            GPUBWTOccValueWithCumu ( bwt, occ, saL, c, inverseSa0, cum_start ) + 1;
        saR =
            GPUBWTOccValueWithCumu ( bwt, occ, saR + 1, c, inverseSa0, cum_end );
        revSaR = revSaR + cum_start - cum_end;
        revSaL = revSaR - ( saR - saL );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
        revForward1MismatchAndForward2MismatchAnd1Mismatch ( query, start2, len2, start3, len3,
                revBwt, revOcc, revInverseSa0,
                saL, saR,
                revSaL, revSaR,
                saCount, output,
                maxSARangesAllowed, strand, accumMismatches );
    }
}

__forceinline__ __device__
void contBackwardSearchAndBackward1MismatchAndForward2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint start3, uint len3,
        uint * bwt, uint * occ, uint inverseSa0,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint saL, uint saR,
        uint revSaL, uint revSaR,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;
    uint cum_start, cum_end;

    for ( i = start + len; i > start && saL <= saR; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        saL =
            GPUBWTOccValueWithCumu ( bwt, occ, saL, c, inverseSa0, cum_start ) + 1;
        saR =
            GPUBWTOccValueWithCumu ( bwt, occ, saR + 1, c, inverseSa0, cum_end );
        revSaR = revSaR + cum_start - cum_end;
        revSaL = revSaR - ( saR - saL );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
        backward1MismatchAndForward2MismatchAnd1Mismatch ( query, start2, len2, start3, len3,
                bwt, occ, inverseSa0,
                revBwt, revOcc, revInverseSa0,
                saL, saR,
                revSaL, revSaR,
                saCount, output,
                maxSARangesAllowed, strand, accumMismatches );
    }
}

__forceinline__ __device__
void backward1MismatchAndForward1MismatchAndForward2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint start3, uint len3,
        uint * bwt, uint * occ, uint inverseSa0,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr && saCount <= maxSARangesAllowed; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            mkR = occCountPEnd[ec];
            revMkR = revPr - occCountP[ec];
            revMkL = revMkR - ( mkR - mkL );

            if ( mkL <= mkR )
            {
                contBackwardSearchAndForward1MismatchAndForward2MismatchAnd1Mismatch ( query, start, i - start - 1,
                        start2, len2,
                        start3, len3,
                        bwt, occ, inverseSa0,
                        revBwt, revOcc, revInverseSa0,
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed, strand, accumMismatches + 1 );
            }

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
        revPr = revPr - occCountP[c];
    }
}



__forceinline__ __device__
void backward1MismatchAndBackward1MismatchAndForward2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint start3, uint len3,
        uint * bwt, uint * occ, uint inverseSa0,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr && saCount <= maxSARangesAllowed; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            mkR = occCountPEnd[ec];
            revMkR = revPr - occCountP[ec];
            revMkL = revMkR - ( mkR - mkL );

            if ( mkL <= mkR )
            {
                contBackwardSearchAndBackward1MismatchAndForward2MismatchAnd1Mismatch ( query, start, i - start - 1,
                        start2, len2,
                        start3, len3,
                        bwt, occ, inverseSa0,
                        revBwt, revOcc, revInverseSa0,
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed, strand, accumMismatches + 1 );
            }

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
        revPr = revPr - occCountP[c];
    }
}

__forceinline__ __device__
void revForward1MismatchAndBackward3MismatchAnd2Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2, // for backward
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint * bwt, uint * occ, uint inverseSa0, // for backward
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start; i < start + len && pl <= pr && saCount <= maxSARangesAllowed; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPl, occCountPStart );
        GPUBWTAllOccValue ( revBwt, revOcc, revInverseSa0, revPr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Forward manner
        for ( ec = 0; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            if ( c == ec ) { continue; } // TODO optimize as before

            revMkL = occCountPStart[ec] + 1;
            revMkR = occCountPEnd[ec];
            mkR = pr - occCountP[ec];
            mkL = mkR - ( revMkR - revMkL );

            if ( mkL <= mkR )
            {
                revContForwardSearchAndBackward3MismatchAnd2Mismatch ( query, i + 1, start + len - i - 1,
                        start2, len2, // for backward
                        revBwt, revOcc, revInverseSa0,
                        bwt, occ, inverseSa0, // for backward
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed, strand, accumMismatches + 1 );
            }
        }

        revPl = occCountPStart[c] + 1;
        revPr = occCountPEnd[c];
        pr = pr - occCountP[c];
        pl = pr - ( revPr - revPl );
    }
}

__forceinline__ __device__
void contBackwardSearchAndBackward3Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint * bwt, uint * occ, uint inverseSa0,
        uint saL, uint saR,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;

    for ( i = start + len; i > start && saL <= saR; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        saL = GPUBWTOccValue ( bwt, occ, saL, c, inverseSa0 ) + 1;
        saR = GPUBWTOccValue ( bwt, occ, saR + 1, c, inverseSa0 );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
        backward3Mismatch ( query, start2, len2,
                            bwt, occ, inverseSa0,
                            saL, saR,
                            saCount, output,
                            maxSARangesAllowed, strand, accumMismatches );
    }
}

__forceinline__ __device__
void backward1MismatchAndBackward3Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint * bwt, uint * occ, uint inverseSa0,
        uint pl, uint pr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr && saCount <= maxSARangesAllowed; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            uint mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            uint mkR = occCountPEnd[ec];

            if ( mkL <= mkR )
            {
                contBackwardSearchAndBackward3Mismatch ( query, start, i - start - 1,
                        start2, len2,
                        bwt, occ, inverseSa0,
                        mkL, mkR,
                        saCount, output,
                        maxSARangesAllowed, strand, accumMismatches + 1 );
            }

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
    }
}

__forceinline__ __device__
void contBackwardSearchAndForward3MismatchAnd2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint * bwt, uint * occ, uint inverseSa0,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint saL, uint saR,
        uint revSaL, uint revSaR,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    unsigned char c;
    uint i;
    uint cum_start, cum_end;

    for ( i = start + len; i > start && saL <= saR; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        saL =
            GPUBWTOccValueWithCumu ( bwt, occ, saL, c, inverseSa0, cum_start ) + 1;
        saR =
            GPUBWTOccValueWithCumu ( bwt, occ, saR + 1, c, inverseSa0, cum_end );
        revSaR = revSaR + cum_start - cum_end;
        revSaL = revSaR - ( saR - saL );
    }

    if ( saL <= saR && saCount <= maxSARangesAllowed )
    {
        revForward3MismatchAnd2MismatchAnd1Mismatch ( query, start2, len2,
                revBwt, revOcc, revInverseSa0,
                saL, saR,
                revSaL, revSaR,
                saCount, output,
                maxSARangesAllowed, strand, accumMismatches );
    }
}

__forceinline__ __device__
void backward2MismatchAndForward2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint * bwt, uint * occ, uint inverseSa0,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr && saCount <= maxSARangesAllowed; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            mkR = occCountPEnd[ec];
            revMkR = revPr - occCountP[ec];
            revMkL = revMkR - ( mkR - mkL );

            if ( mkL <= mkR )
            {
                backward1MismatchAndForward2MismatchAnd1Mismatch ( query, start, i - start - 1,
                        start2, len2,
                        bwt, occ, inverseSa0,
                        revBwt, revOcc, revInverseSa0,
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed, strand, accumMismatches + 1 );
            }

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
        revPr = revPr - occCountP[c];
    }
}

__forceinline__ __device__
void backward1MismatchAndForward3MismatchAnd2MismatchAnd1Mismatch ( uint * query, uint start, uint len,
        uint start2, uint len2,
        uint * bwt, uint * occ, uint inverseSa0,
        uint * revBwt, uint * revOcc, uint revInverseSa0,
        uint pl, uint pr,
        uint revPl, uint revPr,
        uint & saCount, uint * output,
        uint maxSARangesAllowed, uint strand, uint accumMismatches )
{
    uint mkL, mkR, revMkL, revMkR;
    uint __align__(16) occCountPStart[ALPHABET_SIZE];
    uint __align__(16) occCountPEnd[ALPHABET_SIZE];
    uint __align__(16) occCountP[ALPHABET_SIZE];
    unsigned char c;
    unsigned char ec;
    uint i;

    for ( i = start + len; i > start && pl <= pr && saCount <= maxSARangesAllowed; --i )
    {
        // note that we use i-1 here to prevent counter overflow
        c = ( query[ ( ( i - 1 ) / CHAR_PER_WORD ) * 32] >>
              ( ( i - 1 ) % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pl, occCountPStart );
        GPUBWTAllOccValue ( bwt, occ, inverseSa0, pr + 1, occCountPEnd );
        int k;
        occCountP[ALPHABET_SIZE - 1] = 0;

        for ( k = ALPHABET_SIZE - 2; k >= 0; --k )
        {
            occCountP[k] = occCountP[k + 1] + occCountPEnd[k + 1] - occCountPStart[k + 1];
        }

        // Backward manner
        for ( ec = c ? 0 : 1; ec < ALPHABET_SIZE && saCount <= maxSARangesAllowed; ++ec )
        {
            //      if (c == ec) continue; // TODO optimize as before
            mkL = occCountPStart[ec] + 1; // compute SA range if query[i] was ec
            mkR = occCountPEnd[ec];
            revMkR = revPr - occCountP[ec];
            revMkL = revMkR - ( mkR - mkL );

            if ( mkL <= mkR )
            {
                contBackwardSearchAndForward3MismatchAnd2MismatchAnd1Mismatch ( query, start, i - start - 1,
                        start2, len2,
                        bwt, occ, inverseSa0,
                        revBwt, revOcc, revInverseSa0,
                        mkL, mkR,
                        revMkL, revMkR,
                        saCount, output,
                        maxSARangesAllowed, strand, accumMismatches + 1 );
            }

            if ( c == ec + 1 )
            { ec = c; }
        }

        pl = occCountPStart[c] + 1;
        pr = occCountPEnd[c];
        revPr = revPr - occCountP[c];
    }
}

__forceinline__ __device__
void matchQueryCaseA_4mismatch ( uint * query, uint readLength,
                                 uint * bwt, uint * occ, uint inverseSa0,
                                 uint * revBwt, uint * revOcc, uint revInverseSa0,
                                 uint & saCount, uint * output, unsigned int textLength,
                                 uint maxSARangesAllowed, uint strand )
{
    uint l, r;
    int i;
    unsigned char c;
    //======================================================
    // FOR FOUR MISMATCH
    //======================================================
    // Case A   Backward search
    // 1. exact cell4+5
    // 2. 4/3/2/1/0-mismatch cell1+2+3
    //=======================================
    l = 1;
    r = textLength;
    uint cell1 = ( int ) ( readLength * SIZE_A_RATIO );
    uint cell2 = ( int ) ( readLength * SIZE_B_RATIO );
    uint cell3 = ( int ) ( readLength * SIZE_C_RATIO );

    // exact cell4+5
    for ( i = readLength - 1;
            i >= cell1 + cell2 + cell3 && l <= r; --i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        l = GPUBWTOccValue ( bwt, occ, l, c, inverseSa0 ) + 1;
        r = GPUBWTOccValue ( bwt, occ, r + 1, c, inverseSa0 );
    }

    // 4/3/2/1/0-mismatch cell1+2+3
    if ( l <= r )
    {
        // search range = query[0, cell1 + cell2 + cell3 - 1]
        backward4MismatchAnd3MismatchAnd2MismatchAnd1MismatchAndExact ( query, 0, cell1 + cell2 + cell3,
                bwt, occ, inverseSa0,
                l, r,
                saCount, output,
                maxSARangesAllowed, strand, 0 );
    }
}


__forceinline__ __device__
void matchQueryCaseB_4mismatch ( uint * query, uint readLength,
                                 uint * bwt, uint * occ, uint inverseSa0,
                                 uint * revBwt, uint * revOcc, uint revInverseSa0,
                                 uint & saCount, uint * output, unsigned int textLength,
                                 uint maxSARangesAllowed, uint strand )
{
    uint l, r;
    uint revL, revR;
    int i;
    unsigned char c;
    //======================================================
    // FOR FOUR MISMATCH
    //======================================================
    // Case B   Forward search
    // 1. cell1+2+3 exact
    // 2. 4/3/2/1-mismatch cell4+5
    //=======================================
    l = 0;
    r = textLength;
    revL = 0;
    revR = r;
    uint cell1 = ( int ) ( readLength * SIZE_A_RATIO );
    uint cell2 = ( int ) ( readLength * SIZE_B_RATIO );
    uint cell3 = ( int ) ( readLength * SIZE_C_RATIO );
    uint cell4 = ( int ) ( readLength * SIZE_D_RATIO );
    uint cell5 = readLength - cell1 - cell2 - cell3 - cell4;

    // cell1+2+3 exact
    for ( i = 0; i < cell1 + cell2 + cell3 && l <= r; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        uint start, end;
        revL = GPUBWTOccValueWithCumu ( revBwt, revOcc, revL,
                                        c, revInverseSa0, start ) + 1;
        revR = GPUBWTOccValueWithCumu ( revBwt, revOcc, revR + 1,
                                        c, revInverseSa0, end );
        r = r + start - end;
        l = r - ( revR - revL );
    }

    // 4-mismatch cell4+5
    if ( l <= r )
    {
        // search range = query[cell1 + cell2 + cell3, QUERY_LENGTH-1]
        revForward4MismatchAnd3MismatchAnd2MismatchAnd1Mismatch ( query, cell1 + cell2 + cell3, cell4 + cell5,
                revBwt, revOcc, revInverseSa0,
                l, r,
                revL, revR,
                saCount, output,
                maxSARangesAllowed, strand, 0 );
    }
}



__forceinline__ __device__
void matchQueryCaseC_4mismatch ( uint * query, uint readLength,
                                 uint * bwt, uint * occ, uint inverseSa0,
                                 uint * revBwt, uint * revOcc, uint revInverseSa0,
                                 uint & saCount, uint * output, unsigned int textLength,
                                 uint maxSARangesAllowed, uint strand )
{
    uint l, r;
    uint revL, revR;
    int i;
    unsigned char c;
    //======================================================
    // FOR FOUR MISMATCH
    //======================================================
    // Case C
    // 1. exact cell1 (forward)
    // 2. 1-mismatch cell2+3 (forward)
    // 3. 3/2/1-mismatch cell4+5 (forward)
    //=======================================
    l = 0;
    r = textLength;
    revL = 0;
    revR = r;
    uint cell1 = ( int ) ( readLength * SIZE_A_RATIO );
    uint cell2 = ( int ) ( readLength * SIZE_B_RATIO );
    uint cell3 = ( int ) ( readLength * SIZE_C_RATIO );
    uint cell4 = ( int ) ( readLength * SIZE_D_RATIO );
    uint cell5 = readLength - cell1 - cell2 - cell3 - cell4;

    // exact cell1 (forward)
    for ( i = 0; i < cell1 && l <= r; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        uint cum_start, cum_end;
        revL =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revL, c, revInverseSa0, cum_start ) + 1;
        revR =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revR + 1, c, revInverseSa0, cum_end );
        r = r + cum_start - cum_end;
        l = r - ( revR - revL );
    }

    // 1-mismatch cell2+3 (forward)
    // 3/2/1-mismatch cell4+5 (forward)
    if ( l <= r )
    {
        revForward1MismatchAndForward3MismatchAnd2MismatchAnd1Mismatch ( query, cell1, cell2 + cell3,
                cell1 + cell2 + cell3, cell4 + cell5, // for second-step-forward
                revBwt, revOcc, revInverseSa0,
                l, r,
                revL, revR,
                saCount, output,
                maxSARangesAllowed, strand, 0 );
    }
}



__forceinline__ __device__
void matchQueryCaseD_4mismatch ( uint * query, uint readLength,
                                 uint * bwt, uint * occ, uint inverseSa0,
                                 uint * revBwt, uint * revOcc, uint revInverseSa0,
                                 uint & saCount, uint * output, unsigned int textLength,
                                 uint maxSARangesAllowed, uint strand )
{
    uint l, r;
    uint revL, revR;
    int i;
    unsigned char c;
    //=======================================
    //  FOR FOUR MISMATCH
    //=======================================
    // Case D
    // 1. exact cell2+3 (forward)
    // 2. 1-mismatch cell1 (backward)
    // 3. 3/2/1-mismatch cell4+5 (forward)
    //=======================================
    l = 0;
    r = textLength;
    revL = 0;
    revR = r;
    uint cell1 = ( int ) ( readLength * SIZE_A_RATIO );
    uint cell2 = ( int ) ( readLength * SIZE_B_RATIO );
    uint cell3 = ( int ) ( readLength * SIZE_C_RATIO );
    uint cell4 = ( int ) ( readLength * SIZE_D_RATIO );
    uint cell5 = readLength - cell1 - cell2 - cell3 - cell4;

    // exact cell2+3 (forward)
    for ( i = cell1; i < cell1 + cell2 + cell3 && l <= r; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        uint cum_start, cum_end;
        revL =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revL, c, revInverseSa0, cum_start ) + 1;
        revR =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revR + 1, c, revInverseSa0, cum_end );
        r = r + cum_start - cum_end;
        l = r - ( revR - revL );
    }

    // 1-mismatch cell1 (backward)
    // 3/2/1-mismatch cell4+5 (forward)
    if ( l <= r )
    {
        backward1MismatchAndForward3MismatchAnd2MismatchAnd1Mismatch ( query, 0, cell1,
                cell1 + cell2 + cell3, cell4 + cell5,
                bwt, occ, inverseSa0,
                revBwt, revOcc, revInverseSa0,
                l, r,
                revL, revR,
                saCount, output,
                maxSARangesAllowed, strand, 0 );
    }
}



__forceinline__ __device__
void matchQueryCaseE_4mismatch ( uint * query, uint readLength,
                                 uint * bwt, uint * occ, uint inverseSa0,
                                 uint * revBwt, uint * revOcc, uint revInverseSa0,
                                 uint & saCount, uint * output, unsigned int textLength,
                                 uint maxSARangesAllowed, uint strand )
{
    uint l, r;
    uint revL, revR;
    int i;
    unsigned char c;
    //======================================================
    // FOR FOUR MISMATCH
    //======================================================
    // Case E
    // 1. exact cell1 (forward)
    // 2. 2-mismatch cell2+3 (forward)
    // 3. 1/2-mismatch cell4+5 (forward)
    //=======================================
    l = 0;
    r = textLength;
    revL = 0;
    revR = r;
    uint cell1 = ( int ) ( readLength * SIZE_A_RATIO );
    uint cell2 = ( int ) ( readLength * SIZE_B_RATIO );
    uint cell3 = ( int ) ( readLength * SIZE_C_RATIO );
    uint cell4 = ( int ) ( readLength * SIZE_D_RATIO );
    uint cell5 = readLength - cell1 - cell2 - cell3 - cell4;

    // exact cell1 (forward)
    for ( i = 0; i < cell1 && l <= r; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        uint cum_start, cum_end;
        revL =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revL, c, revInverseSa0, cum_start ) + 1;
        revR =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revR + 1, c, revInverseSa0, cum_end );
        r = r + cum_start - cum_end;
        l = r - ( revR - revL );
    }

    // 2-mismatch cell2+3 (forward)
    // 1/2-mismatch cell4+5 (forward)
    if ( l <= r )
    {
        revForward2MismatchAndForward2MismatchAnd1Mismatch ( query, cell1, cell2 + cell3,
                cell1 + cell2 + cell3, cell4 + cell5, // for second-step-forward
                revBwt, revOcc, revInverseSa0,
                l, r,
                revL, revR,
                saCount, output,
                maxSARangesAllowed, strand, 0 );
    }
}


__forceinline__ __device__
void matchQueryCaseF_4mismatch ( uint * query, uint readLength,
                                 uint * bwt, uint * occ, uint inverseSa0,
                                 uint * revBwt, uint * revOcc, uint revInverseSa0,
                                 uint & saCount, uint * output, unsigned int textLength,
                                 uint maxSARangesAllowed, uint strand )
{
    uint l, r;
    uint revL, revR;
    int i;
    unsigned char c;
    //======================================================
    // FOR FOUR MISMATCH
    //======================================================
    // Case F
    // 1. exact cell2+3 (forward)
    // 2. 2-mismatch cell1 (backward)
    // 3. 1/2-mismatch cell4+5 (forward)
    //=======================================
    l = 0;
    r = textLength;
    revL = 0;
    revR = r;
    uint cell1 = ( int ) ( readLength * SIZE_A_RATIO );
    uint cell2 = ( int ) ( readLength * SIZE_B_RATIO );
    uint cell3 = ( int ) ( readLength * SIZE_C_RATIO );
    uint cell4 = ( int ) ( readLength * SIZE_D_RATIO );
    uint cell5 = readLength - cell1 - cell2 - cell3 - cell4;

    // exact cell3 (forward)
    for ( i = cell1; i < cell1 + cell2 + cell3 && l <= r; ++i )
    {
        // TODO compute c here
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        uint cum_start, cum_end;
        revL =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revL, c, revInverseSa0, cum_start ) + 1;
        revR =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revR + 1, c, revInverseSa0, cum_end );
        r = r + cum_start - cum_end;
        l = r - ( revR - revL );
    }

    // 2-mismatch cell1 (backward)
    // 1/2-mismatch cell4+5 (forward)
    if ( l <= r )
    {
        backward2MismatchAndForward2MismatchAnd1Mismatch ( query, 0, cell1,
                cell1 + cell2 + cell3, cell4 + cell5,
                bwt, occ, inverseSa0,
                revBwt, revOcc, revInverseSa0,
                l, r,
                revL, revR,
                saCount, output,
                maxSARangesAllowed, strand, 0 );
    }
}


__forceinline__ __device__
void matchQueryCaseG_4mismatch ( uint * query, uint readLength,
                                 uint * bwt, uint * occ, uint inverseSa0,
                                 uint * revBwt, uint * revOcc, uint revInverseSa0,
                                 uint & saCount, uint * output, unsigned int textLength,
                                 uint maxSARangesAllowed, uint strand )
{
    uint l, r;
    uint revL, revR;
    uint i;
    unsigned char c;
    //======================================================
    // FOR FOUR MISMATCH
    //======================================================
    // Case G
    // 1. exact cell2 (forward)
    // 2. 1-mismatch cell1 (backward)
    // 3. 1-mismatch cell3 (forward)
    // 4. 1/2-mismatch cell4+5 (forward)
    //=======================================
    l = 0;
    r = textLength;
    revL = 0;
    revR = r;
    uint cell1 = ( int ) ( readLength * SIZE_A_RATIO );
    uint cell2 = ( int ) ( readLength * SIZE_B_RATIO );
    uint cell3 = ( int ) ( readLength * SIZE_C_RATIO );
    uint cell4 = ( int ) ( readLength * SIZE_D_RATIO );
    uint cell5 = readLength - cell1 - cell2 - cell3 - cell4;

    // exact cell2 (forward)
    for ( i = cell1; i < cell1 + cell2 && l <= r; ++i )
    {
        // TODO compute c here
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        uint cum_start, cum_end;
        revL =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revL, c, revInverseSa0, cum_start ) + 1;
        revR =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revR + 1, c, revInverseSa0, cum_end );
        r = r + cum_start - cum_end;
        l = r - ( revR - revL );
    }

    // 1-mismatch cell1 (backward)
    // 1-mismatch cell3 (forward)
    // 1/2-mismatch cell4+5 (forward)
    if ( l <= r )
    {
        backward1MismatchAndForward1MismatchAndForward2MismatchAnd1Mismatch ( query, 0, cell1,
                cell1 + cell2, cell3,
                cell1 + cell2 + cell3, cell4 + cell5,
                bwt, occ, inverseSa0,
                revBwt, revOcc, revInverseSa0,
                l, r,
                revL, revR,
                saCount, output,
                maxSARangesAllowed, strand, 0 );
    }
}



__forceinline__ __device__
void matchQueryCaseH_4mismatch ( uint * query, uint readLength,
                                 uint * bwt, uint * occ, uint inverseSa0,
                                 uint * revBwt, uint * revOcc, uint revInverseSa0,
                                 uint & saCount, uint * output, unsigned int textLength,
                                 uint maxSARangesAllowed, uint strand )
{
    uint l, r;
    uint revL, revR;
    uint i;
    unsigned char c;
    //======================================================
    // FOR FOUR MISMATCH
    //======================================================
    // Case H
    // 1. exact cell3 (forward)
    // 2. 1-mismatch cell2 (backward)
    // 3. 1-mismatch cell1 (backward)
    // 4. 1/2-mismatch cell4+5 (forward)
    //=======================================
    l = 0;
    r = textLength;
    revL = 0;
    revR = r;
    uint cell1 = ( int ) ( readLength * SIZE_A_RATIO );
    uint cell2 = ( int ) ( readLength * SIZE_B_RATIO );
    uint cell3 = ( int ) ( readLength * SIZE_C_RATIO );
    uint cell4 = ( int ) ( readLength * SIZE_D_RATIO );
    uint cell5 = readLength - cell1 - cell2 - cell3 - cell4;

    // exact cell3 (forward)
    for ( i = cell1 + cell2; i < cell1 + cell2 + cell3 && l <= r; ++i )
    {
        // TODO compute c here
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        uint cum_start, cum_end;
        revL =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revL, c, revInverseSa0, cum_start ) + 1;
        revR =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revR + 1, c, revInverseSa0, cum_end );
        r = r + cum_start - cum_end;
        l = r - ( revR - revL );
    }

    // 1-mismatch cell2 (backward)
    // 1-mismatch cell1 (backward)
    // 1/2-mismatch cell4+5 (forward)
    if ( l <= r )
    {
        backward1MismatchAndBackward1MismatchAndForward2MismatchAnd1Mismatch ( query, cell1, cell2,
                0, cell1,
                cell1 + cell2 + cell3, cell4 + cell5,
                bwt, occ, inverseSa0,
                revBwt, revOcc, revInverseSa0,
                l, r,
                revL, revR,
                saCount, output,
                maxSARangesAllowed, strand, 0 );
    }
}




__forceinline__ __device__
void matchQueryCaseI_4mismatch ( uint * query, uint readLength,
                                 uint * bwt, uint * occ, uint inverseSa0,
                                 uint * revBwt, uint * revOcc, uint revInverseSa0,
                                 uint & saCount, uint * output, unsigned int textLength,
                                 uint maxSARangesAllowed, uint strand )
{
    uint l, r;
    uint revL, revR;
    uint i;
    unsigned char c;
    //======================================================
    // FOR FOUR MISMATCH
    //======================================================
    // Case I
    // 1. exact cell4 (forward)
    // 2. 1-mismatch cell5 (forward)
    // 3. 3-mismatch cell1+2+3 (backward)
    //=======================================
    l = 0;
    r = textLength;
    revL = 0;
    revR = r;
    uint cell1 = ( int ) ( readLength * SIZE_A_RATIO );
    uint cell2 = ( int ) ( readLength * SIZE_B_RATIO );
    uint cell3 = ( int ) ( readLength * SIZE_C_RATIO );
    uint cell4 = ( int ) ( readLength * SIZE_D_RATIO );
    uint cell5 = readLength - cell1 - cell2 - cell3 - cell4;

    // exact cell4 (forward)
    for ( i = cell1 + cell2 + cell3; i < cell1 + cell2 + cell3 + cell4 && l <= r; ++i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        uint cum_start, cum_end;
        revL =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revL, c, revInverseSa0, cum_start ) + 1;
        revR =
            GPUBWTOccValueWithCumu ( revBwt, revOcc, revR + 1, c, revInverseSa0, cum_end );
        r = r + cum_start - cum_end;
        l = r - ( revR - revL );
    }

    // 1-mismatch cell5 (forward)
    // 3-mismatch cell1+2+3 (backward)
    if ( l <= r )
    {
        revForward1MismatchAndBackward3Mismatch ( query, cell1 + cell2 + cell3 + cell4, cell5,
                0, cell1 + cell2 + cell3, // for backward
                revBwt, revOcc, revInverseSa0,
                bwt,  occ,  inverseSa0, // for backward
                l, r,
                revL, revR,
                saCount, output,
                maxSARangesAllowed, strand, 0 );
    }
}

__forceinline__ __device__
void matchQueryCaseJ_4mismatch ( uint * query, uint readLength,
                                 uint * bwt, uint * occ, uint inverseSa0,
                                 uint * revBwt, uint * revOcc, uint revInverseSa0,
                                 uint & saCount, uint * output, unsigned int textLength,
                                 uint maxSARangesAllowed, uint strand )
{
    uint l, r;
    uint i;
    unsigned char c;
    //======================================================
    // FOR FOUR MISMATCH
    //======================================================
    // Case J
    // 1. exact cell5 (backward)
    // 2. 1-mismatch cell4 (backward)
    // 3. 3-mismatch cell1+2+3 (backward)
    //=======================================
    l = 0;
    r = textLength;
    uint cell1 = ( int ) ( readLength * SIZE_A_RATIO );
    uint cell2 = ( int ) ( readLength * SIZE_B_RATIO );
    uint cell3 = ( int ) ( readLength * SIZE_C_RATIO );
    uint cell4 = ( int ) ( readLength * SIZE_D_RATIO );
    l = 1;
    r = textLength;

    // exact cell5 (backward)
    for ( i = readLength - 1;
            i >= cell1 + cell2 + cell3 + cell4 && l <= r; --i )
    {
        c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
        l = GPUBWTOccValue ( bwt, occ, l, c, inverseSa0 ) + 1;
        r = GPUBWTOccValue ( bwt, occ, r + 1, c, inverseSa0 );
    }

    // 1-mismatch cell4 (backward)
    // 3-mismatch cell1+2+3 (backward)
    if ( l <= r )
    {
        backward1MismatchAndBackward3Mismatch ( query, cell1 + cell2 + cell3, cell4,
                                                0, cell1 + cell2 + cell3,
                                                bwt, occ, inverseSa0,
                                                l, r,
                                                saCount, output, maxSARangesAllowed, strand, 0 );
    }
}



__forceinline__ __device__
void matchQueryCaseA ( uint * query, uint readLength,
                       uint * bwt, uint * occ, uint inverseSa0,
                       uint * revBwt, uint * revOcc, uint revInverseSa0,
                       uint & saCount, uint * output, uint textLength,
                       uint maxSARangesAllowed, uint strand, uint numMismatch,
                       bool isExactNumMismatch )
{
    uint l, r;
    int i;
    unsigned char c;

    if ( numMismatch == 0 )
    {
        //======================================================
        // FOR EXACT MATCH
        //======================================================
        l = 1;
        r = textLength;
        // backward search with BWT
        contBackwardSearch ( query, 0, readLength,
                             bwt, occ, inverseSa0,
                             l, r,
                             saCount, output,
                             maxSARangesAllowed,
                             strand, 0 );
    }

    if ( numMismatch == 1 )
    {
        //======================================================
        // FOR ONE MISMATCH
        //======================================================
        // Case A   Backward search
        // 1. exact cellY
        // 2. 1-mismatch cellX
        //=======================================
        l = 1;
        r = textLength;
        uint sizeX = ( int ) ( readLength * 0.5 );

        // backward search with BWT in cellY
        for ( i = readLength - 1; i >= sizeX && l <= r; --i )
        {
            c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
            l = GPUBWTOccValue ( bwt, occ, l, c, inverseSa0 ) + 1;
            r = GPUBWTOccValue ( bwt, occ, r + 1, c, inverseSa0 );
        }

        // 1/0-mismatch cellX
        if ( l <= r )
        {
            if ( isExactNumMismatch )
                backward1Mismatch ( query, 0, sizeX, bwt, occ, inverseSa0,
                                    l, r, saCount, output, maxSARangesAllowed, strand, 0 );
            else
                backward1MismatchAndExact ( query, 0, sizeX, bwt, occ, inverseSa0,
                                            l, r, saCount, output, maxSARangesAllowed, strand, 0 );
        }
    }
    else if ( numMismatch == 2 )
    {
        //======================================================
        // FOR TWO MISMATCH
        //======================================================
        // Case A   Backward search
        // 1. exact cellZ
        // 2. 2/1/0-mismatch cellX+Y
        //=======================================
        l = 1;
        r = textLength;
        uint sizeX = ( int ) ( readLength * SIZE_X_RATIO );
        uint sizeY = ( int ) ( readLength * SIZE_Y_RATIO );

        // backward search with BWT in cellZ
        for ( i = readLength - 1; i >= sizeX + sizeY && l <= r; --i )
        {
            c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
            l = GPUBWTOccValue ( bwt, occ, l, c, inverseSa0 ) + 1;
            r = GPUBWTOccValue ( bwt, occ, r + 1, c, inverseSa0 );
        }

        // 2. 2/1/0-mismatch cellX+Y
        if ( l <= r )
        {
            backward2MismatchAnd1MismatchAndExact ( query, 0, sizeX + sizeY,
                                                    bwt, occ, inverseSa0,
                                                    l, r,
                                                    saCount, output,
                                                    maxSARangesAllowed, strand, 0 );
        }
    }
    else if ( numMismatch == 3 )
    {
        //======================================================
        // FOR THREE MISMATCH
        //======================================================
        // Case A   Backward search
        // 1. exact cell3+4
        // 2. 3/2/1/0-mismatch cell1+2
        //=======================================
        l = 1;
        r = textLength;
        uint cell1 = ( int ) ( readLength * SIZE_1_RATIO );
        uint cell2 = ( int ) ( readLength * SIZE_2_RATIO );

        // backward search with BWT in cellZ
        for ( i = readLength - 1;
                i >= cell1 + cell2 && l <= r; --i )
        {
            c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
            l = GPUBWTOccValue ( bwt, occ, l, c, inverseSa0 ) + 1;
            r = GPUBWTOccValue ( bwt, occ, r + 1, c, inverseSa0 );
        }

        // 3/2/1/0-mismatch cell1+2
        if ( l <= r )
        {
            // search range = query[0, cell1 + cell2 - 1]
            backward3MismatchAnd2MismatchAnd1MismatchAndExact ( query, 0, cell1 + cell2,
                    bwt, occ, inverseSa0,
                    l, r,
                    saCount, output,
                    maxSARangesAllowed, strand, 0 );
        }
    }
}

__forceinline__ __device__
void matchQueryCaseB ( uint * query, uint readLength,
                       uint * bwt, uint * occ, uint inverseSa0,
                       uint * revBwt, uint * revOcc, uint revInverseSa0,
                       uint & saCount, uint * output, uint textLength,
                       uint maxSARangesAllowed, uint strand, uint numMismatch )
{
    uint l, r;
    uint revL, revR;
    int i;
    unsigned char c;

    if ( numMismatch == 1 )
    {
        //======================================================
        // FOR ONE MISMATCH
        //======================================================
        // Case B   Forward search
        // 1. cellX exact
        // 2. 1-mismatch cellY
        //=======================================
        l = 0;
        r = textLength;
        revL = 0;
        revR = r;
        uint sizeX = ( int ) ( readLength * 0.5 );
        // uint sizeY = (int)(readLength * 0.5);
        uint sizeY = readLength - sizeX;

        // forward search with BWT until the end of forward depth section
        for ( i = 0; i < sizeX && l <= r; ++i )
        {
            c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
            uint cum_start, cum_end;
            revL =
                GPUBWTOccValueWithCumu ( revBwt, revOcc, revL, c, revInverseSa0, cum_start ) + 1;
            revR =
                GPUBWTOccValueWithCumu ( revBwt, revOcc, revR + 1, c, revInverseSa0, cum_end );
            r = r + cum_start - cum_end;
            l = r - ( revR - revL );
        }

        // 2. 1-mismatch cellY
        if ( l <= r )
        {
            revForward1Mismatch ( query, sizeX, sizeY,
                                  revBwt, revOcc, revInverseSa0,
                                  l, r,
                                  revL, revR,
                                  saCount, output,
                                  maxSARangesAllowed, strand, 0 );
        }
    }
    else if ( numMismatch == 2 )
    {
        //======================================================
        // FOR TWO MISMATCH
        //======================================================
        // Case B   Forward search
        // 1. cellX+Y exact
        // 2. 2/1-mismatch cellZ
        //=======================================
        l = 0;
        r = textLength;
        revL = 0;
        revR = r;
        uint sizeX = ( int ) ( readLength * SIZE_X_RATIO );
        uint sizeY = ( int ) ( readLength * SIZE_Y_RATIO );
        uint sizeZ = readLength - sizeX - sizeY;

        // forward search with BWT until the end of forward depth section
        for ( i = 0; i < sizeX + sizeY && l <= r; ++i )
        {
            c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
            uint cum_start, cum_end;
            revL =
                GPUBWTOccValueWithCumu ( revBwt, revOcc, revL, c, revInverseSa0, cum_start ) + 1;
            revR =
                GPUBWTOccValueWithCumu ( revBwt, revOcc, revR + 1, c, revInverseSa0, cum_end );
            r = r + cum_start - cum_end;
            l = r - ( revR - revL );
        }

        // 2 errors in cellZ
        if ( l <= r )
        {
            revForward2MismatchAnd1Mismatch ( query, sizeX + sizeY, sizeZ,
                                              revBwt, revOcc, revInverseSa0,
                                              l, r,
                                              revL, revR,
                                              saCount, output,
                                              maxSARangesAllowed, strand, 0 );
        }
    }
    else if ( numMismatch == 3 )
    {
        //======================================================
        // FOR THREE MISMATCH
        //======================================================
        // Case B   Forward search
        // 1. cell1+2 exact
        // 2. 3/2/1-mismatch cell3+4
        //=======================================
        l = 0;
        r = textLength;
        revL = 0;
        revR = r;
        uint cell1 = ( int ) ( readLength * SIZE_1_RATIO );
        uint cell2 = ( int ) ( readLength * SIZE_2_RATIO );
        uint cell3 = ( int ) ( readLength * SIZE_3_RATIO );
        uint cell4 = readLength - cell1 - cell2 - cell3;

        // forward search with BWT until the end of forward depth section
        for ( i = 0; i < cell1 + cell2 && l <= r; ++i )
        {
            c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
            uint start, end;
            revL = GPUBWTOccValueWithCumu ( revBwt, revOcc, revL,
                                            c, revInverseSa0, start ) + 1;
            revR = GPUBWTOccValueWithCumu ( revBwt, revOcc, revR + 1,
                                            c, revInverseSa0, end );
            r = r + start - end;
            l = r - ( revR - revL );
        }

        // 3/2/1-mismatch cell3+4
        if ( l <= r )
        {
            // search range = query[cell1 + cell2, QUERY_LENGTH-1]
            revForward3MismatchAnd2MismatchAnd1Mismatch ( query, cell1 + cell2, cell3 + cell4,
                    revBwt, revOcc, revInverseSa0,
                    l, r,
                    revL, revR,
                    saCount, output,
                    maxSARangesAllowed, strand, 0 );
        }
    }
}

__forceinline__ __device__
void matchQueryCaseC ( uint * query, uint readLength,
                       uint * bwt, uint * occ, uint inverseSa0,
                       uint * revBwt, uint * revOcc, uint revInverseSa0,
                       uint & saCount, uint * output, uint textLength,
                       uint maxSARangesAllowed, uint strand, uint numMismatch )
{
    uint l, r;
    uint revL, revR;
    uint i;
    unsigned char c;

    if ( numMismatch == 2 )
    {
        //======================================================
        // FOR TWO MISMATCH
        //======================================================
        // Case C
        // 1. cellX (forward)
        // 2. 1-mismatch cellY (forward)
        // 3. 1-mismatch cellZ (forward)
        //=======================================
        l = 0;
        r = textLength;
        revL = 0;
        revR = r;
        uint sizeX = ( int ) ( readLength * SIZE_X_RATIO );
        uint sizeY = ( int ) ( readLength * SIZE_Y_RATIO );
        uint sizeZ = readLength - sizeX - sizeY;

        // forward search with BWT until the end of forward depth section
        for ( i = 0; i < sizeX && l <= r; ++i )
        {
            c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
            uint cum_start, cum_end;
            revL =
                GPUBWTOccValueWithCumu ( revBwt, revOcc, revL, c, revInverseSa0, cum_start ) + 1;
            revR =
                GPUBWTOccValueWithCumu ( revBwt, revOcc, revR + 1, c, revInverseSa0, cum_end );
            r = r + cum_start - cum_end;
            l = r - ( revR - revL );
        }

        // 2. 1-mismatch cellY (forward)
        // 3. 1-mismatch cellZ (forward)
        if ( l <= r )
        {
            revForward1MismatchAndForward1Mismatch ( query, sizeX, sizeY,
                    sizeX + sizeY, sizeZ, // for backward
                    revBwt, revOcc, revInverseSa0,
                    l, r,
                    revL, revR,
                    saCount, output,
                    maxSARangesAllowed, strand, 0 );
        }
    }
    else if ( numMismatch == 3 )
    {
        //======================================================
        // FOR THREE MISMATCH
        //======================================================
        // Case C
        // 1. exact cell1 (forward)
        // 2. 1-mismatch cell2 (forward)
        // 3. 2-mismatch cell3+4 (forward)
        //=======================================
        l = 0;
        r = textLength;
        revL = 0;
        revR = r;
        uint cell1 = ( int ) ( readLength * SIZE_1_RATIO );
        uint cell2 = ( int ) ( readLength * SIZE_2_RATIO );
        uint cell3 = ( int ) ( readLength * SIZE_3_RATIO );
        uint cell4 = readLength - cell1 - cell2 - cell3;

        // exact cell1 (forward)
        for ( i = 0; i < cell1 && l <= r; ++i )
        {
            c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
            uint cum_start, cum_end;
            revL =
                GPUBWTOccValueWithCumu ( revBwt, revOcc, revL, c, revInverseSa0, cum_start ) + 1;
            revR =
                GPUBWTOccValueWithCumu ( revBwt, revOcc, revR + 1, c, revInverseSa0, cum_end );
            r = r + cum_start - cum_end;
            l = r - ( revR - revL );
        }

        // 1-mismatch cell2 (forward)
        // 2-mismatch cell3+4 (forward)

        if ( l <= r )
        {
            revForward1MismatchAndForward2Mismatch ( query, cell1, cell2,
                    cell1 + cell2, cell3 + cell4,
                    revBwt, revOcc, revInverseSa0,
                    l, r,
                    revL, revR,
                    saCount, output,
                    maxSARangesAllowed, strand, 0 );
        }
    }
}

__forceinline__ __device__
void matchQueryCaseD ( uint * query, uint readLength,
                       uint * bwt, uint * occ, uint inverseSa0,
                       uint * revBwt, uint * revOcc, uint revInverseSa0,
                       uint & saCount, uint * output, uint textLength,
                       uint maxSARangesAllowed, uint strand, uint numMismatch )
{
    uint l, r;
    uint revL, revR;
    uint i;
    unsigned char c;

    if ( numMismatch == 2 )
    {
        //======================================================
        // FOR TWO MISMATCH
        //======================================================
        // Case D
        // 1. cellY (forward)
        // 2. 1-mismatch cellZ (forward)
        // 3. 1-mismatch cellX (backward)
        //=======================================
        l = 0;
        r = textLength;
        revL = 0;
        revR = r;
        uint sizeX = ( int ) ( readLength * SIZE_X_RATIO );
        uint sizeY = ( int ) ( readLength * SIZE_Y_RATIO );
        uint sizeZ = readLength - sizeX - sizeY;

        // forward search with BWT until the end of forward depth section
        for ( i = sizeX; i < sizeX + sizeY && l <= r; ++i )
        {
            // TODO compute c here
            c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
            uint cum_start, cum_end;
            revL =
                GPUBWTOccValueWithCumu ( revBwt, revOcc, revL, c, revInverseSa0, cum_start ) + 1;
            revR =
                GPUBWTOccValueWithCumu ( revBwt, revOcc, revR + 1, c, revInverseSa0, cum_end );
            r = r + cum_start - cum_end;
            l = r - ( revR - revL );
        }

        // 2. 1-mismatch cellZ (forward)
        // 3. 1-mismatch cellX (backward)
        if ( l <= r )
        {
            revForward1MismatchAndBackward1Mismatch ( query, sizeX + sizeY, sizeZ,
                    0, sizeX, // for backward
                    revBwt, revOcc, revInverseSa0,
                    bwt, occ, inverseSa0, // for backward
                    l, r,
                    revL, revR,
                    saCount, output,
                    maxSARangesAllowed, strand, 0 );
        }
    }
    else if ( numMismatch == 3 )
    {
        //=======================================
        //  FOR THREE MISMATCH
        //=======================================
        // Case D
        // 1. exact cell3 (forward)
        // 2. 1-mismatch cell4 (forward)
        // 3. 2/1-mismatch cell1+2 (backward)
        //=======================================
        l = 0;
        r = textLength;
        revL = 0;
        revR = r;
        uint cell1 = ( int ) ( readLength * SIZE_1_RATIO );
        uint cell2 = ( int ) ( readLength * SIZE_2_RATIO );
        uint cell3 = ( int ) ( readLength * SIZE_3_RATIO );
        uint cell4 = readLength - cell1 - cell2 - cell3;

        // forward search with BWT until the end of forward depth section
        for ( i = cell1 + cell2; i < cell1 + cell2 + cell3 && l <= r; ++i )
        {
            // TODO compute c here
            c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
            uint cum_start, cum_end;
            revL =
                GPUBWTOccValueWithCumu ( revBwt, revOcc, revL, c, revInverseSa0, cum_start ) + 1;
            revR =
                GPUBWTOccValueWithCumu ( revBwt, revOcc, revR + 1, c, revInverseSa0, cum_end );
            r = r + cum_start - cum_end;
            l = r - ( revR - revL );
        }

        // 1-mismatch cell4 (forward)
        // 2/1-mismatch cell1+2 (backward)
        if ( l <= r )
        {
            revForward1MismatchAndBackward2MismatchAnd1Mismatch ( query, cell1 + cell2 + cell3, cell4,
                    0, cell1 + cell2, // for backward
                    revBwt, revOcc, revInverseSa0,
                    bwt,  occ,  inverseSa0, // for backward
                    l, r,
                    revL, revR,
                    saCount, output,
                    maxSARangesAllowed, strand, 0 );
        }
    }
}

__forceinline__ __device__
void matchQueryCaseE ( uint * query, uint readLength,
                       uint * bwt, uint * occ, uint inverseSa0,
                       uint * revBwt, uint * revOcc, uint revInverseSa0,
                       uint & saCount, uint * output, uint textLength,
                       uint maxSARangesAllowed, uint strand, uint numMismatch )
{
    uint l, r;
    int i;
    unsigned char c;

    if ( numMismatch == 3 )
    {
        //======================================================
        // FOR THREE MISMATCH
        //======================================================
        // Case E
        // 1. exact cell4 (backward)
        // 2. 1-mismatch cell3 (backward)
        // 3. 2/1-mismatch cell1+2 (backward)
        //=======================================
        l = 1;
        r = textLength;
        uint cell1 = ( int ) ( readLength * SIZE_1_RATIO );
        uint cell2 = ( int ) ( readLength * SIZE_2_RATIO );
        uint cell3 = ( int ) ( readLength * SIZE_3_RATIO );

        // backward search with BWT in cell4
        for ( i = readLength - 1;
                i >= cell1 + cell2 + cell3 && l <= r; --i )
        {
            /* this optimization is like 10 ms faster only....
             if (i % 16 == 15) curWord = *(query + i / CHAR_PER_WORD * 32);
             c = (curWord >> (i % CHAR_PER_WORD * 2)) & 3; */
            c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
            l = GPUBWTOccValue ( bwt, occ, l, c, inverseSa0 ) + 1;
            r = GPUBWTOccValue ( bwt, occ, r + 1, c, inverseSa0 );
        }

        // 1-mismatch cell3 (backward)
        // 2/1-mismatch cell1+2 (backward)
        if ( l <= r )
        {
            backward1MismatchAndBackward2MismatchAnd1Mismatch ( query, cell1 + cell2, cell3,
                    0, cell1 + cell2,
                    bwt, occ, inverseSa0,
                    l, r,
                    saCount, output, maxSARangesAllowed, strand, 0 );
        }
    }
}

__forceinline__ __device__
void matchQueryCaseF ( uint * query, uint readLength,
                       uint * bwt, uint * occ, uint inverseSa0,
                       uint * revBwt, uint * revOcc, uint revInverseSa0,
                       uint & saCount, uint * output, uint textLength,
                       uint maxSARangesAllowed, uint strand, uint numMismatch )
{
    uint l, r;
    uint revL, revR;
    uint i;
    unsigned char c;

    if ( numMismatch == 3 )
    {
        //======================================================
        // FOR THREE MISMATCH
        //======================================================
        // Case F
        // 1. exact cell2 (forward)
        // 2. 2-mismatch cell3+4 (forward)
        // 3. 1-mismatch cell1 (backward)
        //=======================================
        l = 0;
        r = textLength;
        revL = 0;
        revR = r;
        uint cell1 = ( int ) ( readLength * SIZE_1_RATIO );
        uint cell2 = ( int ) ( readLength * SIZE_2_RATIO );
        uint cell3 = ( int ) ( readLength * SIZE_3_RATIO );
        uint cell4 = readLength - cell1 - cell2 - cell3;

        // forward search with BWT until the end of forward depth section
        for ( i = cell1; i < cell1 + cell2 && l <= r; ++i )
        {
            // TODO compute c here
            c = ( query[ ( i / CHAR_PER_WORD ) * 32] >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
            uint cum_start, cum_end;
            revL =
                GPUBWTOccValueWithCumu ( revBwt, revOcc, revL, c, revInverseSa0, cum_start ) + 1;
            revR =
                GPUBWTOccValueWithCumu ( revBwt, revOcc, revR + 1, c, revInverseSa0, cum_end );
            r = r + cum_start - cum_end;
            l = r - ( revR - revL );
        }

        // 1-mismatch cell1 (backward)
        // 2-mismatch cell3+4 (forward)

        if ( l <= r )
        {
            backward1MismatchAndForward2Mismatch ( query, 0, cell1,
                                                   cell1 + cell2, cell3 + cell4,
                                                   bwt, occ, inverseSa0,
                                                   revBwt, revOcc, revInverseSa0,
                                                   l, r,
                                                   revL, revR,
                                                   saCount, output,
                                                   maxSARangesAllowed, strand, 0 );
        }
    }
}


// entry point of kernel
__global__ void kernel ( uint whichCase, uint * queries, uint * readLengths, uint numQueries,
                         uint wordPerQuery,
                         uint * bwt, uint * occ, uint inverseSa0,
                         uint * revBwt, uint * revOcc, uint revInverseSa0,
                         uint textLength,
                         uint * answers,
                         bool * isBad, uint round, uint numMismatch,
                         uint sa_range_allowed, uint wordPerAnswer,
                         bool isExactNumMismatch )
{
    uint queryId = ( blockIdx.x * THREADS_PER_BLOCK + threadIdx.x ) *
                   QUERIES_PER_THREAD;

    if ( queryId < numQueries )
    {
#ifdef BGS_OUTPUT_KERNEL_DEBUG_MESSAGE
        printf ( "[KERNEL] Starting..\n" );
#endif
        ///////////////// Process read //////////////////
        ullint idx = queryId % 32;
        ullint queryOffset = queryId / 32 * 32 * wordPerQuery + idx;
        uint * query = queries + queryOffset;
        ullint answerOffset = queryId / 32 * 32 * wordPerAnswer + idx;
        uint * answer = answers + answerOffset;
        uint readLength = readLengths[queryId];

        for ( uint i = 0; i < wordPerAnswer; ++i )
        { answer[i * 32] = 0xFFFFFFFF; }

        uint numSARanges = 0;
#ifndef BGS_DISABLE_NEGATIVE_STRAND
        uint strand = ( round > 0 ) ? 0 : ( whichCase % 2 );
#else
        uint strand = 0;
#endif

        if ( ( round > 0 ) || ( isBad[queryId] == 0 ) )
        {
            // match the positive strand
            if ( whichCase == 0 )
            {
#ifndef BGS_DISABLE_CASE_A
                matchQueryCaseA ( query, readLength,
                                  bwt, occ, inverseSa0,
                                  revBwt, revOcc, revInverseSa0,
                                  numSARanges, answer, textLength,
                                  sa_range_allowed, strand, numMismatch, isExactNumMismatch );
#endif
            }
            else if ( whichCase == 1 )
            {
#ifndef BGS_DISABLE_CASE_B
                matchQueryCaseB ( query, readLength,
                                  bwt, occ, inverseSa0,
                                  revBwt, revOcc, revInverseSa0,
                                  numSARanges, answer, textLength,
                                  sa_range_allowed, strand, numMismatch );
#endif
            }
            else if ( whichCase == 2 )
            {
#ifndef BGS_DISABLE_CASE_C
                matchQueryCaseC ( query, readLength,
                                  bwt, occ, inverseSa0,
                                  revBwt, revOcc, revInverseSa0,
                                  numSARanges, answer, textLength,
                                  sa_range_allowed, strand, numMismatch );
#endif
            }
            else if ( whichCase == 3 )
            {
#ifndef BGS_DISABLE_CASE_D
                matchQueryCaseD ( query, readLength,
                                  bwt, occ, inverseSa0,
                                  revBwt, revOcc, revInverseSa0,
                                  numSARanges, answer, textLength,
                                  sa_range_allowed, strand, numMismatch );
#endif
            }
            else if ( whichCase == 4 )
            {
#ifndef BGS_DISABLE_CASE_E
                matchQueryCaseE ( query, readLength,
                                  bwt, occ, inverseSa0,
                                  revBwt, revOcc, revInverseSa0,
                                  numSARanges, answer, textLength,
                                  sa_range_allowed, strand, numMismatch );
#endif
            }
            else if ( whichCase == 5 )
            {
#ifndef BGS_DISABLE_CASE_F
                matchQueryCaseF ( query, readLength,
                                  bwt, occ, inverseSa0,
                                  revBwt, revOcc, revInverseSa0,
                                  numSARanges, answer, textLength,
                                  sa_range_allowed, strand, numMismatch );
#endif
            }

#ifndef BGS_DISABLE_NEGATIVE_STRAND
            // reverse the read
            uint leftWordIndex = 0, rightWordIndex = ( readLength - 1 ) / CHAR_PER_WORD;
            uint leftWord = query[leftWordIndex * 32];
            uint rightWord = query[rightWordIndex * 32];

            for ( uint i = 0, j = readLength - 1; i <= j; ++i, --j )
            {
                // check if need to move to next word
                if ( i / CHAR_PER_WORD != leftWordIndex )
                {
                    // write back leftword
                    query[leftWordIndex * 32] = leftWord;
                    // load next leftword
                    leftWordIndex++;
                    leftWord = query[leftWordIndex * 32];
                }

                if ( j / CHAR_PER_WORD != rightWordIndex )
                {
                    // write back rightword
                    query[rightWordIndex * 32] = rightWord;
                    // load next rightword
                    rightWordIndex--;
                    rightWord = query[rightWordIndex * 32];
                }

                // swap left and right characters
                unsigned char leftChar = ( leftWord >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
                unsigned char rightChar = ( rightWord >> ( j % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
                leftWord ^= ( leftChar ^ ( _soap3DnaComplement[rightChar] ) ) << ( i % CHAR_PER_WORD * BIT_PER_CHAR );
                rightWord ^= ( ( _soap3DnaComplement[leftChar] ) ^ rightChar ) << ( j % CHAR_PER_WORD * BIT_PER_CHAR );
            }

            // write back
            if ( leftWordIndex == rightWordIndex )
            {
                uint numLeftBits = ( ( readLength - 1 ) / 2 % CHAR_PER_WORD + 1 ) * BIT_PER_CHAR;
                uint numRightBits = 32 - numLeftBits;
                query[leftWordIndex * 32] = ( ( leftWord << numRightBits ) >> numRightBits ) |
                                            ( ( rightWord >> numLeftBits ) << numLeftBits );
            }
            else
            {
                query[leftWordIndex * 32] = leftWord;
                query[rightWordIndex * 32] = rightWord;
            }

            strand = 1 - strand;

            // match the negative strand
            if ( whichCase == 0 )
            {
#ifndef BGS_DISABLE_CASE_A
                matchQueryCaseA ( query, readLength,
                                  bwt, occ, inverseSa0,
                                  revBwt, revOcc, revInverseSa0,
                                  numSARanges, answer, textLength,
                                  sa_range_allowed, strand, numMismatch, isExactNumMismatch );
#endif
            }
            else if ( whichCase == 1 )
            {
#ifndef BGS_DISABLE_CASE_B
                matchQueryCaseB ( query, readLength,
                                  bwt, occ, inverseSa0,
                                  revBwt, revOcc, revInverseSa0,
                                  numSARanges, answer, textLength,
                                  sa_range_allowed, strand, numMismatch );
#endif
            }
            else if ( whichCase == 2 )
            {
#ifndef BGS_DISABLE_CASE_C
                matchQueryCaseC ( query, readLength,
                                  bwt, occ, inverseSa0,
                                  revBwt, revOcc, revInverseSa0,
                                  numSARanges, answer, textLength,
                                  sa_range_allowed, strand, numMismatch );
#endif
            }
            else if ( whichCase == 3 )
            {
#ifndef BGS_DISABLE_CASE_D
                matchQueryCaseD ( query, readLength,
                                  bwt, occ, inverseSa0,
                                  revBwt, revOcc, revInverseSa0,
                                  numSARanges, answer, textLength,
                                  sa_range_allowed, strand, numMismatch );
#endif
            }
            else if ( whichCase == 4 )
            {
#ifndef BGS_DISABLE_CASE_E
                matchQueryCaseE ( query, readLength,
                                  bwt, occ, inverseSa0,
                                  revBwt, revOcc, revInverseSa0,
                                  numSARanges, answer, textLength,
                                  sa_range_allowed, strand, numMismatch );
#endif
            }
            else if ( whichCase == 5 )
            {
#ifndef BGS_DISABLE_CASE_F
                matchQueryCaseF ( query, readLength,
                                  bwt, occ, inverseSa0,
                                  revBwt, revOcc, revInverseSa0,
                                  numSARanges, answer, textLength,
                                  sa_range_allowed, strand, numMismatch );
#endif
            }

#endif

            // write error code if there are too many SA ranges
            if ( numSARanges == 0 )
            {
#ifdef BGS_OUTPUT_KERNEL_DEBUG_MESSAGE
                printf ( "[KERNEL] This query does not return result\n" );
#endif
                * ( answer ) = 0xFFFFFFFD;
            }
            else if ( numSARanges > sa_range_allowed )
            {
#ifdef BGS_OUTPUT_KERNEL_DEBUG_MESSAGE
                printf ( "[KERNEL] This query returns too many result\n" );
#endif
                * ( answer ) = 0xFFFFFFFE;

                if ( round == 0 )
                { isBad[queryId] = 1; }
            }
        }
        else
        {
            numSARanges = sa_range_allowed + 1;
            * ( answer ) = 0xFFFFFFFE;
        }

#ifdef BGS_OUTPUT_KERNEL_DEBUG_MESSAGE
        printf ( "[Global] Address %u\n", answer );
        printf ( "[KERNEL] Sending out %u SA ranges..\n", numSARanges );

        for ( uint i = 0; i < wordPerQuery / 2; ++i )
        {
            printf ( "%d: %u %u\n", i, * ( answer + ( i * 2 ) * 32 ), * ( answer + ( i * 2 + 1 ) * 32 ) );
        }

#endif
#ifdef BGS_OUTPUT_KERNEL_DEBUG_MESSAGE
        printf ( "[KERNEL] Exiting successfully.\n" );
#endif
    }
}

// entry point of kernel
__global__ void kernel_4mismatch_1 ( uint whichCase, uint * queries, uint * readLengths, uint numQueries,
                                     uint wordPerQuery,
                                     uint * bwt, uint * occ, uint inverseSa0,
                                     uint * revBwt, uint * revOcc, uint revInverseSa0,
                                     uint textLength,
                                     uint * answers,
                                     bool * isBad, uint round, uint sa_range_allowed,
                                     uint wordPerAnswer, bool isExactNumMismatch )
{
    uint queryId = ( blockIdx.x * THREADS_PER_BLOCK + threadIdx.x ) *
                   QUERIES_PER_THREAD;

    if ( queryId < numQueries )
    {
#ifdef BGS_OUTPUT_KERNEL_DEBUG_MESSAGE
        printf ( "[KERNEL] Starting..\n" );
#endif
        ///////////////// Process read //////////////////
        ullint idx = queryId % 32;
        ullint queryOffset = queryId / 32 * 32 * wordPerQuery + idx;
        uint * query = queries + queryOffset;
        ullint answerOffset = queryId / 32 * 32 * wordPerAnswer + idx;
        uint * answer = answers + answerOffset;
        uint readLength = readLengths[queryId];

        for ( uint i = 0; i < wordPerAnswer; ++i )
        { answer[i * 32] = 0xFFFFFFFF; }

        uint numSARanges = 0;
#ifndef BGS_DISABLE_NEGATIVE_STRAND
        uint strand = ( round > 0 ) ? 0 : ( whichCase % 2 );
#else
        uint strand = 0;
#endif

        if ( ( round > 0 ) || ( isBad[queryId] == 0 ) )
        {
            // match the positive strand
            if ( whichCase == 0 )
            {
#ifndef BGS_DISABLE_CASE_A
                matchQueryCaseA_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }
            else if ( whichCase == 1 )
            {
#ifndef BGS_DISABLE_CASE_B
                matchQueryCaseB_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }
            else if ( whichCase == 2 )
            {
#ifndef BGS_DISABLE_CASE_C
                matchQueryCaseC_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }
            else if ( whichCase == 3 )
            {
#ifndef BGS_DISABLE_CASE_D
                matchQueryCaseD_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }
            else if ( whichCase == 4 )
            {
#ifndef BGS_DISABLE_CASE_E
                matchQueryCaseE_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }

#ifndef BGS_DISABLE_NEGATIVE_STRAND
            // reverse the read
            uint leftWordIndex = 0, rightWordIndex = ( readLength - 1 ) / CHAR_PER_WORD;
            uint leftWord = query[leftWordIndex * 32];
            uint rightWord = query[rightWordIndex * 32];

            for ( uint i = 0, j = readLength - 1; i <= j; ++i, --j )
            {
                // check if need to move to next word
                if ( i / CHAR_PER_WORD != leftWordIndex )
                {
                    // write back leftword
                    query[leftWordIndex * 32] = leftWord;
                    // load next leftword
                    leftWordIndex++;
                    leftWord = query[leftWordIndex * 32];
                }

                if ( j / CHAR_PER_WORD != rightWordIndex )
                {
                    // write back rightword
                    query[rightWordIndex * 32] = rightWord;
                    // load next rightword
                    rightWordIndex--;
                    rightWord = query[rightWordIndex * 32];
                }

                // swap left and right characters
                unsigned char leftChar = ( leftWord >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
                unsigned char rightChar = ( rightWord >> ( j % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
                leftWord ^= ( leftChar ^ ( _soap3DnaComplement[rightChar] ) ) << ( i % CHAR_PER_WORD * BIT_PER_CHAR );
                rightWord ^= ( ( _soap3DnaComplement[leftChar] ) ^ rightChar ) << ( j % CHAR_PER_WORD * BIT_PER_CHAR );
            }

            // write back
            if ( leftWordIndex == rightWordIndex )
            {
                uint numLeftBits = ( ( readLength - 1 ) / 2 % CHAR_PER_WORD + 1 ) * BIT_PER_CHAR;
                uint numRightBits = 32 - numLeftBits;
                query[leftWordIndex * 32] = ( ( leftWord << numRightBits ) >> numRightBits ) |
                                            ( ( rightWord >> numLeftBits ) << numLeftBits );
            }
            else
            {
                query[leftWordIndex * 32] = leftWord;
                query[rightWordIndex * 32] = rightWord;
            }

            strand = 1 - strand;

            // match the negative strand
            if ( whichCase == 0 )
            {
#ifndef BGS_DISABLE_CASE_A
                matchQueryCaseA_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }
            else if ( whichCase == 1 )
            {
#ifndef BGS_DISABLE_CASE_B
                matchQueryCaseB_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }
            else if ( whichCase == 2 )
            {
#ifndef BGS_DISABLE_CASE_C
                matchQueryCaseC_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }
            else if ( whichCase == 3 )
            {
#ifndef BGS_DISABLE_CASE_D
                matchQueryCaseD_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }
            else if ( whichCase == 4 )
            {
#ifndef BGS_DISABLE_CASE_E
                matchQueryCaseE_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }

#endif

            // write error code if there are too many SA ranges
            if ( numSARanges == 0 )
            {
#ifdef BGS_OUTPUT_KERNEL_DEBUG_MESSAGE
                printf ( "[KERNEL] This query does not return result\n" );
#endif
                * ( answer ) = 0xFFFFFFFD;
            }
            else if ( numSARanges > sa_range_allowed )
            {
#ifdef BGS_OUTPUT_KERNEL_DEBUG_MESSAGE
                printf ( "[KERNEL] This query returns too many result\n" );
#endif
                * ( answer ) = 0xFFFFFFFE;

                if ( round == 0 )
                { isBad[queryId] = 1; }
            }
        }
        else
        {
            numSARanges = sa_range_allowed + 1;
            * ( answer ) = 0xFFFFFFFE;
        }

#ifdef BGS_OUTPUT_KERNEL_DEBUG_MESSAGE
        printf ( "[Global] Address %u\n", answer );
        printf ( "[KERNEL] Sending out %u SA ranges..\n", numSARanges );

        for ( uint i = 0; i < wordPerQuery / 2; ++i )
        {
            printf ( "%d: %u %u\n", i, * ( answer + ( i * 2 ) * 32 ), * ( answer + ( i * 2 + 1 ) * 32 ) );
        }

#endif
#ifdef BGS_OUTPUT_KERNEL_DEBUG_MESSAGE
        printf ( "[KERNEL] Exiting successfully.\n" );
#endif
    }
}


// entry point of kernel
__global__ void kernel_4mismatch_2 ( uint whichCase, uint * queries, uint * readLengths, uint numQueries,
                                     uint wordPerQuery,
                                     uint * bwt, uint * occ, uint inverseSa0,
                                     uint * revBwt, uint * revOcc, uint revInverseSa0,
                                     uint textLength,
                                     uint * answers,
                                     bool * isBad, uint round, uint sa_range_allowed,
                                     uint wordPerAnswer, bool isExactNumMismatch )
{
    uint queryId = ( blockIdx.x * THREADS_PER_BLOCK + threadIdx.x ) *
                   QUERIES_PER_THREAD;

    if ( queryId < numQueries )
    {
#ifdef BGS_OUTPUT_KERNEL_DEBUG_MESSAGE
        printf ( "[KERNEL] Starting..\n" );
#endif
        ///////////////// Process read //////////////////
        ullint idx = queryId % 32;
        ullint queryOffset = queryId / 32 * 32 * wordPerQuery + idx;
        uint * query = queries + queryOffset;
        ullint answerOffset = queryId / 32 * 32 * wordPerAnswer + idx;
        uint * answer = answers + answerOffset;
        uint readLength = readLengths[queryId];

        for ( uint i = 0; i < wordPerAnswer; ++i )
        { answer[i * 32] = 0xFFFFFFFF; }

        uint numSARanges = 0;
#ifndef BGS_DISABLE_NEGATIVE_STRAND
        uint strand = ( round > 0 ) ? 0 : ( whichCase % 2 );
#else
        uint strand = 0;
#endif

        if ( ( round > 0 ) || ( isBad[queryId] == 0 ) )
        {
            // match the positive strand
            if ( whichCase == 5 )
            {
#ifndef BGS_DISABLE_CASE_F
                matchQueryCaseF_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }
            else if ( whichCase == 6 )
            {
#ifndef BGS_DISABLE_CASE_G
                matchQueryCaseG_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }
            else if ( whichCase == 7 )
            {
#ifndef BGS_DISABLE_CASE_H
                matchQueryCaseH_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }
            else if ( whichCase == 8 )
            {
#ifndef BGS_DISABLE_CASE_I
                matchQueryCaseI_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }
            else if ( whichCase == 9 )
            {
#ifndef BGS_DISABLE_CASE_J
                matchQueryCaseJ_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }

#ifndef BGS_DISABLE_NEGATIVE_STRAND
            // reverse the read
            uint leftWordIndex = 0, rightWordIndex = ( readLength - 1 ) / CHAR_PER_WORD;
            uint leftWord = query[leftWordIndex * 32];
            uint rightWord = query[rightWordIndex * 32];

            for ( uint i = 0, j = readLength - 1; i <= j; ++i, --j )
            {
                // check if need to move to next word
                if ( i / CHAR_PER_WORD != leftWordIndex )
                {
                    // write back leftword
                    query[leftWordIndex * 32] = leftWord;
                    // load next leftword
                    leftWordIndex++;
                    leftWord = query[leftWordIndex * 32];
                }

                if ( j / CHAR_PER_WORD != rightWordIndex )
                {
                    // write back rightword
                    query[rightWordIndex * 32] = rightWord;
                    // load next rightword
                    rightWordIndex--;
                    rightWord = query[rightWordIndex * 32];
                }

                // swap left and right characters
                unsigned char leftChar = ( leftWord >> ( i % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
                unsigned char rightChar = ( rightWord >> ( j % CHAR_PER_WORD * BIT_PER_CHAR ) ) & CHAR_MASK;
                leftWord ^= ( leftChar ^ ( _soap3DnaComplement[rightChar] ) ) << ( i % CHAR_PER_WORD * BIT_PER_CHAR );
                rightWord ^= ( ( _soap3DnaComplement[leftChar] ) ^ rightChar ) << ( j % CHAR_PER_WORD * BIT_PER_CHAR );
            }

            // write back
            if ( leftWordIndex == rightWordIndex )
            {
                uint numLeftBits = ( ( readLength - 1 ) / 2 % CHAR_PER_WORD + 1 ) * BIT_PER_CHAR;
                uint numRightBits = 32 - numLeftBits;
                query[leftWordIndex * 32] = ( ( leftWord << numRightBits ) >> numRightBits ) |
                                            ( ( rightWord >> numLeftBits ) << numLeftBits );
            }
            else
            {
                query[leftWordIndex * 32] = leftWord;
                query[rightWordIndex * 32] = rightWord;
            }

            strand = 1 - strand;

            // match the negative strand
            if ( whichCase == 5 )
            {
#ifndef BGS_DISABLE_CASE_F
                matchQueryCaseF_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }
            else if ( whichCase == 6 )
            {
#ifndef BGS_DISABLE_CASE_G
                matchQueryCaseG_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }
            else if ( whichCase == 7 )
            {
#ifndef BGS_DISABLE_CASE_H
                matchQueryCaseH_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }
            else if ( whichCase == 8 )
            {
#ifndef BGS_DISABLE_CASE_I
                matchQueryCaseI_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }
            else if ( whichCase == 9 )
            {
#ifndef BGS_DISABLE_CASE_J
                matchQueryCaseJ_4mismatch ( query, readLength,
                                            bwt, occ, inverseSa0,
                                            revBwt, revOcc, revInverseSa0,
                                            numSARanges, answer, textLength,
                                            sa_range_allowed, strand );
#endif
            }

#endif

            // write error code if there are too many SA ranges
            if ( numSARanges == 0 )
            {
#ifdef BGS_OUTPUT_KERNEL_DEBUG_MESSAGE
                printf ( "[KERNEL] This query does not return result\n" );
#endif
                * ( answer ) = 0xFFFFFFFD;
            }
            else if ( numSARanges > sa_range_allowed )
            {
#ifdef BGS_OUTPUT_KERNEL_DEBUG_MESSAGE
                printf ( "[KERNEL] This query returns too many result\n" );
#endif
                * ( answer ) = 0xFFFFFFFE;

                if ( round == 0 )
                { isBad[queryId] = 1; }
            }
        }
        else
        {
            numSARanges = sa_range_allowed + 1;
            * ( answer ) = 0xFFFFFFFE;
        }

#ifdef BGS_OUTPUT_KERNEL_DEBUG_MESSAGE
        printf ( "[Global] Address %u\n", answer );
        printf ( "[KERNEL] Sending out %u SA ranges..\n", numSARanges );

        for ( uint i = 0; i < wordPerQuery / 2; ++i )
        {
            printf ( "%d: %u %u\n", i, * ( answer + ( i * 2 ) * 32 ), * ( answer + ( i * 2 + 1 ) * 32 ) );
        }

#endif
#ifdef BGS_OUTPUT_KERNEL_DEBUG_MESSAGE
        printf ( "[KERNEL] Exiting successfully.\n" );
#endif
    }
}
